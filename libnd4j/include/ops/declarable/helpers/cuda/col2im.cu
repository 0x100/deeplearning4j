#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// Created by raver119 on 30.11.17.
//

#include <ops/declarable/helpers/col2im.h>

namespace nd4j {
    namespace ops {
        namespace helpers {

//////////////////////////////////////////////////////////////////////////
// [bS, iC, kH, kW, oH, oW] is de-convoluted to [bS, iC, iH, iW]
template<typename T>
__global__ static void col2imCuda(const void *in, void *out, const Nd4jLong *inShapeInfo, const Nd4jLong *outShapeInfo, const int strideY, const int strideX, const int padHeight, const int padWidth, const int imgHeight, const int imgWidth, const int dY, const int dX) {

          const auto dx = reinterpret_cast<const T*>(in);
          auto result = reinterpret_cast<T*>(out);

          auto inShape = shape::shapeOf(const_cast<Nd4jLong *>(inShapeInfo));
          auto inStride = shape::stride(const_cast<Nd4jLong *>(inShapeInfo));

          int strideex = inStride[0];
          int stridech = inStride[1];
          int stridekrow = inStride[2];
          int stridekcol = inStride[3];
          int striderow = inStride[4];
          int stridecol = inStride[5];

          int kernelHeight = inShape[2];
          int kernelWidth = inShape[3];

          auto outShape = shape::shapeOf(const_cast<Nd4jLong *>(outShapeInfo));
          auto resultOrder = shape::order(const_cast<Nd4jLong *>(outShapeInfo));
          auto outStride = shape::stride(const_cast<Nd4jLong *>(outShapeInfo));

          int samples = outShape[0];
          int depth = outShape[1];
          int imgH = outShape[2];
          int imgW = outShape[3];

          int height_col = inShape[4];//(imgHeight + 2 * padHeight - kernelHeight) / strideX + 1;
          int width_col = inShape[5];//(imgWidth + 2 * padWidth - kernelWidth) / strideY + 1;

          int n = samples * depth * imgHeight * imgWidth;

          //Effective kernel size, accounting for dilation
          int kEffectiveW = kernelWidth + (kernelWidth - 1) * (dX - 1);
          int kEffectiveH = kernelHeight + (kernelHeight - 1) * (dY - 1);

          for (int i = (blockDim.x * blockIdx.x) + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
              T val = 0;
              int w_im = i % imgWidth + padWidth;
              int h_im = (i / imgWidth) % imgHeight + padHeight;
              int c_im = i / (imgWidth * imgHeight);

              int num_im = c_im / depth;
              int depth_im = c_im % depth;

              // compute the start and end of the output
              // These are the indexes for dimensions ??? in the 6d col matrix
              int w_col_start = (w_im < kEffectiveW) ? 0 : (w_im - kEffectiveW) / strideX + 1;
              int w_col_end = nd4j::math::nd4j_min<int>(w_im / strideX + 1, width_col);

              int h_col_start = (h_im < kEffectiveH) ? 0 : (h_im - kEffectiveH) / strideY + 1;
              int h_col_end = nd4j::math::nd4j_min<int>(h_im / strideY + 1, height_col);


              //Iterate over col entries in the 6d array... these are added up
              for (int h_col = h_col_start; h_col < h_col_end; h_col += 1) {
                  for (int w_col = w_col_start; w_col < w_col_end; w_col += 1) {
                      int h_k = (h_im - h_col * strideY);
                      int w_k = (w_im - w_col * strideX);

                      if(h_k % dY == 0 && w_k % dX == 0){
                          h_k /= dY;
                          w_k /= dX;

                          int data_col_index = num_im * strideex + depth_im * stridech + h_k * stridekrow + w_k * stridekcol + h_col * striderow + w_col * stridecol;
                          val += dx[data_col_index];
                      }
                  }
              }

              int i_f = 0;
              int i_c = i;
              for (int dim = 3; dim >= 0; dim--) {
                  i_f += (i_c % outShape[dim])  * outStride[dim];
                  i_c = i_c / outShape[dim];
              }

              result[i_f] = val;
          }
}

//////////////////////////////////////////////////////////////////////////
template<typename T>
void col2imCudaLauncher(nd4j::LaunchContext  &context, const void *x, void *z, const Nd4jLong *xShapeInfo, const Nd4jLong *zShapeInfo, const int sH, const int sW, const int pH, const int pW, const int iH, const int iW, const int dH, const int dW) {
    col2imCuda<T><<<512, 512, 1024, *context.getCudaStream()>>>(x, z, xShapeInfo, zShapeInfo, sH, sW, pH, pW, iH, iW, dH, dW);
}

//////////////////////////////////////////////////////////////////////////
void col2im(nd4j::LaunchContext & context, const NDArray& input, NDArray& output, const int sH, const int sW, const int pH, const int pW, const int iH, const int iW, const int dH, const int dW) {
    
    NDArray::prepareSpecialUse({&output}, {&input});

    BUILD_SINGLE_SELECTOR(output.dataType(), col2imCudaLauncher, (context, input.getSpecialBuffer(), output.getSpecialBuffer(), input.getSpecialShapeInfo(), output.getSpecialShapeInfo(), sH, sW, pH, pW, iH, iW, dH, dW), FLOAT_TYPES);

    NDArray::registerSpecialUse({&output}, {&input});
}



BUILD_SINGLE_TEMPLATE(template void col2imCudaLauncher, (nd4j::LaunchContext  &context, const void *x, void *z, const Nd4jLong *xShapeInfo, const Nd4jLong *zShapeInfo, const int sH, const int sW, const int pH, const int pW, const int iH, const int iW, const int dH, const int dW), FLOAT_TYPES);

}
}
}