#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

#include <ops/declarable/helpers/convolutions.h>
#include <ops/declarable/helpers/im2col.h>
#include <ops/declarable/helpers/col2im.h>
#include <NDArrayFactory.h>
#include <MmulHelper.h>

namespace nd4j {
    namespace ops {

        void ConvolutionUtils::conv2d(nd4j::graph::LaunchContext& block, const NDArray* input, const NDArray* weights, const NDArray* bias, NDArray* output, const int kH, const int kW, const int sH, const int sW, int pH, int pW, const int dH, const int dW, const int isSameMode, const int isNCHW) {

        }

        void ConvolutionUtils::conv2d(nd4j::graph::LaunchContext& block, const std::vector<NDArray*>& inArrs, NDArray* output, const std::vector<int>& intArgs) {

        }

        void ConvolutionUtils::conv2dBP(nd4j::graph::LaunchContext& block, const std::vector<NDArray*>& inArrs, const std::vector<NDArray*>& outArrs, const std::vector<int>& intArgs) {

        }

        void ConvolutionUtils::conv2dBP(nd4j::graph::LaunchContext& block, const NDArray* input, const NDArray* weights, const NDArray* bias, const NDArray* gradO, NDArray* gradI, NDArray* gradW, NDArray* gradB, const int kH, const int kW, const int sH, const int sW, int pH, int pW, const int dH, const int dW, const int isSameMode, const int isNCHW) {

        }

        void ConvolutionUtils::depthwiseConv2d(nd4j::graph::LaunchContext& block, const NDArray* input, const NDArray* weights, const NDArray* bias, NDArray* output, const int kH, const int kW, const int sH, const int sW, int pH, int pW, const int dH, const int dW, const int isSameMode, const int isNCHW) {

        }

        void ConvolutionUtils::depthwiseConv2dBP(nd4j::graph::LaunchContext& block, const NDArray* input, const NDArray* weights, const NDArray* bias, const NDArray* gradO, NDArray* gradI, NDArray* gradW, NDArray* gradB, const int kH, const int kW, const int sH, const int sW, int pH, int pW, const int dH, const int dW, const int isSameMode, const int isNCHW) {

        }

        void ConvolutionUtils::sconv2d(nd4j::graph::LaunchContext& block, const NDArray* input, const NDArray* weightsDepth, const NDArray* weightsPoint, const NDArray* bias,  NDArray* output, const int kH, const int kW, const int sH, const int sW, int pH, int pW, const int dH, const int dW, const int isSameMode, const int isNCHW) {

        }

        void ConvolutionUtils::vol2col(nd4j::graph::LaunchContext& block, const NDArray& vol, NDArray& col, const int sD, const int sH, const int sW, const int pD, const int pH, const int pW, const int dD, const int dH, const int dW) {

        }

        void ConvolutionUtils::col2vol(nd4j::graph::LaunchContext& block, const NDArray& col, NDArray& vol, const int sD, const int sH, const int sW, const int pD, const int pH, const int pW, const int dD, const int dH, const int dW) {

        }

        void ConvolutionUtils::upsampling2d(nd4j::graph::LaunchContext& block, const NDArray& input, NDArray& output, const int factorH, const int factorW, const bool isNCHW) {

        }

        void ConvolutionUtils::upsampling3d(nd4j::graph::LaunchContext& block, const NDArray& input, NDArray& output, const int factorD, const int factorH, const int factorW, const bool isNCDHW) {

        }

        void ConvolutionUtils::upsampling2dBP(nd4j::graph::LaunchContext& block, const NDArray& gradO, NDArray& gradI, const bool isNCHW) {

        }

        void ConvolutionUtils::upsampling3dBP(nd4j::graph::LaunchContext& block, const NDArray& gradO, NDArray& gradI, const bool isNCDHW) {

        }

        template <typename T>
        static __global__ void global_avg_pooling2d() {

        }

        template <typename T>
        static void _avg_pooling2d(nd4j::graph::LaunchContext& block, void *vx, Nd4jLong *vxShapeInfo, void *vz, Nd4jLong vzShapeInfo, const int kH, const int kW, const int sH, const int sW, const int pH, const int pW, const int dH, const int dW, const int extraParam0) {

        }
        BUILD_SINGLE_TEMPLATE(template void _avg_pooling2d, (nd4j::graph::LaunchContext& block, void *vx, Nd4jLong *vxShapeInfo, void *vz, Nd4jLong vzShapeInfo, const int kH, const int kW, const int sH, const int sW, const int pH, const int pW, const int dH, const int dW, const int extraParam0), FLOAT_TYPES);

        void ConvolutionUtils::pooling2d(nd4j::graph::LaunchContext& block, const NDArray& input, NDArray& output, const int kH, const int kW, const int sH, const int sW, const int pH, const int pW, const int dH, const int dW, const PoolingType poolingMode, const int extraParam0) {
            switch (poolingMode) {
                case MAX_POOL: {

                    }
                    break;
                case AVG_POOL: {

                    }
                    break;
                case PNORM_POOL: {

                    }
                    break;
                default:
                    throw std::runtime_error("Pooling2D: Unknown PoolingType used");
            }
        }

        void ConvolutionUtils::pooling3d(nd4j::graph::LaunchContext& block, const NDArray& input, NDArray& output, const int kD, const int kH, const int kW, const int sD, const int sH, const int sW, const int pD, const int pH, const int pW, const int dD, const int dH, const int dW, const int poolingMode, const int extraParam0) {

        }

        void ConvolutionUtils::pooling2dBP(nd4j::graph::LaunchContext& block, const NDArray& input, const NDArray& gradO, NDArray& gradI, const int kH, const int kW, const int sH, const int sW, const int pH, const int pW, const int dH, const int dW, const int poolingMode, const int extraParam0) {

        }

        void ConvolutionUtils::pooling3dBP(nd4j::graph::LaunchContext &block, const NDArray& input, const NDArray& gradO, NDArray& gradI, const int kD, const int kH, const int kW, const int sD, const int sH, const int sW, const int pD, const int pH, const int pW, const int dD, const int dH, const int dW, const int poolingMode, const int extraParam0) {

        }
    }
}