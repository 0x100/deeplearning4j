#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 20.04.2018
//


#include<ops/declarable/helpers/transforms.h>
#include <array/ResultSet.h>
#include <helpers/ShapeUtils.h>
#include <numeric>
#include <NDArrayFactory.h>
#include <helpers/TAD.h>
#include <exceptions/cuda_exception.h>
#include <PointersManager.h>
#include <ConstantTadHelper.h>

namespace nd4j 	  {
namespace ops 	  {
namespace helpers {

///////////////////////////////////////////////////////////////////
template<typename T>
__global__ static void concatCuda(const int numOfArrs, void* pVx,  void* pxShapeInfo, void* pVz, void* pzShapeInfo) {
    
    __shared__ int arrIdx, blocksPerArr;
    __shared__ T *x, *z;
    __shared__ Nd4jLong *zShapeInfo, *xShapeInfo, arrLen, arrLenPerBlock, start, end;
    
    if (threadIdx.x == 0) {
            
        blocksPerArr = (gridDim.x + numOfArrs - 1) / numOfArrs;     // ceil 
        arrIdx = blockIdx.x / blocksPerArr;
    
        x = reinterpret_cast<T*>(reinterpret_cast<void**>(pVx)[arrIdx]);
        z = reinterpret_cast<T*>(reinterpret_cast<void**>(pVz)[arrIdx]);        
        xShapeInfo = reinterpret_cast<Nd4jLong**>(pxShapeInfo)[arrIdx];
        zShapeInfo = reinterpret_cast<Nd4jLong**>(pzShapeInfo)[arrIdx];
        arrLen = shape::length(xShapeInfo);

        arrLenPerBlock = (arrLen + blocksPerArr - 1) / blocksPerArr;  // ceil

        start = (blockIdx.x % blocksPerArr) * arrLenPerBlock;
        end   = (start + arrLenPerBlock) > arrLen ? arrLen : (start + arrLenPerBlock);        
    }

    __syncthreads();    

    for (Nd4jLong i = start + threadIdx.x; i < end; i += blockDim.x)
        z[shape::getIndexOffset(i, zShapeInfo, arrLen)] = x[shape::getIndexOffset(i, xShapeInfo, arrLen)];
}

template<typename T>
__host__ static void concatCudaLauncher(const int numOfArrs, const hipStream_t *stream,  void* pVx, void* pxShapeInfo, void* pVz, void* pzShapeInfo) {

    concatCuda<T><<<512, 256, 1024, *stream>>>(numOfArrs, pVx, pxShapeInfo, pVz, pzShapeInfo);
}

    //////////////////////////////////////////////////////////////////////////
    void triu(graph::LaunchContext* context, const NDArray& input, NDArray& output, const int diagonal) {

    }


    //////////////////////////////////////////////////////////////////////////
    template <typename T>
    static void triuBP_(graph::LaunchContext* context, const NDArray& input, const NDArray& gradO, NDArray& gradI, const int diagonal) {

    }

    void triuBP(graph::LaunchContext* context, const NDArray& input, const NDArray& gradO, NDArray& gradI, const int diagonal) {
        BUILD_SINGLE_SELECTOR(gradO.dataType(), triuBP_, (context, input, gradO, gradI, diagonal), LIBND4J_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template void triuBP_, (graph::LaunchContext* context, const NDArray& input, const NDArray& gradO, NDArray& gradI, const int diagonal), LIBND4J_TYPES);

    //////////////////////////////////////////////////////////////////////////
    template <typename T>
    static void trace_(graph::LaunchContext* context, const NDArray& input, NDArray& output) {

    }

    void trace(graph::LaunchContext* context, const NDArray& input, NDArray& output) {
        BUILD_SINGLE_SELECTOR(input.dataType(), trace_, (context, input, output), LIBND4J_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template void trace_, (graph::LaunchContext* context, const NDArray& input, NDArray& output), LIBND4J_TYPES);

    //////////////////////////////////////////////////////////////////////////
    template <typename T>
    void randomShuffle_(graph::LaunchContext* context, NDArray& input, NDArray& output, nd4j::random::RandomBuffer& rng, const bool isInplace) {

    }

    void randomShuffle(graph::LaunchContext* context, NDArray& input, NDArray& output, nd4j::random::RandomBuffer& rng, const bool isInplace) {
        BUILD_SINGLE_SELECTOR(input.dataType(), randomShuffle_, (context, input, output, rng, isInplace), LIBND4J_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template void randomShuffle_, (graph::LaunchContext* context, NDArray& input, NDArray& output, nd4j::random::RandomBuffer& rng, const bool isInplace), LIBND4J_TYPES);

    //////////////////////////////////////////////////////////////////////////
    // Pad kernels
    //
    static __global__ void padFillIndicesKernel(Nd4jLong* outIndices, void* paddingBuffer, Nd4jLong* paddingShape, Nd4jLong* inputShape, bool shortType, int rankBorder) {
        const auto tid = blockIdx.x * gridDim.x + threadIdx.x;
        const auto step = gridDim.x * blockDim.x;
//        if (threadIdx.x == 0) {
//            outIndices[2 * rankBorder + 1] = shortType?reinterpret_cast<int*>(paddingBuffer)[rankBorder]:reinterpret_cast<Nd4jLong*>(paddingBuffer)[pos];
//            outIndices[2 * rankBorder] = rightBorder;
//        }
        __syncthreads();

        for(int i = tid; i < rankBorder + 1; i += step) {
            Nd4jLong coords[2] = {i, 0};
            auto pos = shape::getOffset(0, shape::shapeOf(paddingShape), shape::stride(paddingShape), coords, rankBorder + 1);
            outIndices[2 * i] = shortType?reinterpret_cast<int*>(paddingBuffer)[pos]:reinterpret_cast<Nd4jLong*>(paddingBuffer)[pos];
            outIndices[2 * i + 1] = outIndices[2 * i] + shape::sizeAt(inputShape, i);
        }
    }

    template <typename T>
    static __global__ void padFillValues(void* outputBuffer, Nd4jLong* outputShape, Nd4jLong* paddingBound,
            void* inputBuffer, Nd4jLong* inputShape,
            Nd4jLong* inputTadShape, Nd4jLong* inputTadOffsets, Nd4jLong* outputTadShape, Nd4jLong* outputTadOffsets, const int mode, void* value) {

            __shared__ T* z;
            __shared__ T* x;
            __shared__ T* val;
            __shared__ Nd4jLong inputLen;
            __shared__ Nd4jLong outputLen;
            __shared__ Nd4jLong rank;
            __shared__ Nd4jLong lastInDimSize;
            __shared__ Nd4jLong outTadCount;
            __shared__ Nd4jLong inTadCount;
            if (threadIdx.x == 0) {
                z = reinterpret_cast<T*>(outputBuffer);
                x = reinterpret_cast<T*>(inputBuffer);
                inputLen = shape::length(inputShape);
                outputLen = shape::length(outputShape);
                if (value && mode == 0) // only for CONSTANT mode
                    val = reinterpret_cast<T*>(value);
                else
                    val = nullptr;
                rank = shape::rank(outputShape);
                Nd4jLong lastInDimSize  = shape::sizeAt(inputShape, rank - 1);

                outTadCount = outputLen / shape::length(outputTadShape);
                inTadCount = inputLen / shape::length(inputTadShape);
                printf("%lld, %lld\n", inTadCount, outTadCount);
            }
            __syncthreads();

            const auto tid = blockIdx.x * gridDim.x + threadIdx.x;
            const auto step = gridDim.x * blockDim.x;
            const auto stepY = gridDim.y * blockDim.y;
            Nd4jLong k = rank - 1;
            //for (Nd4jLong k = blockIdx.y * gridDim.y + threadIdx.y; k < rank; k += stepY) {
                for (Nd4jLong i = tid; i < outputLen; i += step) {
                    if (i >= paddingBound[2 * k] && i < paddingBound[2 * k + 1]) {
                        if (k == rank - 1)
                            z[i] = x[i - paddingBound[2 * k]];
                    }
                    else if (val)
                        z[i] = val[0];
                    else if (mode != 0){
                        Nd4jLong startL = mode == 1 ? 1 : 0;                            // REFLECT or SYMMETRIC
                        Nd4jLong startR = mode == 1 ? lastInDimSize - 2 : lastInDimSize - 1;        // REFLECT or SYMMETRIC
                        if (i < paddingBound[2 * k]) {
                            z[i] = x[paddingBound[2 * k] - i - startL];
                        }
                        else {
                            z[i] = x[i - startR - paddingBound[2 * k + 1] + paddingBound[2 * k] + 1];
                        }
                    }
                }
            //}
    }

    template<typename T>
    void pad_(graph::LaunchContext* context, const int mode, const NDArray& input, const NDArray& paddings, NDArray& output, NDArray const& padValue) {
        const int rank = output.rankOf();
        const int rankBorder = rank - 1;
        std::vector<int> dimsToExclude({rankBorder});
        std::iota(dimsToExclude.begin(), dimsToExclude.end(), 0);             // fill with 0, 1, ... rank-1
        //dimsToExclude.pop_back();

//        Nd4jLong numLeft    = paddings.e<Nd4jLong>(rankBorder, 0);
//        Nd4jLong numRight   = paddings.e<Nd4jLong>(rankBorder, 1);
        Nd4jLong inDimSize  = input.sizeAt(rankBorder);
        Nd4jLong outDimSize = output.sizeAt(rankBorder);
        Nd4jLong* outIdx = nullptr;
        hipError_t err = hipMalloc(&outIdx, 2 * rank * sizeof(Nd4jLong));
        if (0 != err) {
            throw cuda_exception::build("Cannot allocate memory for pad indices", err);
        }
        err = hipMemset(outIdx, 0, 2 * rank * sizeof(Nd4jLong));
        if (0 != err) {
            throw cuda_exception::build("Cannot initialize memory for pad indices", err);
        }

       // dim3 launcDim(16, 32, 512);
        auto stream = context->getCudaStream();
        bool shortedType = (paddings.dataType() == DataType::INT32);
        padFillIndicesKernel<<<16, 32, 512, *stream>>>(outIdx, paddings.getSpecialBuffer(), paddings.getSpecialShapeInfo(), input.getSpecialShapeInfo(), shortedType, rankBorder);

        Nd4jLong numOfSubArrs = ShapeUtils::getNumOfSubArrs(input.getShapeInfo(), dimsToExclude);

        //NDArray outSubArr0 = output(outIdx, true);
        dim3 launchDim(128, 256, 2048);
        auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input.getShapeInfo(), dimsToExclude);
        auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output.getShapeInfo(), dimsToExclude);

        padFillValues<T><<<launchDim.x, launchDim.y, launchDim.z, *stream>>>(output.specialBuffer(),
                output.specialShapeInfo(), outIdx, input.getSpecialBuffer(), input.getSpecialShapeInfo(),
                packX.specialShapeInfo(), packX.specialOffsets(), packZ.specialShapeInfo(), packZ.specialOffsets(), mode, padValue.getSpecialBuffer());

//        err = hipFree(outIdx);
//        if (0 != err) {
//            throw cuda_exception::build("Cannot release memory for pad indices", err);
//        }

//#pragma omp parallel for schedule(guided)
//        for(Nd4jLong j = 0; j < numOfSubArrs; ++j) {
//
//            NDArray outSubArr1   = outSubArr0(j, dimsToExclude);
//            NDArray inSubArr     = input(j, dimsToExclude);
//            NDArray outSubArrMid = outSubArr1(outIdx[1]);
//
//            outSubArrMid.assign(inSubArr);      // assign middle
//
//            if(mode == 0)  { // CONSTANT
//                if(numLeft != 0) {
//                    NDArray temp = outSubArr1(outIdx[2]);
//                    temp.assign(padValue);                        // assign left
//                }
//                if(numRight != 0) {
//                    NDArray temp = outSubArr1(outIdx[3]);
//                    temp.assign(padValue);                        // assign right
//                }
//            }
//            else {                                                              // REFLECT or SYMMETRIC
//
//#pragma omp parallel for schedule(guided)
//                for(Nd4jLong k = numLeft-1, e = startL; k >= 0; --k, ++e)     // fill left side
//                    outSubArr1.t<T>(k) = inSubArr.t<T>(e);
//
//#pragma omp parallel for schedule(guided)
//                for(Nd4jLong k = numLeft + inDimSize, e = startR; k < outDimSize; ++k, --e)     // fill right side
//                    outSubArr1.t<T>(k) = inSubArr.t<T>(e);
//            }
//        }
    }

    void pad(graph::LaunchContext* context, const int mode, const NDArray& input, const NDArray& paddings, NDArray& output, NDArray const& padValue) {
        BUILD_SINGLE_SELECTOR(input.dataType(), pad_, (context, mode, input, paddings, output, padValue), LIBND4J_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template void pad_, (graph::LaunchContext* context, const int mode, const NDArray& input, const NDArray& paddings, NDArray& output, NDArray const& padValue), LIBND4J_TYPES);

    ////////////////////////////////////////////////////////////////////////
    void invertPermutation(graph::LaunchContext* context, const NDArray& input, NDArray& output) {

    }

    ////////////////////////////////////////////////////////////////////////
    template<typename T>
    static void gatherND_(graph::LaunchContext* context, NDArray& input, NDArray& indices, NDArray& output) {

    }

    void gatherND(graph::LaunchContext* context, NDArray& input, NDArray& indices, NDArray& output) {
        BUILD_SINGLE_SELECTOR(input.dataType(), gatherND_, (context, input, indices, output), LIBND4J_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template void gatherND_, (graph::LaunchContext* context, NDArray& input, NDArray& indices, NDArray& output), LIBND4J_TYPES);



    //////////////////////////////////////////////////////////////////////////
    void eye(graph::LaunchContext* context, NDArray& output) {

    }

    //////////////////////////////////////////////////////////////////////////
    void scatterUpdate(graph::LaunchContext* context, NDArray& operand, NDArray& updates, const std::vector<int>* intArgs) {

    }

    //////////////////////////////////////////////////////////////////////////
    template <typename T, typename Z>
    static __global__ void global_mergeMaxIndex_(void **inArrs, void **inShapes, const int numArrays, void *voutput, Nd4jLong *outputShape, Nd4jLong length) {
        auto output = reinterpret_cast<Z*>(voutput);

        const auto tid = blockIdx.x * gridDim.x + threadIdx.x;
        const auto step = gridDim.x * blockDim.x;

        for (Nd4jLong e = tid; e < length; e += step) {
            T mVal = -DataTypeUtils::max<T>();
            Z mIdx(0);

            for (int i = 0; i < numArrays; i++) {
                auto x = reinterpret_cast<T*>(inArrs[i]);
                auto xShape = reinterpret_cast<Nd4jLong *>(inShapes[i]);
                auto val = x[shape::getIndexOffset(e, xShape, length)];;
                if (mVal < val)
                    mIdx = static_cast<Z>(e);
            }
            __syncthreads();

            output[shape::getIndexOffset(e, outputShape, length)] = mIdx;
        }
    }

    template <typename T, typename Z>
    static void mergeMaxIndex_(graph::LaunchContext* context, const std::vector<NDArray*>& inArrs, NDArray& output) {
        std::vector<void *> inBuffers(inArrs.size());
        std::vector<void *> inShapes(inArrs.size());

        for (int e = 0; e < inArrs.size(); e++) {
            inBuffers[e] = inArrs[e]->getSpecialBuffer();
            inShapes[e] = inArrs[e]->getSpecialShapeInfo();
        }

        PointersManager manager(context, "mergeMaxIndex");

        auto pInBuffers = reinterpret_cast<void **>(manager.replicatePointer(inBuffers.data(), inBuffers.size() * sizeof(void *)));
        auto pInShapes = reinterpret_cast<void **>(manager.replicatePointer(inShapes.data(), inShapes.size() * sizeof(void *)));
        auto length = output.lengthOf();

        global_mergeMaxIndex_<T,Z><<<512, 512, 512, *context->getCudaStream()>>>(pInBuffers, pInShapes, (int) inArrs.size(), output.getSpecialBuffer(), output.getSpecialShapeInfo(), length);

        manager.synchronize();
    }

    void mergeMaxIndex(graph::LaunchContext* context, const std::vector<NDArray*>& inArrs, NDArray& output) {
        BUILD_DOUBLE_SELECTOR(inArrs[0]->dataType(), output.dataType(), mergeMaxIndex_, (context, inArrs, output), LIBND4J_TYPES, INTEGER_TYPES);
    }

    BUILD_DOUBLE_TEMPLATE(template void mergeMaxIndex_, (graph::LaunchContext* context, const std::vector<NDArray*>& inArrs, NDArray& output), LIBND4J_TYPES, INTEGER_TYPES);

    //////////////////////////////////////////////////////////////////////////
    template <typename T>
    static __global__ void global_mergeMax_(void **inArrs, void **inShapes, const int numArrays, void *voutput, Nd4jLong *outputShape, Nd4jLong length) {
        auto output = reinterpret_cast<T*>(voutput);

        const auto tid = blockIdx.x * gridDim.x + threadIdx.x;
        const auto step = gridDim.x * blockDim.x;

        for (Nd4jLong e = tid; e < length; e += step) {
            T mVal = -DataTypeUtils::max<T>();

            for (int i = 0; i < numArrays; i++) {
                auto x = reinterpret_cast<T*>(inArrs[i]);
                auto xShape = reinterpret_cast<Nd4jLong *>(inShapes[i]);
                auto val = x[shape::getIndexOffset(e, xShape, length)];;
                if (mVal < val)
                    mVal = val;
            }
            __syncthreads();

            output[shape::getIndexOffset(e, outputShape, length)] = mVal;
        }
    }

    template<typename T>
    static void mergeMax_(graph::LaunchContext* context, const std::vector<NDArray*>& inArrs, NDArray& output) {
        std::vector<void *> inBuffers(inArrs.size());
        std::vector<void *> inShapes(inArrs.size());

        for (int e = 0; e < inArrs.size(); e++) {
            inBuffers[e] = inArrs[e]->getSpecialBuffer();
            inShapes[e] = inArrs[e]->getSpecialShapeInfo();
        }

        PointersManager manager(context, "mergeMax");

        auto pInBuffers = reinterpret_cast<void **>(manager.replicatePointer(inBuffers.data(), inBuffers.size() * sizeof(void *)));
        auto pInShapes = reinterpret_cast<void **>(manager.replicatePointer(inShapes.data(), inShapes.size() * sizeof(void *)));
        auto length = output.lengthOf();

        global_mergeMax_<T><<<512, 512, 512, *context->getCudaStream()>>>(pInBuffers, pInShapes, (int) inArrs.size(), output.getSpecialBuffer(), output.getSpecialShapeInfo(), length);

        manager.synchronize();
    }
    BUILD_SINGLE_TEMPLATE(template void mergeMax_, (graph::LaunchContext* context, const std::vector<NDArray*>& inArrs, NDArray& output), LIBND4J_TYPES);

    void mergeMax(graph::LaunchContext* context, const std::vector<NDArray*>& inArrs, NDArray& output) {
        BUILD_SINGLE_SELECTOR(output.dataType(), mergeMax_, (context, inArrs, output), LIBND4J_TYPES);
    }

    //////////////////////////////////////////////////////////////////////////
    template <typename T>
    static __global__ void global_mergeAvg_(void **inArrs, void **inShapes, const int numArrays, void *voutput, Nd4jLong *outputShape, Nd4jLong length) {
        auto output = reinterpret_cast<T*>(voutput);

        const auto tid = blockIdx.x * gridDim.x + threadIdx.x;
        const auto step = gridDim.x * blockDim.x;

        for (Nd4jLong e = tid; e < length; e += step) {
            T sum(0.0f);

            for (int i = 0; i < numArrays; i++) {
                auto x = reinterpret_cast<T*>(inArrs[i]);
                auto xShape = reinterpret_cast<Nd4jLong *>(inShapes[i]);

                sum += x[shape::getIndexOffset(e, xShape, length)];
            }

            output[shape::getIndexOffset(e, outputShape, length)] = sum / numArrays;
        }
    }

    template<typename T>
    static void mergeAvg_(graph::LaunchContext* context, const std::vector<NDArray*>& inArrs, NDArray& output) {
        std::vector<void *> inBuffers(inArrs.size());
        std::vector<void *> inShapes(inArrs.size());

        for (int e = 0; e < inArrs.size(); e++) {
            inBuffers[e] = inArrs[e]->getSpecialBuffer();
            inShapes[e] = inArrs[e]->getSpecialShapeInfo();
        }

        PointersManager manager(context, "mergeAvg");

        auto pInBuffers = reinterpret_cast<void **>(manager.replicatePointer(inBuffers.data(), inBuffers.size() * sizeof(void *)));
        auto pInShapes = reinterpret_cast<void **>(manager.replicatePointer(inShapes.data(), inShapes.size() * sizeof(void *)));
        auto length = output.lengthOf();

        global_mergeAvg_<T><<<512, 512, 512, *context->getCudaStream()>>>(pInBuffers, pInShapes, (int) inArrs.size(), output.getSpecialBuffer(), output.getSpecialShapeInfo(), length);

        manager.synchronize();
    }
    BUILD_SINGLE_TEMPLATE(template void mergeAvg_, (graph::LaunchContext* context, const std::vector<NDArray*>& inArrs, NDArray& output), LIBND4J_TYPES);

    void mergeAvg(graph::LaunchContext* context, const std::vector<NDArray*>& inArrs, NDArray& output) {
        BUILD_SINGLE_SELECTOR(output.dataType(), mergeAvg_, (context, inArrs, output), LIBND4J_TYPES);
    }

    //////////////////////////////////////////////////////////////////////////
    template <typename T>
    static __global__ void global_mergeAdd_(void **inArrs, void **inShapes, const int numArrays, void *voutput, Nd4jLong *outputShape, Nd4jLong length) {
        auto output = reinterpret_cast<T*>(voutput);

        const auto tid = blockIdx.x * gridDim.x + threadIdx.x;
        const auto step = gridDim.x * blockDim.x;

        for (Nd4jLong e = tid; e < length; e += step) {
            T sum(0.0f);

            for (int i = 0; i < numArrays; i++) {
                auto x = reinterpret_cast<T*>(inArrs[i]);
                auto xShape = reinterpret_cast<Nd4jLong *>(inShapes[i]);

                sum += x[shape::getIndexOffset(e, xShape, length)];
            }

            output[shape::getIndexOffset(e, outputShape, length)] = sum;
        }
    }

    template<typename T>
    static void mergeAdd_(graph::LaunchContext* context, const std::vector<NDArray*>& inArrs, NDArray& output) {
        std::vector<void *> inBuffers(inArrs.size());
        std::vector<void *> inShapes(inArrs.size());

        for (int e = 0; e < inArrs.size(); e++) {
            inBuffers[e] = inArrs[e]->getSpecialBuffer();
            inShapes[e] = inArrs[e]->getSpecialShapeInfo();
        }

        PointersManager manager(context, "mergeAdd");

        auto pInBuffers = reinterpret_cast<void **>(manager.replicatePointer(inBuffers.data(), inBuffers.size() * sizeof(void *)));
        auto pInShapes = reinterpret_cast<void **>(manager.replicatePointer(inShapes.data(), inShapes.size() * sizeof(void *)));
        auto length = output.lengthOf();

        global_mergeAdd_<T><<<512, 512, 512, *context->getCudaStream()>>>(pInBuffers, pInShapes, (int) inArrs.size(), output.getSpecialBuffer(), output.getSpecialShapeInfo(), length);

        manager.synchronize();
    }
    BUILD_SINGLE_TEMPLATE(template void mergeAdd_, (graph::LaunchContext* context, const std::vector<NDArray*>& inArrs, NDArray& output), LIBND4J_TYPES);

    void mergeAdd(graph::LaunchContext* context, const std::vector<NDArray*>& inArrs, NDArray& output) {
        BUILD_SINGLE_SELECTOR(output.dataType(), mergeAdd_, (context, inArrs, output), LIBND4J_TYPES);
    }

    //////////////////////////////////////////////////////////////////////////
    template<typename T>
    static void clipByNorm_(graph::LaunchContext* context, NDArray& input, NDArray& output, const std::vector<int>& dimensions, const NDArray& clipNorm, const bool isInplace) {

    }

    void clipByNorm(graph::LaunchContext* context, NDArray& input, NDArray& output, const std::vector<int>& dimensions, const NDArray& clipNorm, const bool isInplace) {
        BUILD_SINGLE_SELECTOR(output.dataType(), clipByNorm_, (context, input, output, dimensions, clipNorm, isInplace), FLOAT_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template void clipByNorm_, (graph::LaunchContext* context, NDArray& input, NDArray& output, const std::vector<int>& dimensions, const NDArray& clipNorm, const bool isInplace), FLOAT_TYPES);

    template <typename T>
    static void clipByGlobalNorm_(graph::LaunchContext* context, std::vector<NDArray*> const& inputs, double clipNorm, nd4j::memory::Workspace* workspace, std::vector<NDArray*>& outputs, bool isInplace) {

    }

    void clipByGlobalNorm(graph::LaunchContext* context, std::vector<NDArray*> const& inputs, double clipNorm, nd4j::memory::Workspace* workspace, std::vector<NDArray*>& outputs, bool isInplace) {
        BUILD_SINGLE_SELECTOR(outputs[0]->dataType(), clipByGlobalNorm_, (context, inputs, clipNorm, workspace, outputs, isInplace), FLOAT_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template void clipByGlobalNorm_, (graph::LaunchContext* context, std::vector<NDArray*> const& inputs, double clipNorm, nd4j::memory::Workspace* workspace, std::vector<NDArray*>& outputs, bool isInplace), FLOAT_TYPES);

    //////////////////////////////////////////////////////////////////////////
    template<typename T>
    static void clipByNormBP_(graph::LaunchContext* context, const NDArray& input, const NDArray& gradO, NDArray& gradI /*output*/, const std::vector<int>& dimensions, const NDArray& clipNorm) {

    }

    void clipByNormBP(graph::LaunchContext* context, const NDArray& input, const NDArray& gradO, NDArray& gradI /*output*/, const std::vector<int>& dimensions, const NDArray& clipNorm) {
        BUILD_SINGLE_SELECTOR(gradI.dataType(), clipByNormBP_, (context, input, gradO, gradI, dimensions, clipNorm), FLOAT_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template void clipByNormBP_, (graph::LaunchContext* context, const NDArray& input, const NDArray& gradO, NDArray& gradI /*output*/, const std::vector<int>& dimensions, const NDArray& clipNorm), FLOAT_TYPES);


    //////////////////////////////////////////////////////////////////////////
    template<typename T>
    static void clipByAveraged_(graph::LaunchContext* context, NDArray& input, NDArray& output, const std::vector<int>& dimensions, const NDArray& clipNorm, const bool isInplace) {
        auto cn = clipNorm.e<T>(0);
        if (dimensions.size() == 0) {
            // all-reduce
            T n2 = input.reduceNumber(reduce::Norm2).e<T>(0) / input.lengthOf();
            if (n2 <= cn) {
                if (!isInplace)
                    output.assign(input);
            }
            else {
                const T factor = cn / n2;
                //auto lambda = LAMBDA_T(_x, factor) { return _x * factor; };
                //input.applyLambda<T>(lambda, &output);
                output.assign(input * factor);
            }
        }
        else {
            // along dimension
            auto norm2 = input.reduceAlongDims(reduce::Norm2, dimensions, false);
            if (!isInplace)
                output.assign(input);
            auto tads = output.allTensorsAlongDimension(dimensions);
            auto outTads = output.allTensorsAlongDimension(dimensions);
            // TODO: make this CUDA-compliant somehow
            for (int e = 0; e < tads->size(); e++) {
                T n2 = norm2.e<T>(e) / tads->at(e)->lengthOf();
                const T factor = cn / n2;
                if (n2 > cn) {
                    //auto lambda = LAMBDA_T(_x, factor) {return _x * factor;};
                    tads->at(e)->applyScalar(scalar::Multiply, factor, outTads->at(e));//applyLambda<T>(lambda, &output);
                }
            }
            delete tads;
            delete outTads;
        }
    }

    void clipByAveraged(graph::LaunchContext* context, NDArray& input, NDArray& output, const std::vector<int>& dimensions, const NDArray& clipNorm, const bool isInplace) {
        BUILD_SINGLE_SELECTOR(input.dataType(), clipByAveraged_, (context, input, output, dimensions, clipNorm, isInplace), FLOAT_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template void clipByAveraged_, (graph::LaunchContext* context, NDArray& input, NDArray& output, const std::vector<int>& dimensions, const NDArray& clipNorm, const bool isInplace), FLOAT_TYPES);

/*
    if (d1 > params[1])
    return params[1];
    else if (d1 < params[0])
    return params[0];
    else return d1;
*/
    template <typename T>
    static void __global__ clipByValueKernel(void* input, Nd4jLong* inputShape, void* output, Nd4jLong* outputShape, double leftBound, double rightBound) {
        __shared__ T* outputBuf;
        __shared__ T* inputBuf;
        __shared__ Nd4jLong length;
        __shared__ bool linearBuffers;
        if (threadIdx.x == 0) {
            outputBuf = reinterpret_cast<T *>(output);
            inputBuf = reinterpret_cast<T *>(input);
            length = shape::length(inputShape);
            linearBuffers = shape::elementWiseStride(inputShape) == shape::elementWiseStride(outputShape) && shape::elementWiseStride(inputShape) == 1;
        }
        __syncthreads();
        const auto tid = blockIdx.x * gridDim.x + threadIdx.x;
        const auto step = gridDim.x * blockDim.x;

        for (Nd4jLong e = tid; e < length; e += step) {
            if (linearBuffers) {
                if (inputBuf[e] > rightBound) outputBuf[e] = (T) rightBound;
                else if (inputBuf[e] < leftBound) outputBuf[e] = (T) leftBound;
                else outputBuf[e] = inputBuf[e];
            }
            else {
                auto inputOffset = shape::getIndexOffset(e, inputShape, length);
                auto outputOffset = shape::getIndexOffset(e, outputShape, length);
                if (inputBuf[inputOffset] > rightBound) outputBuf[outputOffset] = (T) rightBound;
                else if (inputBuf[inputOffset] < leftBound) outputBuf[outputOffset] = (T) leftBound;
                else outputBuf[outputOffset] = inputBuf[outputOffset];
            }
        }
    }

    template <typename T>
    static void clipByValue_(graph::LaunchContext* context, NDArray& input, double leftBound, double rightBound, NDArray& output) {
        auto stream = context->getCudaStream();
        if (!input.isActualOnDeviceSide())
            input.syncToDevice();
        NDArray::prepareSpecialUse({&output}, {&input});
        clipByValueKernel<T><<<256, 512, 8192, *stream>>>(input.specialBuffer(), input.specialShapeInfo(), output.specialBuffer(), output.specialShapeInfo(), leftBound, rightBound);
        NDArray::registerSpecialUse({&output}, {&input});
    }

    void clipByValue(graph::LaunchContext* context, NDArray& input, double leftBound, double rightBound, NDArray& output) {
        BUILD_SINGLE_SELECTOR(input.dataType(), clipByValue_, (context, input, leftBound, rightBound, output), FLOAT_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template void clipByValue_, (graph::LaunchContext* context, NDArray& input, double leftBound, double rightBound, NDArray& output);, FLOAT_TYPES);

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    template<typename T>
    static void mirrorPad_(graph::LaunchContext* context, const NDArray& input, const NDArray& paddings, NDArray& output, const int mode) {

    }

    void mirrorPad(graph::LaunchContext* context, const NDArray& input, const NDArray& paddings, NDArray& output, const int mode) {
        BUILD_SINGLE_SELECTOR(input.dataType(), mirrorPad_, (context, input, paddings, output, mode), LIBND4J_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template void mirrorPad_, (graph::LaunchContext* context, const NDArray& input, const NDArray& paddings, NDArray& output, const int mode), LIBND4J_TYPES);

//////////////////////////////////////////////////////////////////////////
void concat(graph::LaunchContext* context, const std::vector<NDArray*>& inArrs, NDArray& output, const int axis) {

    const int numOfArrs = inArrs.size();
    for(int i = 0; i < numOfArrs; ++i)
        if(!inArrs[i]->isActualOnDeviceSide()) inArrs[i]->syncToDevice();

    const int rank  = inArrs[0]->rankOf();
    const int rank2 = 2*rank;
    std::vector<std::vector<Nd4jLong>> indices(numOfArrs, std::vector<Nd4jLong>(rank2,0));

    // take into account indices for first array
    indices[0][2 * axis + 1] = inArrs[0]->sizeAt(axis);

    // loop through the rest of input arrays
    for(int i = 1; i < numOfArrs; ++i) {
        indices[i][2 * axis]     = indices[i-1][2 * axis + 1];                                // index start from
        indices[i][2 * axis + 1] = indices[i-1][2 * axis + 1] + inArrs[i]->sizeAt(axis);      // index end with (excluding)
    }

    std::vector<NDArray*> outSubArrs(numOfArrs);
    for(int i = 0; i < numOfArrs; ++i) 
        outSubArrs[i] = new NDArray(output(indices[i], true));

    // prepare arrays of pointers on buffers and shapes
    std::vector<void*>     hOutBuffers(numOfArrs), hInBuffers(numOfArrs);
    std::vector<Nd4jLong*> hOutShapeInfo(numOfArrs), hInShapeInfo(numOfArrs);
    for(int i = 0; i < numOfArrs; ++i) {        
        hOutBuffers[i]   = outSubArrs[i]->getSpecialBuffer();
        hInBuffers[i]    =     inArrs[i]->getSpecialBuffer();
        hOutShapeInfo[i] = outSubArrs[i]->getSpecialShapeInfo();
        hInShapeInfo[i]  =     inArrs[i]->getSpecialShapeInfo();
    }

    // allocate and copy all buffers and shapes arrays to global memory    
    PointersManager manager(context, "helpers::concat");
    void* dOutBuffers	= manager.replicatePointer(hOutBuffers.data(),   hOutBuffers.size() * sizeof(void*));
    void* dInBuffers	= manager.replicatePointer(hInBuffers.data(),    hInBuffers.size() * sizeof(void*));
    void* dInShapeInfo  = manager.replicatePointer(hInShapeInfo.data(),  hInShapeInfo.size() * sizeof(Nd4jLong*));
    void* dOutShapeInfo = manager.replicatePointer(hOutShapeInfo.data(), hOutShapeInfo.size() * sizeof(Nd4jLong*));

    BUILD_SINGLE_SELECTOR(inArrs[0]->dataType(), concatCudaLauncher, (numOfArrs, context->getCudaStream(), dInBuffers, dInShapeInfo, dOutBuffers, dOutShapeInfo), LIBND4J_TYPES);

    manager.synchronize();
    
    for(int i = 0; i < numOfArrs; ++i)
        delete outSubArrs[i];

    for(int i = 0; i < numOfArrs; ++i)
        inArrs[i]->tickReadHost();

    output.tickWriteDevice();
}




    //////////////////////////////////////////////////////////////////////////
    template <typename T>
    static void tileBP_(graph::LaunchContext* context, const NDArray& gradO /*input*/, NDArray& gradI /*output*/, const std::vector<Nd4jLong> reps) {

    }

    void tileBP(graph::LaunchContext* context, const NDArray& gradO /*input*/, NDArray& gradI /*output*/, const std::vector<Nd4jLong> reps) {
        BUILD_SINGLE_SELECTOR(gradI.dataType(), tileBP_, (context, gradO, gradI, reps), FLOAT_TYPES);
    }


    BUILD_SINGLE_TEMPLATE(template void tileBP_, (graph::LaunchContext* context, const NDArray& gradO /*input*/, NDArray& gradI /*output*/, const std::vector<Nd4jLong> reps), FLOAT_TYPES);

    void scatterSimple(const int opId, NDArray& input, const NDArray& updates, const NDArray& indices, const std::vector<int>& dimensions) {

    }


BUILD_SINGLE_TEMPLATE(template void concatCudaLauncher, (const int numOfArrs, const hipStream_t *stream, void* pVx, void* pxShapeInfo, void* pVz, void* pzShapeInfo), LIBND4J_TYPES);


}
}
}
