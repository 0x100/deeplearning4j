#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author sgazeos@gmail.com
//
#include <op_boilerplate.h>
#include <NDArray.h>

namespace nd4j {
namespace ops {
namespace helpers {

    template <typename T>
    static __global__ void drawBoundingBoxesKernel(T const* images, Nd4jLong* imagesShape, T const* boxes,
            Nd4jLong* boxesShape, T const* colors, Nd4jLong* colorsShape, T* output, Nd4jLong* outputShape,
            Nd4jLong batchSize, Nd4jLong width, Nd4jLong height, Nd4jLong channels, Nd4jLong colorSetSize) {
        for (auto b = blockIdx.x; b < batchSize; b += gridDim.x) { // loop by batch
            for (auto c = 0; c < colorSetSize; c += blockDim.x) {
                // box with shape
                auto pos = 0;
                auto internalBox = &boxes[pos];//(*boxes)(b, {0})(c, {0});//internalBoxes->at(c);
                auto color = &colors[pos];//colorSet->at(c);
                auto rowStart = nd4j::math::nd4j_max(Nd4jLong (0), Nd4jLong ((height - 1) * internalBox[0]));
                auto rowEnd = nd4j::math::nd4j_min(Nd4jLong (height - 1), Nd4jLong ((height - 1) * internalBox[2]));
                auto colStart = nd4j::math::nd4j_max(Nd4jLong (0), Nd4jLong ((width - 1) * internalBox[1]));
                auto colEnd = nd4j::math::nd4j_min(Nd4jLong(width - 1), Nd4jLong ((width - 1) * internalBox[3]));
                for (auto y = rowStart; y <= rowEnd; y++) {
                    for (auto e = 0; e < channels; ++e) {
                        Nd4jLong yMinPos[] = {b, y, colStart, e};
                        Nd4jLong yMaxPos[] = {b, y, colEnd, e};
                        auto zIndexYmin = shape::getOffset(outputShape, yMinPos, 4);
                        auto zIndexYmax = shape::getOffset(outputShape, yMaxPos, 4);
                        output[zIndexYmin] = color[e];
                        output[zIndexYmax] = color[e];
                    }
                }
                for (auto x = colStart + 1; x < colEnd; x++) {
                    for (auto e = 0; e < channels; ++e) {
                        Nd4jLong xMinPos[] = {b, rowStart, x, e};
                        Nd4jLong xMaxPos[] = {b, rowEnd, x, e};
                        auto zIndexXmin = shape::getOffset(outputShape, xMinPos, 4);
                        auto zIndexXmax = shape::getOffset(outputShape, xMaxPos, 4);
                        output[zIndexXmin] = color[e];
                        output[zIndexXmax] = color[e];
                    }
                }
            }
        }

    }
    template <typename T>
    void drawBoundingBoxesH(nd4j::LaunchContext* context, NDArray const* images, NDArray const* boxes, NDArray const* colors, NDArray* output) {
        auto batchSize = images->sizeAt(0);
        auto height = images->sizeAt(1);
        auto width = images->sizeAt(2);
        auto channels = images->sizeAt(3);
        auto stream = context->getCudaStream();
        auto colorSetSize = colors->sizeAt(0);
//        auto imageList = images->allTensorsAlongDimension({1, 2, 3}); // split images by batch
//        auto boxList = boxes->allTensorsAlongDimension({1, 2}); // split boxes by batch
//        auto colorSet = colors->allTensorsAlongDimension({1});
        auto imagesBuf = images->getDataBuffer()->specialAsT<T>();
        auto boxesBuf = boxes->getDataBuffer()->specialAsT<T>();
        auto colorsBuf = colors->getDataBuffer()->specialAsT<T>();
        auto outputBuf = output->dataBuffer()->specialAsT<T>();
        drawBoundingBoxesKernel<<<128, 256, 1024, *stream>>>(imagesBuf, images->getSpecialShapeInfo(),
                boxesBuf, boxes->getSpecialShapeInfo(), colorsBuf, colors->getSpecialShapeInfo(),
                outputBuf, output->specialShapeInfo(), batchSize, width, height, channels, colorSetSize);
    }

    void drawBoundingBoxesFunctor(nd4j::LaunchContext * context, NDArray* images, NDArray* boxes, NDArray* colors, NDArray* output) {
        // images - batch of 3D images with BW (last dim = 1), RGB (last dim = 3) or RGBA (last dim = 4) channel set
        // boxes - batch of 2D bounds with last dim (y_start, x_start, y_end, x_end) to compute i and j as
        // floor((height - 1 ) * y_start) => rowStart, floor((height - 1) * y_end) => rowEnd
        // floor((width - 1 ) * x_start) => colStart, floor((width - 1) * x_end) => colEnd
        // height = images->sizeAt(1), width = images->sizeAt(2)
        // colors - colors for each box given
        // set up color for each box as frame

        output->assign(images);
        BUILD_SINGLE_SELECTOR(output->dataType(), drawBoundingBoxesH, (context, images, boxes, colors, output), FLOAT_TYPES);
    }
    BUILD_SINGLE_TEMPLATE(template void drawBoundingBoxesH, (nd4j::LaunchContext* context, NDArray const* images, NDArray const* boxes, NDArray const* colors, NDArray* output), FLOAT_TYPES);
}
}
}
