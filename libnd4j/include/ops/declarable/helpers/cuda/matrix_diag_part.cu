#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// Created by GS <sgazeos@gmail.com> on 3/21/2018.
//

#include "ResultSet.h"
#include <ops/declarable/helpers/matrix_diag_part.h>
#include <Status.h>
#include <ShapeUtils.h>
#include <ShapeUtils.h>
#include <TAD.h>
#include <cuda_exception.h>

namespace nd4j {
namespace ops {
namespace helpers {

    template <typename T>
    static __global__ void matrixDiagPartKernel(void const* inputBuffer, void* outputBuffer, Nd4jLong numTads, Nd4jLong inputLength,
                                            Nd4jLong* tadOnlyInputShapeInfo,  Nd4jLong *tadInputOffsets,
                                            Nd4jLong* tadOnlyOutputShapeInfo, Nd4jLong *tadOutputOffsets) {
        int totalThreads = blockDim.x;
        for (Nd4jLong i = blockIdx.x; i < numTads; i += gridDim.x) {
            auto yOffset = tadInputOffsets[i];
            auto xOffset = tadOutputOffsets[i];
            for (Nd4jLong j = threadIdx.x; j < inputLength; j += totalThreads) {
                Nd4jLong coords[2] = {j, j};
                Nd4jLong tadOffset = shape::getOffset(0, shape::shapeOf(tadOnlyInputShapeInfo), shape::stride(tadOnlyInputShapeInfo), coords, 2);
                //shape::getIndexOffset(j, tadOnlyOutputShapeInfo, inputLength)
                *(reinterpret_cast<T*>(outputBuffer) + xOffset + shape::getIndexOffset(j, tadOnlyOutputShapeInfo, inputLength)) = *(reinterpret_cast<T const*>(inputBuffer) + yOffset + tadOffset);
            }
        }
    }

//////////////////////////////////////////////////////////////////////////
// Returns a batched matrix tensor with new batched diagonal values.
// for detailed explanations please take a look on web page: https://www.tensorflow.org/api_docs/python/tf/matrix_set_diag
    template <typename T>
    int _matrixDiagPart(graph::LaunchContext* context, const NDArray* input, NDArray* output) {
        auto stream = context->getCudaStream();
        auto listOut  = output->allTensorsAlongDimension({output->rankOf() - 1});
        auto listDiag = input->allTensorsAlongDimension({input->rankOf() - 2, input->rankOf() - 1});

        if (listOut->size() != listDiag->size()) {
            nd4j_printf("matrix_diag_part: Input matrix has wrong shape.", "");
            return ND4J_STATUS_VALIDATION;
        }
        Nd4jLong lastDimension = nd4j::math::nd4j_min(input->sizeAt(-2), input->sizeAt(-1));

        std::vector<int> dimsToExclude = ShapeUtils::evalDimsToExclude(output->rankOf(), {output->rankOf() - 1});
        const Nd4jLong numTads = ShapeUtils::getNumOfSubArrs(input->getShapeInfo(), dimsToExclude); //this->tensorsAlongDimension({dimension});
        //printf("Repeat delta %lld, numTads %lld\n", repeatDelta, numTads);
        //tadOnlyInputShapeInfo, tadInputOffsets, tadOnlyOutputShapeInfo, tadOutputOffsets;
        std::vector<int> outputDims({output->rankOf() - 1});
        shape::TAD tadOutput(output->getShapeInfo(), outputDims.data(), outputDims.size());
        tadOutput.createTadOnlyShapeInfo();
        tadOutput.createOffsets();
        if (!input->isActualOnDeviceSide())
            input->syncToDevice();

        std::vector<int> inputDims({input->rankOf() - 2, input->rankOf() - 1});
        shape::TAD tadInput(input->getShapeInfo(), inputDims.data(), inputDims.size());
        tadInput.createTadOnlyShapeInfo();
        tadInput.createOffsets();
        if (!input->isActualOnDeviceSide())
            input->syncToDevice();

        // prepare input arrays for prepareDataForCuda function
        std::vector<std::pair<void*,size_t>> hostData;
        hostData.emplace_back(tadInput.tadOnlyShapeInfo, shape::shapeInfoByteLength(tadInput.tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
        hostData.emplace_back(tadInput.tadOffsets, tadInput.numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets
        hostData.emplace_back(tadOutput.tadOnlyShapeInfo, shape::shapeInfoByteLength(tadOutput.tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
        hostData.emplace_back(tadOutput.tadOffsets, tadOutput.numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets
        std::vector<void*> devicePtrs(hostData.size(), nullptr);

        // create cuda stream and LaunchContext
        hipError_t cudaResult;
        //hipStream_t stream;
        //cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
        //hipStream_t* stream = this->getContext()->getCudaStream();
        // allocate required amount of global device memory and copy host data to it
//    cudaResult = allocateDeviceMem(*pLc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
        for(int i = 0; i < devicePtrs.size(); ++i) {
            cudaResult = hipMalloc(reinterpret_cast<void **>(&devicePtrs[i]), hostData[i].second);
            if(cudaResult != 0) throw cuda_exception::build("Cannot allocate memory for tads on device", cudaResult);
            cudaResult = hipMemcpy(devicePtrs[i], hostData[i].first, hostData[i].second, hipMemcpyHostToDevice);
            if(cudaResult != 0) throw cuda_exception::build("Cannot copy memory block for tads on device", cudaResult);
        }

        dim3 launchDims(256, 512, 8192);
        matrixDiagPartKernel<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(input->getSpecialBuffer(), output->getSpecialBuffer(), numTads, lastDimension, (Nd4jLong*)devicePtrs[0], (Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], (Nd4jLong*)devicePtrs[3]);
        for(int i = 0; i < devicePtrs.size(); ++i) {
            cudaResult = hipFree(devicePtrs[i]);
            if(cudaResult != 0) throw cuda_exception::build("Cannot allocate memory for tads on device", cudaResult);
        }
        return Status::OK();
    }

    int matrixDiagPart(graph::LaunchContext* context, const NDArray* input, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), return _matrixDiagPart, (context, input, output), LIBND4J_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template int _matrixDiagPart, (graph::LaunchContext* context, const NDArray* input, NDArray* output), LIBND4J_TYPES);

}
}
}