#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

 //
 // @author raver119@gmail.com
 //

#include "testlayers.h"
#include <NDArray.h>
#include <NDArrayFactory.h>
#include <Context.h>
#include <Node.h>
#include <graph/Variable.h>
#include <graph/VariableSpace.h>
#include <specials_cuda.h>
#include <TAD.h>
#include <MmulHelper.h>

#include <hip/hip_runtime.h>
#include <cuda_launch_config.h>

using namespace nd4j;
using namespace nd4j::graph;

class CudaBasicsTests : public testing::Test {
public:

};


//////////////////////////////////////////////////////////////////////////
static hipError_t allocateDeviceMem(LaunchContext& lc, std::vector<void*>& devicePtrs, const std::vector<std::pair<void*,size_t>>& hostData) { 

	if(devicePtrs.size() != hostData.size())
		throw std::invalid_argument("prepareDataForCuda: two input sts::vectors should same sizes !");

	hipError_t cudaResult;

	void* reductionPointer;
    cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer),  1024*1024);			if(cudaResult != 0) return cudaResult;
    int* allocationPointer;
	cudaResult = hipMalloc(reinterpret_cast<void **>(&allocationPointer), 1024*1024);			if(cudaResult != 0) return cudaResult;

	lc.setReductionPointer(reductionPointer);
	lc.setAllocationPointer(allocationPointer);
	hipStream_t stream = *lc.getCudaStream();

	for(int i = 0; i < devicePtrs.size(); ++i) {
		
		cudaResult = hipMalloc(reinterpret_cast<void **>(&devicePtrs[i]), hostData[i].second); if(cudaResult != 0) return cudaResult;
		hipMemcpyAsync(devicePtrs[i], hostData[i].first, hostData[i].second, hipMemcpyHostToDevice, stream);				
	}
	return cudaResult;
}

//////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, TestPairwise_1) {
	// allocating host-side arrays
	auto x = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
	auto z = NDArrayFactory::create<double>('c', { 5 }, {0,0,0,0,0});

	auto exp = NDArrayFactory::create<double>('c', { 5 }, { 2, 4, 6, 8, 10 });

	// making raw buffers
	Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
	hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
	ASSERT_EQ(0, res);
	res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
	ASSERT_EQ(0, res);
	res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
	ASSERT_EQ(0, res);

	Nd4jPointer nativeStream = (Nd4jPointer)malloc(sizeof(hipStream_t));
	CHECK_ALLOC(nativeStream, "Failed to allocate memory for new CUDA stream", sizeof(hipStream_t));
	hipError_t dZ = hipStreamCreate(reinterpret_cast<hipStream_t *>(&nativeStream));
	auto stream = reinterpret_cast<hipStream_t *>(&nativeStream);

	hipMemcpyAsync(devBufferPtrX, x.buffer(), x.lengthOf() * x.sizeOfT(), hipMemcpyHostToDevice, *stream);
	hipMemcpyAsync(devShapePtrX, x.shapeInfo(), shape::shapeInfoByteLength(x.shapeInfo()), hipMemcpyHostToDevice, *stream);
	
	LaunchContext lc(stream, nullptr, nullptr);
	NativeOpExecutioner::execPairwiseTransform(&lc, pairwise::Add, nullptr, x.shapeInfo(), devBufferPtrX, reinterpret_cast<Nd4jLong*>(devShapePtrX), nullptr, x.shapeInfo(), devBufferPtrX, reinterpret_cast<Nd4jLong*>(devShapePtrX), nullptr, z.shapeInfo(), devBufferPtrZ, reinterpret_cast<Nd4jLong*>(devShapePtrX), nullptr);
	res = hipStreamSynchronize(*stream);
	ASSERT_EQ(0, res);

	hipMemcpyAsync(z.buffer(), devBufferPtrZ, z.lengthOf() * x.sizeOfT(), hipMemcpyDeviceToHost, *stream);
	res = hipStreamSynchronize(*stream);
	ASSERT_EQ(0, res);

	hipFree(devBufferPtrX);
	hipFree(devBufferPtrZ);
	hipFree(devShapePtrX);

	for (int e = 0; e < z.lengthOf(); e++) {
		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
	}
}


////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execIndexReduceScalar_1) {

    NDArray x1('c', {2,2}, {0, 1, 2, 3}, nd4j::DataType::INT32);
    NDArray x2('c', {2,2}, {0.5, 1.5, -4.5, 3.5}, nd4j::DataType::BFLOAT16);    
    NDArray x3('c', {2,2}, {0, -1, 0, 1}, nd4j::DataType::BOOL);
    
    NDArray scalar('c', {0}, {0}, nd4j::DataType::INT64);

    NDArray exp1('c', {0}, {3}, nd4j::DataType::INT64);
    NDArray exp2('c', {0}, {2}, nd4j::DataType::INT64);
    NDArray exp3('c', {0}, {1}, nd4j::DataType::INT64);

    void *dX1, *dX2, *dX3, *dZ; 
    Nd4jLong *dX1ShapeInfo, *dX2ShapeInfo, *dX3ShapeInfo, *dZShapeInfo;

    hipError_t cudaResult;

    cudaResult = hipMalloc(reinterpret_cast<void **>(&dX1), x1.lengthOf() * x1.sizeOfT()); 		   		         	 ASSERT_EQ(0, cudaResult);
    cudaResult = hipMalloc(reinterpret_cast<void **>(&dX2), x2.lengthOf() * x2.sizeOfT()); 		   		         	 ASSERT_EQ(0, cudaResult);    
    cudaResult = hipMalloc(reinterpret_cast<void **>(&dX3), x3.lengthOf() * x3.sizeOfT()); 		   		         	 ASSERT_EQ(0, cudaResult);    
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dZ), scalar.lengthOf() * scalar.sizeOfT()); 				         ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dX1ShapeInfo), shape::shapeInfoByteLength(x1.getShapeInfo()));    ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dX2ShapeInfo), shape::shapeInfoByteLength(x2.getShapeInfo()));    ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dX3ShapeInfo), shape::shapeInfoByteLength(x3.getShapeInfo()));    ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dZShapeInfo), shape::shapeInfoByteLength(scalar.getShapeInfo())); ASSERT_EQ(0, cudaResult);	

    hipStream_t stream;
	cudaResult = hipStreamCreate(&stream); 
	ASSERT_EQ(0, cudaResult);
	
	hipMemcpyAsync(dX1, x1.buffer(), x1.lengthOf() * x1.sizeOfT(), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX2, x2.buffer(), x2.lengthOf() * x2.sizeOfT(), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX3, x3.buffer(), x3.lengthOf() * x3.sizeOfT(), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX1ShapeInfo, x1.getShapeInfo(), shape::shapeInfoByteLength(x1.getShapeInfo()), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX2ShapeInfo, x2.getShapeInfo(), shape::shapeInfoByteLength(x2.getShapeInfo()), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX3ShapeInfo, x3.getShapeInfo(), shape::shapeInfoByteLength(x3.getShapeInfo()), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dZShapeInfo, scalar.getShapeInfo(), shape::shapeInfoByteLength(scalar.getShapeInfo()), hipMemcpyHostToDevice, stream);
	
	void* reductionPointer = nullptr;
	cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer), 1024*1024);
	ASSERT_EQ(0, cudaResult);

	LaunchContext lc(&stream, reductionPointer);

	/***************************************/
	
    NativeOpExecutioner::execIndexReduceScalar(&lc, 
    											nd4j::indexreduce::IndexAbsoluteMax, 
    											x1.buffer(), x1.getShapeInfo(),
    	                                       	dX1, dX1ShapeInfo, 
    	                                       	nullptr, 
    	                                       	scalar.buffer(), scalar.getShapeInfo(),
    	                                       	dZ, dZShapeInfo);

    cudaResult = hipStreamSynchronize(stream); 
    ASSERT_EQ(0, cudaResult);

    hipMemcpyAsync(scalar.buffer(), dZ, scalar.lengthOf() * scalar.sizeOfT(), hipMemcpyDeviceToHost, stream);

    cudaResult = hipStreamSynchronize(stream); 
    ASSERT_EQ(0, cudaResult);

	ASSERT_NEAR(exp1.e<float>(0), scalar.e<float>(0), 1e-5);

    /***************************************/
    
    NativeOpExecutioner::execIndexReduceScalar(&lc,
    											nd4j::indexreduce::IndexAbsoluteMax, 
    											nullptr, x2.getShapeInfo(),
    	                                       	dX2, dX2ShapeInfo, 
    	                                       	nullptr, 
    	                                       	nullptr, scalar.getShapeInfo(),
    	                                       	dZ, dZShapeInfo);

    cudaResult = hipStreamSynchronize(stream); 
    ASSERT_EQ(0, cudaResult);

    hipMemcpyAsync(scalar.buffer(), dZ, scalar.lengthOf() * scalar.sizeOfT(), hipMemcpyDeviceToHost, stream);

    cudaResult = hipStreamSynchronize(stream); 
    ASSERT_EQ(0, cudaResult);

    ASSERT_NEAR(exp2.e<float>(0), scalar.e<float>(0), 1e-5);

    // *************************************

    NativeOpExecutioner::execIndexReduceScalar(&lc, 
    											nd4j::indexreduce::IndexAbsoluteMax, 
    											nullptr, x3.getShapeInfo(),
    	                                       	dX3, dX3ShapeInfo, 
    	                                       	nullptr, 
    	                                       	nullptr, scalar.getShapeInfo(),
    	                                       	dZ, dZShapeInfo);

    cudaResult = hipStreamSynchronize(stream); 
    ASSERT_EQ(0, cudaResult);

    hipMemcpyAsync(scalar.buffer(), dZ, scalar.lengthOf() * scalar.sizeOfT(), hipMemcpyDeviceToHost, stream);

    cudaResult = hipStreamSynchronize(stream); 
    ASSERT_EQ(0, cudaResult);

    ASSERT_NEAR(exp3.e<float>(0), scalar.e<float>(0), 1e-5);
    
	/***************************************/

	hipFree(dX1); 			hipFree(dX2); 			hipFree(dX3); 			hipFree(dZ);
	hipFree(dX1ShapeInfo); hipFree(dX2ShapeInfo); hipFree(dX3ShapeInfo); hipFree(dZShapeInfo); 

	/***************************************/	

	cudaResult = hipStreamDestroy(stream); 
	ASSERT_EQ(0, cudaResult);
	
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execReduce3Scalar_1) {

	 if (!Environment::getInstance()->isExperimentalBuild())
        return;

    NDArray x1('c', {2,2}, {1,2,3,4}, nd4j::DataType::INT32);
    NDArray x2('c', {2,2}, {-1,-2,-3,-4}, nd4j::DataType::INT32);
    NDArray x3('c', {2,2}, {1.5,1.5,1.5,1.5}, nd4j::DataType::DOUBLE);
    NDArray x4('c', {2,2}, {1,2,3,4}, nd4j::DataType::DOUBLE);
    NDArray exp1('c', {0}, {-30}, nd4j::DataType::FLOAT32);
    NDArray exp2('c', {0}, {15}, nd4j::DataType::DOUBLE);
    
	NDArray scalar1('c', {0}, {100}, nd4j::DataType::FLOAT32);
    NDArray scalar2('c', {0}, {100}, nd4j::DataType::DOUBLE);    

    void *dX1, *dX2, *dX3, *dX4, *dZ1, *dZ2; 
    Nd4jLong *dX1ShapeInfo, *dX3ShapeInfo, *dZ1ShapeInfo, *dZ2ShapeInfo;

    hipError_t cudaResult;

    cudaResult = hipMalloc(reinterpret_cast<void **>(&dX1), x1.lengthOf() * x1.sizeOfT()); 		   		         	 	ASSERT_EQ(0, cudaResult);
    cudaResult = hipMalloc(reinterpret_cast<void **>(&dX2), x2.lengthOf() * x2.sizeOfT()); 		   		         	 	ASSERT_EQ(0, cudaResult);
    cudaResult = hipMalloc(reinterpret_cast<void **>(&dX3), x3.lengthOf() * x3.sizeOfT()); 		   		         	 	ASSERT_EQ(0, cudaResult);
    cudaResult = hipMalloc(reinterpret_cast<void **>(&dX4), x4.lengthOf() * x4.sizeOfT()); 		   		         	 	ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dZ1), scalar1.lengthOf() * scalar1.sizeOfT());			         	ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dZ2), scalar2.lengthOf() * scalar2.sizeOfT());			         	ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dX1ShapeInfo), shape::shapeInfoByteLength(x1.getShapeInfo()));    	ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dX3ShapeInfo), shape::shapeInfoByteLength(x3.getShapeInfo()));    	ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dZ1ShapeInfo), shape::shapeInfoByteLength(scalar1.getShapeInfo())); 	ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dZ2ShapeInfo), shape::shapeInfoByteLength(scalar2.getShapeInfo())); 	ASSERT_EQ(0, cudaResult);

    hipStream_t stream;
	cudaResult = hipStreamCreate(&stream); 
	ASSERT_EQ(0, cudaResult);
	
	hipMemcpyAsync(dX1, x1.buffer(), x1.lengthOf() * x1.sizeOfT(), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX2, x2.buffer(), x2.lengthOf() * x2.sizeOfT(), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX3, x3.buffer(), x3.lengthOf() * x3.sizeOfT(), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX4, x4.buffer(), x4.lengthOf() * x4.sizeOfT(), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX1ShapeInfo, x1.getShapeInfo(), shape::shapeInfoByteLength(x1.getShapeInfo()), hipMemcpyHostToDevice, stream);	
	hipMemcpyAsync(dX3ShapeInfo, x3.getShapeInfo(), shape::shapeInfoByteLength(x3.getShapeInfo()), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dZ1ShapeInfo, scalar1.getShapeInfo(), shape::shapeInfoByteLength(scalar1.getShapeInfo()), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dZ2ShapeInfo, scalar2.getShapeInfo(), shape::shapeInfoByteLength(scalar2.getShapeInfo()), hipMemcpyHostToDevice, stream);

	/***************************************/

	void* reductionPointer  = nullptr;
	int*  allocationPointer = nullptr;	

	cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer),  1024*1024);		ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&allocationPointer), 1024*1024);		ASSERT_EQ(0, cudaResult);

	LaunchContext lc(&stream, reductionPointer, nullptr, allocationPointer);

	/***************************************/
	
    NativeOpExecutioner::execReduce3Scalar(&lc, nd4j::reduce3::Dot,nullptr, x1.getShapeInfo(),dX1, dX1ShapeInfo, nullptr, nullptr, x2.getShapeInfo(),dX2, dX1ShapeInfo,nullptr, scalar1.getShapeInfo(),dZ1, dZ1ShapeInfo);

    cudaResult = hipStreamSynchronize(stream);     
    ASSERT_EQ(0, cudaResult);

    hipMemcpyAsync(scalar1.buffer(), dZ1, scalar1.lengthOf() * scalar1.sizeOfT(), hipMemcpyDeviceToHost, stream);

    cudaResult = hipStreamSynchronize(stream); 
    ASSERT_EQ(0, cudaResult);

	ASSERT_NEAR(exp1.e<float>(0), scalar1.e<float>(0), 1e-5);

    /***************************************/
    
    NativeOpExecutioner::execReduce3Scalar(&lc, nd4j::reduce3::Dot,nullptr, x3.getShapeInfo(),dX3, dX3ShapeInfo, nullptr, nullptr, x4.getShapeInfo(),dX4, dX3ShapeInfo,nullptr, scalar2.getShapeInfo(),dZ2, dZ2ShapeInfo);

    cudaResult = hipStreamSynchronize(stream); 
    ASSERT_EQ(0, cudaResult);

    hipMemcpyAsync(scalar2.buffer(), dZ2, scalar2.lengthOf() * scalar2.sizeOfT(), hipMemcpyDeviceToHost, stream);

    cudaResult = hipStreamSynchronize(stream); 
    ASSERT_EQ(0, cudaResult);

	ASSERT_NEAR(exp2.e<float>(0), scalar2.e<float>(0), 1e-5);
    
	/***************************************/

	hipFree(dX1); 			hipFree(dX2); hipFree(dX3); 		   hipFree(dX4); 	hipFree(dZ1); 				hipFree(dZ2);
	hipFree(dX1ShapeInfo); 			   hipFree(dX3ShapeInfo); 					hipFree(dZ1ShapeInfo);		hipFree(dZ2ShapeInfo);

	/***************************************/	

	cudaResult = hipStreamDestroy(stream); 
	ASSERT_EQ(0, cudaResult);
}
 

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execReduce3_1) {

    NDArray x('c', {2,2}, {1,2,3,4}, nd4j::DataType::INT32);
    NDArray y('c', {2,2}, {-1,-2,-3,-4}, nd4j::DataType::INT32);

    NDArray exp('c', {0}, {-30}, nd4j::DataType::FLOAT32);
    NDArray z('c', {0}, {100},  nd4j::DataType::FLOAT32);

    std::vector<int> dimensions = {0, 1};
    
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions	
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

    hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduce3(&lc, nd4j::reduce3::Dot, 
								nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
								nullptr, 
								nullptr, y.getShapeInfo(), y.specialBuffer(), y.specialShapeInfo(), 
								nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
								(int*)devicePtrs[0], dimensions.size(), 
								nullptr, nullptr, nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}


////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execReduce3_2) {
    
	NDArray x('c', {2,2}, {1.5,1.5,1.5,1.5}, nd4j::DataType::DOUBLE);
    NDArray y('c', {2,2}, {1,2,3,4}, nd4j::DataType::DOUBLE);

    NDArray exp('c', {0}, {15}, nd4j::DataType::DOUBLE);
    NDArray z('c', {0}, {100},  nd4j::DataType::DOUBLE);
   
    std::vector<int> dimensions = {0, 1};   

    // prepare input arrays for prepareDataForCuda function       
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions	
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result	
	NativeOpExecutioner::execReduce3(&lc, nd4j::reduce3::Dot, 
								nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
								nullptr, 
								nullptr, y.getShapeInfo(), y.specialBuffer(), y.specialShapeInfo(), 
								nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
								(int*)devicePtrs[0], dimensions.size(), 
								nullptr, nullptr, nullptr, nullptr);


	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execReduce3_3) {
    
	NDArray x('c', {2,3}, {1,2,3,4,5,6}, nd4j::DataType::INT32);
    NDArray y('c', {2,3}, {-6,-5,-4,-3,-2,-1}, nd4j::DataType::INT32);        

    NDArray exp('c', {3}, {-18,-20,-18}, nd4j::DataType::FLOAT32);
    NDArray z('c', {3}, {100,100,100}, nd4j::DataType::FLOAT32);
   
    std::vector<int> dimensions = {0};

    // evaluate xTad data 
    shape::TAD xTad(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // evaluate yTad data
    shape::TAD yTad(y.getShapeInfo(), dimensions.data(), dimensions.size());    	    
    yTad.createTadOnlyShapeInfo();
    yTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function    
    std::vector<std::pair<void*,size_t>> hostData;    
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	hostData.emplace_back(yTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(yTad.tadOnlyShapeInfo));// 3 -- yTadShapeInfo
	hostData.emplace_back(yTad.tadOffsets, yTad.numTads * sizeof(Nd4jLong));						// 4-- yTadOffsets	
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		

	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result	
	NativeOpExecutioner::execReduce3(&lc, nd4j::reduce3::Dot, 
								nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
								nullptr, 
								nullptr, y.getShapeInfo(), y.specialBuffer(), y.specialShapeInfo(), 
								nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
								(int*)devicePtrs[0], dimensions.size(), 
								(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], 
								(Nd4jLong*)devicePtrs[3], (Nd4jLong*)devicePtrs[4]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
	z.syncToHost();
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execReduce3_4) {
    	
    NDArray x('c', {2,3}, {1,2,3,4,5,6}, nd4j::DataType::DOUBLE);
    NDArray y('c', {2,3}, {1.5,1.5,1.5,1.5,1.5,1.5}, nd4j::DataType::DOUBLE);

    NDArray exp('c', {2}, {9,22.5}, nd4j::DataType::DOUBLE);
    NDArray z('c', {2}, {100,100}, nd4j::DataType::DOUBLE);
   
    std::vector<int> dimensions = {1};

    // evaluate xTad data 
    shape::TAD xTad(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // evaluate yTad data
    shape::TAD yTad(y.getShapeInfo(), dimensions.data(), dimensions.size());    	    
    yTad.createTadOnlyShapeInfo();
    yTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	hostData.emplace_back(yTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(yTad.tadOnlyShapeInfo));// 3 -- yTadShapeInfo
	hostData.emplace_back(yTad.tadOffsets, yTad.numTads * sizeof(Nd4jLong));						// 4-- yTadOffsets	
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		

	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result	
	NativeOpExecutioner::execReduce3(&lc, nd4j::reduce3::Dot,
								nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
								nullptr, 
								nullptr, y.getShapeInfo(), y.specialBuffer(), y.specialShapeInfo(), 
								nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
								(int*)devicePtrs[0], dimensions.size(), 
								(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], 
								(Nd4jLong*)devicePtrs[3], (Nd4jLong*)devicePtrs[4]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execReduce3_5) {
    	
    NDArray x('c', {2,2,3}, {1.5,1.5,1.5,1.5,1.5,1.5,1.5,1.5,1.5,1.5,1.5,1.5}, nd4j::DataType::FLOAT32);
    NDArray y('c', {2,2,3}, {1,2,3,4,5,6,7,8,9,10,11,12}, nd4j::DataType::FLOAT32);

    NDArray exp('c', {2,3}, {7.5, 10.5, 13.5, 25.5, 28.5, 31.5}, nd4j::DataType::FLOAT32);
    NDArray z('c', {2,3}, {100,100,100,100,100,100}, nd4j::DataType::FLOAT32);
   
    std::vector<int> dimensions = {1};

    // evaluate xTad data 
    shape::TAD xTad(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // evaluate yTad data
    shape::TAD yTad(y.getShapeInfo(), dimensions.data(), dimensions.size());    	    
    yTad.createTadOnlyShapeInfo();
    yTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	hostData.emplace_back(yTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(yTad.tadOnlyShapeInfo));// 3 -- yTadShapeInfo
	hostData.emplace_back(yTad.tadOffsets, yTad.numTads * sizeof(Nd4jLong));						// 4-- yTadOffsets	
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		

	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduce3(&lc, nd4j::reduce3::Dot,
								nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
								nullptr, 
								nullptr, y.getShapeInfo(), y.specialBuffer(), y.specialShapeInfo(), 
								nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
								(int*)devicePtrs[0], dimensions.size(), 
								(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], 
								(Nd4jLong*)devicePtrs[3], (Nd4jLong*)devicePtrs[4]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execReduce3All_1) {
    	
    NDArray x('c', {2,2}, {1,2,3,4}, nd4j::DataType::INT32);
    NDArray y('c', {2,3}, {-1,1,-1,1,-1,1}, nd4j::DataType::INT32);

    NDArray exp('c', {2,3}, {2,-2,2,2,-2,2}, nd4j::DataType::FLOAT32);
    NDArray z('c', {2,3}, {100,100,100,100,100,100}, nd4j::DataType::FLOAT32);
   
    std::vector<int> dimensions = {0};

    // evaluate xTad data 
    shape::TAD xTad(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // evaluate yTad data
    shape::TAD yTad(y.getShapeInfo(), dimensions.data(), dimensions.size());    	    
    yTad.createTadOnlyShapeInfo();
    yTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function    
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	hostData.emplace_back(yTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(yTad.tadOnlyShapeInfo));// 3 -- yTadShapeInfo
	hostData.emplace_back(yTad.tadOffsets, yTad.numTads * sizeof(Nd4jLong));						// 4 -- yTadOffsets	
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		

	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduce3All(&lc, nd4j::reduce3::Dot, 
										nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
										nullptr, 
										nullptr, y.getShapeInfo(), y.specialBuffer(), y.specialShapeInfo(), 
										nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
										(int*)devicePtrs[0], dimensions.size(), 
										(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], 
										(Nd4jLong*)devicePtrs[3], (Nd4jLong*)devicePtrs[4]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
	z.syncToHost();    
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execReduce3All_2) {
    	
    NDArray x('c', {2,2}, {1,2,3,4}, nd4j::DataType::DOUBLE);
    NDArray y('c', {2,3}, {1.5,1.5,1.5,1.5,1.5,1.5}, nd4j::DataType::DOUBLE);    

    NDArray exp('c', {2,3}, {6,6,6,9,9,9}, nd4j::DataType::DOUBLE);    
    NDArray z('c', {2,3}, {100,100,100,100,100,100,},nd4j::DataType::DOUBLE);    
   
    std::vector<int> dimensions = {0};

    // evaluate xTad data 
    shape::TAD xTad(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // evaluate yTad data
    shape::TAD yTad(y.getShapeInfo(), dimensions.data(), dimensions.size());    	    
    yTad.createTadOnlyShapeInfo();
    yTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function    
    std::vector<std::pair<void*,size_t>> hostData;    
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	hostData.emplace_back(yTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(yTad.tadOnlyShapeInfo));// 3 -- yTadShapeInfo
	hostData.emplace_back(yTad.tadOffsets, yTad.numTads * sizeof(Nd4jLong));						// 4-- yTadOffsets	
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		

	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduce3All(&lc, nd4j::reduce3::Dot, 
										nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
										nullptr, 
										nullptr, y.getShapeInfo(), y.specialBuffer(), y.specialShapeInfo(), 
										nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
										(int*)devicePtrs[0], dimensions.size(), 
										(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], 
										(Nd4jLong*)devicePtrs[3], (Nd4jLong*)devicePtrs[4]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execIndexReduce_1) {
    	
    NDArray x('c', {2,3}, {100,100,100,100,100,100}, nd4j::DataType::DOUBLE);
    x.linspace(-2.); x.syncToDevice();
    NDArray exp('c', {2}, {2, 2}, nd4j::DataType::INT64);
    NDArray z('c', {2}, {100,100}, nd4j::DataType::INT64);
    
    std::vector<int> dimensions = {1};          

    // evaluate xTad data 
    shape::TAD xTad(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function        
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		

	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execIndexReduce(&lc, nd4j::indexreduce::IndexMax, 
										nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
										nullptr, 
										nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
										(int*)devicePtrs[0], dimensions.size(), 
										(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) 
		hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execIndexReduce_2) {
    	
    NDArray x('c', {2,3,4,5}, {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,
    						  	100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,
    							100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,
    							100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,
    							100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,
    							100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100}, nd4j::DataType::FLOAT32);
    x.linspace(-2.f); x.syncToDevice();
    NDArray exp('c', {2,5}, {11,11,11,11,11,11,11,11,11,11}, nd4j::DataType::INT64);    
    NDArray z('c', {2,5}, {100,100,100,100,100,100,100,100,100,100}, nd4j::DataType::INT64);
    
    std::vector<int> dimensions = {1,2};     

    // evaluate xTad data 
    shape::TAD xTad(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function    
    
    std::vector<std::pair<void*,size_t>> hostData;    
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		

	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execIndexReduce(&lc, nd4j::indexreduce::IndexMax, 
										nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
										nullptr, 
										nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
										(int*)devicePtrs[0], dimensions.size(), 
										(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) 
		hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execIndexReduce_3) {
    	
    NDArray x('c', {2,3,4,5}, {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,
    						  	100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,
    							100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,
    							100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,
    							100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,
    							100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100}, nd4j::DataType::DOUBLE);
    x.linspace(-2.); x.syncToDevice();
    NDArray exp('c', {3}, {39, 39, 39}, nd4j::DataType::INT64);    
    NDArray z('c', {3}, {100,100,100}, nd4j::DataType::INT64);
    
    std::vector<int> dimensions = {0,2,3};

    // evaluate xTad data 
    shape::TAD xTad(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function        
    std::vector<std::pair<void*,size_t>> hostData;   
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		

	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execIndexReduce(&lc, nd4j::indexreduce::IndexMax, 
										nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
										nullptr, 
										nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
										(int*)devicePtrs[0], dimensions.size(), 
										(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) 
		hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execScalar_1) {

	if (!Environment::getInstance()->isExperimentalBuild())
        return;
    	
    NDArray x('c', {2,3},  {0,1,2,3,4,5}, nd4j::DataType::INT64); 
    NDArray exp('c',{2,3}, {0,0,1,1,2,2}, nd4j::DataType::INT64);
    NDArray scalar('c',{0}, {2}, nd4j::DataType::FLOAT32);
    NDArray z('c', {2,3}, {100,100,100,100,100,100}, nd4j::DataType::INT64);
    
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
	
	// call cuda kernel which calculates result
	NativeOpExecutioner::execScalar(&lc, nd4j::scalar::Divide, 
									nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
									nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
									nullptr, scalar.getShapeInfo(), scalar.specialBuffer(), scalar.specialShapeInfo(), 
									nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execScalar_2) {

	if (!Environment::getInstance()->isExperimentalBuild())
        return;
    	
    NDArray x('c', {2,3},  {-1,-2,-3,-4,-5,-6}, nd4j::DataType::INT64); 
    NDArray exp('c',{2,3}, {10,10,10,10,10,10}, nd4j::DataType::FLOAT32);
    NDArray scalar('c',{0}, {10}, nd4j::DataType::FLOAT32);
    NDArray z('c', {2,3}, {100,100,100,100,100,100}, nd4j::DataType::FLOAT32);
    
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execScalar(&lc, nd4j::scalar::CopyPws, 
									nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
									nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
									nullptr, scalar.getShapeInfo(), scalar.specialBuffer(), scalar.specialShapeInfo(), 
									nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);


	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execScalar_3) {

	if (!Environment::getInstance()->isExperimentalBuild())
        return;
    	
    NDArray x('c', {2,3,2},  {0,1,2,3,4,5,6,7,8,9,10,11}, nd4j::DataType::INT64); 
    NDArray scalars('c',{2,2}, {1,2,3,4}, nd4j::DataType::FLOAT32);
    NDArray exp('c', {2,3,2},  {0,0,2,1,4,2, 2,1,2,2,3,2}, nd4j::DataType::INT64);     
    NDArray z('c', {2,3,2}, {100,100,100,100,100,100,100,100,100,100,100,100}, nd4j::DataType::INT64);

    std::vector<int> dimensions = {1};

    // evaluate xTad data 
    shape::TAD xTad(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function       
    std::vector<std::pair<void*,size_t>> hostData;   
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));							// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets	
	std::vector<void*> devicePtrs(hostData.size(), nullptr);
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		

	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execScalar(&lc, nd4j::scalar::Divide, 
									nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
									nullptr,
									nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
									nullptr, scalars.getShapeInfo(), scalars.specialBuffer(), scalars.specialShapeInfo(),
									(int*)devicePtrs[0], dimensions.size(), 
									(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], 
									nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5); 		

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) 
		hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execScalarBool_1) {
    	
    NDArray x('c', {2,3},  {-1,-2,0,1,2,3}, nd4j::DataType::BFLOAT16); 
    NDArray scalar('c',{0}, {0}, nd4j::DataType::BFLOAT16);
    NDArray exp('c',{2,3}, {0,0,0,1,1,1}, nd4j::DataType::BOOL);    
    NDArray z('c', {2,3}, {100,100,100,100,100,100,}, nd4j::DataType::BOOL);    
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
		
	// call cuda kernel which calculates result
	// call cuda kernel which calculates result
	NativeOpExecutioner::execScalarBool(&lc, nd4j::scalar::GreaterThan, 
									nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
									nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
									nullptr, scalar.getShapeInfo(), scalar.specialBuffer(), scalar.specialShapeInfo(), 
									nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execScalarBool_2) {
    	
    NDArray x('c', {2,3},  {0,1,2,3,4,5}, nd4j::DataType::FLOAT32); 
    NDArray scalars('c',{2}, {-1,4}, nd4j::DataType::FLOAT32);
    NDArray exp('c', {2,3},  {1,1,1,0,0,1}, nd4j::DataType::BOOL);
    NDArray z('c', {2,3}, {100,100,100,100,100,100}, nd4j::DataType::BOOL);

    std::vector<int> dimensions = {1};

    // evaluate xTad data 
    shape::TAD xTad(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function       
    std::vector<std::pair<void*,size_t>> hostData;   
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));							// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets	
	std::vector<void*> devicePtrs(hostData.size(), nullptr);
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
			
	// call cuda kernel which calculates result
	NativeOpExecutioner::execScalarBool(&lc, nd4j::scalar::GreaterThan, 
									nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
									nullptr,
									nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
									nullptr, scalars.getShapeInfo(), scalars.specialBuffer(), scalars.specialShapeInfo(),
									(int*)devicePtrs[0], dimensions.size(), 
									(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], 
									nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5); 		

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) 
		hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execBroadcast_1) {

	if (!Environment::getInstance()->isExperimentalBuild())
        return;
    	
	NDArray x('c', {2,3,4}, {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100}, nd4j::DataType::INT32);
    NDArray y('c', {3},   {10, 20, 30}, nd4j::DataType::INT64);
    NDArray z('c', {2,3,4}, {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100}, nd4j::DataType::INT32);	
	NDArray exp('c', {2,3,4}, {10, 11, 12, 13,24, 25, 26, 27,38, 39, 40, 41,22, 23, 24, 25,36, 37, 38, 39,50, 51, 52, 53}, nd4j::DataType::INT32);
	x.linspace(0); x.syncToDevice();

    std::vector<int> dimensions = {1};

    // evaluate xTad data 
    shape::TAD xTad(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function       
    std::vector<std::pair<void*,size_t>> hostData;   
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));							// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets	
	std::vector<void*> devicePtrs(hostData.size(), nullptr);
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execBroadcast(&lc, nd4j::broadcast::Add,
										nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
										nullptr, y.getShapeInfo(), y.specialBuffer(), y.specialShapeInfo(),
										nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
										(int*)devicePtrs[0], dimensions.size(), 
										(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], 
										nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5); 		

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) 
		hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execBroadcast_2) {

	if (!Environment::getInstance()->isExperimentalBuild())
        return;
    	
	NDArray x('c', {2,3,4}, {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100}, nd4j::DataType::INT32);
    NDArray y('c', {2,4},   {10,20,30,40,50,60,70,80}, nd4j::DataType::FLOAT32);
    NDArray z('c', {2,3,4}, {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100}, nd4j::DataType::FLOAT32);	
	NDArray exp('c', {2,3,4}, {10., 21., 32., 43., 14., 25., 36., 47., 18., 29., 40., 51., 62., 73., 84., 95., 66., 77., 88., 99., 70., 81., 92., 103}, nd4j::DataType::FLOAT32);
	x.linspace(0); x.syncToDevice();

    std::vector<int> dimensions = {0,2};

    // evaluate xTad data 
    shape::TAD xTad(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function       
    std::vector<std::pair<void*,size_t>> hostData;   
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));							// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets	
	std::vector<void*> devicePtrs(hostData.size(), nullptr);
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execBroadcast(&lc, nd4j::broadcast::Add,
										nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
										nullptr, y.getShapeInfo(), y.specialBuffer(), y.specialShapeInfo(),
										nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
										(int*)devicePtrs[0], dimensions.size(), 
										(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], 
										nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5); 		

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) 
		hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execBroadcastBool_1) {
    	
	NDArray x('c', {2,3,4}, {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100}, nd4j::DataType::INT32);
    NDArray y('c', {3},   {2, 12, 22}, nd4j::DataType::INT32);
    NDArray z('c', {2,3,4}, {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,}, nd4j::DataType::BOOL);	
	NDArray exp('c', {2,3,4}, {0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0}, nd4j::DataType::BOOL);
	x.linspace(1); x.syncToDevice();

    std::vector<int> dimensions = {1};

    // evaluate xTad data 
    shape::TAD xTad(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function       
    std::vector<std::pair<void*,size_t>> hostData;   
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));							// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets	
	std::vector<void*> devicePtrs(hostData.size(), nullptr);
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execBroadcastBool(&lc, nd4j::broadcast::EqualTo,
										nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
										nullptr, y.getShapeInfo(), y.specialBuffer(), y.specialShapeInfo(),
										nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
										(int*)devicePtrs[0], dimensions.size(), 
										(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], 
										nullptr, nullptr);	

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5); 		

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) 
		hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execBroadcastBool_2) {
    	
	NDArray x('c', {2,3,4}, {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100},nd4j::DataType::FLOAT32);
    NDArray y('c', {2,4},   {1,10,10,15,20,20,20,24}, nd4j::DataType::FLOAT32);
    NDArray z('c', {2,3,4}, {100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100,100}, nd4j::DataType::BOOL);	
	NDArray exp('c', {2,3,4}, {1, 0, 0, 0,0, 0, 0, 0,0, 1, 0, 0,0, 0, 0, 0,0, 0, 0, 0,0, 0, 0, 1}, nd4j::DataType::BOOL);
	x.linspace(1); x.syncToDevice();

    std::vector<int> dimensions = {0,2};

    // evaluate xTad data 
    shape::TAD xTad(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function       
    std::vector<std::pair<void*,size_t>> hostData;   
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));							// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets
	
	std::vector<void*> devicePtrs(hostData.size(), nullptr);
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execBroadcastBool(&lc, nd4j::broadcast::EqualTo,
										nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
										nullptr, y.getShapeInfo(), y.specialBuffer(), y.specialShapeInfo(),
										nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
										(int*)devicePtrs[0], dimensions.size(), 
										(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], 
										nullptr, nullptr);	

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5); 		

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) 
		hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execPairwiseTransform_1) {

	if (!Environment::getInstance()->isExperimentalBuild())
        return;
    	
	NDArray x('c', {2,2,2}, {1,5,3,7,2,6,4,8}, nd4j::DataType::INT32);
    NDArray y('c', {4,2}, {0.1,0.2,0.3,0.4,1.5,0.6,0.7,1.8}, nd4j::DataType::DOUBLE);
    NDArray z('c', {8}, {100,100,100,100,100,100,100,100}, nd4j::DataType::INT32);	
	NDArray exp('c', {8}, {0,1,2,3,3,5,6,6}, nd4j::DataType::INT32);
	x.permutei({2,1,0});	// -> {1,2,3,4,5,6,7,8}
    x.syncShape();

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
	
	// call cuda kernel which calculates result
	NativeOpExecutioner::execPairwiseTransform(&lc, nd4j::pairwise::Subtract,
												nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
												nullptr, y.getShapeInfo(), y.specialBuffer(), y.specialShapeInfo(), 
												nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
												nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();
    
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
	
	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execPairwiseBoolTransform_1) {
    	
	NDArray x('c', {2,2,2}, {1,5,3,7,2,6,4,8}, nd4j::DataType::INT64);
    NDArray y('c', {4,2}, {0,2,0,4,0,6,0,8}, nd4j::DataType::INT64);
    NDArray z('c', {8}, {100,100,100,100,100,100,100,100}, nd4j::DataType::BOOL);	
	NDArray exp('c', {8}, {0,1,0,1,0,1,0,1}, nd4j::DataType::BOOL);
	x.permutei({2,1,0});	// -> {1,2,3,4,5,6,7,8}
	x.syncShape();
        
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);	

	// call cuda kernel which calculates result
	NativeOpExecutioner::execPairwiseBoolTransform(&lc, nd4j::pairwise::EqualTo,
													nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
													nullptr, y.getShapeInfo(), y.specialBuffer(), y.specialShapeInfo(), 
													nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
													nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();
    
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
	
	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}


////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execTransformFloat_1) {
    	
	NDArray x('c', {2,2}, {0, 6.25, 2.25, 12.25}, nd4j::DataType::DOUBLE);    
    NDArray z('c', {4}, {100,100,100,100}, nd4j::DataType::FLOAT32);	
	NDArray exp('c', {4}, {0, 1.5, 2.5, 3.5}, nd4j::DataType::FLOAT32);
	x.permutei({1,0});
	x.syncShape();
        
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execTransformFloat(&lc, nd4j::transform::Sqrt,
		nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
		nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
		nullptr, nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();
    
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execTransformFloat_2) {
    	
	NDArray x('c', {1,4}, {0, 4, 9, 16}, nd4j::DataType::INT64);
    NDArray z('c', {2,2}, {100,100,100,100}, nd4j::DataType::DOUBLE);	
	NDArray exp('c', {2,2}, {0, 2, 3, 4}, nd4j::DataType::DOUBLE);	       
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
	
	// call cuda kernel which calculates result
	NativeOpExecutioner::execTransformFloat(&lc, nd4j::transform::Sqrt,
		nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
		nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
		nullptr, nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();
    
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execTransformAny_1) {
    	
	NDArray x('c', {2,2}, {0, 6.25, 2.25, 12.25}, nd4j::DataType::DOUBLE);    
    NDArray z('c', {4,1}, {100,100,100,100}, nd4j::DataType::INT32);	
	NDArray exp('c', {4,1}, {0, 2, 6, 12}, nd4j::DataType::INT32);
	x.permutei({1,0});
	x.syncShape();
        
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execTransformAny(&lc, nd4j::transform::Assign,
		nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
		nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
		nullptr, nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();
    
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execTransformAny_2) {
    	
	NDArray x('c', {1,4}, {0, 6.25, 2.25, 12.25}, nd4j::DataType::BFLOAT16);
    NDArray z('c', {2,2}, {100,100,100,100}, nd4j::DataType::FLOAT32);	
	NDArray exp('c', {2,2}, {0, 6.25, 2.25, 12.25}, nd4j::DataType::FLOAT32);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execTransformAny(&lc, nd4j::transform::Assign,
		nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
		nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
		nullptr, nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();
    
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execTransformStrict_1) {
    	
	NDArray x('c', {2,3}, {0,2,4,1,3,5}, nd4j::DataType::DOUBLE);
    NDArray z('c', {3,2}, {100,100,100,100,100,100}, nd4j::DataType::DOUBLE);	
	NDArray exp('c', {3,2}, {0, 3, 12, 27, 48, 75}, nd4j::DataType::DOUBLE);
	x.permutei({1,0});
	x.syncShape();
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execTransformStrict(&lc, nd4j::transform::CubeDerivative,
		nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
		nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
		nullptr, nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();
    
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execTransformStrict_2) {
    	
	NDArray x('c', {6}, {0,1,2,3,4,5}, nd4j::DataType::FLOAT32);
    NDArray z('c', {3,2}, {100,100,100,100,100,100}, nd4j::DataType::FLOAT32);	
	NDArray exp('c', {3,2}, {0, 3, 12, 27, 48, 75}, nd4j::DataType::FLOAT32);	
    	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execTransformStrict(&lc, nd4j::transform::CubeDerivative,
		nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
		nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
		nullptr, nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
	z.syncToHost();
    
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execTransformSame_1) {
    
	NDArray x('c', {2,3}, {0,2.5,4.5,1.5,3.5,5.5}, nd4j::DataType::DOUBLE);	
    NDArray z('c', {1,6}, {100,100,100,100,100,100}, nd4j::DataType::DOUBLE);	
	NDArray exp('c', {1,6}, {0,2.25,6.25,12.25,20.25,30.25}, nd4j::DataType::DOUBLE);
	x.permutei({1,0});
	x.syncShape();
    	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execTransformSame(&lc, nd4j::transform::Square,
		nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
		nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
		nullptr, nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();
        
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execTransformSame_2) {
    	
	NDArray x('c', {6}, {0,1,2,3,4,5}, nd4j::DataType::INT32);
    NDArray z('c', {3,2}, {100,100,100,100,100,100}, nd4j::DataType::INT32);	
	NDArray exp('c', {3,2}, {0,1,4,9,16,25}, nd4j::DataType::INT32);	
    	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
	
	// call cuda kernel which calculates result
	NativeOpExecutioner::execTransformSame(&lc, nd4j::transform::Square,
		nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
		nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
		nullptr, nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();
    
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execTransformBool_1) {
    
	NDArray x('c', {2,3}, {0,2,4,-1,-3,-5}, nd4j::DataType::DOUBLE);	
    NDArray z('c', {1,6}, {100,100,100,100,100,100}, nd4j::DataType::BOOL);	    
	NDArray exp('c', {1,6}, {0,0,1,0,1,0}, nd4j::DataType::BOOL);
	x.permutei({1,0});
	x.syncShape();
    
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execTransformBool(&lc, nd4j::transform::IsPositive,
		nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
		nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
		nullptr, nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();
         	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execTransformBool_2) {
    	
	NDArray x('c', {6}, {0,-1,2,-3,4,-5}, nd4j::DataType::INT32);
    NDArray z('c', {3,2}, {100,100,100,100,100,100}, nd4j::DataType::BOOL);	
	NDArray exp('c', {3,2}, {0,0,1,0,1,0}, nd4j::DataType::BOOL);
    	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// call cuda kernel which calculates result
	NativeOpExecutioner::execTransformBool(&lc, nd4j::transform::IsPositive,
		nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
		nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
		nullptr, nullptr, nullptr);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();
    
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execReduceFloat_1) {
    	   	
    NDArray x('c', {2,3,4}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18}, nd4j::DataType::INT32);
    NDArray z('c', {3}, {100,100,100}, nd4j::DataType::FLOAT32);
    NDArray exp('c', {3}, {2.5, 6.5, 10.5}, nd4j::DataType::FLOAT32);
    x.permutei({2,1,0});
    x.syncShape();    
    
    std::vector<int> dimensions = {0,2};

    // evaluate xTad data 
    shape::TAD xTad(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function       
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceFloat(&lc, nd4j::reduce::Mean, 
					nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr, 
					nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
					(int*)devicePtrs[0], dimensions.size(), 
					(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)  		
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) 
		hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execReduceFloat_2) {
    	   	
    NDArray x('c', {2,3,4}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18}, nd4j::DataType::INT32);
    NDArray z('c', {2,4}, {100,100,100,100,100,100,100,100}, nd4j::DataType::DOUBLE);
    NDArray exp('c', {2,4}, {-1., 0., 1., 2.,11., 12., 13., 14.}, nd4j::DataType::DOUBLE);
    
    std::vector<int> dimensions = {1};

    // evaluate xTad data 
    shape::TAD xTad(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function       
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceFloat(&lc, nd4j::reduce::Mean, 
					nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr, 
					nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
					(int*)devicePtrs[0], dimensions.size(), 
					(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)  		
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) 
		hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execReduceSame_1) {
    	   	
    NDArray x('c', {2,3,4}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18}, nd4j::DataType::INT32);
    NDArray z('c', {3}, {100,100,100}, nd4j::DataType::INT32);
    NDArray exp('c', {3}, {20, 52, 84}, nd4j::DataType::INT32);
    x.permutei({2,1,0});
    x.syncShape();    
    
    std::vector<int> dimensions = {0,2};

    // evaluate xTad data 
    shape::TAD xTad(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function       
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceSame(&lc, nd4j::reduce::Sum, 
					nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr, 
					nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
					(int*)devicePtrs[0], dimensions.size(), 
					(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)  		
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) 
		hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execReduceSame_2) {
    	   	
    NDArray x('c', {2,3,4}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18}, nd4j::DataType::FLOAT32);
    NDArray z('c', {2,4}, {100,100,100,100,100,100,100,100}, nd4j::DataType::FLOAT32);
    NDArray exp('c', {2,4}, {-3., 0., 3., 6.,33., 36., 39., 42.}, nd4j::DataType::FLOAT32);
    
    std::vector<int> dimensions = {1};

    // evaluate xTad data 
    shape::TAD xTad(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function       
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceSame(&lc, nd4j::reduce::Sum, 
					nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr, 
					nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
					(int*)devicePtrs[0], dimensions.size(), 
					(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)  		
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) 
		hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execReduceBool_1) {
    	   	
    NDArray x('c', {2,3,4}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6,-7,-8,-9,-10,-11,-12,-13,-14,-15,-16,-17,-18}, nd4j::DataType::INT32);
    NDArray z('c', {3}, {100,100,100}, nd4j::DataType::BOOL);
    NDArray exp('c', {3}, {0, 1, 1}, nd4j::DataType::BOOL);
    x.permutei({2,1,0});
    x.syncShape();    
    
    std::vector<int> dimensions = {0,2};

    // evaluate xTad data 
    shape::TAD xTad(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function       
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceBool(&lc, nd4j::reduce::IsPositive, 
					nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr, 
					nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
					(int*)devicePtrs[0], dimensions.size(), 
					(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)  		
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) 
		hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execReduceBool_2) {
    	   	
    NDArray x('c', {2,3,4}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6,-7,-8,-9,-10,-11,-12,-13,-14,-15,-16,-17,-18}, nd4j::DataType::FLOAT32);
    NDArray z('c', {2,4}, {100,100,100,100,100,100,100,100}, nd4j::DataType::BOOL);
    NDArray exp('c', {2,4}, {1, 1, 1, 1, 0, 0, 0, 0}, nd4j::DataType::BOOL);
    
    std::vector<int> dimensions = {1};

    // evaluate xTad data 
    shape::TAD xTad(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function       
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceBool(&lc, nd4j::reduce::IsPositive, 
					nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr, 
					nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
					(int*)devicePtrs[0], dimensions.size(), 
					(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)  		
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) 
		hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execReduceLong_1) {
    	   	
    NDArray x('c', {2,3,4}, {-5,0,-3,0,-1,0,1,2,3,4,5,6,7,0,9,10,11,0,13,14,0,16,0,18}, nd4j::DataType::INT32);
    NDArray z('c', {3}, {100,100,100}, nd4j::DataType::INT64);
    NDArray exp('c', {3}, {5,6,6}, nd4j::DataType::INT64);
    x.permutei({2,1,0});
    x.syncShape();    
    
    std::vector<int> dimensions = {0,2};

    // evaluate xTad data 
    shape::TAD xTad(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function       
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceLong(&lc, nd4j::reduce::CountNonZero, 
					nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr, 
					nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
					(int*)devicePtrs[0], dimensions.size(), 
					(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)  		
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) 
		hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execReduceLong_2) {
    	   	
    NDArray x('c', {2,3,4}, {-5,0,-3,0,-1,0,1,2,3,4,5,6,7,0,9,10,11,0,13,14,0,16,0,18}, nd4j::DataType::FLOAT32);
    NDArray z('c', {2,4}, {100,100,100,100,100,100,100,100}, nd4j::DataType::INT64);
    NDArray exp('c', {2,4}, {3, 1, 3, 2, 2, 1, 2, 3}, nd4j::DataType::INT64);    

    std::vector<int> dimensions = {1};

    // evaluate xTad data 
    shape::TAD xTad(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function       
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets
	std::vector<void*> devicePtrs(hostData.size(), nullptr);
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceLong(&lc, nd4j::reduce::CountNonZero, 
					nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr, 
					nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(),
					(int*)devicePtrs[0], dimensions.size(), 
					(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)  		
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) 
		hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execReduceFloatScalar_1) {
    	   	
    NDArray x('c', {2,3,4}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18}, nd4j::DataType::INT32);
    NDArray z('c', {0}, {100}, nd4j::DataType::FLOAT32);
    NDArray exp('c', {0}, {6.5}, nd4j::DataType::FLOAT32);
    x.permutei({2,1,0});
    x.syncShape();    
       
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
	void* reductionPointer;
    cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer),  1024*1024); ASSERT_EQ(0, cudaResult);
    int* allocationPointer;
	cudaResult = hipMalloc(reinterpret_cast<void **>(&allocationPointer), 1024*1024); ASSERT_EQ(0, cudaResult);
	lc.setReductionPointer(reductionPointer);
	lc.setAllocationPointer(allocationPointer);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceFloatScalar(&lc, nd4j::reduce::Mean, 
					nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr, 
					nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo());
	
	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)  		
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execReduceFloatScalar_2) {
    	   	
    NDArray x('c', {2,3,4}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18}, nd4j::DataType::INT32);
    NDArray z('c', {0}, {100}, nd4j::DataType::DOUBLE);
    NDArray exp('c', {0}, {6.5}, nd4j::DataType::DOUBLE);        
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
	void* reductionPointer;
    cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer),  1024*1024); ASSERT_EQ(0, cudaResult);
    int* allocationPointer;
	cudaResult = hipMalloc(reinterpret_cast<void **>(&allocationPointer), 1024*1024); ASSERT_EQ(0, cudaResult);
	lc.setReductionPointer(reductionPointer);
	lc.setAllocationPointer(allocationPointer);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceFloatScalar(&lc, nd4j::reduce::Mean, 
					nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr, 
					nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo());

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)  		
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execReduceSameScalar_1) {
    	   	
    NDArray x('c', {2,3,4}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18}, nd4j::DataType::INT32);
    NDArray z('c', {0}, {100}, nd4j::DataType::INT32);
    NDArray exp('c', {0}, {156}, nd4j::DataType::INT32);
    x.permutei({2,1,0});
    x.syncShape();    
       
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
	void* reductionPointer;
    cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer),  1024*1024); ASSERT_EQ(0, cudaResult);
    int* allocationPointer;
	cudaResult = hipMalloc(reinterpret_cast<void **>(&allocationPointer), 1024*1024); ASSERT_EQ(0, cudaResult);
	lc.setReductionPointer(reductionPointer);
	lc.setAllocationPointer(allocationPointer);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceSameScalar(&lc, nd4j::reduce::Sum, 
					nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr, 
					nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo());
	
	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)  		
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execReduceSameScalar_2) {
    	   	
    NDArray x('c', {2,3,4}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18}, nd4j::DataType::DOUBLE);
    NDArray z('c', {0}, {100}, nd4j::DataType::DOUBLE);
    NDArray exp('c', {0}, {156}, nd4j::DataType::DOUBLE);        
	
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
	void* reductionPointer;
    cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer),  1024*1024); ASSERT_EQ(0, cudaResult);
    int* allocationPointer;
	cudaResult = hipMalloc(reinterpret_cast<void **>(&allocationPointer), 1024*1024); ASSERT_EQ(0, cudaResult);
	lc.setReductionPointer(reductionPointer);
	lc.setAllocationPointer(allocationPointer);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceSameScalar(&lc, nd4j::reduce::Sum, 
					nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr, 
					nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo());

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)  		
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execReduceBoolScalar_1) {
    	   	
    NDArray x('c', {2,3,4}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6,-7,-8,-9,-10,-11,-12,-13,-14,-15,-16,-17,-18}, nd4j::DataType::INT32);
    NDArray z('c', {0}, {100}, nd4j::DataType::BOOL);
    NDArray exp('c', {0}, {1}, nd4j::DataType::BOOL);
    x.permutei({2,1,0});
    x.syncShape();    
       
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
	void* reductionPointer;
    cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer),  1024*1024); ASSERT_EQ(0, cudaResult);
    int* allocationPointer;
	cudaResult = hipMalloc(reinterpret_cast<void **>(&allocationPointer), 1024*1024); ASSERT_EQ(0, cudaResult);
	lc.setReductionPointer(reductionPointer);
	lc.setAllocationPointer(allocationPointer);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceBoolScalar(&lc, nd4j::reduce::IsPositive, 
					nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr, 
					nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo());
	
	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)  		
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execReduceBoolScalar_2) {
    	   	
    NDArray x('c', {2,3,4}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6,-7,-8,-9,-10,-11,-12,-13,-14,-15,-16,-17,-18}, nd4j::DataType::DOUBLE);
    NDArray z('c', {0}, {100}, nd4j::DataType::BOOL);
    NDArray exp('c', {0}, {1}, nd4j::DataType::BOOL);
    
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
	void* reductionPointer;
    cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer),  1024*1024); ASSERT_EQ(0, cudaResult);
    int* allocationPointer;
	cudaResult = hipMalloc(reinterpret_cast<void **>(&allocationPointer), 1024*1024); ASSERT_EQ(0, cudaResult);
	lc.setReductionPointer(reductionPointer);
	lc.setAllocationPointer(allocationPointer);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceBoolScalar(&lc, nd4j::reduce::IsPositive, 
					nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr, 
					nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo());

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)  		
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execReduceLongScalar_1) {
    	   	
    NDArray x('c', {2,3,4}, {-5,0,-3,0,-1,0,1,2,3,4,5,6,7,0,9,10,11,0,13,14,0,16,0,18}, nd4j::DataType::INT32);
    NDArray z('c', {0}, {100}, nd4j::DataType::INT64);
    NDArray exp('c', {0}, {17}, nd4j::DataType::INT64);
    x.permutei({2,1,0});
    x.syncShape();    
       
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
	void* reductionPointer;
    cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer),  1024*1024); ASSERT_EQ(0, cudaResult);
    int* allocationPointer;
	cudaResult = hipMalloc(reinterpret_cast<void **>(&allocationPointer), 1024*1024); ASSERT_EQ(0, cudaResult);
	lc.setReductionPointer(reductionPointer);
	lc.setAllocationPointer(allocationPointer);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceLongScalar(&lc, nd4j::reduce::CountNonZero, 
					nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr, 
					nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo());
	
	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)  		
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execReduceLongScalar_2) {
    	   	
    NDArray x('c', {2,3,4}, {-5,0,-3,0,-1,0,1,2,3,4,5,6,7,0,9,10,11,0,13,14,0,16,0,18}, nd4j::DataType::DOUBLE);
    NDArray z('c', {0}, {100}, nd4j::DataType::INT64);
    NDArray exp('c', {0}, {17}, nd4j::DataType::INT64);
    
	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
	void* reductionPointer;
    cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer),  1024*1024); ASSERT_EQ(0, cudaResult);
    int* allocationPointer;
	cudaResult = hipMalloc(reinterpret_cast<void **>(&allocationPointer), 1024*1024); ASSERT_EQ(0, cudaResult);
	lc.setReductionPointer(reductionPointer);
	lc.setAllocationPointer(allocationPointer);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduceLongScalar(&lc, nd4j::reduce::CountNonZero, 
					nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(),
					nullptr, 
					nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo());

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost();

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++)  		
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execReduce3TAD_1) {
    	
    NDArray x('c', {2,2,3}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6}, nd4j::DataType::FLOAT32);
    NDArray y('c', {2,2}, {1,2,3,4}, nd4j::DataType::FLOAT32);
    NDArray exp('c', {3}, {10,20,30}, nd4j::DataType::DOUBLE);
    NDArray z('c', {3}, {100,100,100}, nd4j::DataType::DOUBLE);
   
    std::vector<int> dimensions = {0,1};

    // evaluate xTad data 
    shape::TAD xTad(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets	
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		

	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduce3TAD(&lc, nd4j::reduce3::Dot,
								nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
								nullptr, 
								nullptr, y.getShapeInfo(), y.specialBuffer(), y.specialShapeInfo(), 
								nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
								(int*)devicePtrs[0], dimensions.size(), 
								(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost(); 	

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execReduce3TAD_2) {
    	
    NDArray x('c', {2,2,3}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6}, nd4j::DataType::INT64);
    NDArray y('c', {2,3}, {1,2,3,4,5,6}, nd4j::DataType::INT64);
    NDArray exp('c', {2}, {10,73}, nd4j::DataType::FLOAT32);
    NDArray z('c', {2}, {100,100}, nd4j::DataType::FLOAT32);
   
    std::vector<int> dimensions = {0,2};

    // evaluate xTad data 
    shape::TAD xTad(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets	
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		

	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduce3TAD(&lc, nd4j::reduce3::Dot,
								nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
								nullptr, 
								nullptr, y.getShapeInfo(), y.specialBuffer(), y.specialShapeInfo(), 
								nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
								(int*)devicePtrs[0], dimensions.size(), 
								(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost(); 	

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execReduce3TAD_3) {
    	
    NDArray x('c', {2,2,3}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6}, nd4j::DataType::INT64);
    NDArray y('c', {3}, {1,2,3}, nd4j::DataType::INT64);
    NDArray exp('c', {2,2}, {-22,-4,14,32}, nd4j::DataType::FLOAT32);
    NDArray z('c', {2,2}, {100,100,100,100}, nd4j::DataType::FLOAT32);
   
    std::vector<int> dimensions = {2};

    // evaluate xTad data 
    shape::TAD xTad(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets	
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		

	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduce3TAD(&lc, nd4j::reduce3::Dot,
								nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
								nullptr, 
								nullptr, y.getShapeInfo(), y.specialBuffer(), y.specialShapeInfo(), 
								nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
								(int*)devicePtrs[0], dimensions.size(), 
								(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost(); 	

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execReduce3TAD_4) {
    	
    NDArray x('c', {2,2,3}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6}, nd4j::DataType::DOUBLE);
    NDArray y('c', {2,2,3}, {10,20,30,40,50,60,70,80,90,100,110,120}, nd4j::DataType::DOUBLE);
    NDArray exp('c', {0}, {1820}, nd4j::DataType::FLOAT32);
    NDArray z('c', {0}, {100}, nd4j::DataType::FLOAT32);

    std::vector<int> dimensions = {0,1,2};

    // evaluate xTad data 
    shape::TAD xTad(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets	
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execReduce3TAD(&lc, nd4j::reduce3::Dot,
								nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
								nullptr, 
								nullptr, y.getShapeInfo(), y.specialBuffer(), y.specialShapeInfo(), 
								nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
								(int*)devicePtrs[0], dimensions.size(), 
								(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost(); 	

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execSummaryStats_1) {
    	
    NDArray x('c', {2,2,3}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6}, nd4j::DataType::INT64);    
    NDArray exp('c', {0}, {3.605551}, nd4j::DataType::FLOAT32);
    NDArray z('c', {0}, {100}, nd4j::DataType::FLOAT32);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
	void* reductionPointer;
    cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer), 1024*1024); ASSERT_EQ(0, cudaResult);    	
	lc.setReductionPointer(reductionPointer);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execSummaryStats(&lc, nd4j::variance::SummaryStatsStandardDeviation,
								nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
								nullptr, 								
								nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
								true);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost(); 	

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execSummaryStats_2) {
    	
    NDArray x('c', {2,2,3}, {-5,-4,-3,-20,-1,0,1,2,3,4,5,6}, nd4j::DataType::DOUBLE);    
    NDArray exp('c', {2}, {3.405877, 9.715966}, nd4j::DataType::FLOAT32);
    NDArray z('c', {2}, {100,100}, nd4j::DataType::FLOAT32);

    std::vector<int> dimensions = {0,2};

    // evaluate xTad data 
    shape::TAD xTad(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets	
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execSummaryStats(&lc, nd4j::variance::SummaryStatsStandardDeviation,
								nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
								nullptr, 								
								nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
								(int*)devicePtrs[0], dimensions.size(), 
								(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2],
								true);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost(); 	

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execSummaryStats_3) {
    	
    NDArray x('c', {2,2,3}, {-5,-4,-3,-20,-1,0,1,2,3,4,5,6}, nd4j::DataType::DOUBLE);    
    NDArray exp('c', {2}, {10.606602, 2.121320}, nd4j::DataType::FLOAT32);
    NDArray z('c', {2}, {100,100}, nd4j::DataType::FLOAT32);

    std::vector<int> dimensions = {1};

    // evaluate xTad data 
    shape::TAD xTad(x.getShapeInfo(), dimensions.data(), dimensions.size());
    xTad.createTadOnlyShapeInfo();
    xTad.createOffsets();

    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;    	
	hostData.emplace_back(dimensions.data(), dimensions.size() * sizeof(int));						// 0 -- dimensions
	hostData.emplace_back(xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));// 1 -- xTadShapeInfo
	hostData.emplace_back(xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong));						// 2 -- xTadOffsets	
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execSummaryStats(&lc, nd4j::variance::SummaryStatsStandardDeviation,
								nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
								nullptr, 								
								nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
								(int*)devicePtrs[0], dimensions.size(), 
								(Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2],
								true);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost(); 	

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execSummaryStatsScalar_1) {
    	
    NDArray x('c', {2,2,3}, {-5,-4,-3,-2,-1,0,1,2,3,4,5,6}, nd4j::DataType::INT64);
    NDArray exp('c', {0}, {3.605551}, nd4j::DataType::FLOAT32);
    NDArray z('c', {0}, {100}, nd4j::DataType::FLOAT32);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);
	void* reductionPointer;
    cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer), 1024*1024); ASSERT_EQ(0, cudaResult);    	
	lc.setReductionPointer(reductionPointer);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execSummaryStatsScalar(&lc, nd4j::variance::SummaryStatsStandardDeviation,
								nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
								nullptr, 								
								nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
								true);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost(); 	

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

//////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execRandom_1) {
    	   
    NDArray z('c', {10}, {100,0,0,0,0,0,0,0,0,0}, nd4j::DataType::DOUBLE);
    NDArray exp('c', {10}, {0.050942, -0.183229, -0.093921, 0.075469, 0.257166, -0.254838, 0.342227, -0.682188, -0.004345, 0.464633}, nd4j::DataType::DOUBLE);
    
    std::vector<double> extraArguments = {0., 0.5};
    nd4j::graph::RandomGenerator gen(119,5);
    
    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;    		
	hostData.emplace_back(extraArguments.data(), extraArguments.size() * sizeof(double));		// 0 -- dimensions		
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execRandom(&lc, nd4j::random::GaussianDistribution,
								&gen,
								nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
								nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
								nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 
								devicePtrs[0]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost(); 	

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

//////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execRandom_2) {
    	   
    NDArray x('c', {10}, {0.1,0.2,0.3,0.4,0.5,0.6,0.7,0.8,0.9,1}, nd4j::DataType::DOUBLE);    
    NDArray z('c', {2,5}, {100,100,100,100,100,100,100,100,100,100}, nd4j::DataType::DOUBLE);
    NDArray exp('c', {10}, {0., 0., 0.3, 0., 0.5, 0., 0.7, 0., 0., 1.}, nd4j::DataType::DOUBLE);
    
    std::vector<double> extraArguments = {0.7};
    nd4j::graph::RandomGenerator gen(119,5);
    
    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;    		
	hostData.emplace_back(extraArguments.data(), extraArguments.size() * sizeof(double));		// 0 -- dimensions		
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execRandom(&lc, nd4j::random::DropOut,
								&gen,
								nullptr, x.getShapeInfo(), x.specialBuffer(), x.specialShapeInfo(), 
								nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 								
								devicePtrs[0]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost(); 	

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

//////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execRandom_3) {
    	       
    NDArray z('c', {10}, {100,100,100,100,100,100,100,100,100,100}, nd4j::DataType::DOUBLE);    
    NDArray exp('c', {10}, {2.373649, 2.239791, 1.887353, 2.488636, 2.068904, 2.281399, 1.828228, 2.228222, 2.490847, 1.669537}, nd4j::DataType::DOUBLE);
    
    std::vector<double> extraArguments = {1.5, 2.5};
    nd4j::graph::RandomGenerator gen(119,5);
    
    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;    		
	hostData.emplace_back(extraArguments.data(), extraArguments.size() * sizeof(double));		// 0 -- dimensions		
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execRandom(&lc, nd4j::random::UniformDistribution,
								&gen,
								nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 								
								devicePtrs[0]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost(); 	

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

//////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execRandom_4) {
    	       
    NDArray z('c', {2,5}, {1,2,3,4,5,6,7,8,9,10}, nd4j::DataType::DOUBLE);    
    NDArray exp('c', {10}, {2.373649, 2.281399, 2.239791, 1.828228, 1.887353, 2.228222, 2.488636, 2.490847, 2.068904, 1.669537}, nd4j::DataType::DOUBLE);                              
    z.permutei({1,0});        
    
    std::vector<double> extraArguments = {1.5, 2.5};
    nd4j::graph::RandomGenerator gen(119,5);
    
    // prepare input arrays for prepareDataForCuda function
    std::vector<std::pair<void*,size_t>> hostData;    		
	hostData.emplace_back(extraArguments.data(), extraArguments.size() * sizeof(double));		// 0 -- dimensions		
	std::vector<void*> devicePtrs(hostData.size(), nullptr);

	// create cuda stream and LaunchContext
	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
	LaunchContext lc(&stream);

	// allocate required amount of global device memory and copy host data to it 		
	cudaResult = allocateDeviceMem(lc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
		
	// call cuda kernel which calculates result
	NativeOpExecutioner::execRandom(&lc, nd4j::random::UniformDistribution,
								&gen,
								nullptr, z.getShapeInfo(), z.specialBuffer(), z.specialShapeInfo(), 								
								devicePtrs[0]);

	cudaResult = hipStreamSynchronize(stream); ASSERT_EQ(0, cudaResult);
    z.syncToHost(); 	

 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	// free allocated global device memory
	for(int i = 0; i < devicePtrs.size(); ++i) hipFree(devicePtrs[i]);	

	// delete cuda stream
	cudaResult = hipStreamDestroy(stream); ASSERT_EQ(0, cudaResult);
}

//////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, mmulMxM_1) {

	const Nd4jLong M = 3;
	const Nd4jLong K = 4;
	const Nd4jLong N = 5;

	NDArray a('f', {M,K}, {1.2,1.1,1.0,0.9,0.8,0.7,0.5,0.4,0.3,0.2,0.1,0}, nd4j::DataType::DOUBLE);
	NDArray b('f', {K,N}, {1,-2,3,-4,5,-6,7,-8,9,-10,11,-12,13,-14,15,-16,17,-18,19,-20}, nd4j::DataType::DOUBLE);
	NDArray c('f', {M,N}, nd4j::DataType::DOUBLE);

	NDArray exp('f', {M,N}, {0.1, 0.3, 0.5, 2.5, 2.7, 2.9, 4.9, 5.1, 5.3, 7.3, 7.5, 7.7, 9.7, 9.9, 10.1}, nd4j::DataType::DOUBLE);

	nd4j::MmulHelper::mmulMxM<double,double,double>(&a, &b, &c, 1., 0.);	
	// c.printIndexedBuffer();

	ASSERT_TRUE(c.equalsTo(&exp));
}

//////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, mmulMxM_2) {

	const Nd4jLong M = 3;
	const Nd4jLong K = 4;
	const Nd4jLong N = 5;

	NDArray a('c', {M,K}, {1.2,1.1,1.0,0.9,0.8,0.7,0.5,0.4,0.3,0.2,0.1,0}, nd4j::DataType::DOUBLE);
	NDArray b('f', {K,N}, {1,-2,3,-4,5,-6,7,-8,9,-10,11,-12,13,-14,15,-16,17,-18,19,-20}, nd4j::DataType::DOUBLE);
	NDArray c('f', {M,N}, nd4j::DataType::DOUBLE);

	NDArray exp('f', {M,N}, {-1.6, -0.7, 0.2, -0.8, 0.1, 1., -0., 0.9, 1.8, 0.8, 1.7, 2.6, 1.6, 2.5, 3.4}, nd4j::DataType::DOUBLE);

	nd4j::MmulHelper::mmulMxM<double,double,double>(&a, &b, &c, 1., 0.);		

	ASSERT_TRUE(c.equalsTo(&exp));
}

//////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, mmulMxM_3) {

	const Nd4jLong M = 3;
	const Nd4jLong K = 4;
	const Nd4jLong N = 5;

	NDArray a('f', {M,K}, {1.2,1.1,1.0,0.9,0.8,0.7,0.5,0.4,0.3,0.2,0.1,0}, nd4j::DataType::DOUBLE);
	NDArray b('c', {K,N}, {1,-2,3,-4,5,-6,7,-8,9,-10,11,-12,13,-14,15,-16,17,-18,19,-20}, nd4j::DataType::DOUBLE);
	NDArray c('f', {M,N}, nd4j::DataType::DOUBLE);

	NDArray exp('f', {M,N}, {-1.9, -0.9, 0.1, 1.3, 0.3, -0.7, -0.7, 0.3, 1.3, 0.1, -0.9, -1.9, 0.5, 1.5, 2.5}, nd4j::DataType::DOUBLE);

	nd4j::MmulHelper::mmulMxM<double,double,double>(&a, &b, &c, 1., 0.);	

	ASSERT_TRUE(c.equalsTo(&exp));
}



