#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma (iuriish@yahoo.com)
//

#include <exceptions/cuda_exception.h>
#include <hipblas.h>
#include "../MmulHelper.h"
#include <specials_cuda.h>


namespace nd4j { 


//////////////////////////////////////////////////////////////////////////////
// MXK x KxN = MxN
// C array must be in f order
template <typename X, typename Y, typename Z>
static __global__ void usualCudaGemm(const bool transA, const bool transB, const int M, const int N, const int K, const double alpha, const void* vA, const int lda, const void* vB, const int ldb, const double beta, void* vC, const int ldc) {

    X* A = reinterpret_cast<X*>(const_cast<void*>(vA));
    Y* B = reinterpret_cast<Y*>(const_cast<void*>(vB));
    Z* C = reinterpret_cast<Z*>(vC);     

    __shared__ Z alphaZ, betaZ;
    __shared__ Nd4jLong strideArow, strideAcol, strideBrow, strideBcol;

    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row == 0 && col == 0) {        
        
        alphaZ = alpha;
        betaZ  = beta;

        if(transA) { strideArow = lda; strideAcol = 1; } else { strideArow = 1; strideAcol = lda; }
        if(transB) { strideBrow = ldb; strideBcol = 1; } else { strideBrow = 1; strideBcol = ldb; }
    }

    __syncthreads();

    Z val = 0;
    if (row < M && col < N)         
        for (int i = 0; i < K; i++)             
            val = val + A[row * strideArow + i * strideAcol] * B[i * strideBrow + col * strideBcol];
            
    C[row + col * ldc] = alphaZ * val + betaZ * C[row + col * ldc];    
}

////////////////////////////////////////////////////////////////////////
template <typename X, typename Y, typename Z>    
__host__ static void usualGemm(const dim3 &blocksPerGrid, const dim3 &threadsPerBlock, hipStream_t *stream, const bool transA, const bool transB, const int M, const int N, const int K, const double alpha, const void* vA, const int lda, const void* vB, const int ldb, const double beta, void* vC, const int ldc) {
    
    usualCudaGemm<X,Y,Z><<<blocksPerGrid, threadsPerBlock, 1024, *stream>>>(transA, transB, M, N, K, alpha, vA, lda, vB, ldb, beta, vC, ldc);
}

//////////////////////////////////////////////////////////////////////////////
// MXK x KxN = MxN
NDArray* MmulHelper::mmulMxM(const NDArray* A, const NDArray* B, NDArray* C, double alpha, double beta, const char outOrder) {

	if(A->rankOf() != 2)
		throw std::runtime_error("MmulHelper::mmulMxM cuda: rank of A array is not equal 2 !");
	if(B->rankOf() != 2)
		throw std::runtime_error("MmulHelper::mmulMxM cuda: rank of B array is not equal 2 !");
	if(C != nullptr && C->rankOf() != 2)
		throw std::runtime_error("MmulHelper::mmulMxM cuda: rank of C array is not equal 2 !");

	const auto M = A->sizeAt(0);
	const auto K = A->sizeAt(1);
	const auto N = B->sizeAt(1);

	if(B->sizeAt(0) != K)
		throw std::runtime_error("MmulHelper::mmulMxM cuda: B array has wrong number of rows !");
	if(C != nullptr && C->sizeAt(0) != M)
		throw std::runtime_error("MmulHelper::mmulMxM cuda: C array has wrong number of rows !");
	if(C != nullptr && C->sizeAt(1) != N)
		throw std::runtime_error("MmulHelper::mmulMxM cuda: C array has wrong number of columns !");

	if(C == nullptr) 		
        C = new NDArray(outOrder, {M,N}, DataTypeUtils::pickPairwiseResultType(A->dataType(), B->dataType()), A->getContext());

	NDArray *pA(const_cast<NDArray*>(A)), *pB(const_cast<NDArray*>(B)), *pC(const_cast<NDArray*>(C));        

    if(A->ews() != 1)
        pA = pA->dup('f');
    if(B->ews() != 1)
        pB = pB->dup('f');
    if(C->ews() != 1 || C->ordering() != 'f')
        pC = pC->dup('f');

    const auto aOrder = pA->ordering();
    const auto bOrder = pB->ordering();    

    const bool transA = aOrder != 'f';
    const bool transB = bOrder != 'f';
    
    const hipblasOperation_t transAblas = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    const hipblasOperation_t transBblas = transB ? HIPBLAS_OP_T : HIPBLAS_OP_N;

    const int lda = aOrder == 'f' ? M : K;
    const int ldb = bOrder == 'f' ? K : N;
    const int ldc = M; // cOrder == 'f' ? M : N;    

    const auto aType = pA->dataType();
    const auto bType = pB->dataType();
    const auto cType = pC->dataType();

    if(!pA->isActualOnDeviceSide())
        pA->syncToDevice();
    if(!pB->isActualOnDeviceSide())
        pB->syncToDevice();
    if(!pC->isActualOnDeviceSide())
        pC->syncToDevice();    

    hipblasStatus_t status;
    hipblasHandle_t handle;

    hipStream_t* stream = A->getContext()->getCudaStream();

    status = hipblasCreate(&handle); // initialize CUBLAS context
    if (status != HIPBLAS_STATUS_SUCCESS) throw cuda_exception::build("MmulHelper::mmulMxM cuda failed !", status);

    status = hipblasSetStream(handle, *stream);
    if (status != HIPBLAS_STATUS_SUCCESS) throw cuda_exception::build("MmulHelper::mmulMxM cuda failed !", status);

    const bool AB(aType == bType), AC(aType == cType), ABC(AB && AC);

    // choose appropriate cuda gemm api depending on data types    
    if(ABC && aType == DataType::DOUBLE) {
        status = hipblasDgemm(handle, transAblas, transBblas, M, N, K, &alpha, (double*)pA->getSpecialBuffer(), lda, (double*)pB->getSpecialBuffer(), ldb, &beta, (double*)pC->getSpecialBuffer(), ldc);
    }
    else if(ABC && aType == DataType::FLOAT32) {        
        float alphaF(alpha), betaF(beta);
        status = hipblasSgemm(handle, transAblas, transBblas, M, N, K, &alphaF, (float*)pA->getSpecialBuffer(), lda, (float*)pB->getSpecialBuffer(), ldb, &betaF, (float*)pC->getSpecialBuffer(), ldc);
    }
    else if(ABC && aType == DataType::HALF) {
        float16 alphaH(alpha), betaH(beta);
        status = hipblasHgemm(handle, transAblas, transBblas, M, N, K, &alphaH.data, (__half*)pA->getSpecialBuffer(), lda, (__half*)pB->getSpecialBuffer(), ldb, &betaH.data, (__half*)pC->getSpecialBuffer(), ldc);
    }       
    else if(AB && aType == DataType::INT8 && cType == DataType::FLOAT32) {            
           float alphaF(alpha), betaF(beta);
           status = cublasSgemmEx(handle, transAblas, transBblas, M, N, K, &alphaF, pA->getSpecialBuffer(), HIP_R_8I, lda, pB->getSpecialBuffer(), HIP_R_8I, ldb, &betaF, pC->getSpecialBuffer(), HIP_R_32F, ldc);
    }
    else if(AB && aType == DataType::HALF && cType == DataType::FLOAT32) {
        float alphaF(alpha), betaF(beta);
        status = cublasSgemmEx(handle, transAblas, transBblas, M, N, K, &alphaF, pA->getSpecialBuffer(), HIP_R_16F, lda, pB->getSpecialBuffer(), HIP_R_16F, ldb, &betaF, pC->getSpecialBuffer(), HIP_R_32F, ldc);
    }    
    else {        
        dim3 threadsPerBlock(N, M);
        dim3 blocksPerGrid(1, 1);
        if (M*N > 512){
            threadsPerBlock.x = threadsPerBlock.y = 512;             
            blocksPerGrid.x = math::nd4j_ceil<double, int>(static_cast<double>(N) / threadsPerBlock.x);    // cols
            blocksPerGrid.y = math::nd4j_ceil<double, int>(static_cast<double>(M) / threadsPerBlock.y);    // rows
        }

        BUILD_TRIPLE_SELECTOR(aType, bType, cType, usualGemm, (blocksPerGrid, threadsPerBlock, stream, transA, transB, M, N, K, alpha, pA->getSpecialBuffer(), lda, pB->getSpecialBuffer(), ldb, beta, pC->getSpecialBuffer(), ldc), LIBND4J_TYPES, FLOAT_TYPES, FLOAT_TYPES);
    }   
   
    if (status != HIPBLAS_STATUS_SUCCESS) throw cuda_exception::build("MmulHelper::mmulMxM cuda failed !", status);

    auto cudaResult = hipStreamSynchronize(*stream);
    if (cudaResult != 0) throw cuda_exception::build("MmulHelper::mmulMxM cuda failed !", cudaResult);
   
    hipblasDestroy(handle);    

    pA->tickReadDevice();
    pB->tickReadDevice();
    pC->tickWriteDevice();

    if(pC != C) {
    	C->assign(pC);
    	delete pC;
    }
    if(pA != A)
    	delete pA;
    if(pB != B)
    	delete pB;

	return C;
}

////////////////////////////////////////////////////////////////////////////
// static
// MXN x N = M
template <typename X, typename Y, typename Z>
NDArray* MmulHelper::mmulMxV(const NDArray* A, const NDArray* B, nd4j::NDArray* C, const double alpha, const double beta, const char outOrder) {

    int bLenDim, cLenDim;

    if(A->rankOf() != 2)
        throw std::runtime_error("MmulHelper::mmulMxV cuda: rank of A array is not equal 2 !");
    if(!B->isVector() && !shape::isCommonVector(B->getShapeInfo(), bLenDim))
        throw std::runtime_error("MmulHelper::mmulMxV cuda: B array must be vector !");
    if(C != nullptr && !C->isVector() && !shape::isCommonVector(C->getShapeInfo(), cLenDim))
        throw std::runtime_error("MmulHelper::mmulMxV cuda: C array must be vector !");

    const auto M = A->sizeAt(0);    
    const auto N = A->sizeAt(1);

    if(B->lengthOf() != N)
        throw std::runtime_error("MmulHelper::mmulMxV cuda: B vector has wrong length !");
    if(C != nullptr && C->lengthOf() != M)
        throw std::runtime_error("MmulHelper::mmulMxV cuda: C array has wrong length !");    

    if(C == nullptr)        
        C = new NDArray(outOrder, {M}, DataTypeUtils::pickPairwiseResultType(A->dataType(), B->dataType()), A->getContext());
    
    NDArray *pA(const_cast<NDArray*>(A));

    if(A->ews() != 1 || A->ordering() != 'f')
        pA = pA->dup('f');

    // const auto aOrder = pA->ordering();
    // const auto bOrder = pB->ordering();    

    // const bool transA = aOrder != 'f';
    // const bool transB = bOrder != 'f';
    
    // const hipblasOperation_t transAblas = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    // const hipblasOperation_t transBblas = transB ? HIPBLAS_OP_T : HIPBLAS_OP_N;

    // const int lda = aOrder == 'f' ? M : K;
    // const int ldb = bOrder == 'f' ? K : N;
    // const int ldc = M; // cOrder == 'f' ? M : N;    

    // const auto aType = pA->dataType();
    // const auto bType = pB->dataType();
    // const auto cType = pC->dataType();


    //     auto xType = A->dataType();
    //     auto yType = B->dataType();
    //     auto zType = result->dataType();

        // // TODO: strides!!!
        // if (xType == yType && xType == zType && BlasHelper::getInstance()->hasGEMV<X>()) {
        //     nd4j_debug("Using provided GEMV pointer\n","");
        //     auto layout = A->ordering() == 'f' ? CblasColMajor : CblasRowMajor;
        //     if (std::is_same<X, float>::value)
        //         BlasHelper::getInstance()->sgemv()(layout, CblasNoTrans, A->rows(), A->columns(), (float) alpha, reinterpret_cast<float *>(A->getBuffer()), layout == CblasColMajor ? A->rows() : A->columns(), reinterpret_cast<float *>(B->getBuffer()), 1, (float) beta, reinterpret_cast<float *>(result->getBuffer()), 1);
        //     else if (std::is_same<X, double>::value)
        //         BlasHelper::getInstance()->dgemv()(layout, CblasNoTrans, A->rows(), A->columns(), (double) alpha, reinterpret_cast<double *>(A->getBuffer()), layout == CblasColMajor ? A->rows() : A->columns(), reinterpret_cast<double *>(B->getBuffer()), 1, (double) beta, reinterpret_cast<double *>(result->getBuffer()), 1);
        //     else
        //         nd4j::blas::GEMV<X, Y, Z>::op(A->ordering() == 'f' ? CblasTrans : 0, A->rows(), A->columns(), alpha, A->getBuffer(), B->lengthOf(), B->getBuffer(), 1, beta, result->getBuffer(), 1);
        // } else {
        //     nd4j_debug("Using fallback GEMV impl\n","");
        //     nd4j::blas::GEMV<X, Y, Z>::op(A->ordering() == 'f' ? CblasTrans : 0, A->rows(), A->columns(), alpha, A->getBuffer(), B->lengthOf(), B->getBuffer(), 1, beta, result->getBuffer(), 1);
        // }
    return C;
}


BUILD_TRIPLE_TEMPLATE(template void usualGemm, (const dim3 &blocksPerGrid, const dim3 &threadsPerBlock, hipStream_t *stream, const bool transA, const bool transB, const int M, const int N, const int K, const double alpha, const void* vA, const int lda, const void* vB, const int ldb, const double beta, void* vC, const int ldc), LIBND4J_TYPES, FLOAT_TYPES, FLOAT_TYPES);
BUILD_TRIPLE_TEMPLATE(template NDArray* MmulHelper::mmulMxV, (const NDArray* A, const NDArray* B, NDArray* C, const double alpha, const double beta, const char outOrder), LIBND4J_TYPES, FLOAT_TYPES, FLOAT_TYPES);
}
