#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author sgazeos@gmail.com
//

#include <ops/declarable/helpers/fake_quantization.h>
#include <NDArrayFactory.h>

namespace nd4j {
namespace ops {
namespace helpers {
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// fakeQuantWithMinMaxVars_
// input - input tensor
// min - min scalar tensor
// max - max scalar tensor
// numBits - (default 16bit)
// narrowed - shrink is true
// output - output tensor
//
    template <typename T>
    static __host__ __device__ void Nudge(T min, T max, int quant_min, int quant_max, T* scale, T* nudged_min, T* nudged_max) {
        T quant_max_float = static_cast<T>(quant_max);
        T quant_min_float = static_cast<T>(quant_min);
        *scale = (max - min) / (quant_max_float - quant_min_float);
        auto zero_point_from_min = quant_min_float - min / *scale;
        uint16_t const nudged_zero_point = [zero_point_from_min, quant_min, quant_max, quant_max_float, quant_min_float] {
            if (zero_point_from_min < quant_min_float) {
                return static_cast<uint16_t>(quant_min);
            }
            if (zero_point_from_min > quant_max_float) {
                return static_cast<uint16_t>(quant_max);
            }
            return nd4j::math::nd4j_round<T,uint16_t>(zero_point_from_min);
        }();
        *nudged_min = (quant_min_float - nudged_zero_point) * (*scale);
        *nudged_max = (quant_max_float - nudged_zero_point) * (*scale);
    }

    template <typename T>
    void fakeQuantWithMinMaxVars_(NDArray* input, NDArray* min, NDArray* max, int numBits, bool narrowed, NDArray* output) {
        int lowIntBound = narrowed?1:0;
        int upperIntBound = (1 << numBits) - 1;
        min->syncToHost();
        max->syncToHost();
        T scale, nudged_min, nudged_max;
        Nudge(min->t<T>(0), max->t<T>(0), lowIntBound, upperIntBound, &scale, &nudged_min, &nudged_max);

        auto wiseMinMaxAndSoOn = LAMBDA_T(x, nudged_min, nudged_max, scale) {
            T val = x;
            if (x < nudged_min) {
                val = nudged_min;
            }
            else if (x > nudged_max) {
                val = nudged_max;
            }
            else
                val = x;
            return (math::nd4j_floor<T,T>((val - nudged_min) / scale + T(0.5)) * scale + nudged_min);
        };

        input->applyLambda(wiseMinMaxAndSoOn, output);
    }

    template <typename T>
    static __global__ void fakeQuantWithMinMaxKernel(T* input, Nd4jLong* inputShape, T* min, T* max,
            int lowIntBound, int upperIntBound, Nd4jLong channels,
            T* output, Nd4jLong* outputShape, Nd4jLong length) {

        for (auto i = blockIdx.x; i < (int)channels; i += gridDim.x) {
            T scale, nudged_min, nudged_max;
            Nudge(min[i], max[i], lowIntBound, upperIntBound, &scale, &nudged_min, &nudged_max);
            //auto wiseMinMaxAndSoOn = LAMBDA_T(x, nudged_min, nudged_max, scale) {
            for (auto e = threadIdx.x; e < (int)length; e += (int)channels) {
                T val = input[shape::getIndexOffset(e + i, inputShape)];
                if (val < nudged_min) {
                    val = nudged_min;
                } else if (val > nudged_max) {
                    val = nudged_max;
                }
                output[shape::getIndexOffset(e + i, outputShape)] = (math::nd4j_floor<T, T>((val - nudged_min) / scale + T(0.5)) * scale + nudged_min);
            };
        }

    }

    template <typename T>
    void fakeQuantWithMinMaxVarsPerChannel_(LaunchContext* context, NDArray* input, NDArray* min, NDArray* max, int numBits, bool narrowed, NDArray* output) {
        int lowIntBound = narrowed?1:0;
        int upperIntBound = (1 << numBits) - 1;
        auto channels = min->lengthOf();
        auto length = input->lengthOf();
        NDArray::prepareSpecialUse({output}, {min, max, input});
        auto stream = context->getCudaStream();
        T* inputBuf = input->dataBuffer()->specialAsT<T>();
        T* outputBuf = output->dataBuffer()->specialAsT<T>();
        T* minBuf = min->dataBuffer()->specialAsT<T>();
        T* maxBuf = max->dataBuffer()->specialAsT<T>();
        fakeQuantWithMinMaxKernel<<<1, 1, 256, *stream>>>(inputBuf, input->specialShapeInfo(),
                minBuf, maxBuf, lowIntBound, upperIntBound, channels, outputBuf, output->specialShapeInfo(), length);
        NDArray::registerSpecialUse({output}, {min, max, input});

    }

    void fakeQuantWithMinMaxVars(NDArray* input, NDArray* min, NDArray* max, int numBits, bool narrowed, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), fakeQuantWithMinMaxVars_, (input, min, max, numBits, narrowed, output), FLOAT_TYPES);
    }
    void fakeQuantWithMinMaxVarsPerChannel(LaunchContext* context, NDArray* input, NDArray* min, NDArray* max, int numBits, bool narrowed, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), fakeQuantWithMinMaxVarsPerChannel_, (context, input, min, max, numBits, narrowed, output), FLOAT_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template void fakeQuantWithMinMaxVars_, (NDArray* input, NDArray* min, NDArray* max, int numBits, bool narrowed, NDArray* output), FLOAT_TYPES);
    BUILD_SINGLE_TEMPLATE(template void fakeQuantWithMinMaxVarsPerChannel_, (LaunchContext* context, NDArray* input, NDArray* min, NDArray* max, int numBits, bool narrowed, NDArray* output), FLOAT_TYPES);

}
}
}
