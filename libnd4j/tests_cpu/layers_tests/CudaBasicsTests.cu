#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

 //
 // @author raver119@gmail.com
 //

#include "testlayers.h"
#include <NDArray.h>
#include <NDArrayFactory.h>
#include <Context.h>
#include <Node.h>
#include <graph/Variable.h>
#include <graph/VariableSpace.h>
#include <ops/declarable/CustomOperations.h>
#include <ops/declarable/generic/helpers/convolutions.h>
#include <ops/declarable/helpers/col2im.h>

#include <hip/hip_runtime.h>
#include <cuda_launch_config.h>

using namespace nd4j;
using namespace nd4j::graph;

class CudaBasicsTests : public testing::Test {
public:

};


//////////////////////////////////////////////////////////////////////////
static hipError_t prepareDataForCuda(hipStream_t& stream, void* reductionPointer, int* allocationPointer,
										std::vector<NDArray*>& arrs,
										std::vector<void*>& dBuffs,
										std::vector<Nd4jLong*>& dShapes,
										std::vector<int>& dimensions,
										int *dDimensions,
										std::vector<Nd4jLong*>& tadOnlyShapeInfo,
										std::vector<Nd4jLong*>& tadOffsets) { 
    

	hipError_t cudaResult;

    cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer),  1024*1024);		if(cudaResult != 0) return cudaResult;
	cudaResult = hipMalloc(reinterpret_cast<void **>(&allocationPointer), 1024*1024);		if(cudaResult != 0) return cudaResult;

	// allocating device memory for arrays	
	for(int i = 0; i < arrs.size(); ++i) {					

		cudaResult = hipMalloc(reinterpret_cast<void **>(&dBuffs[i]), arrs[i]->lengthOf() * arrs[i]->sizeOfT()); 				if(cudaResult != 0) return cudaResult;
		cudaResult = hipMalloc(reinterpret_cast<void **>(&dShapes[i]), shape::shapeInfoByteLength(arrs[i]->getShapeInfo()));   if(cudaResult != 0) return cudaResult;

		hipMemcpyAsync(dShapes[i], arrs[i]->getShapeInfo(), shape::shapeInfoByteLength(arrs[i]->getShapeInfo()), hipMemcpyHostToDevice, stream);
		if(i != arrs.size()-1)	// do not copy buffer for result last array
			hipMemcpyAsync(dBuffs[i], arrs[i]->buffer(), arrs[i]->lengthOf() * arrs[i]->sizeOfT(),  hipMemcpyHostToDevice, stream);			
	}
	
    // evaluating and allocating device memory for tad
    int dimensionsLength = dimensions.size();    
    for(int i = 0; i < tadOnlyShapeInfo.size(); ++i) {

    	shape::TAD tad(arrs[i]->getShapeInfo(), dimensions.data(), dimensionsLength);    	    
    	tad.createTadOnlyShapeInfo();
    	tad.createOffsets();

    	Nd4jLong* tadShapeInfo = tad.tadOnlyShapeInfo;
		Nd4jLong* tadSteps 	   = tad.tadOffsets;

		cudaResult = hipMalloc(reinterpret_cast<void **>(&tadOnlyShapeInfo[i]), shape::shapeInfoByteLength(tadShapeInfo));	if(cudaResult != 0) return cudaResult;
		cudaResult = hipMalloc(reinterpret_cast<void **>(&tadOffsets[i]), tad.numTads * sizeof(Nd4jLong));					if(cudaResult != 0) return cudaResult;

		hipMemcpyAsync(tadOnlyShapeInfo[i], tadShapeInfo, shape::shapeInfoByteLength(tadShapeInfo), hipMemcpyHostToDevice, stream);
		hipMemcpyAsync(tadOffsets[i], tadSteps, tad.numTads * sizeof(Nd4jLong), hipMemcpyHostToDevice, stream);
    }

    // allocate device memory for dimensions
    if(dimensionsLength != 0) {
		cudaResult = hipMalloc(reinterpret_cast<void **>(&dDimensions), dimensionsLength * sizeof(int));					if(cudaResult != 0) return cudaResult;
    	hipMemcpyAsync(dDimensions, dimensions.data(), dimensionsLength * sizeof(int), hipMemcpyHostToDevice, stream);
    }    

    int temp1[2];
    hipMemcpyAsync(temp1, dBuffs[0], arrs[0]->lengthOf() * arrs[0]->sizeOfT(), hipMemcpyDeviceToHost, stream);    
    for (int i = 0; i < 2; ++i)
    	printf("%i, ", temp1[i]);
    printf("\n");

    hipMemcpyAsync(temp1, dBuffs[1], arrs[1]->lengthOf() * arrs[1]->sizeOfT(), hipMemcpyDeviceToHost, stream);    
    for (int i = 0; i < 2; ++i)
    	printf("%i, ", temp1[i]);
    printf("\n");

	Nd4jLong temp2[8];
    hipMemcpyAsync(temp2, dShapes[0], shape::shapeInfoByteLength(arrs[0]->getShapeInfo()), hipMemcpyDeviceToHost, stream);    
    for (int i = 0; i < 8; ++i)
    	printf("%i, ", temp2[i]);
    printf("\n");

    hipMemcpyAsync(temp2, dShapes[1], shape::shapeInfoByteLength(arrs[1]->getShapeInfo()), hipMemcpyDeviceToHost, stream);
    for (int i = 0; i < 8; ++i)
    	printf("%i, ", temp2[i]);
    printf("\n");

    
    hipMemcpyAsync(temp1, dDimensions, dimensionsLength * sizeof(int), hipMemcpyDeviceToHost, stream);
  	for (int i = 0; i < 2; ++i)
    	printf("%i, ", temp1[i]);
    printf("\n");  

    hipMemcpyAsync(temp2, tadOnlyShapeInfo[0], 8 * sizeof(Nd4jLong), hipMemcpyDeviceToHost, stream);
  	for (int i = 0; i < 8; ++i)
    	printf("%i, ", temp2[i]);
    printf("\n");  

    hipMemcpyAsync(temp2, tadOnlyShapeInfo[1], 8 * sizeof(Nd4jLong), hipMemcpyDeviceToHost, stream);
  	for (int i = 0; i < 8; ++i)
    	printf("%i, ", temp2[i]);
    printf("\n");  
  
  	hipMemcpyAsync(temp2, tadOffsets[0], sizeof(Nd4jLong), hipMemcpyDeviceToHost, stream);
  	for (int i = 0; i < 1; ++i)
    	printf("%i, ", temp2[i]);
    printf("\n");  

    hipMemcpyAsync(temp2, tadOffsets[1], sizeof(Nd4jLong), hipMemcpyDeviceToHost, stream);
  	for (int i = 0; i < 1; ++i)
    	printf("%i, ", temp2[i]);
    printf("\n");  
    
	return cudaResult;
}


//////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, TestPairwise_1) {
	// allocating host-side arrays
	auto x = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
	auto z = NDArrayFactory::create<double>('c', { 5 });

	auto exp = NDArrayFactory::create<double>('c', { 5 }, { 2, 4, 6, 8, 10 });

	// making raw buffers
	Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
	hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
	ASSERT_EQ(0, res);
	res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
	ASSERT_EQ(0, res);
	res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
	ASSERT_EQ(0, res);

	Nd4jPointer nativeStream = (Nd4jPointer)malloc(sizeof(hipStream_t));
	CHECK_ALLOC(nativeStream, "Failed to allocate memory for new CUDA stream");
	hipError_t dZ = hipStreamCreate(reinterpret_cast<hipStream_t *>(&nativeStream));
	auto stream = reinterpret_cast<hipStream_t *>(&nativeStream);

	hipMemcpyAsync(devBufferPtrX, x.buffer(), x.lengthOf() * x.sizeOfT(), hipMemcpyHostToDevice, *stream);
	hipMemcpyAsync(devShapePtrX, x.shapeInfo(), shape::shapeInfoByteLength(x.shapeInfo()), hipMemcpyHostToDevice, *stream);
	
	LaunchContext lc(stream, nullptr, nullptr);
	NativeOpExecutioner::execPairwiseTransform(&lc, pairwise::Add, nullptr, x.shapeInfo(), devBufferPtrX, reinterpret_cast<Nd4jLong*>(devShapePtrX), nullptr, x.shapeInfo(), devBufferPtrX, reinterpret_cast<Nd4jLong*>(devShapePtrX), nullptr, z.shapeInfo(), devBufferPtrZ, reinterpret_cast<Nd4jLong*>(devShapePtrX), nullptr);
	res = hipStreamSynchronize(*stream);
	ASSERT_EQ(0, res);

	hipMemcpyAsync(z.buffer(), devBufferPtrZ, z.lengthOf() * x.sizeOfT(), hipMemcpyDeviceToHost, *stream);
	res = hipStreamSynchronize(*stream);
	ASSERT_EQ(0, res);

	hipFree(devBufferPtrX);
	hipFree(devBufferPtrZ);
	hipFree(devShapePtrX);

	for (int e = 0; e < z.lengthOf(); e++) {
		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
	}
}


////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execIndexReduceScalar_1) {

    NDArray x1('c', {2,2}, {0, 1, 2, 3}, nd4j::DataType::INT32);
    NDArray x2('c', {2,2}, {0.5, 1.5, -4.5, 3.5}, nd4j::DataType::BFLOAT16);    
    NDArray x3('c', {2,2}, {0, -1, 0, 1}, nd4j::DataType::BOOL);
    
    NDArray scalar(nd4j::DataType::INT64);

    NDArray exp1('c', {0}, {3}, nd4j::DataType::INT64);
    NDArray exp2('c', {0}, {2}, nd4j::DataType::INT64);
    NDArray exp3('c', {0}, {1}, nd4j::DataType::INT64);

    void *dX1, *dX2, *dX3, *dZ; 
    Nd4jLong *dX1ShapeInfo, *dX2ShapeInfo, *dX3ShapeInfo, *dZShapeInfo;

    hipError_t cudaResult;

    cudaResult = hipMalloc(reinterpret_cast<void **>(&dX1), x1.lengthOf() * x1.sizeOfT()); 		   		         	 ASSERT_EQ(0, cudaResult);
    cudaResult = hipMalloc(reinterpret_cast<void **>(&dX2), x2.lengthOf() * x2.sizeOfT()); 		   		         	 ASSERT_EQ(0, cudaResult);    
    cudaResult = hipMalloc(reinterpret_cast<void **>(&dX3), x3.lengthOf() * x3.sizeOfT()); 		   		         	 ASSERT_EQ(0, cudaResult);    
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dZ), scalar.lengthOf() * scalar.sizeOfT()); 				         ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dX1ShapeInfo), shape::shapeInfoByteLength(x1.getShapeInfo()));    ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dX2ShapeInfo), shape::shapeInfoByteLength(x2.getShapeInfo()));    ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dX3ShapeInfo), shape::shapeInfoByteLength(x3.getShapeInfo()));    ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dZShapeInfo), shape::shapeInfoByteLength(scalar.getShapeInfo())); ASSERT_EQ(0, cudaResult);	

    hipStream_t stream;
	cudaResult = hipStreamCreate(&stream); 
	ASSERT_EQ(0, cudaResult);
	
	hipMemcpyAsync(dX1, x1.buffer(), x1.lengthOf() * x1.sizeOfT(), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX2, x2.buffer(), x2.lengthOf() * x2.sizeOfT(), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX3, x3.buffer(), x3.lengthOf() * x3.sizeOfT(), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX1ShapeInfo, x1.getShapeInfo(), shape::shapeInfoByteLength(x1.getShapeInfo()), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX2ShapeInfo, x2.getShapeInfo(), shape::shapeInfoByteLength(x2.getShapeInfo()), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX3ShapeInfo, x3.getShapeInfo(), shape::shapeInfoByteLength(x3.getShapeInfo()), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dZShapeInfo, scalar.getShapeInfo(), shape::shapeInfoByteLength(scalar.getShapeInfo()), hipMemcpyHostToDevice, stream);
	
	void* reductionPointer = nullptr;
	cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer), 1024*1024);
	ASSERT_EQ(0, cudaResult);

	LaunchContext lc(&stream, reductionPointer);

	/***************************************/
	
    NativeOpExecutioner::execIndexReduceScalar(&lc, 
    											nd4j::indexreduce::IndexAbsoluteMax, 
    											x1.buffer(), x1.getShapeInfo(),
    	                                       	dX1, dX1ShapeInfo, 
    	                                       	nullptr, 
    	                                       	scalar.buffer(), scalar.getShapeInfo(),
    	                                       	dZ, dZShapeInfo);

    cudaResult = hipStreamSynchronize(stream); 
    ASSERT_EQ(0, cudaResult);

    hipMemcpyAsync(scalar.buffer(), dZ, scalar.lengthOf() * scalar.sizeOfT(), hipMemcpyDeviceToHost, stream);

    cudaResult = hipStreamSynchronize(stream); 
    ASSERT_EQ(0, cudaResult);

	ASSERT_NEAR(exp1.e<float>(0), scalar.e<float>(0), 1e-5);

    /***************************************/
    
    NativeOpExecutioner::execIndexReduceScalar(&lc,
    											nd4j::indexreduce::IndexAbsoluteMax, 
    											nullptr, x2.getShapeInfo(),
    	                                       	dX2, dX2ShapeInfo, 
    	                                       	nullptr, 
    	                                       	nullptr, scalar.getShapeInfo(),
    	                                       	dZ, dZShapeInfo);

    cudaResult = hipStreamSynchronize(stream); 
    ASSERT_EQ(0, cudaResult);

    hipMemcpyAsync(scalar.buffer(), dZ, scalar.lengthOf() * scalar.sizeOfT(), hipMemcpyDeviceToHost, stream);

    cudaResult = hipStreamSynchronize(stream); 
    ASSERT_EQ(0, cudaResult);

    ASSERT_NEAR(exp2.e<float>(0), scalar.e<float>(0), 1e-5);

    // *************************************

    NativeOpExecutioner::execIndexReduceScalar(&lc, 
    											nd4j::indexreduce::IndexAbsoluteMax, 
    											nullptr, x3.getShapeInfo(),
    	                                       	dX3, dX3ShapeInfo, 
    	                                       	nullptr, 
    	                                       	nullptr, scalar.getShapeInfo(),
    	                                       	dZ, dZShapeInfo);

    cudaResult = hipStreamSynchronize(stream); 
    ASSERT_EQ(0, cudaResult);

    hipMemcpyAsync(scalar.buffer(), dZ, scalar.lengthOf() * scalar.sizeOfT(), hipMemcpyDeviceToHost, stream);

    cudaResult = hipStreamSynchronize(stream); 
    ASSERT_EQ(0, cudaResult);

    ASSERT_NEAR(exp3.e<float>(0), scalar.e<float>(0), 1e-5);
    
	/***************************************/

	hipFree(dX1); 			hipFree(dX2); 			hipFree(dX3); 			hipFree(dZ);
	hipFree(dX1ShapeInfo); hipFree(dX2ShapeInfo); hipFree(dX3ShapeInfo); hipFree(dZShapeInfo); 

	/***************************************/	

	cudaResult = hipStreamDestroy(stream); 
	ASSERT_EQ(0, cudaResult);
	
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execReduce3Scalar_1) {

    NDArray x1('c', {2,2}, {1,2,3,4}, nd4j::DataType::INT32);
    NDArray x2('c', {2,2}, {-1,-2,-3,-4}, nd4j::DataType::INT32);
    NDArray x3('c', {2,2}, {1.5,1.5,1.5,1.5}, nd4j::DataType::DOUBLE);
    NDArray x4('c', {2,2}, {1,2,3,4}, nd4j::DataType::DOUBLE);
    NDArray exp1('c', {0}, {-30}, nd4j::DataType::FLOAT32);
    NDArray exp2('c', {0}, {15}, nd4j::DataType::DOUBLE);
    
	NDArray scalar1('c', {0}, nd4j::DataType::FLOAT32);
    NDArray scalar2('c', {0}, nd4j::DataType::DOUBLE);

    void *dX1, *dX2, *dX3, *dX4, *dZ1, *dZ2; 
    Nd4jLong *dX1ShapeInfo, *dX3ShapeInfo, *dZ1ShapeInfo, *dZ2ShapeInfo;

    hipError_t cudaResult;

    cudaResult = hipMalloc(reinterpret_cast<void **>(&dX1), x1.lengthOf() * x1.sizeOfT()); 		   		         	 	ASSERT_EQ(0, cudaResult);
    cudaResult = hipMalloc(reinterpret_cast<void **>(&dX2), x2.lengthOf() * x2.sizeOfT()); 		   		         	 	ASSERT_EQ(0, cudaResult);
    cudaResult = hipMalloc(reinterpret_cast<void **>(&dX3), x3.lengthOf() * x3.sizeOfT()); 		   		         	 	ASSERT_EQ(0, cudaResult);
    cudaResult = hipMalloc(reinterpret_cast<void **>(&dX4), x4.lengthOf() * x4.sizeOfT()); 		   		         	 	ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dZ1), scalar1.lengthOf() * scalar1.sizeOfT());			         	ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dZ2), scalar2.lengthOf() * scalar2.sizeOfT());			         	ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dX1ShapeInfo), shape::shapeInfoByteLength(x1.getShapeInfo()));    	ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dX3ShapeInfo), shape::shapeInfoByteLength(x3.getShapeInfo()));    	ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dZ1ShapeInfo), shape::shapeInfoByteLength(scalar1.getShapeInfo())); 	ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dZ2ShapeInfo), shape::shapeInfoByteLength(scalar2.getShapeInfo())); 	ASSERT_EQ(0, cudaResult);

    hipStream_t stream;
	cudaResult = hipStreamCreate(&stream); 
	ASSERT_EQ(0, cudaResult);
	
	hipMemcpyAsync(dX1, x1.buffer(), x1.lengthOf() * x1.sizeOfT(), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX2, x2.buffer(), x2.lengthOf() * x2.sizeOfT(), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX3, x3.buffer(), x3.lengthOf() * x3.sizeOfT(), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX4, x4.buffer(), x4.lengthOf() * x4.sizeOfT(), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX1ShapeInfo, x1.getShapeInfo(), shape::shapeInfoByteLength(x1.getShapeInfo()), hipMemcpyHostToDevice, stream);	
	hipMemcpyAsync(dX3ShapeInfo, x3.getShapeInfo(), shape::shapeInfoByteLength(x3.getShapeInfo()), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dZ1ShapeInfo, scalar1.getShapeInfo(), shape::shapeInfoByteLength(scalar1.getShapeInfo()), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dZ2ShapeInfo, scalar2.getShapeInfo(), shape::shapeInfoByteLength(scalar2.getShapeInfo()), hipMemcpyHostToDevice, stream);

	/***************************************/

	void* reductionPointer  = nullptr;
	int*  allocationPointer = nullptr;	

	cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer),  1024*1024);		ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&allocationPointer), 1024*1024);		ASSERT_EQ(0, cudaResult);

	LaunchContext lc(&stream, reductionPointer, nullptr, allocationPointer);

	/***************************************/
	
    NativeOpExecutioner::execReduce3Scalar(&lc, nd4j::reduce3::Dot,nullptr, x1.getShapeInfo(),dX1, dX1ShapeInfo, nullptr, nullptr, x2.getShapeInfo(),dX2, dX1ShapeInfo,nullptr, scalar1.getShapeInfo(),dZ1, dZ1ShapeInfo);

    cudaResult = hipStreamSynchronize(stream);     
    ASSERT_EQ(0, cudaResult);

    hipMemcpyAsync(scalar1.buffer(), dZ1, scalar1.lengthOf() * scalar1.sizeOfT(), hipMemcpyDeviceToHost, stream);

    cudaResult = hipStreamSynchronize(stream); 
    ASSERT_EQ(0, cudaResult);

	ASSERT_NEAR(exp1.e<float>(0), scalar1.e<float>(0), 1e-5);

    /***************************************/
    
    NativeOpExecutioner::execReduce3Scalar(&lc, nd4j::reduce3::Dot,nullptr, x3.getShapeInfo(),dX3, dX3ShapeInfo, nullptr, nullptr, x4.getShapeInfo(),dX4, dX3ShapeInfo,nullptr, scalar2.getShapeInfo(),dZ2, dZ2ShapeInfo);

    cudaResult = hipStreamSynchronize(stream); 
    ASSERT_EQ(0, cudaResult);

    hipMemcpyAsync(scalar2.buffer(), dZ2, scalar2.lengthOf() * scalar2.sizeOfT(), hipMemcpyDeviceToHost, stream);

    cudaResult = hipStreamSynchronize(stream); 
    ASSERT_EQ(0, cudaResult);

	ASSERT_NEAR(exp2.e<float>(0), scalar2.e<float>(0), 1e-5);
    
	/***************************************/

	hipFree(dX1); 			hipFree(dX2); hipFree(dX3); 		   hipFree(dX4); 	hipFree(dZ1); 				hipFree(dZ2);
	hipFree(dX1ShapeInfo); 			   hipFree(dX3ShapeInfo); 					hipFree(dZ1ShapeInfo);		hipFree(dZ2ShapeInfo);

	/***************************************/	

	cudaResult = hipStreamDestroy(stream); 
	ASSERT_EQ(0, cudaResult);
}
 

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execReduce3_1) {

    NDArray x('c', {2,2}, {1,2,3,4}, nd4j::DataType::INT32);
    NDArray y('c', {2,2}, {-1,-2,-3,-4}, nd4j::DataType::INT32);

    NDArray exp('c', {0}, {-30}, nd4j::DataType::FLOAT32);
    NDArray z('c', {0},   nd4j::DataType::FLOAT32);
        
    void *reductionPointer;
	int *dDimensions, *allocationPointer;	

	std::vector<NDArray*> arrs = {&x,&y,&z};
	std::vector<void*> dBuffs(3, nullptr);
	std::vector<Nd4jLong*> dShapes(3, nullptr);
	std::vector<int> dimensions = {0,1};
	std::vector<Nd4jLong*> tadOnlyShapeInfo(2, nullptr);
	std::vector<Nd4jLong*> tadOffsets(2, nullptr);

	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);
	ASSERT_EQ(0, cudaResult);

	cudaResult = prepareDataForCuda(stream, reductionPointer, allocationPointer, arrs, dBuffs, dShapes, dimensions, dDimensions, tadOnlyShapeInfo, tadOffsets);
	ASSERT_EQ(0, cudaResult);		

	LaunchContext lc(&stream, reductionPointer, nullptr, allocationPointer);   

	/***************************************/
	NativeOpExecutioner::execReduce3(&lc, nd4j::reduce3::Dot, 
									nullptr, x.getShapeInfo(), dBuffs[0], dShapes[0], 
									nullptr, 
									nullptr, y.getShapeInfo(), dBuffs[1], dShapes[1], 
									nullptr, z.getShapeInfo(), dBuffs[2], dShapes[2], 
									dDimensions, 2, 
									tadOnlyShapeInfo[0], tadOffsets[0], tadOnlyShapeInfo[1], tadOffsets[1]);

	cudaResult = hipStreamSynchronize(stream);
    ASSERT_EQ(0, cudaResult);
    hipMemcpyAsync(z.buffer(), dBuffs[0], z.lengthOf() * z.sizeOfT(), hipMemcpyDeviceToHost, stream);

    cudaResult = hipStreamSynchronize(stream);
    ASSERT_EQ(0, cudaResult);
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	/***************************************/
	for(int i = 0; i < dBuffs.size(); ++i) 			 { hipFree(dBuffs[i]); hipFree(dShapes[i]);	}
	for(int i = 0; i < tadOnlyShapeInfo.size(); ++i) { hipFree(tadOnlyShapeInfo[i]); hipFree(tadOffsets[i]); }
	hipFree(dDimensions);

	/***************************************/	

	cudaResult = hipStreamDestroy(stream); 
	ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execReduce3_2) {

	NDArray x('c', {2,2}, {1.5,1.5,1.5,1.5}, nd4j::DataType::DOUBLE);
    NDArray y('c', {2,2}, {1,2,3,4}, nd4j::DataType::DOUBLE);

    NDArray exp('c', {0}, {15}, nd4j::DataType::DOUBLE);
    NDArray z('c', {0},   nd4j::DataType::DOUBLE);
        
    void *reductionPointer;
	int *dDimensions, *allocationPointer;	

	std::vector<NDArray*> arrs = {&x,&y,&z};
	std::vector<void*> dBuffs(3, nullptr);
	std::vector<Nd4jLong*> dShapes(3, nullptr);
	std::vector<int> dimensions = {0,1};
	std::vector<Nd4jLong*> tadOnlyShapeInfo(2, nullptr);
	std::vector<Nd4jLong*> tadOffsets(2, nullptr);

	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);
	ASSERT_EQ(0, cudaResult);

	cudaResult = prepareDataForCuda(stream, reductionPointer, allocationPointer, arrs, dBuffs, dShapes, dimensions, dDimensions, tadOnlyShapeInfo, tadOffsets);
	ASSERT_EQ(0, cudaResult);		

	LaunchContext lc(&stream, reductionPointer, nullptr, allocationPointer);   

	/***************************************/
	NativeOpExecutioner::execReduce3(&lc, nd4j::reduce3::Dot, 
									nullptr, x.getShapeInfo(), dBuffs[0], dShapes[0], 
									nullptr, 
									nullptr, y.getShapeInfo(), dBuffs[1], dShapes[1], 
									nullptr, z.getShapeInfo(), dBuffs[2], dShapes[2], 
									dDimensions, 2, 
									tadOnlyShapeInfo[0], tadOffsets[0], tadOnlyShapeInfo[1], tadOffsets[1]);

	cudaResult = hipStreamSynchronize(stream);
    ASSERT_EQ(0, cudaResult);
    hipMemcpyAsync(z.buffer(), dBuffs[0], z.lengthOf() * z.sizeOfT(), hipMemcpyDeviceToHost, stream);

    cudaResult = hipStreamSynchronize(stream);
    ASSERT_EQ(0, cudaResult);
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	/***************************************/
	for(int i = 0; i < dBuffs.size(); ++i) 			 { hipFree(dBuffs[i]); hipFree(dShapes[i]);	}
	for(int i = 0; i < tadOnlyShapeInfo.size(); ++i) { hipFree(tadOnlyShapeInfo[i]); hipFree(tadOffsets[i]); }
	hipFree(dDimensions);

	/***************************************/	

	cudaResult = hipStreamDestroy(stream); 
	ASSERT_EQ(0, cudaResult);
}

////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execReduce3_3) {

	NDArray x('c', {2,3}, {1,2,3,4,5,6}, nd4j::DataType::INT32);
    NDArray y('c', {2,3}, {-6,-5,-4,-3,-2,-1}, nd4j::DataType::INT32);        

    NDArray exp('c', {3}, {-18,-20,-18}, nd4j::DataType::FLOAT32);
    NDArray z('c', {3}, nd4j::DataType::FLOAT32);
        
    void *reductionPointer;
	int *dDimensions, *allocationPointer;	

	std::vector<NDArray*> arrs = {&x,&y,&z};
	std::vector<void*> dBuffs(3, nullptr);
	std::vector<Nd4jLong*> dShapes(3, nullptr);
	std::vector<int> dimensions = {0};
	std::vector<Nd4jLong*> tadOnlyShapeInfo(2, nullptr);
	std::vector<Nd4jLong*> tadOffsets(2, nullptr);

	hipError_t cudaResult;
	hipStream_t stream;
	cudaResult = hipStreamCreate(&stream);
	ASSERT_EQ(0, cudaResult);

	cudaResult = prepareDataForCuda(stream, reductionPointer, allocationPointer, arrs, dBuffs, dShapes, dimensions, dDimensions, tadOnlyShapeInfo, tadOffsets);
	ASSERT_EQ(0, cudaResult);			

	LaunchContext lc(&stream, reductionPointer, nullptr, allocationPointer);   

	/***************************************/
	NativeOpExecutioner::execReduce3(&lc, nd4j::reduce3::Dot, 
									nullptr, x.getShapeInfo(), dBuffs[0], dShapes[0], 
									nullptr, 
									nullptr, y.getShapeInfo(), dBuffs[1], dShapes[1], 
									nullptr, z.getShapeInfo(), dBuffs[2], dShapes[2], 
									dDimensions, 2, 
									tadOnlyShapeInfo[0], tadOffsets[0], tadOnlyShapeInfo[1], tadOffsets[1]);

	cudaResult = hipStreamSynchronize(stream);
    ASSERT_EQ(0, cudaResult);
    hipMemcpyAsync(z.buffer(), dBuffs[0], z.lengthOf() * z.sizeOfT(), hipMemcpyDeviceToHost, stream);

    cudaResult = hipStreamSynchronize(stream);
    ASSERT_EQ(0, cudaResult);
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) 
 		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	/***************************************/

 	/***************************************/
	NativeOpExecutioner::execReduce3(&lc, nd4j::reduce3::Dot, 
									nullptr, x.getShapeInfo(), dBuffs[0], dShapes[0], 
									nullptr, 
									nullptr, y.getShapeInfo(), dBuffs[1], dShapes[1], 
									nullptr, z.getShapeInfo(), dBuffs[2], dShapes[2], 
									dDimensions, 2, 
									tadOnlyShapeInfo[0], tadOffsets[0], tadOnlyShapeInfo[1], tadOffsets[1]);

	cudaResult = hipStreamSynchronize(stream);
    ASSERT_EQ(0, cudaResult);
    hipMemcpyAsync(z.buffer(), dBuffs[0], z.lengthOf() * z.sizeOfT(), hipMemcpyDeviceToHost, stream);

    cudaResult = hipStreamSynchronize(stream);
    ASSERT_EQ(0, cudaResult);
 	
 	// verify results
 	for (int e = 0; e < z.lengthOf(); e++) ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);

	/***************************************/
	for(int i = 0; i < dBuffs.size(); ++i) 			 { hipFree(dBuffs[i]); hipFree(dShapes[i]);	}
	for(int i = 0; i < tadOnlyShapeInfo.size(); ++i) { hipFree(tadOnlyShapeInfo[i]); hipFree(tadOffsets[i]); }
	hipFree(dDimensions);

	/***************************************/	

	cudaResult = hipStreamDestroy(stream); 
	ASSERT_EQ(0, cudaResult);
}

// ////////////////////////////////////////////////////////////////////////////
// TEST_F(CudaBasicsTests, execReduce3_1) {

//     NDArray x1('c', {2,2}, {1,2,3,4}, nd4j::DataType::INT32);
//     NDArray x2('c', {2,2}, {-1,-2,-3,-4}, nd4j::DataType::INT32);    
//     NDArray x3('c', {2,2}, {1.5,1.5,1.5,1.5}, nd4j::DataType::DOUBLE);
//     NDArray x4('c', {2,2}, {1,2,3,4}, nd4j::DataType::DOUBLE);
    
//     NDArray x5('c', {2,3}, {1,2,3,4,5,6}, nd4j::DataType::INT32);
//     NDArray x6('c', {2,3}, {-6,-5,-4,-3,-2,-1}, nd4j::DataType::INT32);    
    
//     NDArray x7('c', {2,3}, {1.5,1.5,1.5,1.5,1.5,1.5}, nd4j::DataType::DOUBLE);
//     NDArray x8('c', {2,3}, {1,2,3,4,5,6}, nd4j::DataType::DOUBLE);
//     NDArray x9('c', {2,2,3}, {1.5,1.5,1.5,1.5,1.5,1.5,1.5,1.5,1.5,1.5,1.5,1.5}, nd4j::DataType::FLOAT32);
//     NDArray x10('c', {2,2,3}, {1,2,3,4,5,6,7,8,9,10,11,12}, nd4j::DataType::FLOAT32);

//     NDArray exp1('c', {0}, {-30}, nd4j::DataType::FLOAT32);
//     NDArray exp2('c', {0}, {15}, nd4j::DataType::DOUBLE);
//     NDArray exp3('c', {3}, {-18,-20,-18}, nd4j::DataType::FLOAT32);
//     NDArray exp4('c', {2}, {-28,-28}, nd4j::DataType::FLOAT32);
//     NDArray exp5('c', {3}, {7.5,10.5,13.5}, nd4j::DataType::DOUBLE);
//     NDArray exp6('c', {2}, {9,22.5}, nd4j::DataType::DOUBLE);        
//     NDArray exp7('c', {2,3}, {7.5, 10.5, 13.5, 25.5, 28.5, 31.5}, nd4j::DataType::FLOAT32);

//     NDArray res1('c', {0},   nd4j::DataType::FLOAT32);
//     NDArray res2('c', {0},   nd4j::DataType::FLOAT32);
//     NDArray res3('c', {0},   nd4j::DataType::DOUBLE);
//     NDArray res4('c', {0},   nd4j::DataType::DOUBLE);
//     NDArray res5('c', {3},   nd4j::DataType::FLOAT32);
//     NDArray res6('c', {2},   nd4j::DataType::FLOAT32);
//     NDArray res7('c', {3},   nd4j::DataType::DOUBLE);
//     NDArray res8('c', {2},   nd4j::DataType::DOUBLE);    
//     NDArray res9('c', {2,3}, nd4j::DataType::FLOAT32);
//     NDArray res10('c', {2,3}, nd4j::DataType::FLOAT32);
   
//    	const int N = 10;
// 	NDArray x[N] = {x1, x2, x3, x4, x5, x6, x7, x8, x9, x10};
// 	NDArray exp[] = {exp1, exp1, exp2, exp2, exp3, exp4, exp5, exp6, exp7, exp7};
// 	NDArray res[] = {res1, res2, res3, res4, res5, res6, res7, res8, res9, res10};
// 	std::vector<std::vector<int>> dimensions = {{0,1},{0,1},  {0,1},{0,1},  {0},{1},   {0},{1},   {1},{1}};

//     for(int i = 2; i < 3; ++i) {
		
// 		void *dX1, *dX2, *dZ1, *dZ2;    	
// 		int *dDimensions, int 
// 		Nd4jLong *dXShapeInfo, *dZ1ShapeInfo, *dZ2ShapeInfo;

// 		hipError_t cudaResult;

//     	cudaResult = hipMalloc(reinterpret_cast<void **>(&dX1), x[2*i].lengthOf()     * x[2*i].sizeOfT()); 		   		    	ASSERT_EQ(0, cudaResult);
//     	cudaResult = hipMalloc(reinterpret_cast<void **>(&dX2), x[2*i+1].lengthOf()   * x[2*i+1].sizeOfT()); 		   				ASSERT_EQ(0, cudaResult);
//     	cudaResult = hipMalloc(reinterpret_cast<void **>(&dZ1), res[2*i].lengthOf()   * res[2*i].sizeOfT());			        	ASSERT_EQ(0, cudaResult);
//     	cudaResult = hipMalloc(reinterpret_cast<void **>(&dZ2), res[2*i+1].lengthOf() * res[2*i+1].sizeOfT());			     		ASSERT_EQ(0, cudaResult);
// 		cudaResult = hipMalloc(reinterpret_cast<void **>(&dXShapeInfo), shape::shapeInfoByteLength(x[2*i].getShapeInfo()));   		ASSERT_EQ(0, cudaResult);
// 		cudaResult = hipMalloc(reinterpret_cast<void **>(&dZ1ShapeInfo), shape::shapeInfoByteLength(res[2*i].getShapeInfo())); 	ASSERT_EQ(0, cudaResult);
// 		cudaResult = hipMalloc(reinterpret_cast<void **>(&dZ2ShapeInfo), shape::shapeInfoByteLength(res[2*i+1].getShapeInfo()));	ASSERT_EQ(0, cudaResult);
// 		cudaResult = hipMalloc(reinterpret_cast<void **>(&dDimensions), shape::shapeInfoByteLength(res[2*i+1].getShapeInfo()));	ASSERT_EQ(0, cudaResult);

// 		hipStream_t stream;
// 		cudaResult = hipStreamCreate(&stream); 		
// 		ASSERT_EQ(0, cudaResult);

// 		hipMemcpyAsync(dX1, x[2*i].buffer(),   x[2*i].lengthOf()   * x[2*i].sizeOfT(),   hipMemcpyHostToDevice, stream);
// 		hipMemcpyAsync(dX2, x[2*i+1].buffer(), x[2*i+1].lengthOf() * x[2*i+1].sizeOfT(), hipMemcpyHostToDevice, stream);
// 		hipMemcpyAsync(dXShapeInfo, x[2*i].getShapeInfo(), shape::shapeInfoByteLength(x[2*i].getShapeInfo()), hipMemcpyHostToDevice, stream);	
// 		hipMemcpyAsync(dZ1ShapeInfo, res[2*i].getShapeInfo(), shape::shapeInfoByteLength(res[2*i].getShapeInfo()), hipMemcpyHostToDevice, stream);
// 		hipMemcpyAsync(dZ2ShapeInfo, res[2*i+1].getShapeInfo(), shape::shapeInfoByteLength(res[2*i+1].getShapeInfo()), hipMemcpyHostToDevice, stream);
		
// 		void*  reductionPointer = nullptr;
// 		int*   allocationPointer = nullptr;		
// 		cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer), 1024*1024);		ASSERT_EQ(0, cudaResult);
// 		cudaResult = hipMalloc(reinterpret_cast<void **>(&allocationPointer), 1024*1024);		ASSERT_EQ(0, cudaResult);
		

// 		LaunchContext lc(&stream, reductionPointer, nullptr, allocationPointer);

// 		NativeOpExecutioner::execReduce3(&lc, 
//     								nd4j::reduce3::Dot,
//     								nullptr, x[2*i].getShapeInfo(),
//     	                            dX1, dXShapeInfo, 
//     	                            nullptr, 
//     	                            nullptr, x[2*i+1].getShapeInfo(),
//     	                            dX2, dXShapeInfo,
//     	                            nullptr, res[2*i].getShapeInfo(),
//     	                            dZ1, dZ1ShapeInfo,
//     	                            dimensions[2*i].data(), dimensions[2*i].size());

// 		cudaResult = hipStreamSynchronize(stream);
//     	ASSERT_EQ(0, cudaResult);
//     	hipMemcpyAsync(res[2*i].buffer(), dZ1, res[2*i].lengthOf() * res[2*i].sizeOfT(), hipMemcpyDeviceToHost, stream);

//     	cudaResult = hipStreamSynchronize(stream);
//     	ASSERT_EQ(0, cudaResult);
 		
//  		for (int e = 0; e < res[2*i].lengthOf(); e++)
// 			ASSERT_NEAR(exp[2*i].e<double>(e), res[2*i].e<double>(e), 1e-5);

// 		/***************************************/

// 		NativeOpExecutioner::execReduce3(&lc, 
//     								nd4j::reduce3::Dot,
//     								nullptr, x[2*i+1].getShapeInfo(),
//     	                            dX2, dXShapeInfo, 
//     	                            nullptr, 
//     	                            nullptr, x[2*i].getShapeInfo(),
//     	                            dX1, dXShapeInfo,    	                            
//     	                            nullptr, res[2*i+1].getShapeInfo(),
//     	                            dZ2, dZ2ShapeInfo,
//     	                            dimensions[2*i+1].data(), dimensions[2*i+1].size());

// 		cudaResult = hipStreamSynchronize(stream);     
//     	ASSERT_EQ(0, cudaResult);
//     	hipMemcpyAsync(res[2*i+1].buffer(), dZ2, res[2*i+1].lengthOf() * res[2*i+1].sizeOfT(), hipMemcpyDeviceToHost, stream);

//     	cudaResult = hipStreamSynchronize(stream); 
//     	ASSERT_EQ(0, cudaResult);
 		
//  		for (int e = 0; e < res[2*i+1].lengthOf(); e++)
// 			ASSERT_NEAR(exp[2*i+1].e<double>(e), res[2*i+1].e<double>(e), 1e-5);

// 		/***************************************/
// 		hipFree(dX1); 			hipFree(dX2);  hipFree(dZ1); 			hipFree(dZ2);
// 		hipFree(dXShapeInfo);					hipFree(dZ1ShapeInfo);	hipFree(dZ2ShapeInfo);

// 		/***************************************/	

// 		cudaResult = hipStreamDestroy(stream); 
// 		ASSERT_EQ(0, cudaResult);
//     }
// }
