#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

#ifndef NDARRAY_CPP
#define NDARRAY_CPP

#include "../NDArray.h"
#include "../NDArrayFactory.h"
#include "NativeOpExecutioner.h"
#include <memory/Workspace.h>
#include <memory/MemoryRegistrator.h>
#include <ops.h>
#include <ops/gemm.h>
#include <pointercast.h>
#include <stdexcept>
#include <memory>
#include <helpers/logger.h>
#include <loops/pairwise_transform.h>
#include <loops/transform_same.h>
#include <loops/random.h>
#include <loops/broadcasting.h>
#include <indexing/NDIndex.h>
#include <indexing/IndicesList.h>
#include <helpers/ShapeUtils.h>
#include <sstream>
#include <helpers/ArrayUtils.h>
#include <MmulHelper.h>
#include <helpers/threshold.h>
#include <graph/exceptions/datatype_exception.h>

#include "../NDArray.hpp"

namespace nd4j {

    void* NDArray::operator new(size_t i) {
        if (nd4j::memory::MemoryRegistrator::getInstance()->hasWorkspaceAttached()) {
            nd4j::memory::Workspace* ws = nd4j::memory::MemoryRegistrator::getInstance()->getWorkspace();

            return ws->allocateBytes((Nd4jLong) i);
        } else {
            auto p = malloc(i);
            
            CHECK_ALLOC(p, "Failed to allocate new NDArray");

            return p;
        }
    }

    void NDArray::operator delete(void* p) {
        if (!nd4j::memory::MemoryRegistrator::getInstance()->hasWorkspaceAttached()) {
            free(p);
        }
    }


////////////////////////////////////////////////////////////////////////
// copy constructor
NDArray::NDArray(const NDArray& other) {

    _length = other._length;
    _context = other._context;
    _dataType = other._dataType;
    if (other._isBuffAlloc)
        ALLOCATE(_buffer, other._context->getWorkspace(), _length * other.sizeOfT(), int8_t);
    hipMalloc(&_bufferD, _length * other.sizeOfT());
    _shapeInfo = ShapeBuilders::copyShapeInfo(other._shapeInfo, false, _context->getWorkspace());
    hipMalloc(&_shapeInfoD, shape::shapeInfoByteLength(_shapeInfo));
    syncShape();
    _isBuffAlloc = other._isBuffAlloc;
    _isShapeAlloc = true;

    this->assign(&other);
}

////////////////////////////////////////////////////////////////////////
// do not allocate memory, memory for array is passed from outside
NDArray::NDArray(void *buffer, Nd4jLong *shapeInfo, graph::LaunchContext* context, const bool isBuffAlloc, const bool isShapeAlloc) {
    _shapeInfo = shapeInfo;
    _isBuffAlloc = isBuffAlloc;                                  // indicate that memory for array is passed from outside
    _isShapeAlloc = isShapeAlloc;
    _context = context == nullptr ? nd4j::graph::LaunchContext::defaultContext() : context;
    if (shapeInfo != nullptr) {
        _length = shape::length(shapeInfo);
        _dataType = ArrayOptions::dataType(shapeInfo);
    } else
        throw std::runtime_error("NDArray can't be initalized without shapeinfo");
    if (_isBuffAlloc)
        _buffer = reinterpret_cast<int8_t *>(buffer);

    hipMalloc(&_bufferD, _length * sizeOfT());
    hipMalloc(&_shapeInfoD, shape::shapeInfoByteLength(_shapeInfo));
    syncShape();
    hipMemcpy(_bufferD, buffer, _length * sizeOfT(), hipMemcpyHostToDevice);
}

////////////////////////////////////////////////////////////////////////
NDArray::NDArray(const char order, const std::vector<Nd4jLong> &shape, nd4j::DataType dtype, nd4j::graph::LaunchContext* context) {

    if ((int) shape.size() > MAX_RANK)
        throw std::invalid_argument("Rank of NDArray can't exceed 32");

    setShapeInfo(ShapeBuilders::createShapeInfo(dtype, order, shape, context->getWorkspace()));
//    ALLOCATE(_buffer, context->getWorkspace(), _length * DataTypeUtils::sizeOf(dtype), int8_t);
//    memset(_buffer, 0, _length * DataTypeUtils::sizeOf(dtype));
    _context = context == nullptr ? nd4j::graph::LaunchContext::defaultContext() : context;
    triggerAllocationFlag(true, true);
    hipMalloc(&_bufferD, _length * sizeOfT());
    hipMalloc(&_shapeInfoD, shape::shapeInfoByteLength(_shapeInfo));
    syncShape();
    tickWriteDevice();
//    syncToDevice();

}


//////////////////////////////////////////////////////////////////////////
// perform array transformation
    // void NDArray::applyTransform(nd4j::transform::FloatOps op, void *extraParams) {
    //     applyTransform(op, this, extraParams);
    // }

    // void NDArray::applyTransform(nd4j::transform::AnyOps op, void *extraParams) {
    //     applyTransform(op, this, extraParams);
    // }

    // void NDArray::applyTransform(nd4j::transform::SameOps op, void *extraParams) {
    //     applyTransform(op, this, extraParams);
    // }

    // void NDArray::applyTransform(nd4j::transform::BoolOps op, void *extraParams) {
    //     applyTransform(op, this, extraParams);
    // }

    // void NDArray::applyTransform(nd4j::transform::StrictOps op, void *extraParams) {
    //     applyTransform(op, this, extraParams);
    // }

    // perform array transformation

/*
    template<typename T>
    template<typename OpName>
    void NDArray<T>::applyRandom(nd4j::random::RandomBuffer *buffer, NDArray<T>* y, NDArray<T>* z, T* extraArgs) {
        Nd4jPointer state = (Nd4jPointer) buffer;
        if (y == nullptr && z == nullptr) {
            // we're executing indexed z here
            functions::random::RandomFunction<T>::template execTransform<OpName>(state, this->buffer(), this->shapeInfo(), extraArgs);
        } else if (y == nullptr && z != nullptr) {
            // XZ case
            functions::random::RandomFunction<T>::template execTransform<OpName>(state, this->buffer(), this->shapeInfo(), z->buffer(), z->shapeInfo(), extraArgs);
        } else if (y != nullptr && z != nullptr) {
            // XYZ case
            functions::random::RandomFunction<T>::template execTransform<OpName>(state, this->buffer(), this->shapeInfo(), y->buffer(), y->shapeInfo(), z->buffer(), z->shapeInfo(), extraArgs);
        }
    }
    */

    //////////////////////////////////////////////////////////////////////////
    void NDArray::applyTrueBroadcast(nd4j::BroadcastBoolOpsTuple op, const NDArray* other, NDArray* target, const bool checkTargetShape, ExtraArguments *extraArgs) const {
        if (isS())
            throw std::runtime_error("NDArray::applyTrueBroadcast bool: you can't use this method on String array!");
        if(target == nullptr || other == nullptr)
            throw std::runtime_error("NDArray::applyTrueBroadcast bool method: target or other = nullptr !");
        
        if (isScalar()) {
            NDArray temp(target->_shapeInfo, _dataType, false, _context);
            temp.assign(this);
            temp.applyPairwiseTransform(op.p, other, target,  extraArgs);
            return;
        }
        if (other->isScalar()) {
            this->applyScalarArr(op.s, other, target, extraArgs);
            return;
        }

        const NDArray* min(nullptr), *max(nullptr);
        if(this->rankOf() >= other->rankOf()) {
            max = this;
            min = other;
        }
        else {
            max = other;
            min = this;
        }

        if(checkTargetShape) {
            Nd4jLong* newShapeInfo = nullptr;
            if(!ShapeUtils::evalBroadcastShapeInfo(*max, *min, false, newShapeInfo, _context->getWorkspace()))          // the rank of target array must be equal to max->rankOf)()
                throw std::runtime_error("NDArray::applyTrueBroadcast method: the shapes of this and other arrays are not suitable for broadcast operation !");
            if(!shape::equalsSoft(target->_shapeInfo, newShapeInfo) || target->_dataType != DataType::BOOL)
                throw std::runtime_error("NDArray::applyTrueBroadcast bool method: the shape or type of target array is wrong !");
            if(_dataType != other->_dataType)
                throw std::invalid_argument("NDArray::applyTrueBroadcast bool method: this and other arrays must have the same type !");

            // if workspace is not null - do not call delete.
            if (_context->getWorkspace() == nullptr)
                delete[] newShapeInfo;
        }

        NDArray* pTarget = (max->_dataType == target->_dataType) ? target : new NDArray(target->ordering(), target->getShapeAsVector(), max->_dataType, target->_context);
        // check whether max array has to be tiled
        if(!max->isSameShape(target)) {
            // evaluate repeating dimensions for tile operation
            std::vector<Nd4jLong> repeatMax(max->rankOf());
            for(int i = 1; i <= max->rankOf(); ++i)
                repeatMax[i-1] = (target->_shapeInfo[i] / max->_shapeInfo[i]);
            max->tile(repeatMax, *pTarget);
        }
        else
            pTarget->assign(max);

        // check whether min array has to be tiled
        std::vector<Nd4jLong> repeatMin(min->rankOf());
        int product = 1;
        for(int i = min->rankOf(); i >=1 ; --i) {
            repeatMin[i-1] = (target->_shapeInfo[target->rankOf() - min->rankOf() + i] / min->_shapeInfo[i]);
            product *= repeatMin[i-1];
        }

        auto pMin = const_cast<NDArray *>(min);
        if(product != 1 )
            pMin = new NDArray(min->tile(repeatMin));


        std::vector<int> sameDims = ShapeUtils::getDimsWithSameShape(*target, *pMin);

        if(max == this) {
            pTarget->applyBroadcast(op.b, sameDims, pMin, target, extraArgs);
        }
        else {
            auto dimsToExclude = ShapeUtils::evalDimsToExclude(target->rankOf(), sameDims);
            const auto numOfSubArrs = ShapeUtils::getNumOfSubArrs(target->_shapeInfo, dimsToExclude);

            for(Nd4jLong i = 0; i < numOfSubArrs; ++i) {
                NDArray targetSubArr = (*target)(i, dimsToExclude);
                if (pTarget == target)
                    pMin->applyPairwiseTransform(op.p, &targetSubArr, &targetSubArr, extraArgs);
                else {
                    NDArray pTargetSubArr = (*pTarget)(i, dimsToExclude);
                    pMin->applyPairwiseTransform(op.p, &pTargetSubArr, &targetSubArr, extraArgs);
                }
            }
        }

        if(pMin != min)
            delete pMin;
        if(pTarget != target)
            delete pTarget;
    }

    //////////////////////////////////////////////////////////////////////////
    void NDArray::applyTrueBroadcast(nd4j::BroadcastOpsTuple op, const NDArray* other, NDArray* target, const bool checkTargetShape, ExtraArguments *extraArgs) const {
        if (isS())
            throw std::runtime_error("NDArray::applyTrueBroadcast: you can't use this method on String array!");
        if(target == nullptr || other == nullptr)
            throw std::runtime_error("NDArray::applyTrueBroadcast method: target or other = nullptr !");
        if(((op.s == scalar::Divide || op.s == scalar::FloorDiv || op.s == scalar::FloorMod) && other->isB()) || (op.s == scalar::ReverseDivide && this->isB()))
            throw std::runtime_error("NDArray::applyTrueBroadcast method: you can't divide by bool array !");

        if (isScalar()) {
            target->assign(this);
            target->applyPairwiseTransform(op.p, *other, extraArgs);
            return;
        }
        if (other->isScalar()) {
            const_cast<NDArray*>(this)->applyScalarArr(op.s, other, target, extraArgs);
            return;
        }

        const NDArray* min(nullptr), *max(nullptr);
        if(this->rankOf() >= other->rankOf()) {
            max = this;
            min = other;
        }
        else {
            max = other;
            min = this;
        }

        if(checkTargetShape) {
            Nd4jLong* newShapeInfo = nullptr;
            if(!ShapeUtils::evalBroadcastShapeInfo(*max, *min, false, newShapeInfo, _context->getWorkspace()))          // the rank of target array must be equal to max->rankOf)()
                throw std::runtime_error("NDArray::applyTrueBroadcast method: the shapes of this and other arrays are not suitable for broadcast operation !");
            if(!shape::equalsTypesAndShapesSoft(target->getShapeInfo(), newShapeInfo))
                throw std::runtime_error("NDArray::applyTrueBroadcast method: the shape or type of target array is wrong !");

            // if workspace is not null - do not call delete.
            if (_context->getWorkspace() == nullptr)
                delete[] newShapeInfo;
        }

        NDArray* pTarget = (max->_dataType == target->_dataType) ? target : new NDArray(target->ordering(), target->getShapeAsVector(), max->_dataType, target->_context);
        // check whether max array has to be tiled
        if(!max->isSameShape(target)) {
            // evaluate repeating dimensions for tile operation
            std::vector<Nd4jLong> repeatMax(max->rankOf());
            for(int i = 1; i <= max->rankOf(); ++i)
                repeatMax[i-1] = (target->_shapeInfo[i] / max->_shapeInfo[i]);
            max->tile(repeatMax, *pTarget);
        }
        else
            pTarget->assign(max);


        // check whether min array has to be tiled
        std::vector<Nd4jLong> repeatMin(min->rankOf());
        int product = 1;
        for(int i = min->rankOf(); i >=1 ; --i) {
            repeatMin[i-1] = (target->_shapeInfo[target->rankOf() - min->rankOf() + i] / min->_shapeInfo[i]);
            product *= repeatMin[i-1];
        }

        auto pMin = const_cast<NDArray *>(min);
        if(product != 1 )
            pMin = new NDArray(min->tile(repeatMin));

        std::vector<int> sameDims = ShapeUtils::getDimsWithSameShape(*target, *pMin);

        if(max == this) {
            pTarget->applyBroadcast(op.b, sameDims, pMin, target, extraArgs);
        }
        else {
            auto dimsToExclude = ShapeUtils::evalDimsToExclude(target->rankOf(), sameDims);
            const auto numOfSubArrs = ShapeUtils::getNumOfSubArrs(target->_shapeInfo, dimsToExclude);

            for(Nd4jLong i = 0; i < numOfSubArrs; ++i) {
                auto targetSubArr = (*target)(i, dimsToExclude);
                if(pTarget == target)
                    pMin->applyPairwiseTransform(op.p, &targetSubArr, &targetSubArr, extraArgs);
                else {
                    auto pTargetSubArr = (*pTarget)(i, dimsToExclude);
                    pMin->applyPairwiseTransform(op.p, &pTargetSubArr, &targetSubArr, extraArgs);
                }
            }
        }

        if(pMin != min)
            delete pMin;
         if(pTarget != target)
            delete pTarget;
    }

    //////////////////////////////////////////////////////////////////////////
    // return array which is broadcasted from this and argument array
    NDArray* NDArray::broadcast(const NDArray& other) {
	    // the orders must be the same
	    char order = ordering();
	    if(order != other.ordering())
		    throw std::runtime_error("Broadcast method: arrays have different orders!");

	    // recognize shapes with smaller and bigger rank
	    Nd4jLong* biggerShapeInfo = nullptr;
	    Nd4jLong* smallerShapeInfo = nullptr;
	    int smallerRank, biggerRank;
	    if (rankOf() > other.rankOf()) {
		    biggerShapeInfo = _shapeInfo;
		    biggerRank = shape::rank(_shapeInfo);
		    smallerShapeInfo = other._shapeInfo;
		    smallerRank = shape::rank(other._shapeInfo);
	    }
	    else {
		    biggerShapeInfo = other._shapeInfo;
		    biggerRank = shape::rank(other._shapeInfo);
		    smallerShapeInfo = _shapeInfo;
		    smallerRank = shape::rank(_shapeInfo);
	    }

	    // check shapes on consistency
	    int diff = biggerRank - smallerRank;
	    for (int i = smallerRank; i<=1; --i)
		    if(biggerShapeInfo[diff+i] != smallerShapeInfo[i] && biggerShapeInfo[i] != 1 && smallerShapeInfo[i] != 1)
			    throw std::runtime_error("Broadcast method: arrays have incompatible shapes !");

		// create and fill ret shapeInfo
	    auto shapeInfoNew = new Nd4jLong[shape::shapeInfoLength(biggerRank)];
	    memcpy(shapeInfoNew, biggerShapeInfo, shape::shapeInfoByteLength(biggerRank));
	    for (int i = smallerRank; i>=1; --i)
		    if(shapeInfoNew[diff+i] == 1 || smallerShapeInfo[i] == 1)
			    shapeInfoNew[diff+i] *= smallerShapeInfo[i];

	    auto ret = new NDArray(shapeInfoNew, true, _context);
        ShapeUtils::updateStridesAndType(ret->getShapeInfo(), DataTypeUtils::pickPairwiseResultType(_dataType, other._dataType), order);
	    delete []shapeInfoNew;

    	return ret;
    }


    //////////////////////////////////////////////////////////////////////////
    // check whether array's rows (arg=0) or columns (arg=1) create orthogonal basis
    bool NDArray::hasOrthonormalBasis(const int arg) {
        if (isS())
            throw std::runtime_error("NDArray::hasOrthonormalBasis: you can't use this method on String array!");
	    if(rankOf() !=2 )
		    throw std::runtime_error("NDArray::hasOrthBasis method: rank of ndarray is not equal 2 !");

	    if(arg!=0  && arg!=1)
		    throw std::runtime_error("NDArray::hasOrthBasis method: input argument is not equal to 0 or 1 !");

	    const double eps = 1e-5;
        double dot = 0.f;

        if(arg) {					// check whether columns create orthogonal basis
		    for(int j=0; j<columns()-1; ++j)
			    for(int k=j+1; k<columns(); ++k) {
				    for(int i=0; i<rows(); ++i)
					    dot += e<double>(i,j)*e<double>(i,k);

				    if(nd4j::math::nd4j_abs(dot) > eps )
					    return false;

				    dot = 0.f;
			    }

			    for(int j=0; j<columns(); ++j)	{	// check whether norm of column vector = 1
			        for(int i=0; i<rows(); ++i)
				        dot += e<double>(i,j)*e<double>(i,j);
			    if(dot != 0.f && nd4j::math::nd4j_abs(nd4j::math::nd4j_sqrt<double, double>(dot) - 1.f) > eps)
				    return false;

			    dot = 0.f;
		    }
	    }
	    else {						// check whether rows create orthogonal basis
		    for(int i=0; i<rows()-1; ++i)
			    for(int k=i+1; k<rows(); ++k) {
				    for(int j=0; j<columns(); ++j)
					    dot += e<double>(i,j)*e<double>(k,j);

				    if(nd4j::math::nd4j_abs(dot) > eps )
					    return false;

				    dot = 0.;
			    }

		        for(int i=0; i<rows(); ++i) {		// check whether norm of row vector = 1
			        for(int j=0; j<columns(); ++j)
					    dot += e<double>(i,j)*e<double>(i,j);

			        if(dot!= 0. && nd4j::math::nd4j_abs(nd4j::math::nd4j_sqrt<double, double>(dot) - 1.) > eps)
				        return false;
			        dot = 0.;
		        }
	        }
	    return true;
    }

    template <typename T>
    std::vector<T> NDArray::asVectorT() {
        std::vector<T> result(this->lengthOf());

#pragma omp parallel for simd
        for (int e = 0; e < this->lengthOf(); e++)
            result[e] = this->e<T>(e);

        return result;
    }
    BUILD_SINGLE_TEMPLATE(template std::vector, NDArray::asVectorT(), LIBND4J_TYPES);


    ////////////////////////////////////////////////////////////////////////
    template<typename T>
    void NDArray::setValueInDiagMatrix(const T& value, const int diag, const char direction) {
        if (isS())
            throw std::runtime_error("NDArray::setValueInDiagMatrix: you can't use this method on String array!");
        if(rankOf() != 2)
           throw std::string("NDArray::setValueInDiagMatrix method: array must have rank = 2, but got " + toStringValue(rankOf()) + " instead !");
    }
    template void NDArray::setValueInDiagMatrix(const double& value, const int diag, const char direction);
    template void NDArray::setValueInDiagMatrix(const float& value, const int diag, const char direction);
    template void NDArray::setValueInDiagMatrix(const float16& value, const int diag, const char direction);
    template void NDArray::setValueInDiagMatrix(const bfloat16& value, const int diag, const char direction);
    template void NDArray::setValueInDiagMatrix(const Nd4jLong& value, const int diag, const char direction);
    template void NDArray::setValueInDiagMatrix(const int& value, const int diag, const char direction);
    template void NDArray::setValueInDiagMatrix(const int16_t& value, const int diag, const char direction);
    template void NDArray::setValueInDiagMatrix(const uint8_t& value, const int diag, const char direction);
    template void NDArray::setValueInDiagMatrix(const int8_t& value, const int diag, const char direction);
    template void NDArray::setValueInDiagMatrix(const bool& value, const int diag, const char direction);

    ////////////////////////////////////////////////////////////////////////
    // default destructor
    NDArray::~NDArray() noexcept {
        if (_isBuffAlloc && _context->getWorkspace() == nullptr && _buffer != nullptr) {
            if (!isS()) {
                delete[] _buffer;
            } else {
                for (int e = 0; e < lengthOf(); e++) {
                    auto t = reinterpret_cast<utf8string**>(_buffer);
                    delete t[e];
                };

                delete[] _buffer;
            }
        }

        if (_isShapeAlloc  && _context->getWorkspace() == nullptr && _shapeInfo != nullptr)
            delete[] _shapeInfo;
        if (_shapeInfoD)
            hipFree(_shapeInfoD);
        if (_bufferD)
            hipFree(_bufferD);
    }


    //////////////////////////////////////////////////////////////////////////
// set new order and shape in case of suitable array length
    bool NDArray::reshapei(const char order, const std::vector<Nd4jLong>& cshape) {

        // check firstly whether cshape is identical to shape of array, if yes then reshape is unnecessary
        if(order == ordering() && rankOf() == cshape.size()) {
            bool areShapesSame = true;
            for(int i = 0; i < cshape.size(); ++i)
                if(cshape[i] != sizeAt(i)) {
                    areShapesSame = false;
                    break;
                }
            if(areShapesSame)
                return areShapesSame;
        }

        std::vector<Nd4jLong> shape(cshape);
        int rank = shape.size();

        // looking for negative in shape

        int numberNegativesOnes = 0;

        Nd4jLong* shape_ = shape.data();
        for (int i = 0; i < (int) shape.size(); i++) {
            if (shape[i] < 0) {
                if (numberNegativesOnes >= 1)
                    throw std::runtime_error("Only one dimension can be negative at once");

                numberNegativesOnes++;

                int shapeLength = 1;
                for (int j = 0; j < (int) shape.size(); j++)
                    if (i != j)
                        shapeLength *= shape_[j];

                Nd4jLong realShape = nd4j::math::nd4j_abs<int>(lengthOf() / shapeLength);
                auto thisNewShape = new Nd4jLong[shape.size()];

                for (int j = 0; j < (int) shape.size(); j++)
                    if (i != j)
                        thisNewShape[j] = shape_[j];
                    else
                        thisNewShape[j] = realShape;

                shape_ = thisNewShape;
            }
        }

        for (int e = 0; e < (int) shape.size(); e++)
            shape[e] = shape_[e];

        if (numberNegativesOnes > 0)
            delete[] shape_;

        int arrLength = 1;
        for(const auto& item : shape)
            arrLength *= item;

        if(_buffer==nullptr || arrLength != this->lengthOf()) {
            this->printShapeInfo("Mismatched shape");
            nd4j::Logger::printv("Shape requested: ", shape);
            nd4j_debug("Requested length in reshape: %i; Existing length: %i;\n", arrLength, this->lengthOf());
            throw std::runtime_error("Bad shape!");
        }

        int shapeLength = shape::shapeInfoLength(rank);
        // remember old values

        // we can do this only if there was no permute applied, or there are no weird strides
        if (shape::canReshape(this->rankOf(), this->_shapeInfo, shape.size(), shape.data(), order == 'f')) {
            Nd4jLong *shapeInfoNew;
            ALLOCATE(shapeInfoNew, _context->getWorkspace(), shape::shapeInfoLength(rank), Nd4jLong);

            shape::reshapeCF(this->rankOf(), this->_shapeInfo, shape.size(), shape.data(), order == 'f', shapeInfoNew);

            if (_isShapeAlloc)
                RELEASE(_shapeInfo, _context->getWorkspace());

            ArrayOptions::setDataType(shapeInfoNew, this->dataType());
            _shapeInfo = shapeInfoNew;
            _isShapeAlloc = true;
        } else {
            Nd4jLong *shapeInfoNew;
            ALLOCATE(shapeInfoNew, _context->getWorkspace(), shape::shapeInfoLength(rank), Nd4jLong);

            if (order == 'c')
                shape::shapeBuffer(shape.size(), dataType(), shape.data(), shapeInfoNew);
            else
                shape::shapeBufferFortran(shape.size(), dataType(), shape.data(), shapeInfoNew);

            int8_t *newBuffer;
            ALLOCATE(newBuffer, _context->getWorkspace(), this->lengthOf() * sizeOfT(), int8_t);

            NativeOpExecutioner::execTransformSame(nullptr, transform::Copy, _buffer, _shapeInfo, _bufferD, _shapeInfoD, newBuffer, shapeInfoNew, nullptr, nullptr, nullptr, nullptr, nullptr);

            if (_isBuffAlloc)
                RELEASE(_buffer, _context->getWorkspace());


            if (_isShapeAlloc)
                RELEASE(_shapeInfo, _context->getWorkspace());

            _buffer = newBuffer;
            _shapeInfo = shapeInfoNew;
            _isShapeAlloc = true;
            _isBuffAlloc = true;
        }

        return true;
    }

    ////////////////////////////////////////////////////////////////////////
    void NDArray::setIdentity() {
        if (isS())
            throw std::runtime_error("NDArray::setIdentity: you can't use this method on String array!");

        this->assign(0.);

        int  rank    = rankOf();
        auto shape   = shapeOf();
        auto strides = stridesOf();
        int  minDim  = 100000000;
        Nd4jLong indices[MAX_RANK];
        for(int j = 0; j < rank; ++j)
            indices[j] = 1;

        Nd4jLong offset = shape::getOffset(0, shape, strides, indices, rank);

        for(int i = 0; i < rank; ++i)
            if(minDim > shape[i])
                minDim = shape[i];

        float v = 1.0f;
#pragma omp parallel for if(minDim > Environment::getInstance()->elementwiseThreshold()) schedule(guided)
        for(int i = 0; i < minDim; ++i)
            templatedSet<float>(_buffer, i*offset, this->dataType(), &v);
    }

    template <typename T>
    void NDArray::templatedSet(void *buffer, const Nd4jLong xOfsset, nd4j::DataType dtype, const void *value) {
        BUILD_SINGLE_PARTIAL_SELECTOR(dtype, templatedSet< , T>(buffer, xOfsset, value), LIBND4J_TYPES);
    }
    BUILD_SINGLE_TEMPLATE(template void NDArray::templatedSet, (void *buffer, const Nd4jLong xOfsset, nd4j::DataType dtype, const void *value), LIBND4J_TYPES);



    template <typename T>
    void NDArray::templatedSwap(void *xBuffer, void *yBuffer, Nd4jLong length) {
        auto x = reinterpret_cast<T *>(xBuffer);
        auto y = reinterpret_cast<T *>(yBuffer);

#pragma omp parallel for simd schedule(static)
        for (int i = 0; i < length; ++i) {
            auto temp = x[i];
            x[i] = y[i];
            y[i] = temp;
        }
    }
    BUILD_SINGLE_TEMPLATE(template void NDArray::templatedSwap, (void *xBuffer, void *yBuffer, Nd4jLong length), LIBND4J_TYPES);

    ////////////////////////////////////////////////////////////////////////
    void NDArray::swapUnsafe(NDArray& other) {
        auto xType = this->dataType();

        if (xType != other.dataType())
            throw std::runtime_error("NDArray::swapUnsage method: both arrays must have the same data type");

        if(_buffer == nullptr || other._buffer == nullptr)
            throw std::runtime_error("NDArray::swapUnsafe method: input array should not be empty!");

        // if(_buffer == other._buffer)
        //     throw std::runtime_error("NDArray::swapUnsafe method: the buffers of input arrays should not point on the same address!");

        if(lengthOf() != other.lengthOf())
            throw std::runtime_error("NDArray::swapUnsafe method: input arrays should have the same length!");

        BUILD_SINGLE_SELECTOR(xType, templatedSwap, (this->_buffer, other.buffer(), this->lengthOf()), LIBND4J_TYPES);
    }

    ////////////////////////////////////////////////////////////////////////
    NDArray* NDArray::diagonal(const char type) const {

        if (isS())
            throw std::runtime_error("NDArray::diagonal: you can't use this method on String array!");

        const char order = ordering();
        const int  rank  = rankOf();
        Nd4jLong *outShapeInfo;
        ALLOCATE(outShapeInfo, _context->getWorkspace(), 8, Nd4jLong);
        outShapeInfo[0] = 2;
        outShapeInfo[5] = 0;

        if(isVector() || isScalar()) {

            outShapeInfo[1] = outShapeInfo[2] = outShapeInfo[3] = outShapeInfo[4] = 1;
            outShapeInfo[6] = 1;
            outShapeInfo[7] = (int)order;
        }
        else {

            int diagSize  = 100000000;
            Nd4jLong indices[MAX_RANK];

            for(int i = 0; i < rank; ++i) {
                if(diagSize > shapeOf()[i])
                    diagSize = shapeOf()[i];
                indices[i] = 1;
            }

            auto step = shape::getOffset(0, shapeOf(), stridesOf(), indices, rank);

            if(type == 'c') {
                outShapeInfo[1] = diagSize;
                outShapeInfo[2] = 1;
            }
            else {
                outShapeInfo[1] = 1;
                outShapeInfo[2] = diagSize;
            }
            shape::updateStrides(outShapeInfo, order);

            outShapeInfo[3] *= step;
            outShapeInfo[4] *= step;
            outShapeInfo[6] =  -1;
        }

        ArrayOptions::setDataType(outShapeInfo, this->dataType());

        auto result = new NDArray(this->_buffer, outShapeInfo, this->_context);
        result->_isShapeAlloc = true;
        return result;
    }

    void NDArray::streamline(char o) {
        char order = o == 'a' ? this->ordering() : o;

        Nd4jLong *newShape;
        ALLOCATE(newShape, this->_context->getWorkspace(), shape::shapeInfoLength(this->rankOf()), Nd4jLong);

        int8_t *newBuffer;
        ALLOCATE(newBuffer, this->_context->getWorkspace(), this->lengthOf() * sizeOfT(), int8_t);

        std::vector<Nd4jLong> shape(this->rankOf());
        for (int e = 0; e < this->rankOf(); e++)
            shape[e] = this->sizeAt(e);

        if (order == 'c')
            shape::shapeBuffer(this->rankOf(),dataType(),  shape.data(), newShape);
        else
            shape::shapeBufferFortran(this->rankOf(), dataType(), shape.data(), newShape);

        if (!isView()) {
            NativeOpExecutioner::execTransformSame(nullptr, transform::Copy, _buffer, _shapeInfo, nullptr, nullptr, newBuffer, newShape, nullptr, nullptr, nullptr, nullptr, nullptr);
            memcpy(_buffer, newBuffer, this->lengthOf() * sizeOfT());

            //if (_isBuffAlloc)
            //    RELEASE(this->_buffer, this->_workspace);
            if (_isShapeAlloc)
                RELEASE(this->_shapeInfo, this->_context->getWorkspace());

            //this->_buffer = newBuffer;
            //this->_isBuffAlloc = true;

            RELEASE(newBuffer, this->_context->getWorkspace());

            this->_shapeInfo = newShape;
            this->_isShapeAlloc = true;
        } else {
            NativeOpExecutioner::execTransformSame(nullptr, transform::Copy, _buffer, _shapeInfo, nullptr, nullptr, newBuffer, newShape, nullptr, nullptr, nullptr, nullptr, nullptr);

            if (_isBuffAlloc)
                RELEASE(this->_buffer, this->_context->getWorkspace());
            if (_isShapeAlloc)
                RELEASE(this->_shapeInfo, this->_context->getWorkspace());

            this->_buffer = newBuffer;
            this->_isBuffAlloc = true;

            this->_shapeInfo = newShape;
            this->_isShapeAlloc = true;
        }
    }

    void NDArray::applyPairwiseTransform(nd4j::pairwise::Ops op, const NDArray* other, NDArray *target, ExtraArguments *extraParams) const{
        if (isS())
            throw std::runtime_error("NDArray::applyPairwiseTransform: you can't use this method on String array!");
        if (other->lengthOf() != target->lengthOf())
            throw std::invalid_argument("NDArray::applyPairwiseTransform method - lengths of arrays are mismatched");
        if (target->_dataType != this->_dataType && target->_dataType != other->_dataType)
            throw std::invalid_argument("NDArray::applyPairwiseTransform method - type of target array must be the same as type of this or other array !");
        if (_context == nullptr)
            throw std::runtime_error("Launch context cannot be NULL!!!");
        if (_context->getCudaStream() == nullptr)
            throw std::runtime_error("CUDA stream cannot be NULL!!!");

        //Nd4jPointer nativeStream = (Nd4jPointer)malloc(sizeof(hipStream_t));
        //CHECK_ALLOC(nativeStream, "Failed to allocate memory for new CUDA stream");
        //if (nativeStream == nullptr) throw std::runtime_error("Failed to allocate memory for new CUDA stream");
        //hipError_t err = hipStreamCreate(reinterpret_cast<hipStream_t *>(&nativeStream));
        auto stream = _context->getCudaStream(); //reinterpret_cast<hipStream_t *>(&nativeStream);
        //_context->setCudaStream(stream);

        NativeOpExecutioner::execPairwiseTransform(_context, op, this->_buffer, this->_shapeInfo, this->_bufferD, this->_shapeInfoD, other->_buffer, other->_shapeInfo, other->_bufferD, other->_shapeInfoD, target->_buffer, target->_shapeInfo, target->_bufferD, target->_shapeInfoD, extraParams != nullptr ? extraParams->argumentAsT(target->dataType()) : nullptr);

        //auto res = hipStreamSynchronize(*stream);
        //if (res != 0) {
        //    nd4j_printf("Error: %i\n", res);
        //    throw std::runtime_error("Operation failed.");
        //}

        //target->syncToHost();
        if (extraParams != nullptr)
            this->synchronize();
    }

    void
    NDArray::syncToHost() {
        auto res = hipStreamSynchronize(*_context->getCudaStream());
        if (this->_buffer == nullptr) {
            ALLOCATE(this->_buffer, this->_context->getWorkspace(), this->lengthOf() * this->sizeOfT(), int8_t);
            triggerAllocationFlag(true, true);
        }
        hipMemcpy(this->_buffer, this->_bufferD, this->lengthOf() * this->sizeOfT(), hipMemcpyDeviceToHost);
    }

    void
    NDArray::syncToDevice() {
        hipMemcpy(this->_bufferD, this->_buffer, this->lengthOf() * this->sizeOfT(), hipMemcpyHostToDevice);
    }

    void
    NDArray::syncShape() {
        hipMemcpy(_shapeInfoD, _shapeInfo, shape::shapeInfoByteLength(_shapeInfo), hipMemcpyHostToDevice);
    }

    template <typename X, typename Y>
    void NDArray::templatedDoubleAssign(void *xBuffer, const Nd4jLong xOffset, const void *yBuffer, const Nd4jLong yOffset) const {
        auto x = reinterpret_cast<X *>(xBuffer);
        const auto y = reinterpret_cast<const Y *>(yBuffer);

        x[xOffset] = static_cast<X>(y[yOffset]);
    }
    BUILD_DOUBLE_TEMPLATE(template void NDArray::templatedDoubleAssign, (void *xBuffer, const Nd4jLong xOffset, const void *yBuffer, const Nd4jLong yOffset) const, LIBND4J_TYPES, LIBND4J_TYPES);

    // This method assigns values of given NDArray to this one
    void NDArray::assign(const NDArray& other) {

        if (this == &other)
            return;

        if (!Environment::getInstance()->isExperimentalBuild() && (this->dataType() != other.dataType() && other.dataType() != DataType::BOOL)) {
            throw datatype_exception::build("NDArray::assign: cannot assign array of different types", this->dataType(), other.dataType());
        }

        if (other.isScalar()) {
            if(this->isScalar()) {
                if (!this->isEmpty() && !other.isEmpty()) {
                    BUILD_DOUBLE_SELECTOR(_dataType, other._dataType, templatedDoubleAssign,
                                          (_buffer, 0, other._buffer, 0), LIBND4J_TYPES, LIBND4J_TYPES);
                }
                else if (this->isEmpty() != other.isEmpty()) { // need assign non-empty scalar to empty
                    if (other.isEmpty()) {
                        ArrayOptions::setPropertyBit(this->_shapeInfo, ARRAY_EMPTY);
                        syncShape();
                    }
                    else
                        *this = other;
                }
            }
            else {
                NativeOpExecutioner::execScalar(_context, scalar::CopyPws, _buffer, _shapeInfo, _bufferD, _shapeInfoD, _buffer, _shapeInfo, _bufferD, _shapeInfoD, other._buffer, other._shapeInfo, other._bufferD, other._shapeInfoD, nullptr);
            }
            return;
        }

        if (other._length != _length) {
            auto shapeThis = ShapeUtils::shapeAsString(this);
            auto shapeThat = ShapeUtils::shapeAsString(&other);
            nd4j_printf("Can't assign new value to the array: this shape %s; other shape: %s\n", shapeThis.c_str(), shapeThat.c_str());
            throw std::runtime_error("Lengths of arrays are mismatched");
        }

        // memcpy is allowed only for same order && same ews (being equal to 1)
        if (ordering() == other.ordering() && _dataType == other._dataType && ews() == 1 && other.ews() == 1)
            hipMemcpy(_bufferD, other._bufferD, _length * sizeOfT(), hipMemcpyDeviceToDevice);
        else if(_dataType == other._dataType)
            NativeOpExecutioner::execTransformSame(_context, transform::Copy, other._buffer, other._shapeInfo, other._bufferD, other._shapeInfoD, _buffer, _shapeInfo, _bufferD, _shapeInfoD, nullptr, nullptr, nullptr);
        else
            NativeOpExecutioner::execPairwiseTransform(_context, pairwise::CopyPws, _buffer, _shapeInfo, _bufferD, _shapeInfoD, other._buffer, other._shapeInfo, other._bufferD, other._shapeInfoD, _buffer, _shapeInfo, _bufferD, _shapeInfoD, nullptr);

    }

    ////////////////////////////////////////////////////////////////////////
// This method returns new copy of this NDArray, optionally in different order
    NDArray* NDArray::dup(const char newOrder) {

        char order = newOrder == 'a' ? ordering() : newOrder;

        auto outShapeInfo = ShapeBuilders::createShapeInfo(_dataType, order, getShapeAsVector(), _context->getWorkspace());
        void* outBuffer = nullptr;
        int8_t* outBufferD = nullptr;
        Nd4jLong* outShapeD = nullptr;
        ALLOCATE(outBuffer, _context->getWorkspace(), _length * sizeOfT(), int8_t);
        //hipMalloc(&outBufferD, _length * sizeOfT());
        //hipMalloc(&outShapeD, shape::shapeInfoByteLength(outShapeInfo));
        auto result = new NDArray(outBuffer, outShapeInfo, _context, true, true);
        result->setSpecialBuffers(outBufferD, outShapeD);
        result->assign(*this);

        return result;
    }

    void NDArray::synchronize() const {
        auto res = hipStreamSynchronize(*(_context->getCudaStream()));
        if (res != 0)
            throw std::runtime_error("Synchronization failed");
    }

    void NDArray::registerSpecialUse(std::initializer_list<NDArray*> writeList, std::initializer_list<NDArray*> readList) {
        // no-op
        for (auto p:writeList) {
            if (!p->isActualOnDeviceSide())
                p->syncToDevice();

            p->tickWriteDevice();
        }

        for (auto p:readList) {
            if (!p->isActualOnDeviceSide())
                p->syncToDevice();

            p->tickReadDevice();
        }
    }

    ////////////////////////////////////////////////////////////////////////
    NDArray::NDArray(const char order, const std::vector<Nd4jLong> &shape, const std::vector<double>& data, nd4j::DataType dtype, nd4j::graph::LaunchContext* context) {

        if ((int) shape.size() > MAX_RANK)
            throw std::invalid_argument("Rank of NDArray can't exceed 32");

        setShapeInfo(ShapeBuilders::createShapeInfo(dtype, order, shape, context->getWorkspace()));

        if (_length != data.size()) {
            nd4j_printf("NDArray constructor: data size [%i] doesn't match shape length [%i]\n", data.size(), _length);
            throw std::runtime_error("Data size doesn't match shape");
        }

        ALLOCATE(_buffer, context->getWorkspace(), _length * DataTypeUtils::sizeOf(dtype), int8_t);
        hipMalloc(&_bufferD, _length * DataTypeUtils::sizeOf(dtype));
        hipMalloc(&_shapeInfoD, shape::shapeInfoByteLength(_shapeInfo));
        syncShape();
        _context = context == nullptr ? nd4j::graph::LaunchContext::defaultContext() : context;
        triggerAllocationFlag(true, true);

        for(Nd4jLong i=0; i < _length; ++i) {
            BUILD_SINGLE_PARTIAL_SELECTOR(dtype, templatedDoubleAssign<, double>(_buffer, i, reinterpret_cast<const void *>(data.data()), i), LIBND4J_TYPES);
        }
        syncToDevice();
    }

////////////////////////////////////////////////////////////////////////
    NDArray::NDArray(const NDArray *other, const bool copyStrides, nd4j::graph::LaunchContext* context) {

        ALLOCATE(_buffer, context->getWorkspace(), other->_length * DataTypeUtils::sizeOf(other->dataType()), int8_t);
        setShapeInfo(ShapeBuilders::copyShapeInfo(other->_shapeInfo, copyStrides, context->getWorkspace()));
        if (_context == nullptr)
            _context = graph::LaunchContext::defaultContext();

        _context = context == nullptr ? nd4j::graph::LaunchContext::defaultContext() : context;

        triggerAllocationFlag(true, true);
    }

////////////////////////////////////////////////////////////////////////
    NDArray::NDArray(void* buffer, const char order, const std::vector<Nd4jLong> &shape,  nd4j::DataType dtype, nd4j::graph::LaunchContext* context) {

        if ((int) shape.size() > MAX_RANK)
            throw std::invalid_argument("Rank of NDArray can't exceed 32");

        setShapeInfo(ShapeBuilders::createShapeInfo(dtype, order, shape, context->getWorkspace()));

        _buffer = reinterpret_cast<int8_t *>(buffer);
        _context = context == nullptr ? nd4j::graph::LaunchContext::defaultContext() : context;
        triggerAllocationFlag(false, true);
    }

////////////////////////////////////////////////////////////////////////
// creates new NDArray using shape information from "shapeInfo" array, set all elements in new array to be zeros
    NDArray::NDArray(Nd4jLong* shapeInfo, const bool copyStrides, nd4j::graph::LaunchContext* context, const bool isShapeAlloc) {

        if ((int) shapeInfo[0] > MAX_RANK)
            throw std::invalid_argument("Rank of NDArray can't exceed 32");

        if(isShapeAlloc) {
            setShapeInfo(shapeInfo);
            if(!copyStrides)
                shape::updateStrides(_shapeInfo, shape::order(shapeInfo));
        }
        else
            setShapeInfo(ShapeBuilders::copyShapeInfo(shapeInfo, copyStrides, context->getWorkspace()));

        if (ArrayOptions::hasPropertyBitSet(shapeInfo, ARRAY_EMPTY)) {
            _buffer = nullptr;
            _length = 0;
            triggerAllocationFlag(false, true);
        }
        else {
            ALLOCATE(_buffer, context->getWorkspace(), _length * DataTypeUtils::sizeOfElement(_dataType), int8_t);

            memset(_buffer, 0, _length * DataTypeUtils::sizeOfElement(_dataType));

            triggerAllocationFlag(true, true);
        }
        _context = context == nullptr ? nd4j::graph::LaunchContext::defaultContext() : context;
    }

////////////////////////////////////////////////////////////////////////
// creates new NDArray using shape information from "shapeInfo" array, set all elements in new array to be zeros, set dtype as array type
    NDArray::NDArray(Nd4jLong* shapeInfo, const nd4j::DataType dtype, const bool copyStrides, nd4j::graph::LaunchContext* context, const bool isShapeAlloc) {

        if (shapeInfo == nullptr || (int) shapeInfo[0] > MAX_RANK)
            throw std::invalid_argument("NDArray constructor: input shapeInfo is nullptr or its rank exceeds 32");

        if(isShapeAlloc) {
            _shapeInfo = shapeInfo;
            if(!copyStrides)
                shape::updateStrides(_shapeInfo, shape::order(shapeInfo));
        }
        else
            _shapeInfo = ShapeBuilders::copyShapeInfo(shapeInfo, copyStrides, context->getWorkspace());

        _dataType = dtype;
        _length = shape::length(_shapeInfo);
        _context = context == nullptr ? nd4j::graph::LaunchContext::defaultContext() : context;
        ArrayOptions::setDataType(_shapeInfo, _dataType);

        ALLOCATE(_buffer, _context->getWorkspace(), _length * sizeOfT() , int8_t);

        memset(_buffer, 0, _length * DataTypeUtils::sizeOfElement(_dataType));

        triggerAllocationFlag(true, true);
    }

////////////////////////////////////////////////////////////////////////
    NDArray::NDArray(nd4j::DataType dtype, nd4j::graph::LaunchContext* context) {

        setShapeInfo(ShapeBuilders::createScalarShapeInfo(dtype, context->getWorkspace()));
        ALLOCATE(_buffer, context->getWorkspace(), DataTypeUtils::sizeOfElement(dtype), int8_t);
        memset(_buffer, 0, DataTypeUtils::sizeOfElement(dtype));
        _context = context == nullptr ? nd4j::graph::LaunchContext::defaultContext() : context;
        triggerAllocationFlag(true, true);
    }

    //BUILD_DOUBLE_TEMPLATE(template void NDArray::templatedSet, (void *buffer, const Nd4jLong *indices, Y value), LIBND4J_TYPES, LIBND4J_TYPES);
/*
#ifndef __CLION_IDE__
#include "NDArray.macro"
#endif
 */
}



#endif

