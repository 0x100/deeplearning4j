#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author raver119@gmail.com
//

#include <ops/declarable/helpers/scatter.h>
#include <numeric>
#include <helpers/ShapeUtils.h>
#include <helpers/PointersManager.h>
#include <TAD.h>
#include <ConstantShapeHelper.h>


namespace nd4j {
    namespace ops {
        namespace helpers {
            template<typename T, bool locking>
            __global__ static void scatterCuda(const int opCode, const int numOfSubArrs,
                                                     void* vx, const Nd4jLong *xShapeInfo, const Nd4jLong *xOffsets,
                                                     void* vy, const Nd4jLong *yShapeInfo, const Nd4jLong *yOffsets,
                                                     const int* indexes, unsigned int arrLenX, unsigned int arrLenY) {

                __shared__ T *x, *y;

                if (locking) {

                    for (int e = 0; e < numOfSubArrs; e++) {

                        const auto xIndex = indexes[e];
                        const bool isOwner = xIndex < gridDim.x ? blockIdx.x == xIndex : blockIdx.x == xIndex % gridDim.x;

                        if (!isOwner)
                            continue;

                        if (threadIdx.x == 0) {
                            x = reinterpret_cast<T *>(vx) + xOffsets[xIndex];
                            y = reinterpret_cast<T *>(vy) + yOffsets[e];
                        }
                        __syncthreads();

                        for (Nd4jLong i = threadIdx.x; i < arrLenX; i += blockDim.x) {

                            const auto xOffset = shape::getIndexOffset(i, xShapeInfo, arrLenX);
                            const auto yOffset = shape::getIndexOffset(i, yShapeInfo, arrLenY);

                            switch (opCode) {
                                case pairwise::Add:
                                    x[xOffset] += y[yOffset];
                                    break;
                                case pairwise::Subtract:
                                    x[xOffset] -= y[yOffset];
                                    break;
                                case pairwise::Multiply:
                                    x[xOffset] *= y[yOffset];
                                    break;
                                case pairwise::Divide:
                                    x[xOffset] /= y[yOffset];
                                    break;
                                case pairwise::ReverseSubtract:
                                    x[xOffset] = y[yOffset] - x[xOffset];
                                    break;
                                case pairwise::ReverseDivide:
                                    x[xOffset] = y[yOffset] / x[xOffset];
                                    break;
                                case pairwise::Copy2:
                                case pairwise::CopyPws:
                                    x[xOffset] = y[yOffset];
                                    break;
                                default:
                                    continue;
                            }
                        }
                        __syncthreads();
                    }
                } else {
                    for (int e = blockIdx.x; e < numOfSubArrs; e+= gridDim.x) {

                        if (threadIdx.x == 0) {
                            const auto xIndex = indexes[e];
                            x = reinterpret_cast<T *>(vx) + xOffsets[xIndex];
                            y = reinterpret_cast<T *>(vy) + yOffsets[e];
                        }
                        __syncthreads();

                        for (Nd4jLong i = threadIdx.x; i < arrLenX; i += blockDim.x) {
                            const auto xOffset = shape::getIndexOffset(i, xShapeInfo, arrLenX);
                            const auto yOffset = shape::getIndexOffset(i, yShapeInfo, arrLenY);

                            switch (opCode) {
                                case pairwise::Add:
                                    x[xOffset] += y[yOffset];
                                    break;
                                case pairwise::Subtract:
                                    x[xOffset] -= y[yOffset];
                                    break;
                                case pairwise::Multiply:
                                    x[xOffset] *= y[yOffset];
                                    break;
                                case pairwise::Divide:
                                    x[xOffset] /= y[yOffset];
                                    break;
                                case pairwise::ReverseSubtract:
                                    x[xOffset] = y[yOffset] - x[xOffset];
                                    break;
                                case pairwise::ReverseDivide:
                                    x[xOffset] = y[yOffset] / x[xOffset];
                                    break;
                                case pairwise::Copy2:
                                case pairwise::CopyPws:
                                    x[xOffset] = y[yOffset];
                                    break;
                                default:
                                    continue;
                            }
                        }
                        __syncthreads();
                    }
                }
            }


            template <typename T>
            void scatter_(graph::LaunchContext *context, pairwise::Ops op, const NDArray& indices, const NDArray& updates, NDArray& output, const bool lock) {
                std::vector<int> dims = {0};
                auto inverted = ShapeUtils::evalDimsToExclude(output.rankOf(), dims);

                shape::TAD tadX;
                tadX.init(output.getShapeInfo(), inverted.data(), inverted.size());
                tadX.createTadOnlyShapeInfo();
                tadX.createOffsets();

                shape::TAD tadY;
                tadY.init(updates.getShapeInfo(), inverted.data(), inverted.size());
                tadY.createTadOnlyShapeInfo();
                tadY.createOffsets();

                auto bX = ConstantShapeHelper::getInstance()->bufferForShapeInfo(tadX.tadOnlyShapeInfo);
                auto bY = ConstantShapeHelper::getInstance()->bufferForShapeInfo(tadY.tadOnlyShapeInfo);
                auto psX = reinterpret_cast<Nd4jLong *>(bX.special());
                auto psY = reinterpret_cast<Nd4jLong *>(bY.special());

                PointersManager manager(context, "scatter");
                //auto psX = reinterpret_cast<Nd4jLong *>(manager.replicatePointer(tadX.tadOnlyShapeInfo, shape::shapeInfoByteLength(tadX.tadOnlyShapeInfo)));
                //auto psY = reinterpret_cast<Nd4jLong *>(manager.replicatePointer(tadY.tadOnlyShapeInfo, shape::shapeInfoByteLength(tadY.tadOnlyShapeInfo)));

                auto poX = reinterpret_cast<Nd4jLong *>(manager.replicatePointer(tadX.tadOffsets, tadX.numTads * sizeof(Nd4jLong)));
                auto poY = reinterpret_cast<Nd4jLong *>(manager.replicatePointer(tadY.tadOffsets, tadY.numTads * sizeof(Nd4jLong)));


                NDArray::prepareSpecialUse({&output}, {&updates, &indices});

                unsigned int tadLengthX = shape::length(tadX.tadOnlyShapeInfo);
                unsigned int tadLengthY = shape::length(tadY.tadOnlyShapeInfo);
                if (tadLengthX != tadLengthY)
                    throw std::runtime_error("scatter: Lengths of TADs must be equal");

                auto blockSize = nd4j::math::nd4j_max<int>(32, nd4j::math::nd4j_min<int>(tadLengthX, 1024));

                if (lock)
                    scatterCuda<T, true><<<512, blockSize, 1024, *context->getCudaStream()>>>(op, indices.lengthOf(), output.getSpecialBuffer(), psX, poX, updates.getSpecialBuffer(), psY, poY, reinterpret_cast<int *>(indices.getSpecialBuffer()), tadLengthX, tadLengthY);
                else
                    scatterCuda<T, false><<<512, blockSize, 1024, *context->getCudaStream()>>>(op, indices.lengthOf(), output.getSpecialBuffer(), psX, poX, updates.getSpecialBuffer(), psY, poY, reinterpret_cast<int *>(indices.getSpecialBuffer()), tadLengthX, tadLengthY);

                NDArray::registerSpecialUse({&output}, {&updates, &indices});
                manager.synchronize();
            }

            void scatter(graph::LaunchContext *context, pairwise::Ops op, const NDArray& indices, const NDArray& updates, NDArray& output, const bool lock) {
                BUILD_SINGLE_SELECTOR(output.dataType(), scatter_, (context, op, indices, updates, output, lock), LIBND4J_TYPES);
            }

            void scatterND(graph::LaunchContext *context, pairwise::Ops op, const NDArray& indices, const NDArray& updates, NDArray& output, const bool lock) {

            }

            void scatterForLoss(graph::LaunchContext *context, const NDArray& indices, const NDArray& updates, NDArray& output, const bool calcGrad) {

            }
        }
    }
}