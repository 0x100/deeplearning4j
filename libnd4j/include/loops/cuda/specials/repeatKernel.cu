#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author GS <sgazeos@gmail.com>, created on 17.01.2019
//

#include <loops/special_kernels.h>

namespace nd4j {


    template <typename T>
    static __global__ void repeatKernel(void const* inputBuffer, void* outputBuffer, Nd4jLong numTads, Nd4jLong inputLength,
                                        Nd4jLong* tadOnlyInputShapeInfo,  Nd4jLong *tadInputOffsets,
                                        Nd4jLong* tadOnlyOutputShapeInfo, Nd4jLong *tadOutputOffsets) {
        //auto tid = blockIdx.x * blockDim.x; // + threadIdx.x;
        int totalThreads = gridDim.x * blockDim.x;
        //const auto resultLength = shape::length(outputShape);
        for (Nd4jLong i = blockIdx.x; i < numTads; i += gridDim.x) {
            auto yOffset = tadInputOffsets[i];
            auto xOffset = tadOutputOffsets[i];
            for (Nd4jLong j = threadIdx.x; j < inputLength; j += totalThreads) {
                *(reinterpret_cast<T*>(outputBuffer) + xOffset + shape::getIndexOffset(j, tadOnlyOutputShapeInfo, inputLength)) = *(reinterpret_cast<T const*>(inputBuffer) + yOffset + shape::getIndexOffset(j, tadOnlyInputShapeInfo, inputLength));
            }
        }
    }
    BUILD_SINGLE_TEMPLATE(template __global__ void repeatKernel, (void const* inputBuffer, void* outputBuffer,
            Nd4jLong numTads, Nd4jLong inputLength, Nd4jLong* tadOnlyInputShapeInfo,  Nd4jLong *tadInputOffsets,
            Nd4jLong* tadOnlyOutputShapeInfo, Nd4jLong *tadOutputOffsets), LIBND4J_TYPES);

    template <typename X, typename Y>
    static __global__ void repeatKernelDouble(void const* inputBuffer, void* outputBuffer, Nd4jLong numTads, Nd4jLong inputLength,
                                              Nd4jLong* tadOnlyInputShapeInfo,  Nd4jLong *tadInputOffsets,
                                              Nd4jLong* tadOnlyOutputShapeInfo, Nd4jLong *tadOutputOffsets) {
        //auto tid = blockIdx.x * blockDim.x; // + threadIdx.x;
        int totalThreads = gridDim.x * blockDim.x;
        //const auto resultLength = shape::length(outputShape);
        for (Nd4jLong i = blockIdx.x; i < numTads; i += gridDim.x) {
            auto yOffset = tadInputOffsets[i];
            auto xOffset = tadOutputOffsets[i];
            for (Nd4jLong j = threadIdx.x; j < inputLength; j += totalThreads) {
                *(reinterpret_cast<X*>(outputBuffer) + xOffset + shape::getIndexOffset(j, tadOnlyOutputShapeInfo, inputLength)) = static_cast<X>(*(reinterpret_cast<Y const*>(inputBuffer) + yOffset + shape::getIndexOffset(j, tadOnlyInputShapeInfo, inputLength)));
            }
        }
    }
    BUILD_DOUBLE_TEMPLATE(template __global__ void repeatKernelDouble, (void const* inputBuffer, void* outputBuffer,
            Nd4jLong numTads, Nd4jLong inputLength, Nd4jLong* tadOnlyInputShapeInfo,  Nd4jLong *tadInputOffsets,
            Nd4jLong* tadOnlyOutputShapeInfo, Nd4jLong *tadOutputOffsets), LIBND4J_TYPES, LIBND4J_TYPES);

    template <typename T>
    void repeatKernelH(void const* inputBuffer, void* outputBuffer, Nd4jLong numTads, Nd4jLong inputLength,
                              Nd4jLong *tadOnlyInputShapeInfo, Nd4jLong *tadInputOffsets,
                              Nd4jLong *tadOnlyOutputShapeInfo,Nd4jLong *tadOutputOffsets,
                              hipStream_t stream) {
        dim3 launchDims(256, 512, 8192);
        repeatKernel<T><<<launchDims.x, launchDims.y, launchDims.z, stream>>>(inputBuffer, outputBuffer, numTads, inputLength, tadOnlyInputShapeInfo, tadInputOffsets, tadOnlyOutputShapeInfo, tadOutputOffsets);
    }
    BUILD_SINGLE_TEMPLATE(template void repeatKernelH, (void const* inputBuffer, void* outputBuffer, Nd4jLong numTads, Nd4jLong inputLength,
            Nd4jLong* tadOnlyInputShapeInfo,  Nd4jLong *tadInputOffsets,
            Nd4jLong* tadOnlyOutputShapeInfo, Nd4jLong *tadOutputOffsets,
            hipStream_t stream), LIBND4J_TYPES);


    template <typename X, typename Y>
    void repeatKernelHH(void const* inputBuffer, void* outputBuffer, Nd4jLong numTads, Nd4jLong inputLength,
                               Nd4jLong *tadOnlyInputShapeInfo, Nd4jLong *tadInputOffsets,
                               Nd4jLong *tadOnlyOutputShapeInfo,Nd4jLong *tadOutputOffsets,
                               hipStream_t stream) {
        dim3 launchDims(256, 512, 8192);
        repeatKernelDouble<X,Y><<<launchDims.x, launchDims.y, launchDims.z, stream>>>(inputBuffer, outputBuffer, numTads, inputLength, tadOnlyInputShapeInfo, tadInputOffsets, tadOnlyOutputShapeInfo, tadOutputOffsets);
    }
    BUILD_DOUBLE_TEMPLATE(template void repeatKernelHH, (void const* inputBuffer, void* outputBuffer, Nd4jLong numTads, Nd4jLong inputLength,
            Nd4jLong* tadOnlyInputShapeInfo,  Nd4jLong *tadInputOffsets,
            Nd4jLong* tadOnlyOutputShapeInfo, Nd4jLong *tadOutputOffsets,
            hipStream_t stream), LIBND4J_TYPES, LIBND4J_TYPES);


}