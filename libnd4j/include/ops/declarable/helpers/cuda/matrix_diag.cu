#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// Created by GS <sgazeos@gmail.com> on 3/21/2018.
//

#include "ResultSet.h"
#include <ops/declarable/helpers/matrix_diag.h>
#include <Status.h>
#include <ShapeUtils.h>
#include <ShapeUtils.h>
#include <TAD.h>
#include <cuda_exception.h>

namespace nd4j {
namespace ops {
namespace helpers {


    template <typename T>
    static __global__ void matrixDiagKernel(void const* inputBuffer, void* outputBuffer, Nd4jLong numTads, Nd4jLong inputLength,
                                       Nd4jLong* tadOnlyInputShapeInfo,  Nd4jLong *tadInputOffsets,
                                       Nd4jLong* tadOnlyOutputShapeInfo, Nd4jLong *tadOutputOffsets) {
        int totalThreads = blockDim.x;
        for (Nd4jLong i = blockIdx.x; i < numTads; i += gridDim.x) {
            auto yOffset = tadInputOffsets[i];
            auto xOffset = tadOutputOffsets[i];
            for (Nd4jLong j = threadIdx.x; j < inputLength; j += totalThreads) {
                Nd4jLong coords[2] = {j, j};
                Nd4jLong tadOffset = shape::getOffset(0, shape::shapeOf(tadOnlyOutputShapeInfo), shape::stride(tadOnlyOutputShapeInfo), coords, 2);
                //shape::getIndexOffset(j, tadOnlyOutputShapeInfo, inputLength)
                *(reinterpret_cast<T*>(outputBuffer) + xOffset + tadOffset) = *(reinterpret_cast<T const*>(inputBuffer) + yOffset + shape::getIndexOffset(j, tadOnlyInputShapeInfo, inputLength));
            }
        }
    }
    //////////////////////////////////////////////////////////////////////////
    // Returns a batched matrix tensor with new batched diagonal values.
    // for detailed explanations please take a look on web page: https://www.tensorflow.org/api_docs/python/tf/matrix_set_diag

    template <typename T>
    static int _matrixDiag(graph::LaunchContext* context, const NDArray* input, NDArray* output) {
        hipStream_t* stream = context->getCudaStream();
        //auto listOut  = output->allTensorsAlongDimension({output->rankOf() - 2, output->rankOf() - 1});
        //auto listDiag = input->allTensorsAlongDimension({input->rankOf() - 1});

        //auto repeatDelta = shape::prodLong(newShape.data(), rank) / this->lengthOf();
        std::vector<int> dimsToExclude = ShapeUtils::evalDimsToExclude(input->rankOf(), {input->rankOf() - 1});
        const Nd4jLong numTads = ShapeUtils::getNumOfSubArrs(input->getShapeInfo(), dimsToExclude); //this->tensorsAlongDimension({dimension});
        //printf("Repeat delta %lld, numTads %lld\n", repeatDelta, numTads);
        //tadOnlyInputShapeInfo, tadInputOffsets, tadOnlyOutputShapeInfo, tadOutputOffsets;
        std::vector<int> inputDims({input->rankOf() - 1});
        shape::TAD tadInput(input->getShapeInfo(), inputDims.data(), inputDims.size());
        tadInput.createTadOnlyShapeInfo();
        tadInput.createOffsets();
        if (!input->isActualOnDeviceSide())
            input->syncToDevice();

        std::vector<int> outputDims({output->rankOf() - 2, output->rankOf() - 1});
        shape::TAD tadOutput(output->getShapeInfo(), outputDims.data(), outputDims.size());
        tadOutput.createTadOnlyShapeInfo();
        tadOutput.createOffsets();
        if (!input->isActualOnDeviceSide())
            input->syncToDevice();

        // prepare input arrays for prepareDataForCuda function
        std::vector<std::pair<void*,size_t>> hostData;
        hostData.emplace_back(tadInput.tadOnlyShapeInfo, shape::shapeInfoByteLength(tadInput.tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
        hostData.emplace_back(tadInput.tadOffsets, tadInput.numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets
        hostData.emplace_back(tadOutput.tadOnlyShapeInfo, shape::shapeInfoByteLength(tadOutput.tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
        hostData.emplace_back(tadOutput.tadOffsets, tadOutput.numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets
        std::vector<void*> devicePtrs(hostData.size(), nullptr);

        // create cuda stream and LaunchContext
        hipError_t cudaResult;
        //hipStream_t stream;
        //cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
        //hipStream_t* stream = this->getContext()->getCudaStream();
        // allocate required amount of global device memory and copy host data to it
//    cudaResult = allocateDeviceMem(*pLc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
        for(int i = 0; i < devicePtrs.size(); ++i) {
            cudaResult = hipMalloc(reinterpret_cast<void **>(&devicePtrs[i]), hostData[i].second);
            if(cudaResult != 0) throw cuda_exception::build("Cannot allocate memory for tads on device", cudaResult);
            cudaResult = hipMemcpy(devicePtrs[i], hostData[i].first, hostData[i].second, hipMemcpyHostToDevice);
            if(cudaResult != 0) throw cuda_exception::build("Cannot copy memory block for tads on device", cudaResult);
        }

        dim3 launchDims(256, 512, 8192);
        matrixDiagKernel<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(input->getSpecialBuffer(), output->getSpecialBuffer(), numTads, input->sizeAt(-1), (Nd4jLong*)devicePtrs[0], (Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], (Nd4jLong*)devicePtrs[3]);
        for(int i = 0; i < devicePtrs.size(); ++i) {
            cudaResult = hipFree(devicePtrs[i]);
            if(cudaResult != 0) throw cuda_exception::build("Cannot allocate memory for tads on device", cudaResult);
        }

        return Status::OK();
    }

    int matrixDiag(graph::LaunchContext* context, const NDArray* input, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), return _matrixDiag, (context, input, output), LIBND4J_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template int _matrixDiag, (graph::LaunchContext* context, const NDArray* input, NDArray* output), LIBND4J_TYPES);

}
}
}