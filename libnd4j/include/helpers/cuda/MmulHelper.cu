#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma (iuriish@yahoo.com)
//

#include <exceptions/cuda_exception.h>
#include <hipblas.h>
#include "../MmulHelper.h"


namespace nd4j { 

//////////////////////////////////////////////////////////////////////////////
// MXK x KxN = MxN
void MmulHelper::basicGemm(const NDArray* A, const NDArray* B, NDArray* C, double alpha, double beta) {

	const int M = A->sizeAt(0);
	const int K = A->sizeAt(1);
	const int N = B->sizeAt(1);

	const auto aType = A->dataType();
    const auto bType = B->dataType();
    const auto cType = C->dataType();
     
    hipblasStatus_t status;
    hipblasHandle_t handle;

    status = hipblasCreate(&handle); // initialize CUBLAS context
    if (status != HIPBLAS_STATUS_SUCCESS) throw cuda_exception::build("MmulHelper::basicGemm cuda failed !", status);

    status = hipblasSetStream(handle, *A->getContext()->getCudaStream());
    if (status != HIPBLAS_STATUS_SUCCESS) throw cuda_exception::build("MmulHelper::basicGemm cuda failed !", status);

    const bool AB(aType == bType), AC(aType == cType), ABC(AB && AC);

    // // choose appropriate cuda gemm api depending on data types
    // if(ABC && aType != DataType::DOUBLE && aType != DataType::FLOAT32 && aType != DataType::HALF) {
    //     throw std::runtime_error("MmulHelper::basicGemm cuda: not implemented yet for given data types !");
    // }
    if(ABC && aType == DataType::DOUBLE) {
    	status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, (double*)A->getSpecialBuffer(), M, (double*)B->getSpecialBuffer(), K, &beta, (double*)C->getSpecialBuffer(), M);
    }
    else if(ABC && aType == DataType::FLOAT32) {        
    	float alphaF(alpha), betaF(beta);
    	status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alphaF, (float*)A->getSpecialBuffer(), M, (float*)B->getSpecialBuffer(), K, &betaF, (float*)C->getSpecialBuffer(), M);
    }
    else if(ABC && aType == DataType::HALF) {
    	float16 alphaH(alpha), betaH(beta);
    	status = hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alphaH.data, (__half*)A->getSpecialBuffer(), M, (__half*)B->getSpecialBuffer(), K, &betaH.data, (__half*)C->getSpecialBuffer(), M);
    }    	
    else if(AB && aType == DataType::INT8 && cType == DataType::FLOAT32) {            
           float alphaF(alpha), betaF(beta);
           status = cublasSgemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alphaF, A->getSpecialBuffer(), HIP_R_8I, M, B->getSpecialBuffer(), HIP_R_8I, K, &betaF, C->getSpecialBuffer(), HIP_R_32F, M);            
    }
    else if(AB && aType == DataType::HALF && cType == DataType::FLOAT32) {
        float alphaF(alpha), betaF(beta);
        status = cublasSgemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alphaF, A->getSpecialBuffer(), HIP_R_16F, M, B->getSpecialBuffer(), HIP_R_16F, K, &betaF, C->getSpecialBuffer(), HIP_R_32F, M);
    }    
    else {
        auto systemFloatType = Environment::getInstance()->defaultFloatDataType();        
        NDArray *pA(const_cast<NDArray*>(A)), *pB(const_cast<NDArray*>(B)), *pC(const_cast<NDArray*>(C));
        if(aType != systemFloatType) {pA = new NDArray(A->getShapeInfo(), systemFloatType, true, A->getContext()); pA->assign(A); }
        if(bType != systemFloatType) {pB = new NDArray(B->getShapeInfo(), systemFloatType, true, B->getContext()); pB->assign(B); }
        if(cType != systemFloatType) {pC = new NDArray(C->getShapeInfo(), systemFloatType, true, C->getContext()); }
        
        MmulHelper::basicGemm(pA, pB, pC, alpha, beta);        

        if(pA != A) 
            delete pA;
        if(pB != B)
            delete pB;
        if(pC != C) {
            C->assign(pC);
            delete pC;
        }
    }	
   
   	if (status != HIPBLAS_STATUS_SUCCESS) throw cuda_exception::build("MmulHelper::basicGemm cuda failed !", status);

   	auto cudaResult = hipStreamSynchronize(*A->getContext()->getCudaStream());
   	if (cudaResult != 0) throw cuda_exception::build("MmulHelper::basicGemm cuda failed !", cudaResult);
   
    hipblasDestroy(handle);    

    A->tickReadDevice();
    B->tickReadDevice();
    C->tickWriteDevice();
}

//////////////////////////////////////////////////////////////////////////////
// MXK x KxN = MxN
template<typename X, typename Y, typename Z>
NDArray* MmulHelper::mmulMxM(const NDArray* A, const NDArray* B, NDArray* C, double alpha, double beta) {

	if(A->rankOf() != 2)
		throw std::runtime_error("MmulHelper::mmulMxM cuda: rank of A array is not equal 2 !");
	if(B->rankOf() != 2)
		throw std::runtime_error("MmulHelper::mmulMxM cuda: rank of B array is not equal 2 !");
	if(C != nullptr && C->rankOf() != 2)
		throw std::runtime_error("MmulHelper::mmulMxM cuda: rank of C array is not equal 2 !");

	const auto M = A->sizeAt(0);
	const auto K = A->sizeAt(1);
	const auto N = B->sizeAt(1);

	if(B->sizeAt(0) != K)
		throw std::runtime_error("MmulHelper::mmulMxM cuda: B array has wrong number of rows !");
	if(C != nullptr && C->sizeAt(0) != M)
		throw std::runtime_error("MmulHelper::mmulMxM cuda: C array has wrong number of rows !");
	if(C != nullptr && C->sizeAt(1) != N)
		throw std::runtime_error("MmulHelper::mmulMxM cuda: C array has wrong number of columns !");

	if(C == nullptr)
		C = new NDArray('f', {M,N}, DataTypeUtils::fromT<Z>(), A->getContext());

	if(!A->isActualOnDeviceSide())
		A->syncToDevice();
	if(!B->isActualOnDeviceSide())
		B->syncToDevice();
	if(!C->isActualOnDeviceSide())
		C->syncToDevice();

	NDArray *pA(const_cast<NDArray*>(A)), *pB(const_cast<NDArray*>(B)), *pC(const_cast<NDArray*>(C));

	if(A->ews() != 1 || A->ordering() == 'c')
		pA = pA->dup('f');
	if(B->ews() != 1 || B->ordering() == 'c')
		pB = pB->dup('f');
	if(C->ews() != 1 || C->ordering() == 'c')
		pC = pC->dup('f');

	MmulHelper::basicGemm(pA, pB, pC, alpha, beta);

    if(pC != C) {
    	C->assign(pC);
    	delete pC;
    }
    if(pA != A)
    	delete pA;
    if(pB != B)
    	delete pB;

	return C;
}


BUILD_TRIPLE_TEMPLATE(template nd4j::NDArray* MmulHelper::mmulMxM, (const NDArray* A, const NDArray* B, NDArray* C, double alpha, double beta), LIBND4J_TYPES, FLOAT_TYPES, FLOAT_TYPES);


}
