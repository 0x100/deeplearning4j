#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma, created on 16.04.2018
//

#include <ops/declarable/helpers/reverse.h>
#include <helpers/ShapeUtils.h>
#include <array/ResultSet.h>
#include <TAD.h>
#include <PointersManager.h>
#include <ConstantTadHelper.h>


namespace nd4j    {
namespace ops     {
namespace helpers {

    template <typename T>
    inline void __device__ indexSwap(T* arr, Nd4jLong idx1, Nd4jLong idx2) {
        T tmp = arr[idx1];
        arr[idx1] = arr[idx2];
        arr[idx2] = tmp;
    }
//    template <typename T>
//    void reverseArray(graph::LaunchContext* context, void* inArr, Nd4jLong *inShapeBuffer, void *result, Nd4jLong *zShapeBuffer, int numOfElemsToReverse = 0);

    /////////////////////////////////////////////////////////////////////////////////////
    template <typename T>
    static __global__ void reverseArrayInplaceKernel(void *input, Nd4jLong *inputShape, Nd4jLong numOfElemsToReverse) {
        const auto tid = blockIdx.x * gridDim.x + threadIdx.x;
        const auto step = gridDim.x * blockDim.x;
        __shared__ Nd4jLong length;
        __shared__ int linearStatus;
        __shared__ T* inputArr;
        if (threadIdx.x == 0) {
            length = shape::length(inputShape);
            linearStatus = shape::elementWiseStride(inputShape);
            inputArr = reinterpret_cast<T*>(input);
        }

        for (Nd4jLong e = tid; e < numOfElemsToReverse / 2; e += step) {
            if (linearStatus == 1) {
                auto idx = numOfElemsToReverse - e - 1;
                indexSwap(inputArr, e, idx);
            }
            else if (linearStatus > 1) {
                auto idx1 = (numOfElemsToReverse - e - 1) * linearStatus;
                Nd4jLong idx2 =  e * linearStatus;
                indexSwap(inputArr, idx1, idx2);
            }
            else {
                auto inOffset  = shape::getIndexOffset(e, inputShape, length);
                auto outOffset = shape::getIndexOffset(numOfElemsToReverse - e - 1, inputShape, length);
                indexSwap(inputArr, inOffset, outOffset);
            }
        }
    }

    template <typename T>
    static __global__ void reverseArrayKernel(void* input, Nd4jLong *inputShape, void* output, Nd4jLong *outputShape, Nd4jLong numOfElemsToReverse) {
        const auto tid = blockIdx.x * gridDim.x + threadIdx.x;
        const auto step = gridDim.x * blockDim.x;
        __shared__ Nd4jLong length;
        __shared__ int linearStatus;
        __shared__ T* inputArr;
        __shared__ T* outputArr;
        __shared__ char inputOrder, outputOrder;

        if (threadIdx.x == 0) {
            length = shape::length(inputShape);
            linearStatus = (shape::elementWiseStride(inputShape) == shape::elementWiseStride(outputShape)) && (inputOrder == outputOrder)? shape::elementWiseStride(inputShape):0;

            char inputOrder = shape::order(inputShape);
            char outputOrder = shape::order(outputShape);
            inputArr = reinterpret_cast<T*>(input);
            outputArr = reinterpret_cast<T*>(output);
        }
        __syncthreads();

        for (Nd4jLong e = tid; e < numOfElemsToReverse; e += step) {
            if (linearStatus == 1) {
                auto idx = numOfElemsToReverse - e - 1;
                outputArr[idx] = inputArr[e];
            }
            else if (linearStatus > 1) {
                auto idx1 = (numOfElemsToReverse - e - 1) * linearStatus;
                Nd4jLong idx2 =  e * linearStatus;
                outputArr[idx1] = inputArr[idx2];
            }
            else {
                auto inOffset  = shape::getIndexOffset(e, inputShape, length);
                auto outOffset = shape::getIndexOffset(numOfElemsToReverse - e - 1, outputShape, length);
                outputArr[outOffset] = inputArr[inOffset];
            }
        }
        //printf("\n");
    }

    template<typename T>
    static void reverseArray(graph::LaunchContext* context, NDArray* input, NDArray* output, int numOfElemsToReverse) {
        auto stream = context->getCudaStream();
        Nd4jLong numOfReverse = numOfElemsToReverse;
        if (numOfElemsToReverse == 0)
            numOfReverse = input->lengthOf();
        if (input == output) {
            reverseArrayInplaceKernel<T><<<256, 512, 8192, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), numOfReverse);
        }
        else {
            reverseArrayKernel<T><<<256, 512, 8192, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(), numOfReverse);
        }
    }


    ///////////////////////////////////////////////////////////////////
    template <typename T>
    static void _reverseSequence(const NDArray* input, const NDArray* seqLengths, NDArray* output, int seqDim, const int batchDim){

    }

    void reverseSequence(graph::LaunchContext* context, const NDArray* input, const NDArray* seqLengths, NDArray* output, int seqDim, const int batchDim) {
        BUILD_SINGLE_SELECTOR(input->dataType(), _reverseSequence, (input, seqLengths, output, seqDim, batchDim), LIBND4J_TYPES);
    }

    //////////////////////////////////////////////////////////////////////////
    void reverse(graph::LaunchContext* context, const NDArray* input, NDArray* output, const std::vector<int>* intArgs, bool isBackProp) {
        // we need to reverse axis only if that's new op
        std::vector<int> dimensions = isBackProp ? ShapeUtils::evalDimsToExclude(input->rankOf(), *intArgs) : *intArgs;
        std::vector<int> axis = ShapeUtils::evalDimsToExclude(input->rankOf(), dimensions);
        auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(input->getShapeInfo(), axis);
        auto packZ = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output->getShapeInfo(), axis);

        auto listOut = output->allTensorsAlongDimension(dimensions);
        auto listIn  = input->allTensorsAlongDimension(dimensions);

        NDArray *subArrIn, *subArrOut;

        for(int i = 0; i < listIn->size(); ++i) {               // listIn->size() = listOut->size()
            subArrIn   = listIn->at(i);
            subArrOut  = listOut->at(i);
            BUILD_SINGLE_SELECTOR(input->dataType(), reverseArray, (context, subArrIn, subArrOut, 0), LIBND4J_TYPES);
        }
        //BUILD_SINGLE_SELECTOR(input->dataType(), reverseArray, (context, const_cast<NDArray*>(input), output, (int)0), LIBND4J_TYPES);
        input->tickReadDevice();
        output->tickWriteDevice();
        delete listOut;
        delete listIn;
    }

BUILD_SINGLE_TEMPLATE(template void reverseArray, (graph::LaunchContext* context, NDArray *inArr, NDArray *outArr, int numOfElemsToReverse), LIBND4J_TYPES);

}
}
}

