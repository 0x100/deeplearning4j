#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// Created by Yurii Shyrma on 02.01.2018
//

#include <ops/declarable/helpers/stack.h>
#include <helpers/ShapeUtils.h>
#include <array/ResultSet.h>
#include <cuda_exception.h>
#include <TAD.h>

namespace nd4j {
namespace ops {
namespace helpers {

//	Nd4jLong* tadOnlyInputShapeInfo,  Nd4jLong *tadInputOffsets,
//	Nd4jLong* tadOnlyOutputShapeInfo, Nd4jLong *tadOutputOffsets

//	template <typename T>
//	static __global__ void stackKernelScalar(void const* inputList[], void* outputBuffer, Nd4jLong* outputShape, Nd4jLong outputLength) {
//		auto tid = blockIdx.x * blockDim.x + threadIdx.x;
//		int totalThreads = gridDim.x * blockDim.x;
//		//const auto resultLength = shape::length(outputShape);
//		for (Nd4jLong i = tid; i < outputLength; i += totalThreads) {
//			//auto yOffset = shape::subArrayOffset(i, outputShape, inputShape);
//			//printf(">> %lld\n", i);
//			auto xOffset = shape::getIndexOffset(i, outputShape, outputLength);
//			printf(">> %lld\n", xOffset);
//
//			//*(reinterpret_cast<T *>(outputBuffer) + xOffset) = *(reinterpret_cast<T const *>(inputList[xOffset]));
//		}
//	}

	template <typename T>
	static __global__ void stackKernel(void** inputList, void** inputShapeList, int inputListLength, Nd4jLong arrLen, void* outputBuffer, Nd4jLong* outputShape) {  //, Nd4jLong* tadShape, Nd4jLong* tadOffsets) {

		__shared__ int arrIdx, blocksPerArr;
		__shared__ T *x, *z;
		__shared__ Nd4jLong *zShapeInfo, *xShapeInfo, arrLenPerBlock, start, end, offsetZ;

		if (threadIdx.x == 0) {

			blocksPerArr = (gridDim.x + inputListLength - 1) / inputListLength;     // ceil
			arrIdx = blockIdx.x / blocksPerArr;

			x = reinterpret_cast<T*>(inputList[arrIdx]);
			z = reinterpret_cast<T*>(outputBuffer);
			xShapeInfo = reinterpret_cast<Nd4jLong*>(inputShapeList[arrIdx]);
			zShapeInfo = reinterpret_cast<Nd4jLong*>(outputShape);
			//arrLen = shape::length(xShapeInfo);

			arrLenPerBlock = (arrLen + blocksPerArr - 1) / blocksPerArr;  // ceil

			start = (blockIdx.x % blocksPerArr) * arrLenPerBlock;
			end   = (start + arrLenPerBlock) > arrLen ? arrLen : (start + arrLenPerBlock);
            offsetZ = arrLen * arrIdx;
			//if (blockIdx.x == 0)
			    //printf("Block: [%i]; arrLen: %i\n", blockIdx.x, arrLen);
			    printf("Block: [%i]; arrIdx: [%i]; start: [%lld]; end: [%lld], arrLen: [%lld], arrLenPerBlock: [%lld]; blocksPerArr: [%i]; \n", blockIdx.x, arrIdx, start, end, arrLen, arrLenPerBlock, blocksPerArr);
		}

		__syncthreads();

//		auto z = reinterpret_cast<T*>(outputBuffer);
//        for (Nd4jLong arrIdx = blockIdx.x; arrIdx < inputListLength; arrIdx += gridDim.x) {
//			auto x = reinterpret_cast<T*>(inputList[arrIdx]);
//			auto xShapeInfo = reinterpret_cast<Nd4jLong*>(inputShapeList[arrIdx]);
//			printf("Block: [%i]; arrIdx: [%lld]; arrLen: [%lld]; TAD offset: [%lld]\n", blockIdx.x, arrIdx, arrLen, tadOffsets[arrIdx]);

			for (Nd4jLong i = start + threadIdx.x; i < end; i += blockDim.x)
				z[offsetZ + shape::getIndexOffset(i, outputShape, arrLen)] = x[shape::getIndexOffset(i, xShapeInfo, arrLen)];
//		}

	}
	///////////////////////////////////////////////////////////////////
	template <typename T>
	static void stack_(graph::LaunchContext* context, const std::vector<NDArray*>& inArrs, NDArray* outArr, const int dim) {
		if(inArrs[0]->isScalar()) {
            outArr->lazyAllocateBuffer();

//#pragma omp parallel for
			for (size_t i = 0; i < inArrs.size(); ++i) {
                inArrs[i]->syncToHost();

                outArr->p(i, inArrs[i]->e<T>(0));
            }
			outArr->syncToDevice();
		}
		else {
			Nd4jLong **dInShapeInfo;
			void **dInBuffers;
			std::vector<void const*> inputList(inArrs.size());
			std::vector<Nd4jLong const*> inputShapeList(inArrs.size());
			auto stream = context->getCudaStream();

			for (size_t i = 0; i < inputList.size(); ++i) {
				inputList[i] = inArrs[i]->getSpecialBuffer();
				inputShapeList[i] = inArrs[i]->getSpecialShapeInfo();
			}
            std::vector<int> axis({dim});
            shape::TAD tadOutput(outArr->shapeInfo(), axis.data(), axis.size());
            tadOutput.createTadOnlyShapeInfo();
            tadOutput.createOffsets();

            // prepare input arrays for prepareDataForCuda function
//            Nd4jLong* dTadShape; //,size_t>> hostData;
//            Nd4jLong* dTadOffsets;

            // create cuda stream and LaunchContext
            hipError_t cudaResult;
//            cudaResult = hipMalloc(&dTadShape, shape::shapeInfoByteLength(tadOutput.tadOnlyShapeInfo));
//            if(cudaResult != 0) throw cuda_exception::build("helpers::stack_: Cannot allocate memory for tads on device", cudaResult);
//            cudaResult = hipMemcpy(dTadShape, tadOutput.tadOnlyShapeInfo, shape::shapeInfoByteLength(tadOutput.tadOnlyShapeInfo), hipMemcpyHostToDevice);
//            if(cudaResult != 0) throw cuda_exception::build("helpers::stack_: Cannot copy memory block for tads on device", cudaResult);
//            cudaResult = hipMalloc(&dTadOffsets, tadOutput.numTads * sizeof(Nd4jLong));
//            if(cudaResult != 0) throw cuda_exception::build("helpers::stack_: Cannot allocate memory for tad offsets on device", cudaResult);
//            cudaResult = hipMemcpy(dTadOffsets, tadOutput.tadOffsets, tadOutput.numTads * sizeof(Nd4jLong), hipMemcpyHostToDevice);
//            if(cudaResult != 0) throw cuda_exception::build("helpers::stack_: Cannot copy memory block for tads on device", cudaResult);

			cudaResult = hipMalloc(reinterpret_cast<void **>(&dInBuffers), inputList.size() * sizeof(void*));
			if(cudaResult != 0) throw cuda_exception::build("helpers::stack_: cannot allocate global memory on device", cudaResult);
			cudaResult = hipMalloc(reinterpret_cast<void **>(&dInShapeInfo), inputShapeList.size() * sizeof(Nd4jLong*));
			if(cudaResult != 0) throw cuda_exception::build("helpers::stack_: cannot allocate global memory on device", cudaResult);

			hipMemcpyAsync(dInBuffers,    inputList.data(),    inputList.size()  * sizeof(void*),       hipMemcpyHostToDevice, *stream);
			hipMemcpyAsync(dInShapeInfo,  inputShapeList.data(),  inputShapeList.size() * sizeof(Nd4jLong*),  hipMemcpyHostToDevice, *stream);

            dim3 launchDims(inArrs.size(), inArrs[0]->lengthOf(), 1024);
			//Nd4jLong lastDim = outArr->sizeAt(dim);

            //nd4j_printf("length: %i\n", inArrs[0]->lengthOf());
			stackKernel<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>((void**)dInBuffers, (void**)dInShapeInfo, inputList.size(), inArrs[0]->lengthOf(), outArr->specialBuffer(), outArr->specialShapeInfo()); //, dTadShape, dTadOffsets);
			cudaResult = hipFree(dInBuffers);
			if(cudaResult != 0)
				throw cuda_exception::build("helpers::stack_: cannot deallocate global memory on device for buffer list", cudaResult);
			cudaResult = hipFree(dInShapeInfo);
			if(cudaResult != 0)
				throw cuda_exception::build("helpers::stack_: cannot deallocate global memory on device for shape list", cudaResult);

//            cudaResult = hipFree(dTadShape);
//            if(cudaResult != 0)
//                throw cuda_exception::build("helpers::stack_: cannot deallocate global memory on device for tad shape", cudaResult);
//            cudaResult = hipFree(dTadOffsets);
//            if(cudaResult != 0)
//                throw cuda_exception::build("helpers::stack_: cannot deallocate global memory on device for tad offsets", cudaResult);

		}
	}

	void stack(graph::LaunchContext* context, const std::vector<NDArray*>& inArrs, NDArray* outArr, const int dim) {
		BUILD_SINGLE_SELECTOR(outArr->dataType(), stack_, (context, inArrs, outArr, dim), LIBND4J_TYPES);
	}

	BUILD_SINGLE_TEMPLATE(template void stack_ , (graph::LaunchContext* context, const std::vector<NDArray*>& inArrs, NDArray* outArr, const int dim), LIBND4J_TYPES);

}
}
}

