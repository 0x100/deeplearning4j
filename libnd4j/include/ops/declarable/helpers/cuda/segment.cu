#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author GS <sgazeos@gmail.com>
//

#include <ops/declarable/helpers/segment.h>
#include <NDArrayFactory.h>

namespace nd4j {
namespace ops {
namespace helpers {

    template <typename T, typename I>
    static __global__ void segmentMaxLinearKernel(void* input, Nd4jLong* inputShape, int* starts, int* lengths, Nd4jLong numOfClasses, void* output, Nd4jLong* outputShape) {
         __shared__ T* val;
         __shared__ Nd4jLong xLen, zLen, segment, zIndex;
         __shared__ T* x;
         __shared__ T* z;
         __shared__ int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
            segment = blockIdx.x / threadsPerSegment;
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);
            extern __shared__ unsigned char shmem[];
            val = reinterpret_cast<T*>(shmem);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);

            //[zIndex] =
            if (segment < numOfClasses) {
                zIndex = shape::getIndexOffset(segment, outputShape, zLen);
                start = starts[segment];
                finish = start + lengths[segment];
                //val[segment] = ;
                z[zIndex] = x[shape::getIndexOffset(start, inputShape, xLen)];
                val[segment] = z[zIndex];
            }

        }
        __syncthreads();
//         auto tid = threadIdx.x + blockIdx.x * blockDim.x;
//         auto step = blockDim.x * gridDim.x;

         for (auto e = start + threadIdx.x + 1; e < finish; e += blockDim.x) {
             auto xIndex = shape::getIndexOffset(e, inputShape, xLen);
             //val[segment] = nd4j::math::nd4j_max<T>(x[xIndex], val[segment]);
             if (val[segment] < x[xIndex])
                 val[segment] = x[xIndex];

         }
        __syncthreads();
        for (auto e = start + threadIdx.x + 1; e < finish; e += blockDim.x) {
            auto xIndex = shape::getIndexOffset(e, inputShape, xLen);
            //val[segment] = nd4j::math::nd4j_max<T>(x[xIndex], val[segment]);
            if (val[segment] < x[xIndex])
                val[segment] = x[xIndex];
        }
        __syncthreads();

        if (threadIdx.x == 0) {
            z[zIndex] = val[segment];
        }

    }

    template <typename T, typename I>
    static __global__ void segmentMinLinearKernel(void* input, Nd4jLong* inputShape, int* starts, int* lengths, Nd4jLong numOfClasses, void* output, Nd4jLong* outputShape) {
        __shared__ T* val;
        __shared__ Nd4jLong xLen, zLen, segment, zIndex;
        __shared__ T* x;
        __shared__ T* z;
        __shared__ int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
            segment = blockIdx.x / threadsPerSegment;
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);
            extern __shared__ unsigned char shmem[];
            val = reinterpret_cast<T*>(shmem);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);

            //[zIndex] =
            if (segment < numOfClasses) {
                zIndex = shape::getIndexOffset(segment, outputShape, zLen);
                start = starts[segment];
                finish = start + lengths[segment];
                //val[segment] = ;
                z[zIndex] = x[shape::getIndexOffset(start, inputShape, xLen)];
                val[segment] = z[zIndex];
            }

        }
        __syncthreads();
//         auto tid = threadIdx.x + blockIdx.x * blockDim.x;
//         auto step = blockDim.x * gridDim.x;

        for (auto e = start + threadIdx.x + 1; e < finish; e += blockDim.x) {
            auto xIndex = shape::getIndexOffset(e, inputShape, xLen);
            //val[segment] = nd4j::math::nd4j_max<T>(x[xIndex], val[segment]);
            if (val[segment] > x[xIndex])
                val[segment] = x[xIndex];

        }
        __syncthreads();
        for (auto e = start + threadIdx.x + 1; e < finish; e += blockDim.x) {
            auto xIndex = shape::getIndexOffset(e, inputShape, xLen);
            //val[segment] = nd4j::math::nd4j_max<T>(x[xIndex], val[segment]);
            if (val[segment] > x[xIndex])
                val[segment] = x[xIndex];
        }
        __syncthreads();

        if (threadIdx.x == 0) {
            z[zIndex] = val[segment];
        }

    }
    template <typename T, typename I>
    static __global__ void segmentSumLinearKernel(void* input, Nd4jLong* inputShape, int* starts, int* lengths, Nd4jLong numOfClasses, void* output, Nd4jLong* outputShape) {
        __shared__ T* val;
        __shared__ Nd4jLong xLen, zLen, segment, zIndex;
        __shared__ T* x;
        __shared__ T* z;
        __shared__ int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
            segment = blockIdx.x / threadsPerSegment;
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);
            extern __shared__ unsigned char shmem[];
            val = reinterpret_cast<T*>(shmem);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);

            //[zIndex] =
            if (segment < numOfClasses) {
                zIndex = shape::getIndexOffset(segment, outputShape, zLen);
                start = starts[segment];
                finish = start + lengths[segment];
                //val[segment] = ;
                z[zIndex] = x[shape::getIndexOffset(start, inputShape, xLen)];
                val[segment] = z[zIndex];
            }

        }
        __syncthreads();
//         auto tid = threadIdx.x + blockIdx.x * blockDim.x;
//         auto step = blockDim.x * gridDim.x;

        for (auto e = start + threadIdx.x + 1; e < finish; e += blockDim.x) {
            auto xIndex = shape::getIndexOffset(e, inputShape, xLen);
            //val[segment] = nd4j::math::nd4j_max<T>(x[xIndex], val[segment]);
            nd4j::math::atomics::nd4j_atomicAdd(&val[segment], x[xIndex]);
            //atomicAdd(&val[segment], x[xIndex]);

        }
//        __syncthreads();
//        for (auto e = start + threadIdx.x + 1; e < finish; e += blockDim.x) {
//            auto xIndex = shape::getIndexOffset(e, inputShape, xLen);
//            //val[segment] = nd4j::math::nd4j_max<T>(x[xIndex], val[segment]);
//            val[segment] += x[xIndex];
//        }
        __syncthreads();

        if (threadIdx.x == 0) {
            z[zIndex] = val[segment];
        }

    }
    template <typename T, typename I>
    static __global__ void segmentMeanLinearKernel(void* input, Nd4jLong* inputShape, int* starts, int* lengths, Nd4jLong numOfClasses, void* output, Nd4jLong* outputShape) {
        __shared__ T* val;
        __shared__ Nd4jLong xLen, zLen, segment, zIndex;
        __shared__ T* x;
        __shared__ T* z;
        __shared__ int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
            segment = blockIdx.x / threadsPerSegment;
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);
            extern __shared__ unsigned char shmem[];
            val = reinterpret_cast<T*>(shmem);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);

            //[zIndex] =
            if (segment < numOfClasses) {
                zIndex = shape::getIndexOffset(segment, outputShape, zLen);
                start = starts[segment];
                finish = start + lengths[segment];
                //val[segment] = ;
                z[zIndex] = x[shape::getIndexOffset(start, inputShape, xLen)];
                val[segment] = z[zIndex];
            }

        }
        __syncthreads();
//         auto tid = threadIdx.x + blockIdx.x * blockDim.x;
//         auto step = blockDim.x * gridDim.x;

        for (auto e = start + threadIdx.x + 1; e < finish; e += blockDim.x) {
            auto xIndex = shape::getIndexOffset(e, inputShape, xLen);
            //val[segment] = nd4j::math::nd4j_max<T>(x[xIndex], val[segment]);
            nd4j::math::atomics::nd4j_atomicAdd(&val[segment], x[xIndex]);
        }
//        __syncthreads();
//        for (auto e = start + threadIdx.x + 1; e < finish; e += blockDim.x) {
//            auto xIndex = shape::getIndexOffset(e, inputShape, xLen);
//            //val[segment] = nd4j::math::nd4j_max<T>(x[xIndex], val[segment]);
//            val[segment] += x[xIndex];
//        }
        __syncthreads();

        if (threadIdx.x == 0) {
            z[zIndex] = val[segment] / lengths[segment];
        }

    }

    template <typename I>
    static __global__ void fillUpSegmentsKernel(void* indices, Nd4jLong* indexShape, int numClasses, int* classesRangesStart, int* classesRangesLenghts) {
        __shared__ I* idxBuf;
        __shared__ Nd4jLong idxLen;
        __shared__ int* result;
        if (threadIdx.x == 0) {
            idxBuf = reinterpret_cast<I*>(indices);
            idxLen = shape::length(indexShape);
            extern __shared__ unsigned char shmem[];
            result = reinterpret_cast<int*>(shmem);
            result[0] = 0; //idxBuf[0];
        }
        __syncthreads();

        auto tid = threadIdx.x + blockDim.x * blockIdx.x;
        auto step = blockDim.x * gridDim.x;

        for (auto j = tid; j < idxLen; j += step) {
            auto pos = idxBuf[j];
             if (classesRangesStart[pos] == idxLen)
                 classesRangesStart[pos] = j;
            result[pos] = nd4j::math::nd4j_min<int>(classesRangesStart[pos], j);

            classesRangesStart[pos] = nd4j::math::nd4j_min<int>(classesRangesStart[pos], result[pos]);
            nd4j::math::atomics::nd4j_atomicAdd(&classesRangesLenghts[pos], 1);
        }
    }
    // segment max
    template <typename T, typename I>
    static void segmentMaxFunctor_(LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output) {
        //int numClasses = output->sizeAt(0);
        // if input is a vector: (as if in doc sample)
        //Nd4jLong idx = indices->e<Nd4jLong>(0);
        auto stream = context->getCudaStream();
        Nd4jLong numClasses = indices->e<Nd4jLong>(indices->lengthOf() - 1) + 1;
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numClasses});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numClasses});

        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);

        dim3 dims(256, 512, 256);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        fillUpSegmentsKernel<I><<<dims.x, dims.y, dims.z, *stream>>>(indices->specialBuffer(), indices->specialShapeInfo(), numClasses, begins, lengths);

        if (input->isVector()) {
            segmentMaxLinearKernel<T,I><<<numClasses, input->lengthOf(), numClasses * 32 + 32, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
//            std::vector<int> restDims(input->rankOf() - 1);
//            Nd4jLong loop_size = input->rankOf();
//            PRAGMA_OMP_PARALLEL_FOR
//            for (Nd4jLong e = 1; e < loop_size; e++)
//                restDims[e - 1] = e;
//
//            auto listOfTensors = input->allTensorsAlongDimension(restDims);
//            auto listOfOutTensors = output->allTensorsAlongDimension(restDims);
//
//            auto numOfClasses = output->sizeAt(0); // number of classes
//            std::vector<std::pair<NDArray*, int>> outputs(numOfClasses);
//            auto maxT = listOfOutTensors->at(idx);
//
//            //int pos = 0;
//            maxT->assign(listOfTensors->at(0));
//
//            for (Nd4jLong i = 1; i < indices->lengthOf(); i++) {
//                if (indices->e<int>(i) == idx) {
//
//                    for (Nd4jLong e = 0; e < maxT->lengthOf(); e++) {
//                        maxT->t<T>(e) = nd4j::math::nd4j_max(maxT->t<T>(e), listOfTensors->at(i)->t<T>(e));
//                    }
//                }
//                else {
//                    idx = indices->e<Nd4jLong>(i);
//                    maxT = listOfOutTensors->at(idx);
//                    maxT->assign(listOfTensors->at(i));
//                }
//
//            }
//            delete listOfTensors;
//            delete listOfOutTensors;
        }
    }

    // segmen min 
    template <typename T, typename I>
    static void segmentMinFunctor_(LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output) {
        auto stream = context->getCudaStream();
        Nd4jLong numClasses = indices->e<Nd4jLong>(indices->lengthOf() - 1) + 1;
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numClasses});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numClasses});

        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);

        dim3 dims(numClasses, indices->lengthOf(), numClasses * 32 + 32);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        fillUpSegmentsKernel<I><<<dims.x, dims.y, dims.z, *stream>>>(indices->specialBuffer(), indices->specialShapeInfo(), numClasses, begins, lengths);

        if (input->isVector()) {
            segmentMinLinearKernel<T,I><<<numClasses, input->lengthOf(), numClasses * 32 + 32, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {

        }
    }

    // segmen mean
    template <typename T, typename I>
    static void segmentMeanFunctor_(LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output) {
        auto stream = context->getCudaStream();
        Nd4jLong numClasses = indices->e<Nd4jLong>(indices->lengthOf() - 1) + 1;
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numClasses});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numClasses});

        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);

        dim3 dims(numClasses, indices->lengthOf(), numClasses * 32 + 32);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        fillUpSegmentsKernel<I><<<dims.x, dims.y, dims.z, *stream>>>(indices->specialBuffer(), indices->specialShapeInfo(), numClasses, begins, lengths);

        if (input->isVector()) {
            segmentMeanLinearKernel<T,I><<<numClasses, input->lengthOf(), numClasses * 32 + 32, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {

        }

    }

    template <typename T, typename I>
    static void segmentSumFunctor_(nd4j::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output) {
        auto stream = context->getCudaStream();
        Nd4jLong numClasses = indices->e<Nd4jLong>(indices->lengthOf() - 1) + 1;
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numClasses});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numClasses});

        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);

        dim3 dims(numClasses, indices->lengthOf(), numClasses * 32 + 32);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        fillUpSegmentsKernel<I><<<dims.x, dims.y, dims.z, *stream>>>(indices->specialBuffer(), indices->specialShapeInfo(), numClasses, begins, lengths);

        if (input->isVector()) {
            segmentSumLinearKernel<T,I><<<numClasses, input->lengthOf(), numClasses * 32 + 32, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {

        }

    }

    template <typename T>
    static void segmentProdFunctor_(NDArray* input, NDArray* indices, NDArray* output) {

    }

    template <typename T, typename I>
    static bool segmentIndicesValidate_(NDArray* indices, NDArray& aexpected, NDArray& aoutput) {
        return true;
    }

    void segmentMaxFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), segmentMaxFunctor_, (context, input, indices, output), NUMERIC_TYPES, INTEGER_TYPES);
    }

    void segmentMinFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), segmentMinFunctor_, (context, input, indices, output), NUMERIC_TYPES, INTEGER_TYPES);
    }

    void segmentMeanFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), segmentMeanFunctor_, (context, input, indices, output), NUMERIC_TYPES, INTEGER_TYPES);
    }

    void segmentSumFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), segmentSumFunctor_, (context, input, indices, output), NUMERIC_TYPES, INTEGER_TYPES);
    }

    void segmentProdFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), segmentProdFunctor_, (input, indices, output), NUMERIC_TYPES);
    }

    bool segmentIndicesValidate(nd4j::LaunchContext * context, NDArray* indices, NDArray& expected, NDArray& output) {
        BUILD_DOUBLE_SELECTOR(output.dataType(), indices->dataType(), return segmentIndicesValidate_, (indices, expected, output), NUMERIC_TYPES, INTEGER_TYPES);
    }

    BUILD_DOUBLE_TEMPLATE(template bool segmentIndicesValidate_, (NDArray*, NDArray&, NDArray&), NUMERIC_TYPES, INTEGER_TYPES);
    BUILD_SINGLE_TEMPLATE(template void segmentProdFunctor_, (NDArray* input, NDArray* indices, NDArray* output), NUMERIC_TYPES);
    BUILD_DOUBLE_TEMPLATE(template void segmentSumFunctor_, (nd4j::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output), NUMERIC_TYPES, INTEGER_TYPES);
    BUILD_DOUBLE_TEMPLATE(template void segmentMeanFunctor_, (nd4j::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output), NUMERIC_TYPES, INTEGER_TYPES);
    BUILD_DOUBLE_TEMPLATE(template void segmentMinFunctor_, (nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* output), NUMERIC_TYPES, INTEGER_TYPES);
    BUILD_DOUBLE_TEMPLATE(template void segmentMaxFunctor_, (LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output), NUMERIC_TYPES, INTEGER_TYPES);
    // -------------------------------------------------------------------------------------------------------------- //
    // Unsorted segment ops
    // -------------------------------------------------------------------------------------------------------------- //

    bool unsortedSegmentIndicesValidate(nd4j::LaunchContext * context, NDArray* indices, Nd4jLong expected, Nd4jLong& output) {
        return true;
    }

    template <typename T>
    static void unsortedSegmentMaxFunctor_(NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {

    }

    void unsortedSegmentMaxFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), unsortedSegmentMaxFunctor_, (input, indices, numOfClasses, output), NUMERIC_TYPES);
    }
    BUILD_SINGLE_TEMPLATE(template void unsortedSegmentMaxFunctor_, (NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output), NUMERIC_TYPES);

    template <typename T>
    static void unsortedSegmentMinFunctor_(NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {

    }

    void unsortedSegmentMinFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), unsortedSegmentMinFunctor_, (input, indices, numOfClasses, output),
                              NUMERIC_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template void unsortedSegmentMinFunctor_, (NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output), NUMERIC_TYPES);

    void unsortedSegmentMeanFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {

    }

    void unsortedSegmentSumFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {

    }

    void unsortedSegmentProdFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
      //  BUILD_SINGLE_SELECTOR(input->dataType(), unsortedSegmentProdFunctor_, (input, indices, numOfClasses, output), NUMERIC_TYPES);
    }
    //BUILD_SINGLE_TEMPLATE(template void unsortedSegmentProdFunctor_, (NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output), NUMERIC_TYPES);

    void unsortedSegmentSqrtNFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {

    }

    // -------------------------------------------------------------------------------------------------------------- //
    // Backpropagate ops helpers
    // -------------------------------------------------------------------------------------------------------------- //
    // Sorted backpropagate ops
    //

    // segment max
    template <typename T>
    int segmentMaxFunctorBP_(NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        return Status::OK();
    }

    int segmentMaxFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        BUILD_SINGLE_SELECTOR(output->dataType(), return segmentMaxFunctorBP_, (input, indices, gradOut, output), NUMERIC_TYPES);
    }
    BUILD_SINGLE_TEMPLATE(template int segmentMaxFunctorBP_, (NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output), NUMERIC_TYPES);

    // segmen min
    int segmentMinFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        return Status::OK();
    }

    // segmen mean
    int segmentMeanFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        return Status::OK();
    }

    int segmentSumFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        return Status::OK();
    }

    // -------------------------------------------------------------------------------------------------------------- //
    // Unsorted backpropagate segment ops
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T>
    static int unsortedSegmentMaxFunctorBP_(NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        return Status::OK();
    }

    int unsortedSegmentMaxFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        BUILD_SINGLE_SELECTOR(output->dataType(), return unsortedSegmentMaxFunctorBP_, (input, indices, gradOut, numOfClasses, output), NUMERIC_TYPES);
    }
    BUILD_SINGLE_TEMPLATE(template int unsortedSegmentMaxFunctorBP_, (NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output), NUMERIC_TYPES);

    template <typename T>
    static int unsortedSegmentMinFunctorBP_(NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        return Status::OK();
    }

    int unsortedSegmentMinFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        BUILD_SINGLE_SELECTOR(output->dataType(), return unsortedSegmentMinFunctorBP_, (input, indices, gradOut, numOfClasses, output), NUMERIC_TYPES);
    }
    BUILD_SINGLE_TEMPLATE(template int unsortedSegmentMinFunctorBP_, (NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output), NUMERIC_TYPES);

    int unsortedSegmentMeanFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        return Status::OK();
    }

    int unsortedSegmentSumFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        return Status::OK();
    }

    int unsortedSegmentProdFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        return Status::OK();
    }

//    template <typename T>
    int unsortedSegmentSqrtNFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        return Status::OK();
    }

//    int unsortedSegmentSqrtNFunctorBP(NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
//        BUILD_SINGLE_SELECTOR(output->dataType(), return unsortedSegmentSqrtNFunctorBP_, (input, indices, gradOut, numOfClasses, output), FLOAT_TYPES);
//    }
//    BUILD_SINGLE_TEMPLATE(template int unsortedSegmentSqrtNFunctorBP_, (NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output), FLOAT_TYPES);
}
}
}