#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma, created on 21.09.2018
// @author raver119@gmail.com
//


#include <helpers/TAD.h>
#include<ops/declarable/helpers/ismax.h>
#include<loops/special_kernels.h>
#include <helpers/DebugHelper.h>
#include <cuda_exception.h>

namespace nd4j 	  {
namespace ops 	  {
namespace helpers {

template <typename T>
static void ismax_(graph::LaunchContext* context, const NDArray* input, NDArray* output, const std::vector<int>& dimensions) {
    void* extraParams = nullptr;
    bool scalarCheat = false;
    if (extraParams == nullptr) {
        scalarCheat = true;
    }
    auto stream = context->getCudaStream();

    auto xRank = input->rankOf();
    auto zRank = output->rankOf();
    auto xType = input->dataType();
    auto zType = output->dataType();

    Nd4jLong* special = nullptr;
    if (dimensions.size() == 0) {
//        auto scalarShape = ShapeBuilders::createScalarShapeInfo(nd4j::DataType::INT64);
        /**
        * In case of vector-input for IsMax, it just turns into IndexReduce call + further filler call
        */
        auto indexMax = input->applyIndexReduce(indexreduce::IndexMax, dimensions);
        //NativeOpExecutioner::execIndexReduceScalar(context, indexreduce::IndexMax, nullptr, input->getShapeInfo(), input->getSpecialBuffer(), input->getSpecialShapeInfo(), extraParams, nullptr, scalarShape, special, nullptr);
        //Nd4jLong maxIdx = -119;
        //checkCudaErrors(hipStreamSynchronize(*stream));
        //hipMemcpyAsync(&maxIdx, special, sizeof(Nd4jLong), hipMemcpyDeviceToHost, *stream);
        //checkCudaErrors(hipStreamSynchronize(*stream));
        int targetIdx = 0;

        if (input->ordering() == 'c' || input->ordering() == 'f' && indexMax->e<Nd4jLong>(0) * shape::stride(input->getShapeInfo())[input->rankOf() - 1] >= input->lengthOf())
            targetIdx = indexMax->e<Nd4jLong>(0);
        else
            targetIdx = indexMax->e<Nd4jLong>(0) * shape::stride(input->getShapeInfo())[input->rankOf() - 1];

        dim3 launchDims(1, 512, 1024);
        BUILD_SINGLE_SELECTOR(zType, fillIsMaxGeneric, (launchDims, stream, output->specialBuffer(), output->lengthOf(), targetIdx), LIBND4J_TYPES);

        nd4j::DebugHelper::checkErrorCode(stream, "Legacy IsMax(...) failed");

        //delete[] scalarShape;
        delete indexMax;
    } else {
        Nd4jLong* hostYShapeInfo  = nullptr;
        Nd4jLong* hostTShapeInfo  = nullptr;
        Nd4jLong* tadMaxShapeInfo = nullptr;
        Nd4jLong* tadMaxOffsets   = nullptr;
        int* dimension = nullptr;
        int dimensionLength = 0;
        std::vector<int> copy(dimensions);
        shape::TAD tadOutput(input->getShapeInfo(), copy.data(), copy.size());
        tadOutput.createTadOnlyShapeInfo();
        tadOutput.createOffsets();
        hipError_t err = hipMalloc(&tadMaxShapeInfo, shape::shapeInfoByteLength(tadOutput.tadOnlyShapeInfo));
        if (0 != err)
            throw cuda_exception::build("helpers::ismax_: Cannot allocate memory for tad shape.", err);
        err = hipMalloc(&tadMaxOffsets, tadOutput.numTads * sizeof(Nd4jLong));
        if (0 != err)
            throw cuda_exception::build("helpers::ismax_: Cannot allocate memory for tad offset.", err);
        err = hipMemcpy(tadMaxShapeInfo, tadOutput.tadOnlyShapeInfo, shape::shapeInfoByteLength(tadOutput.tadOnlyShapeInfo), hipMemcpyHostToDevice);
        if (0 != err)
            throw cuda_exception::build("helpers::ismax_: Cannot copy memory for tad shape.", err);
        err = hipMemcpy(tadMaxOffsets, tadOutput.tadOffsets, tadOutput.numTads * sizeof(Nd4jLong), hipMemcpyHostToDevice);
        if (0 != err)
            throw cuda_exception::build("helpers::ismax_: Cannot copy memory for tad offset.", err);
        auto indexMaxArr = input->applyIndexReduce(indexreduce::IndexMax, dimensions);
        indexMaxArr->printIndexedBuffer("Index max!!!");
        // we call for IMax on specified dimension
        //NativeOpExecutioner::execIndexReduce(context, indexreduce::IndexMax, nullptr, input->getShapeInfo(), input->getSpecialBuffer(), input->getSpecialShapeInfo(), extraParams, nullptr, hostTShapeInfo, special, hostYShapeInfo, const_cast<int*>(dimensions.data()), (int)dimensions.size(), nullptr, nullptr);

        //DEBUG_KERNEL(stream, opNum);

        dim3 launchDims(256, 256, 16384);

        // at this point, all IMax indexes are gathered, and we execute filler
        BUILD_SINGLE_SELECTOR(zType, fillDimensionalIsMaxGeneric, (launchDims, stream, indexMaxArr->specialBuffer(), output->specialBuffer(), output->specialShapeInfo(), tadMaxShapeInfo, const_cast<int*>(dimensions.data()), (int)dimensions.size(), tadMaxOffsets), LIBND4J_TYPES);
        err = hipFree(tadMaxOffsets);
        if (0 != err)
            throw cuda_exception::build("helpers::ismax_: Cannot deallocate memory for tad offset.", err);
        err = hipFree(tadMaxShapeInfo);
        if (0 != err)
            throw cuda_exception::build("helpers::ismax_: Cannot deallocate memory for tad shape.", err);

        nd4j::DebugHelper::checkErrorCode(stream, "Legacy IsMax(...) failed");
        delete indexMaxArr;
    }
}


void ismax(graph::LaunchContext* context, const NDArray *input, NDArray *output, const std::vector<int>& dimensions) {
    BUILD_SINGLE_SELECTOR(input->dataType(), ismax_, (context, input, output, dimensions), LIBND4J_TYPES);
}

BUILD_SINGLE_TEMPLATE(template void ismax_, (graph::LaunchContext* context, const NDArray *input, NDArray *output, const std::vector<int>& dimensions), LIBND4J_TYPES);

}
}
}

