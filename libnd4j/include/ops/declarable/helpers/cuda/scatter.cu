#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author raver119@gmail.com
//

#include <ops/declarable/helpers/scatter.h>
#include <numeric>
#include <helpers/ShapeUtils.h>
#include <helpers/PointersManager.h>
#include <TAD.h>
#include <helpers/ConstantShapeHelper.h>
#include <helpers/ConstantTadHelper.h>


namespace nd4j {
    namespace ops {
        namespace helpers {
            template<typename T, bool locking>
            __global__ static void scatterCuda(const int opCode, const int numOfSubArrs,
                                                     void* vx, const Nd4jLong *xShapeInfo, const Nd4jLong *xOffsets,
                                                     void* vy, const Nd4jLong *yShapeInfo, const Nd4jLong *yOffsets,
                                                     const int* indexes, unsigned int arrLenX, unsigned int arrLenY) {

                __shared__ T *x, *y;

                if (locking) {

                    for (int e = 0; e < numOfSubArrs; e++) {

                        const auto xIndex = indexes[e];
                        const bool isOwner = xIndex < gridDim.x ? blockIdx.x == xIndex : blockIdx.x == xIndex % gridDim.x;

                        if (!isOwner)
                            continue;

                        if (threadIdx.x == 0) {
                            x = reinterpret_cast<T *>(vx) + xOffsets[xIndex];
                            y = reinterpret_cast<T *>(vy) + yOffsets[e];
                        }
                        __syncthreads();

                        for (Nd4jLong i = threadIdx.x; i < arrLenX; i += blockDim.x) {

                            const auto xOffset = shape::getIndexOffset(i, xShapeInfo, arrLenX);
                            const auto yOffset = shape::getIndexOffset(i, yShapeInfo, arrLenY);

                            switch (opCode) {
                                case pairwise::Add:
                                    x[xOffset] += y[yOffset];
                                    break;
                                case pairwise::Subtract:
                                    x[xOffset] -= y[yOffset];
                                    break;
                                case pairwise::Multiply:
                                    x[xOffset] *= y[yOffset];
                                    break;
                                case pairwise::Divide:
                                    x[xOffset] /= y[yOffset];
                                    break;
                                case pairwise::ReverseSubtract:
                                    x[xOffset] = y[yOffset] - x[xOffset];
                                    break;
                                case pairwise::ReverseDivide:
                                    x[xOffset] = y[yOffset] / x[xOffset];
                                    break;
                                case pairwise::Copy2:
                                case pairwise::CopyPws:
                                    x[xOffset] = y[yOffset];
                                    break;
                                default:
                                    continue;
                            }
                        }
                        __syncthreads();
                    }
                } else {
                    for (int e = blockIdx.x; e < numOfSubArrs; e+= gridDim.x) {

                        if (threadIdx.x == 0) {
                            const auto xIndex = indexes[e];
                            x = reinterpret_cast<T *>(vx) + xOffsets[xIndex];
                            y = reinterpret_cast<T *>(vy) + yOffsets[e];
                        }
                        __syncthreads();

                        for (Nd4jLong i = threadIdx.x; i < arrLenX; i += blockDim.x) {
                            const auto xOffset = shape::getIndexOffset(i, xShapeInfo, arrLenX);
                            const auto yOffset = shape::getIndexOffset(i, yShapeInfo, arrLenY);

                            switch (opCode) {
                                case pairwise::Add:
                                    x[xOffset] += y[yOffset];
                                    break;
                                case pairwise::Subtract:
                                    x[xOffset] -= y[yOffset];
                                    break;
                                case pairwise::Multiply:
                                    x[xOffset] *= y[yOffset];
                                    break;
                                case pairwise::Divide:
                                    x[xOffset] /= y[yOffset];
                                    break;
                                case pairwise::ReverseSubtract:
                                    x[xOffset] = y[yOffset] - x[xOffset];
                                    break;
                                case pairwise::ReverseDivide:
                                    x[xOffset] = y[yOffset] / x[xOffset];
                                    break;
                                case pairwise::Copy2:
                                case pairwise::CopyPws:
                                    x[xOffset] = y[yOffset];
                                    break;
                                default:
                                    continue;
                            }
                        }
                        __syncthreads();
                    }
                }
            }


            template <typename T>
            void scatter_(graph::LaunchContext *context, pairwise::Ops op, const NDArray& indices, const NDArray& updates, NDArray& output, const bool lock) {
                std::vector<int> dims = {0};
                auto inverted = ShapeUtils::evalDimsToExclude(output.rankOf(), dims);

                auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(output.getShapeInfo(), inverted);
                auto packY = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(updates.getShapeInfo(), inverted);

                auto psX = packX.specialShapeInfo();
                auto psY = packY.specialShapeInfo();

                PointersManager manager(context, "scatter");

                auto poX = packX.specialOffsets();
                auto poY = packY.specialOffsets();

                NDArray::prepareSpecialUse({&output}, {&updates, &indices});

                unsigned int tadLengthX = shape::length(packX.primaryShapeInfo());
                unsigned int tadLengthY = shape::length(packY.primaryShapeInfo());
                if (tadLengthX != tadLengthY)
                    throw std::runtime_error("scatter: Lengths of TADs must be equal");

                auto blockSize = nd4j::math::nd4j_max<int>(32, nd4j::math::nd4j_min<int>(tadLengthX, 1024));

                if (lock)
                    scatterCuda<T, true><<<512, blockSize, 1024, *context->getCudaStream()>>>(op, indices.lengthOf(), output.getSpecialBuffer(), psX, poX, updates.getSpecialBuffer(), psY, poY, reinterpret_cast<int *>(indices.getSpecialBuffer()), tadLengthX, tadLengthY);
                else
                    scatterCuda<T, false><<<512, blockSize, 1024, *context->getCudaStream()>>>(op, indices.lengthOf(), output.getSpecialBuffer(), psX, poX, updates.getSpecialBuffer(), psY, poY, reinterpret_cast<int *>(indices.getSpecialBuffer()), tadLengthX, tadLengthY);

                 NDArray::registerSpecialUse({&output}, {&updates, &indices});
                manager.synchronize();
            }

            void scatter(graph::LaunchContext *context, pairwise::Ops op, const NDArray& indices, const NDArray& updates, NDArray& output, const bool lock) {
                BUILD_SINGLE_SELECTOR(output.dataType(), scatter_, (context, op, indices, updates, output, lock), LIBND4J_TYPES);
            }

            void scatterND(graph::LaunchContext *context, pairwise::Ops op, const NDArray& indices, const NDArray& updates, NDArray& output, const bool lock) {

            }

            void scatterForLoss(graph::LaunchContext *context, const NDArray& indices, const NDArray& updates, NDArray& output, const bool calcGrad) {

            }
        }
    }
}