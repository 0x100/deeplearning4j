/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 06.02.2019
//

#include <PointersManager.h>
#include <exceptions/cuda_exception.h>
#include <logger.h>
#include <memory/Workspace.h>

namespace nd4j {

//////////////////////////////////////////////////////////////////////////
PointersManager::PointersManager(nd4j::graph::LaunchContext *context)  {
        _context = context;
}

//////////////////////////////////////////////////////////////////////////
void* PointersManager::replicatePointer(const void* src, const size_t numberOfBytes, const std::string& message) {

	void* dst = nullptr;
	if (_context->getWorkspace() == nullptr) {
        hipError_t cudaResult = hipMalloc(reinterpret_cast<void **>(&dst), numberOfBytes);
        if (cudaResult != 0)
            throw cuda_exception::build(message + ": cannot allocate of global memory on device!", cudaResult);
    } else {
	    dst = _context->getWorkspace()->allocateBytes(nd4j::memory::MemoryType::DEVICE, numberOfBytes);
	}

    if (_context != nullptr)
        hipMemcpyAsync(dst, src, numberOfBytes, hipMemcpyHostToDevice, *_context->getCudaStream());
    else
        hipMemcpy(dst, src, numberOfBytes, hipMemcpyHostToDevice);

    _pOnGlobMem.emplace_back(dst);
    
    return dst;
}

//////////////////////////////////////////////////////////////////////////
void PointersManager::synchronize(const std::string& message) const {
    if (_context != nullptr) {
        hipError_t cudaResult = hipStreamSynchronize(*_context->getCudaStream());
        if (cudaResult != 0)
            throw cuda_exception::build(message + ": cuda stream synchronization failed !", cudaResult);
    } else {
        nd4j_printf("<%s> syncStream isn't possible: no stream set!", message.c_str());
    }
}

//////////////////////////////////////////////////////////////////////////
PointersManager::~PointersManager() {
    
    for (auto& p :_pOnGlobMem)
        hipFree(p);
}


}
