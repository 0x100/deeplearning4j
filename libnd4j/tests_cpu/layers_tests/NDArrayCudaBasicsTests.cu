#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

 //
 // @author raver119@gmail.com
 //

#include "testlayers.h"
#include <NDArray.h>
#include <NDArrayFactory.h>
#include <Context.h>
#include <Node.h>
#include <graph/Variable.h>
#include <graph/VariableSpace.h>
#include <graph/LaunchContext.h>
#include <specials_cuda.h>
#include <TAD.h>

#include <hip/hip_runtime.h>
#include <cuda_launch_config.h>

using namespace nd4j;
using namespace nd4j::graph;

class NDArrayCudaBasicsTests : public testing::Test {
public:

};


//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestAdd_1) {
    // allocating host-side arrays
    auto x = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto z = NDArrayFactory::create<double>('c', { 5 }, {10, 10, 10, 10, 10});

    auto exp = NDArrayFactory::create<double>('c', { 5 }, { 2, 4, 6, 8, 10 });

    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);

    Nd4jPointer nativeStream = (Nd4jPointer)malloc(sizeof(hipStream_t));
    CHECK_ALLOC(nativeStream, "Failed to allocate memory for new CUDA stream");
    hipError_t dZ = hipStreamCreate(reinterpret_cast<hipStream_t *>(&nativeStream));
    auto stream = reinterpret_cast<hipStream_t *>(&nativeStream);

    //hipMemcpyAsync(devBufferPtrX, x.buffer(), x.lengthOf() * x.sizeOfT(), hipMemcpyHostToDevice, *stream);
    //hipMemcpyAsync(devShapePtrX, x.shapeInfo(), shape::shapeInfoByteLength(x.shapeInfo()), hipMemcpyHostToDevice, *stream);

    LaunchContext lc(stream, nullptr, nullptr);
    NativeOpExecutioner::execPairwiseTransform(&lc, pairwise::Add, x.buffer(), x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(), y.buffer(), y.shapeInfo(), y.specialBuffer(), y.specialShapeInfo(), z.buffer(), z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(), nullptr);
    auto res = hipStreamSynchronize(*stream);
    ASSERT_EQ(0, res);
    //double* localBuffer = ;
    hipMemcpy(z.buffer(), z.specialBuffer(), z.lengthOf() * z.sizeOfT(), hipMemcpyDeviceToHost);
    res = hipStreamSynchronize(*stream);
    ASSERT_EQ(0, res);
    x.printBuffer("X = ");
    y.printBuffer("Y = ");
    z.printBuffer("Result out");

    //
    // hipFree(devBufferPtrX);
    //hipFree(devBufferPtrZ);
    //hipFree(devShapePtrX);

    for (int e = 0; e < z.lengthOf(); e++) {
        ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
    }
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestAdd_2) {
    // allocating host-side arrays
    auto x = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto z = NDArrayFactory::create<double>('c', { 5 });

    auto exp = NDArrayFactory::create<double>('c', { 5 }, { 2, 4, 6, 8, 10 });

    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);

    Nd4jPointer nativeStream = (Nd4jPointer)malloc(sizeof(hipStream_t));
    CHECK_ALLOC(nativeStream, "Failed to allocate memory for new CUDA stream");
    hipError_t dZ = hipStreamCreate(reinterpret_cast<hipStream_t *>(&nativeStream));
    auto stream = reinterpret_cast<hipStream_t *>(&nativeStream);

    //hipMemcpyAsync(devBufferPtrX, x.buffer(), x.lengthOf() * x.sizeOfT(), hipMemcpyHostToDevice, *stream);
    //hipMemcpyAsync(devShapePtrX, x.shapeInfo(), shape::shapeInfoByteLength(x.shapeInfo()), hipMemcpyHostToDevice, *stream);

    LaunchContext lc(stream, *stream, nullptr, nullptr);
    NativeOpExecutioner::execPairwiseTransform(&lc, pairwise::Add, nullptr, x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(), nullptr, y.shapeInfo(), y.specialBuffer(), y.specialShapeInfo(), nullptr, z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(), nullptr);
    auto res = hipStreamSynchronize(*stream);
    ASSERT_EQ(0, res);

    hipMemcpyAsync(z.buffer(), z.specialBuffer(), z.lengthOf() * z.sizeOfT(), hipMemcpyDeviceToHost, *stream);
    res = hipStreamSynchronize(*stream);
    ASSERT_EQ(0, res);
    z.printBuffer("2Result out");
    //hipFree(devBufferPtrX);
    //hipFree(devBufferPtrZ);
    //hipFree(devShapePtrX);

    for (int e = 0; e < z.lengthOf(); e++) {
        ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
    }
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestAdd_3) {
    // allocating host-side arrays
    auto x = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto z = NDArrayFactory::create<double>('c', { 5 }, {10, 10, 10, 10, 10});

    auto exp = NDArrayFactory::create<double>('c', { 5 }, { 2, 4, 6, 8, 10 });

    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);

    Nd4jPointer nativeStream = (Nd4jPointer)malloc(sizeof(hipStream_t));
    CHECK_ALLOC(nativeStream, "Failed to allocate memory for new CUDA stream");
    hipError_t dZ = hipStreamCreate(reinterpret_cast<hipStream_t *>(&nativeStream));
    auto stream = reinterpret_cast<hipStream_t *>(&nativeStream);

    //hipMemcpyAsync(devBufferPtrX, x.buffer(), x.lengthOf() * x.sizeOfT(), hipMemcpyHostToDevice, *stream);
    //hipMemcpyAsync(devShapePtrX, x.shapeInfo(), shape::shapeInfoByteLength(x.shapeInfo()), hipMemcpyHostToDevice, *stream);

    LaunchContext lc(stream, *stream, nullptr, nullptr);
    NativeOpExecutioner::execPairwiseTransform(&lc, pairwise::Add, x.buffer(), x.shapeInfo(), x.specialBuffer(), x.specialShapeInfo(), y.buffer(), y.shapeInfo(), y.specialBuffer(), y.specialShapeInfo(), z.buffer(), z.shapeInfo(), z.specialBuffer(), z.specialShapeInfo(), nullptr);
    auto res = hipStreamSynchronize(*stream);
    ASSERT_EQ(0, res);
    //double* localBuffer = ;
    hipMemcpy(z.buffer(), z.specialBuffer(), z.lengthOf() * z.sizeOfT(), hipMemcpyDeviceToHost);
    res = hipStreamSynchronize(*stream);
    ASSERT_EQ(0, res);
    x.printBuffer("3X = ");
    y.printBuffer("3Y = ");
    z.printBuffer("3Result out");

    //
    // hipFree(devBufferPtrX);
    //hipFree(devBufferPtrZ);
    //hipFree(devShapePtrX);

    for (int e = 0; e < z.lengthOf(); e++) {
        ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
    }
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestAdd_4) {
    // allocating host-side arrays
    auto x = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto z = NDArrayFactory::create<double>('c', { 5 });

    auto exp = NDArrayFactory::create<double>('c', { 5 }, { 2, 4, 6, 8, 10 });

    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);
    x.applyPairwiseTransform(pairwise::Add, &y, &z, nullptr);
    z.syncToHost();
    x.printBuffer("3X = ");
    y.printBuffer("3Y = ");
    z.printBuffer("3Result out");

    //
    // hipFree(devBufferPtrX);
    //hipFree(devBufferPtrZ);
    //hipFree(devShapePtrX);

    for (int e = 0; e < z.lengthOf(); e++) {
        ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
    }
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestMultiply_1) {
    // allocating host-side arrays
    auto x = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto z = NDArrayFactory::create<double>('c', { 5 });

    auto exp = NDArrayFactory::create<double>('c', { 5 }, { 1, 4, 9, 16, 25 });

    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);
    x.applyPairwiseTransform(pairwise::Multiply, &y, &z, nullptr);
    z.syncToHost();
    x.printBuffer("3X = ");
    y.printBuffer("3Y = ");
    z.printBuffer("3Result out");

    //
    // hipFree(devBufferPtrX);
    //hipFree(devBufferPtrZ);
    //hipFree(devShapePtrX);

    for (int e = 0; e < z.lengthOf(); e++) {
        ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
    }
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestMultiply_2) {
    // allocating host-side arrays
    auto x = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    auto y = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
    NDArray z('c', { 5 }, nd4j::DataType::DOUBLE);

    auto exp = NDArrayFactory::create<double>('c', { 5 }, { 1, 4, 9, 16, 25 });

    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);
    x.applyPairwiseTransform(pairwise::Multiply, &y, &z, nullptr);
    x.printBuffer("3X = ");
    y.printBuffer("3Y = ");
    z.syncToHost();
    z.printBuffer("3Result out");

    //
    // hipFree(devBufferPtrX);
    //hipFree(devBufferPtrZ);
    //hipFree(devShapePtrX);

    for (int e = 0; e < z.lengthOf(); e++) {
        ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
    }
}

//////////////////////////////////////////////////////////////////////////
TEST_F(NDArrayCudaBasicsTests, TestMultiply_3) {
    // allocating host-side arrays
    NDArray x('c', { 5 }, { 1, 2, 3, 4, 5}, nd4j::DataType::DOUBLE);
    NDArray y('c', { 5 }, { 1., 2., 3., 4., 5.}, nd4j::DataType::DOUBLE);
    auto z = NDArrayFactory::create<double>('c', { 5 });

    auto exp = NDArrayFactory::create<double>('c', { 5 }, { 1, 4, 9, 16, 25 });

    // making raw buffers
    //Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
    //hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
    //ASSERT_EQ(0, res);
    //res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
    //ASSERT_EQ(0, res);
    x.applyPairwiseTransform(pairwise::Multiply, &y, &z, nullptr);
    //x.printBuffer("23X = ");
    //y.printBuffer("23Y = ");
    z.syncToHost();
    z.printBuffer("23Result out");

    //
    // hipFree(devBufferPtrX);
    //hipFree(devBufferPtrZ);
    //hipFree(devShapePtrX);

    for (int e = 0; e < z.lengthOf(); e++) {
        ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
    }
}
