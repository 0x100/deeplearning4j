#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

 //
 // @author raver119@gmail.com
 //

#include "testlayers.h"
#include <NDArray.h>
#include <NDArrayFactory.h>
#include <Context.h>
#include <Node.h>
#include <graph/Variable.h>
#include <graph/VariableSpace.h>
#include <specials_cuda.h>
#include <TAD.h>
#include <MmulHelper.h>

#include <hip/hip_runtime.h>
#include <cuda_launch_config.h>

using namespace nd4j;
using namespace nd4j::graph;

class CudaBasicsTests2 : public testing::Test {
public:

};

//////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests2, mmulMxM_1) {

	const Nd4jLong M = 3;
	const Nd4jLong K = 4;
	const Nd4jLong N = 5;

	NDArray a('f', {M,K}, {1.2,1.1,1.0,0.9,0.8,0.7,0.5,0.4,0.3,0.2,0.1,0}, nd4j::DataType::DOUBLE);
	NDArray b('f', {K,N}, {1,-2,3,-4,5,-6,7,-8,9,-10,11,-12,13,-14,15,-16,17,-18,19,-20}, nd4j::DataType::DOUBLE);
	NDArray c('f', {M,N}, nd4j::DataType::DOUBLE);

	NDArray exp('f', {M,N}, {0.1, 0.3, 0.5, 2.5, 2.7, 2.9, 4.9, 5.1, 5.3, 7.3, 7.5, 7.7, 9.7, 9.9, 10.1}, nd4j::DataType::DOUBLE);

	nd4j::MmulHelper::mmulMxM<double,double,double>(&a, &b, &c, 1., 0.);	
	// c.printIndexedBuffer();

	ASSERT_TRUE(c.equalsTo(&exp));
}

//////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests2, mmulMxM_2) {

	const Nd4jLong M = 3;
	const Nd4jLong K = 4;
	const Nd4jLong N = 5;

	NDArray a('c', {M,K}, {1.2,1.1,1.0,0.9,0.8,0.7,0.5,0.4,0.3,0.2,0.1,0}, nd4j::DataType::DOUBLE);
	NDArray b('f', {K,N}, {1,-2,3,-4,5,-6,7,-8,9,-10,11,-12,13,-14,15,-16,17,-18,19,-20}, nd4j::DataType::DOUBLE);
	NDArray c('f', {M,N}, nd4j::DataType::DOUBLE);

	NDArray exp('f', {M,N}, {-1.6, -0.7, 0.2, -0.8, 0.1, 1., -0., 0.9, 1.8, 0.8, 1.7, 2.6, 1.6, 2.5, 3.4}, nd4j::DataType::DOUBLE);

	nd4j::MmulHelper::mmulMxM<double,double,double>(&a, &b, &c, 1., 0.);		

	ASSERT_TRUE(c.equalsTo(&exp));
}

//////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests2, mmulMxM_3) {

	const Nd4jLong M = 3;
	const Nd4jLong K = 4;
	const Nd4jLong N = 5;

	NDArray a('f', {M,K}, {1.2,1.1,1.0,0.9,0.8,0.7,0.5,0.4,0.3,0.2,0.1,0}, nd4j::DataType::DOUBLE);
	NDArray b('c', {K,N}, {1,-2,3,-4,5,-6,7,-8,9,-10,11,-12,13,-14,15,-16,17,-18,19,-20}, nd4j::DataType::DOUBLE);
	NDArray c('f', {M,N}, nd4j::DataType::DOUBLE);

	NDArray exp('f', {M,N}, {-1.9, -0.9, 0.1, 1.3, 0.3, -0.7, -0.7, 0.3, 1.3, 0.1, -0.9, -1.9, 0.5, 1.5, 2.5}, nd4j::DataType::DOUBLE);

	nd4j::MmulHelper::mmulMxM<double,double,double>(&a, &b, &c, 1., 0.);	

	ASSERT_TRUE(c.equalsTo(&exp));
}

//////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests2, mmulMxM_4) {

	const Nd4jLong M = 3;
	const Nd4jLong K = 4;
	const Nd4jLong N = 5;

	NDArray a('f', {M,K}, {1.2,1.1,1.0,0.9,0.8,0.7,0.5,0.4,0.3,0.2,0.1,0}, nd4j::DataType::DOUBLE);
	NDArray b('f', {K,N}, {1,-2,3,-4,5,-6,7,-8,9,-10,11,-12,13,-14,15,-16,17,-18,19,-20}, nd4j::DataType::DOUBLE);
	NDArray c('c', {M,N}, nd4j::DataType::DOUBLE);

	NDArray exp('c', {M,N}, {0.1, 2.5, 4.9, 7.3, 9.7,0.3, 2.7, 5.1, 7.5, 9.9,0.5, 2.9, 5.3, 7.7, 10.1}, nd4j::DataType::DOUBLE);

	nd4j::MmulHelper::mmulMxM<double,double,double>(&a, &b, &c, 1., 0.);	

	ASSERT_TRUE(c.equalsTo(&exp));
}

//////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests2, mmulMxM_5) {

	const Nd4jLong M = 3;
	const Nd4jLong K = 4;
	const Nd4jLong N = 5;

	NDArray a('c', {M,K}, {1.2,1.1,1.0,0.9,0.8,0.7,0.5,0.4,0.3,0.2,0.1,0}, nd4j::DataType::DOUBLE);
	NDArray b('c', {K,N}, {1,-2,3,-4,5,-6,7,-8,9,-10,11,-12,13,-14,15,-16,17,-18,19,-20}, nd4j::DataType::DOUBLE);
	NDArray c('f', {M,N}, nd4j::DataType::DOUBLE);

	NDArray exp('f', {M,N}, {-8.8, -4.3, 0.2, 8.6, 4.1, -0.4, -8.4, -3.9, 0.6, 8.2, 3.7, -0.8, -8.0, -3.5, 1.}, nd4j::DataType::DOUBLE);

	nd4j::MmulHelper::mmulMxM<double,double,double>(&a, &b, &c, 1., 0.);	

	ASSERT_TRUE(c.equalsTo(&exp));
}

//////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests2, mmulMxM_6) {

	const Nd4jLong M = 3;
	const Nd4jLong K = 4;
	const Nd4jLong N = 5;

	NDArray a('c', {M,K}, {1.2,1.1,1.0,0.9,0.8,0.7,0.5,0.4,0.3,0.2,0.1,0}, nd4j::DataType::DOUBLE);
	NDArray b('f', {K,N}, {1,-2,3,-4,5,-6,7,-8,9,-10,11,-12,13,-14,15,-16,17,-18,19,-20}, nd4j::DataType::DOUBLE);
	NDArray c('c', {M,N}, nd4j::DataType::DOUBLE);

	NDArray exp('c', {M,N}, {-1.6, -0.8, -0.0, 0.8, 1.6, -0.7, 0.1, 0.9, 1.7, 2.5, 0.2, 1.0, 1.8, 2.6, 3.4}, nd4j::DataType::DOUBLE);

	nd4j::MmulHelper::mmulMxM<double,double,double>(&a, &b, &c, 1., 0.);	

	ASSERT_TRUE(c.equalsTo(&exp));
}

//////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests2, mmulMxM_7) {

	const Nd4jLong M = 3;
	const Nd4jLong K = 4;
	const Nd4jLong N = 5;

	NDArray a('f', {M,K}, {1.2,1.1,1.0,0.9,0.8,0.7,0.5,0.4,0.3,0.2,0.1,0}, nd4j::DataType::DOUBLE);
	NDArray b('c', {K,N}, {1,-2,3,-4,5,-6,7,-8,9,-10,11,-12,13,-14,15,-16,17,-18,19,-20}, nd4j::DataType::DOUBLE);
	NDArray c('c', {M,N}, nd4j::DataType::DOUBLE);

	NDArray exp('c', {M,N}, {-1.9, 1.3, -0.7, 0.1, 0.5, -0.9, 0.3, 0.3, -0.9, 1.5, 0.1, -0.7, 1.3, -1.9, 2.5}, nd4j::DataType::DOUBLE);

	nd4j::MmulHelper::mmulMxM<double,double,double>(&a, &b, &c, 1., 0.);	

	ASSERT_TRUE(c.equalsTo(&exp));
}

//////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests2, mmulMxM_8) {

	const Nd4jLong M = 3;
	const Nd4jLong K = 4;
	const Nd4jLong N = 5;

	NDArray a('c', {M,K}, {1.2,1.1,1.0,0.9,0.8,0.7,0.5,0.4,0.3,0.2,0.1,0}, nd4j::DataType::DOUBLE);
	NDArray b('c', {K,N}, {1,-2,3,-4,5,-6,7,-8,9,-10,11,-12,13,-14,15,-16,17,-18,19,-20}, nd4j::DataType::DOUBLE);
	NDArray c('c', {M,N}, nd4j::DataType::DOUBLE);

	NDArray exp('c', {M,N}, {-8.8, 8.6, -8.4, 8.2, -8.0, -4.3, 4.1, -3.9, 3.7, -3.5, 0.2, -0.4, 0.6, -0.8, 1.}, nd4j::DataType::DOUBLE);

	nd4j::MmulHelper::mmulMxM<double,double,double>(&a, &b, &c, 1., 0.);	

	ASSERT_TRUE(c.equalsTo(&exp));
}

//////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests2, mmulMxM_9) {

	const Nd4jLong M = 3;
	const Nd4jLong K = 4;
	const Nd4jLong N = 5;

	NDArray a('c', {M,K}, {1.2,1.1,1.0,0.9,0.8,0.7,0.5,0.4,0.3,0.2,0.1,0}, nd4j::DataType::FLOAT32);
	NDArray b('c', {K,N}, {1,-2,3,-4,5,-6,7,-8,9,-10,11,-12,13,-14,15,-16,17,-18,19,-20}, nd4j::DataType::FLOAT32);
	NDArray c('c', {M,N}, nd4j::DataType::FLOAT32);

	NDArray exp('c', {M,N}, {-8.8, 8.6, -8.4, 8.2, -8.0, -4.3, 4.1, -3.9, 3.7, -3.5, 0.2, -0.4, 0.6, -0.8, 1.}, nd4j::DataType::FLOAT32);

	nd4j::MmulHelper::mmulMxM<float,float,float>(&a, &b, &c, 1., 0.);	

	ASSERT_TRUE(c.equalsTo(&exp));
}

//////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests2, mmulMxM_10) {

	const Nd4jLong M = 3;
	const Nd4jLong K = 4;
	const Nd4jLong N = 5;

	NDArray a('f', {M,K}, {1.2,1.1,1.0,0.9,0.8,0.7,0.5,0.4,0.3,0.2,0.1,0}, nd4j::DataType::FLOAT32);
	NDArray b('f', {K,N}, {1,-2,3,-4,5,-6,7,-8,9,-10,11,-12,13,-14,15,-16,17,-18,19,-20}, nd4j::DataType::FLOAT32);
	NDArray c('f', {M,N}, nd4j::DataType::FLOAT32);

	NDArray exp('f', {M,N}, {0.1, 0.3, 0.5, 2.5, 2.7, 2.9, 4.9, 5.1, 5.3, 7.3, 7.5, 7.7, 9.7, 9.9, 10.1}, nd4j::DataType::FLOAT32);

	nd4j::MmulHelper::mmulMxM<float,float,float>(&a, &b, &c, 1., 0.);	
	// c.printIndexedBuffer();

	ASSERT_TRUE(c.equalsTo(&exp));
}

//////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests2, mmulMxM_11) {

	const Nd4jLong M = 3;
	const Nd4jLong K = 4;
	const Nd4jLong N = 5;

	NDArray a('f', {M,K}, {1.2,1.1,1.0,0.9,0.8,0.7,0.5,0.4,0.3,0.2,0.1,0}, nd4j::DataType::FLOAT32);
	NDArray b('c', {K,N}, {1,-2,3,-4,5,-6,7,-8,9,-10,11,-12,13,-14,15,-16,17,-18,19,-20}, nd4j::DataType::FLOAT32);
	NDArray c('f', {M,N}, nd4j::DataType::FLOAT32);

	NDArray exp('f', {M,N}, {-1.9, -0.9, 0.1, 1.3, 0.3, -0.7, -0.7, 0.3, 1.3, 0.1, -0.9, -1.9, 0.5, 1.5, 2.5}, nd4j::DataType::FLOAT32);

	nd4j::MmulHelper::mmulMxM<float,float,float>(&a, &b, &c, 1., 0.);	

	ASSERT_TRUE(c.equalsTo(&exp));
}

// //////////////////////////////////////////////////////////////////////////
// TEST_F(CudaBasicsTests2, mmulMxM_12) {

// 	const Nd4jLong M = 3;
// 	const Nd4jLong K = 4;
// 	const Nd4jLong N = 5;

// 	NDArray a('c', {M,K}, {1.2,1.1,1.0,0.9,0.8,0.7,0.5,0.4,0.3,0.2,0.1,0}, nd4j::DataType::HALF);
// 	NDArray b('c', {K,N}, {1,-2,3,-4,5,-6,7,-8,9,-10,11,-12,13,-14,15,-16,17,-18,19,-20}, nd4j::DataType::HALF);
// 	NDArray c('c', {M,N}, nd4j::DataType::HALF);

// 	NDArray exp('c', {M,N}, {-8.8, 8.6, -8.4, 8.2, -8.0, -4.3, 4.1, -3.9, 3.7, -3.5, 0.2, -0.4, 0.6, -0.8, 1.}, nd4j::DataType::HALF);

// 	nd4j::MmulHelper::mmulMxM<float16,float16,float16>(&a, &b, &c, 1., 0.);	

// 	ASSERT_TRUE(c.equalsTo(&exp));
// }

// //////////////////////////////////////////////////////////////////////////
// TEST_F(CudaBasicsTests2, mmulMxM_13) {

// 	const Nd4jLong M = 3;
// 	const Nd4jLong K = 4;
// 	const Nd4jLong N = 5;

// 	NDArray a('f', {M,K}, {1.2,1.1,1.0,0.9,0.8,0.7,0.5,0.4,0.3,0.2,0.1,0}, nd4j::DataType::HALF);
// 	NDArray b('f', {K,N}, {1,-2,3,-4,5,-6,7,-8,9,-10,11,-12,13,-14,15,-16,17,-18,19,-20}, nd4j::DataType::HALF);
// 	NDArray c('f', {M,N}, nd4j::DataType::HALF);

// 	NDArray exp('f', {M,N}, {0.1, 0.3, 0.5, 2.5, 2.7, 2.9, 4.9, 5.1, 5.3, 7.3, 7.5, 7.7, 9.7, 9.9, 10.1}, nd4j::DataType::HALF);

// 	nd4j::MmulHelper::mmulMxM<float16,float16,float16>(&a, &b, &c, 1., 0.);	
// 	// c.printIndexedBuffer();

// 	ASSERT_TRUE(c.equalsTo(&exp));
// }

// //////////////////////////////////////////////////////////////////////////
// TEST_F(CudaBasicsTests2, mmulMxM_14) {

// 	const Nd4jLong M = 3;
// 	const Nd4jLong K = 4;
// 	const Nd4jLong N = 5;

// 	NDArray a('f', {M,K}, {1.2,1.1,1.0,0.9,0.8,0.7,0.5,0.4,0.3,0.2,0.1,0}, nd4j::DataType::HALF);
// 	NDArray b('c', {K,N}, {1,-2,3,-4,5,-6,7,-8,9,-10,11,-12,13,-14,15,-16,17,-18,19,-20}, nd4j::DataType::HALF);
// 	NDArray c('f', {M,N}, nd4j::DataType::HALF);

// 	NDArray exp('f', {M,N}, {-1.9, -0.9, 0.1, 1.3, 0.3, -0.7, -0.7, 0.3, 1.3, 0.1, -0.9, -1.9, 0.5, 1.5, 2.5}, nd4j::DataType::HALF);

// 	nd4j::MmulHelper::mmulMxM<float16,float16,float16>(&a, &b, &c, 1., 0.);	

// 	ASSERT_TRUE(c.equalsTo(&exp));
// }