#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

 //
 // @author raver119@gmail.com
 //

#include "testlayers.h"
#include <NDArray.h>
#include <NDArrayFactory.h>
#include <Context.h>
#include <Node.h>
#include <graph/Variable.h>
#include <graph/VariableSpace.h>
#include <ops/declarable/CustomOperations.h>
#include <ops/declarable/generic/helpers/convolutions.h>
#include <ops/declarable/helpers/col2im.h>

#include <hip/hip_runtime.h>
#include <cuda_launch_config.h>

using namespace nd4j;
using namespace nd4j::graph;

class CudaBasicsTests : public testing::Test {
public:

};



TEST_F(CudaBasicsTests, TestPairwise_1) {
	// allocating host-side arrays
	auto x = NDArrayFactory::create<double>('c', { 5 }, { 1, 2, 3, 4, 5});
	auto z = NDArrayFactory::create<double>('c', { 5 });

	auto exp = NDArrayFactory::create<double>('c', { 5 }, { 2, 4, 6, 8, 10 });

	// making raw buffers
	Nd4jPointer devBufferPtrX, devBufferPtrZ, devShapePtrX;
	hipError_t res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrX), x.lengthOf() * x.sizeOfT());
	ASSERT_EQ(0, res);
	res = hipMalloc(reinterpret_cast<void **>(&devBufferPtrZ), x.lengthOf() * x.sizeOfT());
	ASSERT_EQ(0, res);
	res = hipMalloc(reinterpret_cast<void **>(&devShapePtrX), shape::shapeInfoByteLength(x.shapeInfo()));
	ASSERT_EQ(0, res);

	Nd4jPointer nativeStream = (Nd4jPointer)malloc(sizeof(hipStream_t));
	CHECK_ALLOC(nativeStream, "Failed to allocate memory for new CUDA stream");
	hipError_t dZ = hipStreamCreate(reinterpret_cast<hipStream_t *>(&nativeStream));
	auto stream = reinterpret_cast<hipStream_t *>(&nativeStream);

	hipMemcpyAsync(devBufferPtrX, x.buffer(), x.lengthOf() * x.sizeOfT(), hipMemcpyHostToDevice, *stream);
	hipMemcpyAsync(devShapePtrX, x.shapeInfo(), shape::shapeInfoByteLength(x.shapeInfo()), hipMemcpyHostToDevice, *stream);
	
	LaunchContext lc(stream, nullptr, nullptr);
	NativeOpExecutioner::execPairwiseTransform(&lc, pairwise::Add, nullptr, x.shapeInfo(), devBufferPtrX, reinterpret_cast<Nd4jLong*>(devShapePtrX), nullptr, x.shapeInfo(), devBufferPtrX, reinterpret_cast<Nd4jLong*>(devShapePtrX), nullptr, z.shapeInfo(), devBufferPtrZ, reinterpret_cast<Nd4jLong*>(devShapePtrX), nullptr);
	res = hipStreamSynchronize(*stream);
	ASSERT_EQ(0, res);


	hipMemcpyAsync(z.buffer(), devBufferPtrZ, z.lengthOf() * x.sizeOfT(), hipMemcpyDeviceToHost, *stream);
	res = hipStreamSynchronize(*stream);
	ASSERT_EQ(0, res);

	hipFree(devBufferPtrX);
	hipFree(devBufferPtrZ);
	hipFree(devShapePtrX);

	for (int e = 0; e < z.lengthOf(); e++) {
		ASSERT_NEAR(exp.e<double>(e), z.e<double>(e), 1e-5);
	}
}


////////////////////////////////////////////////////////////////////////////
TEST_F(CudaBasicsTests, execIndexReducescalar_1) {

    NDArray x1('c', {2,2}, {0, 1, 2, 3}, nd4j::DataType::INT32);
    NDArray x2('c', {2,2}, {0.5, 1.5, -4.5, 3.5}, nd4j::DataType::HALF);    
    NDArray x3('c', {2,2}, {0, -1, 0, 1}, nd4j::DataType::BOOL);
    
    NDArray scalar(nd4j::DataType::INT64);

    NDArray exp1('c', {0}, {3}, nd4j::DataType::INT64);
    NDArray exp2('c', {0}, {2}, nd4j::DataType::INT64);
    NDArray exp3('c', {0}, {1}, nd4j::DataType::INT64);

    void *dX1, *dX2, *dX3, *dZ; 
    Nd4jLong *dX1ShapeInfo, *dX2ShapeInfo, *dX3ShapeInfo, *dZShapeInfo;

    hipError_t cudaResult;

    cudaResult = hipMalloc(reinterpret_cast<void **>(&dX1), x1.lengthOf() * x1.sizeOfT()); 		   		         	 ASSERT_EQ(0, cudaResult);
    cudaResult = hipMalloc(reinterpret_cast<void **>(&dX2), x2.lengthOf() * x2.sizeOfT()); 		   		         	 ASSERT_EQ(0, cudaResult);    
    cudaResult = hipMalloc(reinterpret_cast<void **>(&dX3), x3.lengthOf() * x3.sizeOfT()); 		   		         	 ASSERT_EQ(0, cudaResult);    
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dZ), scalar.lengthOf() * scalar.sizeOfT()); 				         ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dX1ShapeInfo), shape::shapeInfoByteLength(x1.getShapeInfo()));    ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dX2ShapeInfo), shape::shapeInfoByteLength(x2.getShapeInfo()));    ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dX3ShapeInfo), shape::shapeInfoByteLength(x3.getShapeInfo()));    ASSERT_EQ(0, cudaResult);
	cudaResult = hipMalloc(reinterpret_cast<void **>(&dZShapeInfo), shape::shapeInfoByteLength(scalar.getShapeInfo())); ASSERT_EQ(0, cudaResult);	

    hipStream_t stream;
	cudaResult = hipStreamCreate(&stream); 
	ASSERT_EQ(0, cudaResult);
	
	hipMemcpyAsync(dX1, x1.buffer(), x1.lengthOf() * x1.sizeOfT(), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX2, x2.buffer(), x2.lengthOf() * x2.sizeOfT(), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX3, x3.buffer(), x3.lengthOf() * x3.sizeOfT(), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX1ShapeInfo, x1.getShapeInfo(), shape::shapeInfoByteLength(x1.getShapeInfo()), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX2ShapeInfo, x2.getShapeInfo(), shape::shapeInfoByteLength(x2.getShapeInfo()), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dX3ShapeInfo, x3.getShapeInfo(), shape::shapeInfoByteLength(x3.getShapeInfo()), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(dZShapeInfo, scalar.getShapeInfo(), shape::shapeInfoByteLength(scalar.getShapeInfo()), hipMemcpyHostToDevice, stream);
	
	void* reductionPointer = nullptr;	
	cudaResult = hipMalloc(reinterpret_cast<void **>(&reductionPointer), 1024*1024);
	ASSERT_EQ(0, cudaResult);

	LaunchContext lc(&stream, reductionPointer);

	/***************************************/
	
    NativeOpExecutioner::execIndexReduceScalar(&lc, 
    											nd4j::indexreduce::IndexAbsoluteMax, 
    											x1.buffer(), x1.getShapeInfo(),
    	                                       	dX1, dX1ShapeInfo, 
    	                                       	nullptr, 
    	                                       	scalar.buffer(), scalar.getShapeInfo(),
    	                                       	dZ, dZShapeInfo);

    cudaResult = hipStreamSynchronize(stream); 
    ASSERT_EQ(0, cudaResult);

    hipMemcpyAsync(scalar.buffer(), dZ, scalar.lengthOf() * scalar.sizeOfT(), hipMemcpyDeviceToHost, stream);

    cudaResult = hipStreamSynchronize(stream); 
    ASSERT_EQ(0, cudaResult);

	ASSERT_NEAR(exp1.e<float>(0), scalar.e<float>(0), 1e-5);

    /***************************************/
    
    NativeOpExecutioner::execIndexReduceScalar(&lc,
    											nd4j::indexreduce::IndexAbsoluteMax, 
    											nullptr, x2.getShapeInfo(),
    	                                       	dX2, dX2ShapeInfo, 
    	                                       	nullptr, 
    	                                       	nullptr, scalar.getShapeInfo(),
    	                                       	dZ, dZShapeInfo);

    cudaResult = hipStreamSynchronize(stream); 
    ASSERT_EQ(0, cudaResult);

    hipMemcpyAsync(scalar.buffer(), dZ, scalar.lengthOf() * scalar.sizeOfT(), hipMemcpyDeviceToHost, stream);

    cudaResult = hipStreamSynchronize(stream); 
    ASSERT_EQ(0, cudaResult);

    ASSERT_NEAR(exp2.e<float>(0), scalar.e<float>(0), 1e-5);

    // *************************************

    NativeOpExecutioner::execIndexReduceScalar(&lc, 
    											nd4j::indexreduce::IndexAbsoluteMax, 
    											nullptr, x3.getShapeInfo(),
    	                                       	dX3, dX3ShapeInfo, 
    	                                       	nullptr, 
    	                                       	nullptr, scalar.getShapeInfo(),
    	                                       	dZ, dZShapeInfo);

    cudaResult = hipStreamSynchronize(stream); 
    ASSERT_EQ(0, cudaResult);

    hipMemcpyAsync(scalar.buffer(), dZ, scalar.lengthOf() * scalar.sizeOfT(), hipMemcpyDeviceToHost, stream);

    cudaResult = hipStreamSynchronize(stream); 
    ASSERT_EQ(0, cudaResult);

    ASSERT_NEAR(exp3.e<float>(0), scalar.e<float>(0), 1e-5);
    
	/***************************************/

	hipFree(dX1); 			hipFree(dX2); 			hipFree(dX3); 			hipFree(dZ);
	hipFree(dX1ShapeInfo); hipFree(dX2ShapeInfo); hipFree(dX3ShapeInfo); hipFree(dZShapeInfo); 

	/***************************************/	

	cudaResult = hipStreamDestroy(stream); 
	ASSERT_EQ(0, cudaResult);
	
}