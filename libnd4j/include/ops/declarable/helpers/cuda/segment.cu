#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author GS <sgazeos@gmail.com>
//

#include <ops/declarable/helpers/segment.h>

namespace nd4j {
namespace ops {
namespace helpers {

    // segment max
    template <typename T>
    static void segmentMaxFunctor_(NDArray* input, NDArray* indices, NDArray* output) {

    }

    // segmen min 
    template <typename T>
    static void segmentMinFunctor_(NDArray* input, NDArray* indices, NDArray* output) {

    }

    // segmen mean
    template <typename T>
    static void segmentMeanFunctor_(NDArray* input, NDArray* indices, NDArray* output) {

    }

    template <typename T>
    static void segmentSumFunctor_(NDArray* input, NDArray* indices, NDArray* output) {

    }

    template <typename T>
    static void segmentProdFunctor_(NDArray* input, NDArray* indices, NDArray* output) {

    }

    template <typename T>
    static bool segmentIndicesValidate_(NDArray* indices, NDArray& aexpected, NDArray& aoutput) {
        return true;
    }

    void segmentMaxFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), segmentMaxFunctor_, (input, indices, output), LIBND4J_TYPES);
    }

    void segmentMinFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), segmentMinFunctor_, (input, indices, output), LIBND4J_TYPES);
    }

    void segmentMeanFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), segmentMeanFunctor_, (input, indices, output), LIBND4J_TYPES);
    }

    void segmentSumFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), segmentSumFunctor_, (input, indices, output), LIBND4J_TYPES);
    }

    void segmentProdFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), segmentProdFunctor_, (input, indices, output), LIBND4J_TYPES);
    }

    bool segmentIndicesValidate(nd4j::LaunchContext * context, NDArray* indices, NDArray& expected, NDArray& output) {
        BUILD_SINGLE_SELECTOR(output.dataType(), return segmentIndicesValidate_, (indices, expected, output), LIBND4J_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template bool segmentIndicesValidate_, (NDArray*, NDArray&, NDArray&), LIBND4J_TYPES);
    BUILD_SINGLE_TEMPLATE(template void segmentProdFunctor_, (NDArray* input, NDArray* indices, NDArray* output), LIBND4J_TYPES);
    BUILD_SINGLE_TEMPLATE(template void segmentSumFunctor_, (NDArray* input, NDArray* indices, NDArray* output), LIBND4J_TYPES);
    BUILD_SINGLE_TEMPLATE(template void segmentMeanFunctor_, (NDArray* input, NDArray* indices, NDArray* output), LIBND4J_TYPES);
    BUILD_SINGLE_TEMPLATE(template void segmentMinFunctor_, (NDArray* input, NDArray* indices, NDArray* output), LIBND4J_TYPES);
    BUILD_SINGLE_TEMPLATE(template void segmentMaxFunctor_, (NDArray* input, NDArray* indices, NDArray* output), LIBND4J_TYPES);
    // -------------------------------------------------------------------------------------------------------------- //
    // Unsorted segment ops
    // -------------------------------------------------------------------------------------------------------------- //

    bool unsortedSegmentIndicesValidate(nd4j::LaunchContext * context, NDArray* indices, Nd4jLong expected, Nd4jLong& output) {
        return true;
    }

    template <typename T>
    static void unsortedSegmentMaxFunctor_(NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {

    }

    void unsortedSegmentMaxFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), unsortedSegmentMaxFunctor_, (input, indices, numOfClasses, output), NUMERIC_TYPES);
    }
    BUILD_SINGLE_TEMPLATE(template void unsortedSegmentMaxFunctor_, (NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output), NUMERIC_TYPES);

    template <typename T>
    static void unsortedSegmentMinFunctor_(NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {

    }

    void unsortedSegmentMinFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
        BUILD_SINGLE_SELECTOR(input->dataType(), unsortedSegmentMinFunctor_, (input, indices, numOfClasses, output),
                              NUMERIC_TYPES);
    }

    BUILD_SINGLE_TEMPLATE(template void unsortedSegmentMinFunctor_, (NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output), NUMERIC_TYPES);

    void unsortedSegmentMeanFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {

    }

    void unsortedSegmentSumFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {

    }

    void unsortedSegmentProdFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {
      //  BUILD_SINGLE_SELECTOR(input->dataType(), unsortedSegmentProdFunctor_, (input, indices, numOfClasses, output), NUMERIC_TYPES);
    }
    //BUILD_SINGLE_TEMPLATE(template void unsortedSegmentProdFunctor_, (NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output), NUMERIC_TYPES);

    void unsortedSegmentSqrtNFunctor(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, Nd4jLong numOfClasses, NDArray* output) {

    }

    // -------------------------------------------------------------------------------------------------------------- //
    // Backpropagate ops helpers
    // -------------------------------------------------------------------------------------------------------------- //
    // Sorted backpropagate ops
    //

    // segment max
    template <typename T>
    int segmentMaxFunctorBP_(NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        return Status::OK();
    }

    int segmentMaxFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        BUILD_SINGLE_SELECTOR(output->dataType(), return segmentMaxFunctorBP_, (input, indices, gradOut, output), NUMERIC_TYPES);
    }
    BUILD_SINGLE_TEMPLATE(template int segmentMaxFunctorBP_, (NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output), NUMERIC_TYPES);

    // segmen min
    int segmentMinFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        return Status::OK();
    }

    // segmen mean
    int segmentMeanFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        return Status::OK();
    }

    int segmentSumFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        return Status::OK();
    }

    // -------------------------------------------------------------------------------------------------------------- //
    // Unsorted backpropagate segment ops
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T>
    static int unsortedSegmentMaxFunctorBP_(NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        return Status::OK();
    }

    int unsortedSegmentMaxFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        BUILD_SINGLE_SELECTOR(output->dataType(), return unsortedSegmentMaxFunctorBP_, (input, indices, gradOut, numOfClasses, output), NUMERIC_TYPES);
    }
    BUILD_SINGLE_TEMPLATE(template int unsortedSegmentMaxFunctorBP_, (NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output), NUMERIC_TYPES);

    template <typename T>
    static int unsortedSegmentMinFunctorBP_(NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        return Status::OK();
    }

    int unsortedSegmentMinFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        BUILD_SINGLE_SELECTOR(output->dataType(), return unsortedSegmentMinFunctorBP_, (input, indices, gradOut, numOfClasses, output), NUMERIC_TYPES);
    }
    BUILD_SINGLE_TEMPLATE(template int unsortedSegmentMinFunctorBP_, (NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output), NUMERIC_TYPES);

    int unsortedSegmentMeanFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        return Status::OK();
    }

    int unsortedSegmentSumFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        return Status::OK();
    }

    int unsortedSegmentProdFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        return Status::OK();
    }

//    template <typename T>
    int unsortedSegmentSqrtNFunctorBP(nd4j::LaunchContext * context, NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
        return Status::OK();
    }

//    int unsortedSegmentSqrtNFunctorBP(NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output) {
//        BUILD_SINGLE_SELECTOR(output->dataType(), return unsortedSegmentSqrtNFunctorBP_, (input, indices, gradOut, numOfClasses, output), FLOAT_TYPES);
//    }
//    BUILD_SINGLE_TEMPLATE(template int unsortedSegmentSqrtNFunctorBP_, (NDArray* input, NDArray* indices, NDArray* gradOut, Nd4jLong numOfClasses, NDArray* output), FLOAT_TYPES);
}
}
}