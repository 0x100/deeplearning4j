#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

#ifndef NDARRAY_CPP
#define NDARRAY_CPP

#include "../NDArray.h"
#include "../NDArrayFactory.h"
#include "NativeOpExecutioner.h"
#include <memory/Workspace.h>
#include <memory/MemoryRegistrator.h>
#include <ops.h>
#include <ops/gemm.h>
#include <pointercast.h>
#include <stdexcept>
#include <memory>
#include <helpers/logger.h>
#include <loops/pairwise_transform.h>
#include <loops/transform_same.h>
#include <loops/random.h>
#include <loops/broadcasting.h>
#include <indexing/NDIndex.h>
#include <indexing/IndicesList.h>
#include <helpers/ShapeUtils.h>
#include <sstream>
#include <helpers/ArrayUtils.h>
#include <MmulHelper.h>
#include <helpers/threshold.h>
#include <exceptions/datatype_exception.h>
#include <exceptions/cuda_exception.h>
#include <specials_cuda.h>

#include "../NDArray.hpp"

namespace nd4j {

////////////////////////////////////////////////////////////////////////
void* NDArray::operator new(size_t i) {
    if (nd4j::memory::MemoryRegistrator::getInstance()->hasWorkspaceAttached()) {
        nd4j::memory::Workspace* ws = nd4j::memory::MemoryRegistrator::getInstance()->getWorkspace();
        return ws->allocateBytes((Nd4jLong) i);
    } else {
        auto p = malloc(i);
        
        CHECK_ALLOC(p, "Failed to allocate new NDArray", i);
        return p;
    }
}

////////////////////////////////////////////////////////////////////////
void NDArray::operator delete(void* p) {
    
    if (!nd4j::memory::MemoryRegistrator::getInstance()->hasWorkspaceAttached())
        free(p);
}


////////////////////////////////////////////////////////////////////////
// copy constructor
NDArray::NDArray(const NDArray& other) {
    
    _context = other._context;    

    setShapeInfo(ShapeBuilders::copyShapeInfo(other._shapeInfo, false, _context->getWorkspace()));
    _isShapeAlloc = true;

    ALLOCATE_SPECIAL(_bufferD, _context->getWorkspace(), _length * sizeOfT(), int8_t);
    _isBuffDAlloc = true;

    if(other.isActualOnHostSide()) {
        auto res = hipMemcpy(_bufferD, other._buffer, _length * sizeOfT(), hipMemcpyHostToDevice);
        if (res != 0)
            throw cuda_exception::build("hipMemcpy failed", res);
    } else {
        auto res = hipMemcpy(_bufferD, other._bufferD, _length * sizeOfT(), hipMemcpyDeviceToDevice);
        if (res != 0)
            throw cuda_exception::build("hipMemcpy failed", res);
    }        

    tickWriteDevice();
}

////////////////////////////////////////////////////////////////////////
void NDArray::lazyAllocateBuffer() const {
    if (_buffer == nullptr && !this->isEmpty()) {
        int8_t* pB = const_cast<int8_t*>(_buffer);
        ALLOCATE(pB, _context->getWorkspace(), this->lengthOf() * this->sizeOfT(), int8_t);
        syncToHost();
    }
}

////////////////////////////////////////////////////////////////////////
// scalar constructor
NDArray::NDArray(nd4j::DataType dtype, nd4j::graph::LaunchContext* context) {

    setShapeInfo(ShapeBuilders::createScalarShapeInfo(dtype, context->getWorkspace()));

    ALLOCATE_SPECIAL(_bufferD, context->getWorkspace(), sizeOfT(), int8_t);
    _isBuffDAlloc = true;
    hipMemset(_bufferD, 0, sizeOfT());    
    
    tickWriteDevice();
}

////////////////////////////////////////////////////////////////////////
// creates new NDArray using shape information from "shapeInfo" array, set all elements in new array to be zeros
NDArray::NDArray(Nd4jLong* shapeInfo, const bool copyStrides, nd4j::graph::LaunchContext* context, const bool isShapeAlloc) {
    
    if (shapeInfo == nullptr)
        throw std::runtime_error("NDArray constructor: can't be initalized without shapeinfo");

    if ((int) shapeInfo[0] > MAX_RANK)
        throw std::invalid_argument("Rank of NDArray can't exceed 32");

    _context = context;        

    if(!isShapeAlloc) {
        setShapeInfo(ShapeBuilders::copyShapeInfo(shapeInfo, copyStrides, _context->getWorkspace()));       
    }
    else {
        setShapeInfo(shapeInfo);
        if(!copyStrides)
            shape::updateStrides(_shapeInfo, shape::order(shapeInfo));         
    }

    _isShapeAlloc = true;
   
    ALLOCATE_SPECIAL(_bufferD, _context->getWorkspace(), _length * sizeOfT(), int8_t);
    hipMemset(_bufferD, 0, _length * sizeOfT());
    _isBuffDAlloc = true;        
   
    tickWriteDevice();  
}

////////////////////////////////////////////////////////////////////////
NDArray::NDArray(const char order, const std::vector<Nd4jLong> &shape, const std::vector<double>& data, nd4j::DataType dtype, nd4j::graph::LaunchContext* context) {

    if (shape.empty())
        throw std::runtime_error("NDArray constructor: input shape is empty !");

    if ((int) shape.size() > MAX_RANK)
        throw std::invalid_argument("Rank of NDArray can't exceed 32");

    _context = context;

    setShapeInfo(ShapeBuilders::createShapeInfo(dtype, order, shape, _context->getWorkspace()));
    _isShapeAlloc = true;

    if (_length != data.size()) {
        nd4j_printf("NDArray constructor: data size [%i] doesn't match shape length [%i]\n", data.size(), _length);
        throw std::runtime_error("Data size doesn't match shape");
    }

    ALLOCATE(_buffer, _context->getWorkspace(), _length * DataTypeUtils::sizeOf(dtype), int8_t);
    _isBuffAlloc = true;
    
    ALLOCATE_SPECIAL(_bufferD, _context->getWorkspace(), _length * DataTypeUtils::sizeOf(dtype), int8_t);
    _isBuffDAlloc = true;    

    for(Nd4jLong i=0; i < _length; ++i) {
        BUILD_SINGLE_PARTIAL_SELECTOR(dtype, templatedDoubleAssign<, double>(_buffer, i, reinterpret_cast<const void *>(data.data()), i), LIBND4J_TYPES);
    }
        
    syncToDevice();
    tickReadHost();
}

////////////////////////////////////////////////////////////////////////
// do not allocate memory, memory for array is passed from outside
NDArray::NDArray(void *buffer, Nd4jLong *shapeInfo, graph::LaunchContext* context, const bool isBuffAlloc, const bool isShapeAlloc) {

    if (shapeInfo == nullptr)
        throw std::runtime_error("NDArray constructor: can't be initalized without shapeinfo");

    if ((int) shapeInfo[0] > MAX_RANK)
        throw std::invalid_argument("NDArray constructor: rank of NDArray can't exceed 32");

     if(!isShapeAlloc) 
        setShapeInfo(ShapeBuilders::copyShapeInfo(shapeInfo, true, _context->getWorkspace()));
    else 
        setShapeInfo(shapeInfo);
    
    _context = context;
    _isShapeAlloc = true;    
    
    _buffer = reinterpret_cast<int8_t *>(buffer);        
    ALLOCATE_SPECIAL(_bufferD, _context->getWorkspace(), _length * sizeOfT(), int8_t);
    _isBuffDAlloc = true;
    
    if(_buffer != nullptr)
        hipMemcpy(_bufferD, _buffer, _length * sizeOfT(), hipMemcpyHostToDevice);                    
    
    _isBuffAlloc = isBuffAlloc;
    
    tickWriteDevice();
    tickReadHost();
}

////////////////////////////////////////////////////////////////////////
NDArray::NDArray(const char order, const std::vector<Nd4jLong> &shape, nd4j::DataType dtype, nd4j::graph::LaunchContext* context) {

    if (shape.empty())
        throw std::runtime_error("NDArray constructor: input shape is empty !");

    if ((int) shape.size() > MAX_RANK)
        throw std::invalid_argument("Rank of NDArray can't exceed 32");

    _context = context;

    setShapeInfo(ShapeBuilders::createShapeInfo(dtype, order, shape, _context->getWorkspace()));
    _isShapeAlloc = true;

    ALLOCATE_SPECIAL(_bufferD, _context->getWorkspace(), _length * sizeOfT(), int8_t);
    hipMemset(_bufferD, '\0', _length * sizeOfT()); // zero all memory
    _isBuffDAlloc = true;    

    tickWriteDevice();
}

////////////////////////////////////////////////////////////////////////
NDArray::NDArray(const NDArray *other, const bool copyStrides, nd4j::graph::LaunchContext* context) {

    _context = context;
    
    setShapeInfo(ShapeBuilders::copyShapeInfo(other->_shapeInfo, copyStrides, _context->getWorkspace()));
    _isShapeAlloc = true;
    
    ALLOCATE_SPECIAL(_bufferD, _context->getWorkspace(), _length * sizeOfT(), int8_t);
    _isBuffDAlloc = true;    

    tickWriteDevice();
}

////////////////////////////////////////////////////////////////////////
NDArray::NDArray(void* buffer, const char order, const std::vector<Nd4jLong> &shape,  nd4j::DataType dtype, nd4j::graph::LaunchContext* context) {
    
    if (shape.empty())
        throw std::runtime_error("NDArray constructor: input shape is empty !");
        
    if ((int) shape.size() > MAX_RANK)
        throw std::invalid_argument("Rank of NDArray can't exceed 32");
    
    _context = context;

    setShapeInfo(ShapeBuilders::createShapeInfo(dtype, order, shape, _context->getWorkspace()));
    _isShapeAlloc = true;

    _buffer = reinterpret_cast<int8_t *>(buffer);
    
    ALLOCATE_SPECIAL(_bufferD, _context->getWorkspace(), _length * sizeOfT(), int8_t);
    _isBuffDAlloc = true;
    
    if(_buffer != nullptr)
        hipMemcpy(_bufferD, _buffer, _length * sizeOfT(), hipMemcpyHostToDevice);
        
    tickWriteDevice();
    tickReadHost(); 
}

////////////////////////////////////////////////////////////////////////
// assignment operator
    NDArray& NDArray::operator=(const NDArray& other) {

    if (this == &other)
        return *this;    

    if (shape::equalsSoft(_shapeInfo, other._shapeInfo) && _dataType == other._dataType) {
        if(!isEmpty())
            this->assign(&other);
    }
    else {
        
        if(_isBuffAlloc && _context->getWorkspace() == nullptr) 
            delete []_buffer;

        if(_isShapeAlloc && _context->getWorkspace() == nullptr) 
            delete []_shapeInfo;
        
        if(_isBuffDAlloc)
            hipFree(_bufferD);

        if(_isShapeDAlloc)
            hipFree(_shapeInfoD);
               
        _context= other._context;
        _buffer = nullptr;
              
        setShapeInfo(ShapeBuilders::copyShapeInfo(other._shapeInfo, false, _context->getWorkspace()));    
        _isShapeAlloc = true;

        ALLOCATE_SPECIAL(_bufferD, _context->getWorkspace(), _length * sizeOfT(), int8_t);
        _isBuffDAlloc = true;        
                
        this->assign(&other);
    }

    return *this;
}

//////////////////////////////////////////////////////////////////////////
// perform array transformation
    // void NDArray::applyTransform(nd4j::transform::FloatOps op, void *extraParams) {
    //     applyTransform(op, this, extraParams);
    // }

    // void NDArray::applyTransform(nd4j::transform::AnyOps op, void *extraParams) {
    //     applyTransform(op, this, extraParams);
    // }

    // void NDArray::applyTransform(nd4j::transform::SameOps op, void *extraParams) {
    //     applyTransform(op, this, extraParams);
    // }

    // void NDArray::applyTransform(nd4j::transform::BoolOps op, void *extraParams) {
    //     applyTransform(op, this, extraParams);
    // }

    // void NDArray::applyTransform(nd4j::transform::StrictOps op, void *extraParams) {
    //     applyTransform(op, this, extraParams);
    // }

    // perform array transformation

/*
    template<typename T>
    template<typename OpName>
    void NDArray<T>::applyRandom(nd4j::random::RandomBuffer *buffer, NDArray<T>* y, NDArray<T>* z, T* extraArgs) {
        Nd4jPointer state = (Nd4jPointer) buffer;
        if (y == nullptr && z == nullptr) {
            // we're executing indexed z here
            functions::random::RandomFunction<T>::template execTransform<OpName>(state, this->buffer(), this->shapeInfo(), extraArgs);
        } else if (y == nullptr && z != nullptr) {
            // XZ case
            functions::random::RandomFunction<T>::template execTransform<OpName>(state, this->buffer(), this->shapeInfo(), z->buffer(), z->shapeInfo(), extraArgs);
        } else if (y != nullptr && z != nullptr) {
            // XYZ case
            functions::random::RandomFunction<T>::template execTransform<OpName>(state, this->buffer(), this->shapeInfo(), y->buffer(), y->shapeInfo(), z->buffer(), z->shapeInfo(), extraArgs);
        }
    }
    */

    //////////////////////////////////////////////////////////////////////////
    void NDArray::applyTrueBroadcast(nd4j::BroadcastBoolOpsTuple op, const NDArray* other, NDArray* target, const bool checkTargetShape, ExtraArguments *extraArgs) const {
        if (isS())
            throw std::runtime_error("NDArray::applyTrueBroadcast bool: you can't use this method on String array!");
        if(target == nullptr || other == nullptr)
            throw std::runtime_error("NDArray::applyTrueBroadcast bool method: target or other = nullptr !");
        
        if (isScalar()) {
            NDArray temp(target->_shapeInfo, _dataType, false, _context);
            temp.assign(this);
            temp.applyPairwiseTransform(op.p, other, target,  extraArgs);
            return;
        }
        if (other->isScalar()) {
            this->applyScalarArr(op.s, other, target, extraArgs);
            return;
        }

        const NDArray* min(nullptr), *max(nullptr);
        if(this->rankOf() >= other->rankOf()) {
            max = this;
            min = other;
        }
        else {
            max = other;
            min = this;
        }

        if(checkTargetShape) {
            Nd4jLong* newShapeInfo = nullptr;
            if(!ShapeUtils::evalBroadcastShapeInfo(*max, *min, false, newShapeInfo, _context->getWorkspace()))          // the rank of target array must be equal to max->rankOf)()
                throw std::runtime_error("NDArray::applyTrueBroadcast method: the shapes of this and other arrays are not suitable for broadcast operation !");
            if(!shape::equalsSoft(target->_shapeInfo, newShapeInfo) || target->_dataType != DataType::BOOL)
                throw std::runtime_error("NDArray::applyTrueBroadcast bool method: the shape or type of target array is wrong !");
            if(_dataType != other->_dataType)
                throw std::invalid_argument("NDArray::applyTrueBroadcast bool method: this and other arrays must have the same type !");

            // if workspace is not null - do not call delete.
            if (_context->getWorkspace() == nullptr)
                delete[] newShapeInfo;
        }

        NDArray* pTarget = (max->_dataType == target->_dataType) ? target : new NDArray(target->ordering(), target->getShapeAsVector(), max->_dataType, target->_context);
        // check whether max array has to be tiled
        if(!max->isSameShape(target)) {
            // evaluate repeating dimensions for tile operation
            std::vector<Nd4jLong> repeatMax(max->rankOf());
            for(int i = 1; i <= max->rankOf(); ++i)
                repeatMax[i-1] = (target->_shapeInfo[i] / max->_shapeInfo[i]);
            max->tile(repeatMax, *pTarget);
        }
        else
            pTarget->assign(max);

        // check whether min array has to be tiled
        std::vector<Nd4jLong> repeatMin(min->rankOf());
        int product = 1;
        for(int i = min->rankOf(); i >=1 ; --i) {
            repeatMin[i-1] = (target->_shapeInfo[target->rankOf() - min->rankOf() + i] / min->_shapeInfo[i]);
            product *= repeatMin[i-1];
        }

        auto pMin = const_cast<NDArray *>(min);
        if(product != 1 )
            pMin = new NDArray(min->tile(repeatMin));


        std::vector<int> sameDims = ShapeUtils::getDimsWithSameShape(*target, *pMin);

        if(max == this) {
            pTarget->applyBroadcast(op.b, sameDims, pMin, target, extraArgs);
        }
        else {
            auto dimsToExclude = ShapeUtils::evalDimsToExclude(target->rankOf(), sameDims);
            const auto numOfSubArrs = ShapeUtils::getNumOfSubArrs(target->_shapeInfo, dimsToExclude);

            for(Nd4jLong i = 0; i < numOfSubArrs; ++i) {
                NDArray targetSubArr = (*target)(i, dimsToExclude);
                if (pTarget == target)
                    pMin->applyPairwiseTransform(op.p, &targetSubArr, &targetSubArr, extraArgs);
                else {
                    NDArray pTargetSubArr = (*pTarget)(i, dimsToExclude);
                    pMin->applyPairwiseTransform(op.p, &pTargetSubArr, &targetSubArr, extraArgs);
                }
            }
        }

        if(pMin != min)
            delete pMin;
        if(pTarget != target)
            delete pTarget;
    }

    //////////////////////////////////////////////////////////////////////////
    void NDArray::applyTrueBroadcast(nd4j::BroadcastOpsTuple op, const NDArray* other, NDArray* target, const bool checkTargetShape, ExtraArguments *extraArgs) const {
        if (isS())
            throw std::runtime_error("NDArray::applyTrueBroadcast: you can't use this method on String array!");
        if(target == nullptr || other == nullptr)
            throw std::runtime_error("NDArray::applyTrueBroadcast method: target or other = nullptr !");
        if(((op.s == scalar::Divide || op.s == scalar::FloorDiv || op.s == scalar::FloorMod) && other->isB()) || (op.s == scalar::ReverseDivide && this->isB()))
            throw std::runtime_error("NDArray::applyTrueBroadcast method: you can't divide by bool array !");
        //NDArray::registerSpecialUse({target}, {this,other});
        if (isScalar()) {
            target->assign(this);
            target->applyPairwiseTransform(op.p, *other, extraArgs);
            return;
        }
        if (other->isScalar()) {
            const_cast<NDArray*>(this)->applyScalarArr(op.s, other, target, extraArgs);
            return;
        }

        const NDArray* min(nullptr), *max(nullptr);
        if(this->rankOf() >= other->rankOf()) {
            max = this;
            min = other;
        }
        else {
            max = other;
            min = this;
        }

        if(checkTargetShape) {
            Nd4jLong* newShapeInfo = nullptr;
            if(!ShapeUtils::evalBroadcastShapeInfo(*max, *min, false, newShapeInfo, _context->getWorkspace()))          // the rank of target array must be equal to max->rankOf)()
                throw std::runtime_error("NDArray::applyTrueBroadcast method: the shapes of this and other arrays are not suitable for broadcast operation !");
            if(!shape::equalsTypesAndShapesSoft(target->getShapeInfo(), newShapeInfo))
                throw std::runtime_error("NDArray::applyTrueBroadcast method: the shape or type of target array is wrong !");

            // if workspace is not null - do not call delete.
            if (_context->getWorkspace() == nullptr)
                delete[] newShapeInfo;
        }

        NDArray* pTarget = (max->_dataType == target->_dataType) ? target : new NDArray(target->ordering(), target->getShapeAsVector(), max->_dataType, target->_context);
        // check whether max array has to be tiled
        if(!max->isSameShape(target)) {
            // evaluate repeating dimensions for tile operation
            std::vector<Nd4jLong> repeatMax(max->rankOf());
            for(int i = 1; i <= max->rankOf(); ++i)
                repeatMax[i-1] = (target->_shapeInfo[i] / max->_shapeInfo[i]);
            max->tile(repeatMax, *pTarget);
        }
        else
            pTarget->assign(max);


        // check whether min array has to be tiled
        std::vector<Nd4jLong> repeatMin(min->rankOf());
        int product = 1;
        for(int i = min->rankOf(); i >=1 ; --i) {
            repeatMin[i-1] = (target->_shapeInfo[target->rankOf() - min->rankOf() + i] / min->_shapeInfo[i]);
            product *= repeatMin[i-1];
        }

        auto pMin = const_cast<NDArray *>(min);
        if(product != 1 )
            pMin = new NDArray(min->tile(repeatMin));

        std::vector<int> sameDims = ShapeUtils::getDimsWithSameShape(*target, *pMin);

        if(max == this) {
            pTarget->applyBroadcast(op.b, sameDims, pMin, target, extraArgs);
        }
        else {
            auto dimsToExclude = ShapeUtils::evalDimsToExclude(target->rankOf(), sameDims);
            const auto numOfSubArrs = ShapeUtils::getNumOfSubArrs(target->_shapeInfo, dimsToExclude);

            for(Nd4jLong i = 0; i < numOfSubArrs; ++i) {
                auto targetSubArr = (*target)(i, dimsToExclude);
                if(pTarget == target)
                    pMin->applyPairwiseTransform(op.p, &targetSubArr, &targetSubArr, extraArgs);
                else {
                    auto pTargetSubArr = (*pTarget)(i, dimsToExclude);
                    pMin->applyPairwiseTransform(op.p, &pTargetSubArr, &targetSubArr, extraArgs);
                }
            }
        }

        if(pMin != min)
            delete pMin;
         if(pTarget != target)
            delete pTarget;
    }

    //////////////////////////////////////////////////////////////////////////
    // return array which is broadcasted from this and argument array
    NDArray* NDArray::broadcast(const NDArray& other) {
	    // the orders must be the same
	    char order = ordering();
	    if(order != other.ordering())
		    throw std::runtime_error("Broadcast method: arrays have different orders!");

	    // recognize shapes with smaller and bigger rank
	    Nd4jLong* biggerShapeInfo = nullptr;
	    Nd4jLong* smallerShapeInfo = nullptr;
	    int smallerRank, biggerRank;
	    if (rankOf() > other.rankOf()) {
		    biggerShapeInfo = _shapeInfo;
		    biggerRank = shape::rank(_shapeInfo);
		    smallerShapeInfo = other._shapeInfo;
		    smallerRank = shape::rank(other._shapeInfo);
	    }
	    else {
		    biggerShapeInfo = other._shapeInfo;
		    biggerRank = shape::rank(other._shapeInfo);
		    smallerShapeInfo = _shapeInfo;
		    smallerRank = shape::rank(_shapeInfo);
	    }

	    // check shapes on consistency
	    int diff = biggerRank - smallerRank;
	    for (int i = smallerRank; i<=1; --i)
		    if(biggerShapeInfo[diff+i] != smallerShapeInfo[i] && biggerShapeInfo[i] != 1 && smallerShapeInfo[i] != 1)
			    throw std::runtime_error("Broadcast method: arrays have incompatible shapes !");

		// create and fill ret shapeInfo
	    auto shapeInfoNew = new Nd4jLong[shape::shapeInfoLength(biggerRank)];
	    memcpy(shapeInfoNew, biggerShapeInfo, shape::shapeInfoByteLength(biggerRank));
	    for (int i = smallerRank; i>=1; --i)
		    if(shapeInfoNew[diff+i] == 1 || smallerShapeInfo[i] == 1)
			    shapeInfoNew[diff+i] *= smallerShapeInfo[i];

	    auto ret = new NDArray(shapeInfoNew, true, _context);
        ShapeUtils::updateStridesAndType(ret->getShapeInfo(), DataTypeUtils::pickPairwiseResultType(_dataType, other._dataType), order);
	    delete []shapeInfoNew;

    	return ret;
    }


    //////////////////////////////////////////////////////////////////////////
    // check whether array's rows (arg=0) or columns (arg=1) create orthogonal basis
    bool NDArray::hasOrthonormalBasis(const int arg) {
        if (isS())
            throw std::runtime_error("NDArray::hasOrthonormalBasis: you can't use this method on String array!");
	    if(rankOf() !=2 )
		    throw std::runtime_error("NDArray::hasOrthBasis method: rank of ndarray is not equal 2 !");

	    if(arg!=0  && arg!=1)
		    throw std::runtime_error("NDArray::hasOrthBasis method: input argument is not equal to 0 or 1 !");

	    const double eps = 1e-5;
        double dot = 0.f;

        if(arg) {					// check whether columns create orthogonal basis
		    for(int j=0; j<columns()-1; ++j)
			    for(int k=j+1; k<columns(); ++k) {
				    for(int i=0; i<rows(); ++i)
					    dot += e<double>(i,j)*e<double>(i,k);

				    if(nd4j::math::nd4j_abs(dot) > eps )
					    return false;

				    dot = 0.f;
			    }

			    for(int j=0; j<columns(); ++j)	{	// check whether norm of column vector = 1
			        for(int i=0; i<rows(); ++i)
				        dot += e<double>(i,j)*e<double>(i,j);
			    if(dot != 0.f && nd4j::math::nd4j_abs(nd4j::math::nd4j_sqrt<double, double>(dot) - 1.f) > eps)
				    return false;

			    dot = 0.f;
		    }
	    }
	    else {						// check whether rows create orthogonal basis
		    for(int i=0; i<rows()-1; ++i)
			    for(int k=i+1; k<rows(); ++k) {
				    for(int j=0; j<columns(); ++j)
					    dot += e<double>(i,j)*e<double>(k,j);

				    if(nd4j::math::nd4j_abs(dot) > eps )
					    return false;

				    dot = 0.;
			    }

		        for(int i=0; i<rows(); ++i) {		// check whether norm of row vector = 1
			        for(int j=0; j<columns(); ++j)
					    dot += e<double>(i,j)*e<double>(i,j);

			        if(dot!= 0. && nd4j::math::nd4j_abs(nd4j::math::nd4j_sqrt<double, double>(dot) - 1.) > eps)
				        return false;
			        dot = 0.;
		        }
	        }
	    return true;
    }

    template <typename T>
    std::vector<T> NDArray::asVectorT() {
        std::vector<T> result(this->lengthOf());

#pragma omp parallel for simd
        for (int e = 0; e < this->lengthOf(); e++)
            result[e] = this->e<T>(e);

        return result;
    }
    BUILD_SINGLE_TEMPLATE(template std::vector, NDArray::asVectorT(), LIBND4J_TYPES);


    ////////////////////////////////////////////////////////////////////////
    template<typename T>
    void NDArray::setValueInDiagMatrix(const T& value, const int diag, const char direction) {
        if (isS())
            throw std::runtime_error("NDArray::setValueInDiagMatrix: you can't use this method on String array!");
        if(rankOf() != 2)
           throw std::string("NDArray::setValueInDiagMatrix method: array must have rank = 2, but got " + toStringValue(rankOf()) + " instead !");
    }
    template void NDArray::setValueInDiagMatrix(const double& value, const int diag, const char direction);
    template void NDArray::setValueInDiagMatrix(const float& value, const int diag, const char direction);
    template void NDArray::setValueInDiagMatrix(const float16& value, const int diag, const char direction);
    template void NDArray::setValueInDiagMatrix(const bfloat16& value, const int diag, const char direction);
    template void NDArray::setValueInDiagMatrix(const Nd4jLong& value, const int diag, const char direction);
    template void NDArray::setValueInDiagMatrix(const int& value, const int diag, const char direction);
    template void NDArray::setValueInDiagMatrix(const int16_t& value, const int diag, const char direction);
    template void NDArray::setValueInDiagMatrix(const uint8_t& value, const int diag, const char direction);
    template void NDArray::setValueInDiagMatrix(const int8_t& value, const int diag, const char direction);
    template void NDArray::setValueInDiagMatrix(const bool& value, const int diag, const char direction);

    


    //////////////////////////////////////////////////////////////////////////
// set new order and shape in case of suitable array length
    bool NDArray::reshapei(const char order, const std::vector<Nd4jLong>& cshape) {

        // check firstly whether cshape is identical to shape of array, if yes then reshape is unnecessary
        if(order == ordering() && rankOf() == cshape.size()) {
            bool areShapesSame = true;
            for(int i = 0; i < cshape.size(); ++i)
                if(cshape[i] != sizeAt(i)) {
                    areShapesSame = false;
                    break;
                }
            if(areShapesSame)
                return areShapesSame;
        }

        std::vector<Nd4jLong> shape(cshape);
        int rank = shape.size();

        // looking for negative in shape

        int numberNegativesOnes = 0;

        Nd4jLong* shape_ = shape.data();
        for (int i = 0; i < (int) shape.size(); i++) {
            if (shape[i] < 0) {
                if (numberNegativesOnes >= 1)
                    throw std::runtime_error("Only one dimension can be negative at once");

                numberNegativesOnes++;

                int shapeLength = 1;
                for (int j = 0; j < (int) shape.size(); j++)
                    if (i != j)
                        shapeLength *= shape_[j];

                Nd4jLong realShape = nd4j::math::nd4j_abs<int>(lengthOf() / shapeLength);
                auto thisNewShape = new Nd4jLong[shape.size()];

                for (int j = 0; j < (int) shape.size(); j++)
                    if (i != j)
                        thisNewShape[j] = shape_[j];
                    else
                        thisNewShape[j] = realShape;

                shape_ = thisNewShape;
            }
        }

        for (int e = 0; e < (int) shape.size(); e++)
            shape[e] = shape_[e];

        if (numberNegativesOnes > 0)
            delete[] shape_;

        int arrLength = 1;
        for(const auto& item : shape)
            arrLength *= item;

        if(_buffer==nullptr || arrLength != this->lengthOf()) {
            this->printShapeInfo("Mismatched shape");
            nd4j::Logger::printv("Shape requested: ", shape);
            nd4j_debug("Requested length in reshape: %i; Existing length: %i;\n", arrLength, this->lengthOf());
            throw std::runtime_error("Bad shape!");
        }

        int shapeLength = shape::shapeInfoLength(rank);
        // remember old values

        // we can do this only if there was no permute applied, or there are no weird strides
        if (shape::canReshape(this->rankOf(), this->_shapeInfo, shape.size(), shape.data(), order == 'f')) {
            Nd4jLong *shapeInfoNew;
            ALLOCATE(shapeInfoNew, _context->getWorkspace(), shape::shapeInfoLength(rank), Nd4jLong);

            shape::reshapeCF(this->rankOf(), this->_shapeInfo, shape.size(), shape.data(), order == 'f', shapeInfoNew);

            if (_isShapeAlloc)
                RELEASE(_shapeInfo, _context->getWorkspace());

            ArrayOptions::setDataType(shapeInfoNew, this->dataType());
            _shapeInfo = shapeInfoNew;
            _isShapeAlloc = true;
        } else {
            Nd4jLong *shapeInfoNew;
            ALLOCATE(shapeInfoNew, _context->getWorkspace(), shape::shapeInfoLength(rank), Nd4jLong);

            if (order == 'c')
                shape::shapeBuffer(shape.size(), dataType(), shape.data(), shapeInfoNew);
            else
                shape::shapeBufferFortran(shape.size(), dataType(), shape.data(), shapeInfoNew);

            int8_t *newBuffer;
            ALLOCATE(newBuffer, _context->getWorkspace(), this->lengthOf() * sizeOfT(), int8_t);

            NativeOpExecutioner::execTransformSame(nullptr, transform::Copy, _buffer, _shapeInfo, _bufferD, _shapeInfoD, newBuffer, shapeInfoNew, nullptr, nullptr, nullptr, nullptr, nullptr);

            if (_isBuffAlloc)
                RELEASE(_buffer, _context->getWorkspace());


            if (_isShapeAlloc)
                RELEASE(_shapeInfo, _context->getWorkspace());

            _buffer = newBuffer;
            _shapeInfo = shapeInfoNew;
            _isShapeAlloc = true;
            _isBuffAlloc = true;
        }

        return true;
    }

    ////////////////////////////////////////////////////////////////////////
    void NDArray::setIdentity() {
        if (isS())
            throw std::runtime_error("NDArray::setIdentity: you can't use this method on String array!");

        this->assign(0.);

        int  rank    = rankOf();
        auto shape   = shapeOf();
        auto strides = stridesOf();
        int  minDim  = 100000000;
        Nd4jLong indices[MAX_RANK];
        for(int j = 0; j < rank; ++j)
            indices[j] = 1;

        Nd4jLong offset = shape::getOffset(0, shape, strides, indices, rank);

        for(int i = 0; i < rank; ++i)
            if(minDim > shape[i])
                minDim = shape[i];

        float v = 1.0f;
#pragma omp parallel for if(minDim > Environment::getInstance()->elementwiseThreshold()) schedule(guided)
        for(int i = 0; i < minDim; ++i)
            templatedSet<float>(_buffer, i*offset, this->dataType(), &v);
    }

    template <typename T>
    void NDArray::templatedSet(void *buffer, const Nd4jLong xOfsset, nd4j::DataType dtype, const void *value) {
        BUILD_SINGLE_PARTIAL_SELECTOR(dtype, templatedSet< , T>(buffer, xOfsset, value), LIBND4J_TYPES);
    }
    BUILD_SINGLE_TEMPLATE(template void NDArray::templatedSet, (void *buffer, const Nd4jLong xOfsset, nd4j::DataType dtype, const void *value), LIBND4J_TYPES);



    template <typename T>
    void NDArray::templatedSwap(void *xBuffer, void *yBuffer, Nd4jLong length) {
        auto x = reinterpret_cast<T *>(xBuffer);
        auto y = reinterpret_cast<T *>(yBuffer);

#pragma omp parallel for simd schedule(static)
        for (int i = 0; i < length; ++i) {
            auto temp = x[i];
            x[i] = y[i];
            y[i] = temp;
        }
    }
    BUILD_SINGLE_TEMPLATE(template void NDArray::templatedSwap, (void *xBuffer, void *yBuffer, Nd4jLong length), LIBND4J_TYPES);

    ////////////////////////////////////////////////////////////////////////
    void NDArray::swapUnsafe(NDArray& other) {
        auto xType = this->dataType();

        if (xType != other.dataType())
            throw std::runtime_error("NDArray::swapUnsage method: both arrays must have the same data type");

        if(_buffer == nullptr || other._buffer == nullptr)
            throw std::runtime_error("NDArray::swapUnsafe method: input array should not be empty!");

        // if(_buffer == other._buffer)
        //     throw std::runtime_error("NDArray::swapUnsafe method: the buffers of input arrays should not point on the same address!");

        if(lengthOf() != other.lengthOf())
            throw std::runtime_error("NDArray::swapUnsafe method: input arrays should have the same length!");

        BUILD_SINGLE_SELECTOR(xType, templatedSwap, (this->_buffer, other.buffer(), this->lengthOf()), LIBND4J_TYPES);
    }

    ////////////////////////////////////////////////////////////////////////
    NDArray* NDArray::diagonal(const char type) const {

        if (isS())
            throw std::runtime_error("NDArray::diagonal: you can't use this method on String array!");

        const char order = ordering();
        const int  rank  = rankOf();
        Nd4jLong *outShapeInfo;
        ALLOCATE(outShapeInfo, _context->getWorkspace(), 8, Nd4jLong);
        outShapeInfo[0] = 2;
        outShapeInfo[5] = 0;

        if(isVector() || isScalar()) {

            outShapeInfo[1] = outShapeInfo[2] = outShapeInfo[3] = outShapeInfo[4] = 1;
            outShapeInfo[6] = 1;
            outShapeInfo[7] = (int)order;
        }
        else {

            int diagSize  = 100000000;
            Nd4jLong indices[MAX_RANK];

            for(int i = 0; i < rank; ++i) {
                if(diagSize > shapeOf()[i])
                    diagSize = shapeOf()[i];
                indices[i] = 1;
            }

            auto step = shape::getOffset(0, shapeOf(), stridesOf(), indices, rank);

            if(type == 'c') {
                outShapeInfo[1] = diagSize;
                outShapeInfo[2] = 1;
            }
            else {
                outShapeInfo[1] = 1;
                outShapeInfo[2] = diagSize;
            }
            shape::updateStrides(outShapeInfo, order);

            outShapeInfo[3] *= step;
            outShapeInfo[4] *= step;
            outShapeInfo[6] =  -1;
        }

        ArrayOptions::setDataType(outShapeInfo, this->dataType());

        auto result = new NDArray(this->_buffer, outShapeInfo, this->_context);
        result->_isShapeAlloc = true;
        return result;
    }

    void NDArray::streamline(char o) {
        char order = o == 'a' ? this->ordering() : o;

        Nd4jLong *newShape;
        ALLOCATE(newShape, this->_context->getWorkspace(), shape::shapeInfoLength(this->rankOf()), Nd4jLong);

        int8_t *newBuffer;
        ALLOCATE(newBuffer, this->_context->getWorkspace(), this->lengthOf() * sizeOfT(), int8_t);

        std::vector<Nd4jLong> shape(this->rankOf());
        for (int e = 0; e < this->rankOf(); e++)
            shape[e] = this->sizeAt(e);

        if (order == 'c')
            shape::shapeBuffer(this->rankOf(),dataType(),  shape.data(), newShape);
        else
            shape::shapeBufferFortran(this->rankOf(), dataType(), shape.data(), newShape);

        if (!isView()) {
            NativeOpExecutioner::execTransformSame(nullptr, transform::Copy, _buffer, _shapeInfo, nullptr, nullptr, newBuffer, newShape, nullptr, nullptr, nullptr, nullptr, nullptr);
            memcpy(_buffer, newBuffer, this->lengthOf() * sizeOfT());

            //if (_isBuffAlloc)
            //    RELEASE(this->_buffer, this->_workspace);
            if (_isShapeAlloc)
                RELEASE(this->_shapeInfo, this->_context->getWorkspace());

            //this->_buffer = newBuffer;
            //this->_isBuffAlloc = true;

            RELEASE(newBuffer, this->_context->getWorkspace());

            this->_shapeInfo = newShape;
            this->_isShapeAlloc = true;
        } else {
            NativeOpExecutioner::execTransformSame(nullptr, transform::Copy, _buffer, _shapeInfo, nullptr, nullptr, newBuffer, newShape, nullptr, nullptr, nullptr, nullptr, nullptr);

            if (_isBuffAlloc)
                RELEASE(this->_buffer, this->_context->getWorkspace());
            if (_isShapeAlloc)
                RELEASE(this->_shapeInfo, this->_context->getWorkspace());

            this->_buffer = newBuffer;
            this->_isBuffAlloc = true;

            this->_shapeInfo = newShape;
            this->_isShapeAlloc = true;
        }
    }

    void NDArray::applyPairwiseTransform(nd4j::pairwise::Ops op, const NDArray* other, NDArray *target, ExtraArguments *extraParams) const{
        if (isS())
            throw std::runtime_error("NDArray::applyPairwiseTransform: you can't use this method on String array!");
        if (other->lengthOf() != target->lengthOf())
            throw std::invalid_argument("NDArray::applyPairwiseTransform method - lengths of arrays are mismatched");
        if (target->_dataType != this->_dataType && target->_dataType != other->_dataType)
            throw std::invalid_argument("NDArray::applyPairwiseTransform method - type of target array must be the same as type of this or other array !");
        if (_context == nullptr)
            throw std::runtime_error("Launch context cannot be NULL!!!");
        if (_context->getCudaStream() == nullptr)
            throw std::runtime_error("CUDA stream cannot be NULL!!!");


        if (!this->isActualOnDeviceSide())
            this->syncToDevice();

        if (!other->isActualOnDeviceSide())
            other->syncToDevice();

        NativeOpExecutioner::execPairwiseTransform(_context, op, this->_buffer, this->_shapeInfo, this->_bufferD, this->_shapeInfoD, other->_buffer, other->_shapeInfo, other->_bufferD, other->_shapeInfoD, target->_buffer, target->_shapeInfo, target->_bufferD, target->_shapeInfoD, extraParams != nullptr ? extraParams->argumentAsT(target->dataType()) : nullptr);

        if(target)
            target->tickWriteDevice();
        else
            this->tickWriteDevice();

        if (extraParams != nullptr)
            this->synchronize();
    }

    void NDArray::syncToHost() const {
        hipStreamSynchronize(*_context->getCudaStream());
        if (this->_buffer == nullptr) {
            NDArray* constThis =  const_cast<NDArray*>(this); // not recommended solution
            ALLOCATE(constThis->_buffer, constThis->_context->getWorkspace(), constThis->lengthOf() * constThis->sizeOfT(), int8_t);
            constThis->_isBuffAlloc = true;
        }
        hipMemcpy(this->_buffer, this->_bufferD, this->lengthOf() * this->sizeOfT(), hipMemcpyDeviceToHost);
        tickReadHost();
    }

    void NDArray::syncToDevice() const {
        hipMemcpy(this->_bufferD, this->_buffer, this->lengthOf() * this->sizeOfT(), hipMemcpyHostToDevice);
        tickReadDevice();
    }

    void NDArray::syncShape() const {
        hipMemcpy(_shapeInfoD, _shapeInfo, shape::shapeInfoByteLength(_shapeInfo), hipMemcpyHostToDevice);
    }

    template <typename X, typename Y>
    void NDArray::templatedDoubleAssign(void *xBuffer, const Nd4jLong xOffset, const void *yBuffer, const Nd4jLong yOffset) const {
        auto x = reinterpret_cast<X *>(xBuffer);
        const auto y = reinterpret_cast<const Y *>(yBuffer);

        x[xOffset] = static_cast<X>(y[yOffset]);
    }
    BUILD_DOUBLE_TEMPLATE(template void NDArray::templatedDoubleAssign, (void *xBuffer, const Nd4jLong xOffset, const void *yBuffer, const Nd4jLong yOffset) const, LIBND4J_TYPES, LIBND4J_TYPES);

    // This method assigns values of given NDArray to this one
    void NDArray::assign(const NDArray& other) {

        if (this == &other)
            return;

        if (!Environment::getInstance()->isExperimentalBuild() && (this->dataType() != other.dataType() && other.dataType() != DataType::BOOL)) {
            throw datatype_exception::build("NDArray::assign: cannot assign array of different types", this->dataType(), other.dataType());
        }

        if (other.isScalar()) {
            if(this->isScalar()) {
                if (!this->isEmpty() && !other.isEmpty()) {
                    BUILD_DOUBLE_SELECTOR(_dataType, other._dataType, templatedDoubleAssign,
                                          (_buffer, 0, other._buffer, 0), LIBND4J_TYPES, LIBND4J_TYPES);
                    syncToDevice();
                }
                else if (this->isEmpty() != other.isEmpty()) { // need assign non-empty scalar to empty
                    if (other.isEmpty()) {
                        ArrayOptions::setPropertyBit(this->_shapeInfo, ARRAY_EMPTY);
                        syncShape();
                    }
                    else
                        *this = other;
                }
            }
            else {
                syncToDevice();
                NativeOpExecutioner::execScalar(_context, scalar::CopyPws, _buffer, _shapeInfo, _bufferD, _shapeInfoD, _buffer, _shapeInfo, _bufferD, _shapeInfoD, other._buffer, other._shapeInfo, other._bufferD, other._shapeInfoD, nullptr);
            }
            tickWriteDevice();
            return;
        }

        if (other._length != _length) {
            auto shapeThis = ShapeUtils::shapeAsString(this);
            auto shapeThat = ShapeUtils::shapeAsString(&other);
            nd4j_printf("Can't assign new value to the array: this shape %s; other shape: %s\n", shapeThis.c_str(), shapeThat.c_str());
            throw std::runtime_error("Lengths of arrays are mismatched");
        }

        //syncToDevice();

        // memcpy is allowed only for same order && same ews (being equal to 1)
        if (ordering() == other.ordering() && _dataType == other._dataType && ews() == 1 && other.ews() == 1) {
            hipMemcpy(_bufferD, other._bufferD, _length * sizeOfT(), hipMemcpyDeviceToDevice);
        }
        else if(_dataType == other._dataType)
            NativeOpExecutioner::execTransformSame(_context, transform::Copy, other._buffer, other._shapeInfo, other._bufferD, other._shapeInfoD, _buffer, _shapeInfo, _bufferD, _shapeInfoD, nullptr, nullptr, nullptr);
        else
            NativeOpExecutioner::execPairwiseTransform(_context, pairwise::CopyPws, _buffer, _shapeInfo, _bufferD, _shapeInfoD, other._buffer, other._shapeInfo, other._bufferD, other._shapeInfoD, _buffer, _shapeInfo, _bufferD, _shapeInfoD, nullptr);
        syncToHost();

        tickWriteDevice();
    }

    ////////////////////////////////////////////////////////////////////////
// This method returns new copy of this NDArray, optionally in different order
    NDArray* NDArray::dup(const char newOrder) {

        char order = newOrder == 'a' ? ordering() : newOrder;

        auto outShapeInfo = ShapeBuilders::createShapeInfo(_dataType, order, getShapeAsVector(), _context->getWorkspace());
        void* outBuffer = nullptr;
        //int8_t* outBufferD = nullptr;
        //Nd4jLong* outShapeD = nullptr;
        ALLOCATE(outBuffer, _context->getWorkspace(), _length * sizeOfT(), int8_t);
        auto result = new NDArray(outBuffer, outShapeInfo, _context, true, true);
        //result->setSpecialBuffers(outBufferD, outShapeD);
        //syncToDevice();
        result->assign(*this);

        return result;
    }

    void NDArray::synchronize() const {
        auto res = hipStreamSynchronize(*(_context->getCudaStream()));
        if (res != 0)
            throw std::runtime_error("Synchronization failed");
    }

    void NDArray::registerSpecialUse(const std::initializer_list<const NDArray*>& writeList, const std::initializer_list<const NDArray*>& readList) {
        // no-op
        for (auto p:writeList) {
            if (!p->isActualOnDeviceSide())
                p->syncToDevice();

            p->tickWriteDevice();
        }

        for (auto p:readList) {
            if (!p->isActualOnDeviceSide())
                p->syncToDevice();

            p->tickReadDevice();
        }
    }
    

//////////////////////////////////////////////////////////////////////////
    template <>
    utf8string NDArray::e(const Nd4jLong i) const {
        if (i >= _length)
            throw std::invalid_argument("NDArray::e(i): input index is out of array length !");

        if (!isS())
            throw std::runtime_error("This method is available for String arrays only");

        lazyAllocateBuffer();
        if(!isActualOnHostSide()) 
            syncToHost();

        auto rp = getOffset(i);
        tickReadHost();
        return *(reinterpret_cast<utf8string**>(_buffer)[rp]);
    }

//////////////////////////////////////////////////////////////////////////
    template <>
    std::string NDArray::e(const Nd4jLong i) const {
        
        lazyAllocateBuffer();
        if(!isActualOnHostSide())
            syncToHost();

        auto u = e<utf8string>(i);
        std::string r(u._buffer);
        tickReadHost();
        return r;
    }

//////////////////////////////////////////////////////////////////////////
    template <typename T>
    T NDArray::e(const Nd4jLong i) const {

        if (i >= _length)
            throw std::invalid_argument("NDArray::e(i): input index is out of array length !");

        lazyAllocateBuffer();
        if(!isActualOnHostSide())
            syncToHost();

        auto rp = getOffset(i);

        BUILD_SINGLE_PARTIAL_SELECTOR(this->dataType(), return templatedGet<, T>(this->_buffer, rp), LIBND4J_TYPES);

        tickReadHost();
//        return static_cast<T>(119);
    }
    BUILD_SINGLE_UNCHAINED_TEMPLATE(template , NDArray::e(const Nd4jLong) const, LIBND4J_TYPES);
    //BUILD_DOUBLE_TEMPLATE(template void NDArray::templatedSet, (void *buffer, const Nd4jLong *indices, Y value), LIBND4J_TYPES, LIBND4J_TYPES);


//////////////////////////////////////////////////////////////////////////
// Returns value from 2D matrix by coordinates/indexes
    template <typename T>
    T NDArray::e(const Nd4jLong i, const Nd4jLong j) const {
        if (rankOf() != 2 || i >= shapeOf()[0] || j >= shapeOf()[1])
            throw std::invalid_argument("NDArray::e(i,j): one of input indexes is out of array length or rank!=2 !");

        lazyAllocateBuffer();
        if(!isActualOnHostSide()) 
            syncToHost();

        auto xType = this->dataType();
        Nd4jLong coords[2] = {i, j};
        auto xOffset = shape::getOffset(0, shapeOf(), stridesOf(), coords, rankOf());
        //return (*this)(i, j);
        BUILD_SINGLE_PARTIAL_SELECTOR(xType, return templatedGet<, T>(this->_buffer, xOffset), LIBND4J_TYPES);
        
        tickReadHost();
        return static_cast<T>(119);
    }
    BUILD_SINGLE_UNCHAINED_TEMPLATE(template , NDArray::e(const Nd4jLong, const Nd4jLong) const, LIBND4J_TYPES);

//////////////////////////////////////////////////////////////////////////
// returns value from 3D tensor by coordinates
    template <typename T>
    T NDArray::e(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k) const {
        //return (*this)(i, j, k);
        if (rankOf() != 3 || i >= shapeOf()[0] || j >= shapeOf()[1] || k >= shapeOf()[2])
            throw std::invalid_argument("NDArray::e(i,j,k): one of input indexes is out of array length or rank!=3 !");

        lazyAllocateBuffer();
        if(!isActualOnHostSide()) 
            syncToHost();

        auto xType = this->dataType();
        Nd4jLong coords[3] = {i, j, k};
        auto xOffset = shape::getOffset(0, shapeOf(), stridesOf(), coords, rankOf());
        BUILD_SINGLE_PARTIAL_SELECTOR(xType, return templatedGet<, T>(this->_buffer, xOffset), LIBND4J_TYPES);
        
        tickReadHost();
        return static_cast<T>(119);
    }
    BUILD_SINGLE_UNCHAINED_TEMPLATE(template , NDArray::e(const Nd4jLong, const Nd4jLong, const Nd4jLong) const, LIBND4J_TYPES);

//////////////////////////////////////////////////////////////////////////
    // returns value from 3D tensor by coordinates
    template <typename T>
    T NDArray::e(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const Nd4jLong l) const {
        //return (*this)(i, j, k);
        if (rankOf() != 4 || i >= shapeOf()[0] || j >= shapeOf()[1] || k >= shapeOf()[2] || l >= shapeOf()[3])
            throw std::invalid_argument("NDArray::e(i,j,k,l): one of input indexes is out of array length or rank!=4 !");

        lazyAllocateBuffer();
        if(!isActualOnHostSide()) 
            syncToHost();

        auto xType = this->dataType();
        Nd4jLong coords[4] = {i, j, k, l};
        auto xOffset = shape::getOffset(0, shapeOf(), stridesOf(), coords, rankOf());
        BUILD_SINGLE_PARTIAL_SELECTOR(xType, return templatedGet<, T>(this->_buffer, xOffset), LIBND4J_TYPES);

        tickReadHost();
        return static_cast<T>(119);
    }
    BUILD_SINGLE_UNCHAINED_TEMPLATE(template , NDArray::e(const Nd4jLong, const Nd4jLong, const Nd4jLong, const Nd4jLong) const, LIBND4J_TYPES);

//////////////////////////////////////////////////////////////////////////
NDArray NDArray::e(const Nd4jLong i) const {
    if (i >= _length)
        throw std::invalid_argument("scalar NDArray::e(i): input index is out of array length !");
    
    NDArray scalar(_dataType, _context);
    
    if(isActualOnHostSide()) {
        hipMemcpy(scalar._bufferD, bufferWithOffset(getOffset(i)), sizeOfT(), hipMemcpyHostToDevice);
        tickReadHost();
    }
    else {
        hipMemcpy(scalar._bufferD, specialBufferWithOffset(getOffset(i)), sizeOfT(), hipMemcpyDeviceToDevice);
        tickReadDevice();
    }

    scalar.tickWriteDevice();
    return scalar;
}    

////////////////////////////////////////////////////////////////////////
#ifndef __JAVACPP_HACK__

    template<typename T>
    void NDArray::applyTriplewiseLambda(NDArray* second, NDArray *third, const std::function<T(T, T, T)>& func, NDArray* target) {
        if (target == nullptr)
            target = this;

        if (second == nullptr) {
            nd4j_printf("applyTriplewiseLambda requires three operands to be valid NDArrays, but Second is NULL\n","");
            throw std::runtime_error("second is null");
        }

        if (third == nullptr) {
            nd4j_printf("applyTriplewiseLambda requires three operands to be valid NDArrays, but Third is NULL\n","");
            throw std::runtime_error("third is null");
        }
        if(_dataType != DataTypeUtils::fromT<T>())
            throw std::runtime_error("NDArray::applyTriplewiseLambda<T> method: wrong template parameter T, its type should be the same as type of this array!");
        if(_dataType != second->_dataType || _dataType != third->_dataType || _dataType != target->_dataType)
            throw std::runtime_error("NDArray::applyTriplewiseLambda<T> method: bother four arrays (this, second, third, target) should have the same type !");

        if (this->lengthOf() != second->lengthOf() || this->lengthOf() != third->lengthOf() || !this->isSameShape(second) || !this->isSameShape(third)) {
            nd4j_printf("applyPairwiseLambda requires both operands to have the same shape\n","");
            throw std::runtime_error("Shapes mismach");
        }

        auto f = this->bufferAsT<T>();
        auto s = second->bufferAsT<T>();
        auto t = third->bufferAsT<T>();
        auto z = target->bufferAsT<T>();

        if (this->ordering() == second->ordering() && this->ordering() == third->ordering()  && this->ordering() == target->ordering() && (this->ews() == 1 && target->ews() == 1) && this->ews() == second->ews() && this->ews() == third->ews()) {
#pragma omp parallel for simd schedule(static)
            for (Nd4jLong e = 0; e < this->lengthOf(); e++)
                z[e] = func(f[e], s[e], t[e]);
        } else {
            if (f == z) {

#pragma omp parallel for schedule(guided)
                for (int e = 0; e < this->lengthOf(); e++) {

                    auto tOffset = this->getOffset(e);
                    auto uOffset = second->getOffset(e);
                    auto vOffset = third->getOffset(e);

                    f[tOffset] = func(f[tOffset], s[uOffset], t[vOffset]);
                }
            } else {

#pragma omp parallel for schedule(guided)
                for (int e = 0; e < this->lengthOf(); e++) {

                    auto tOffset = this->getOffset(e);
                    auto uOffset = second->getOffset(e);
                    auto vOffset = third->getOffset(e);
                    auto zOffset = target->getOffset(e);

                    z[zOffset] = func(f[tOffset], s[uOffset], t[vOffset]);
                }
            }
        }
        target->tickWriteDevice();
    }
    template void NDArray::applyTriplewiseLambda(NDArray* second, NDArray *third, const std::function<double (double, double, double)>& func, NDArray* target);
    template void NDArray::applyTriplewiseLambda(NDArray* second, NDArray *third, const std::function<float (float, float, float)>& func, NDArray* target);
    template void NDArray::applyTriplewiseLambda(NDArray* second, NDArray *third, const std::function<float16 (float16, float16, float16)>& func, NDArray* target);
    template void NDArray::applyTriplewiseLambda(NDArray* second, NDArray *third, const std::function<bfloat16 (bfloat16, bfloat16, bfloat16)>& func, NDArray* target);
    template void NDArray::applyTriplewiseLambda(NDArray* second, NDArray *third, const std::function<Nd4jLong (Nd4jLong, Nd4jLong, Nd4jLong)>& func, NDArray* target);
    template void NDArray::applyTriplewiseLambda(NDArray* second, NDArray *third, const std::function<int (int, int, int)>& func, NDArray* target);
    template void NDArray::applyTriplewiseLambda(NDArray* second, NDArray *third, const std::function<int16_t (int16_t, int16_t, int16_t)>& func, NDArray* target);
    template void NDArray::applyTriplewiseLambda(NDArray* second, NDArray *third, const std::function<uint8_t (uint8_t, uint8_t, uint8_t)>& func, NDArray* target);
    template void NDArray::applyTriplewiseLambda(NDArray* second, NDArray *third, const std::function<int8_t (int8_t, int8_t, int8_t)>& func, NDArray* target);
    template void NDArray::applyTriplewiseLambda(NDArray* second, NDArray *third, const std::function<bool (bool, bool, bool)>& func, NDArray* target);


    template<typename T>
    void NDArray::applyPairwiseLambda(NDArray* other, const std::function<T(T, T)>& func, NDArray* target) {
        if (target == nullptr)
            target = this;

        if (other == nullptr) {
            nd4j_printf("applyPairwiseLambda requires both operands to be valid NDArrays, but Y is NULL\n","");
            throw std::runtime_error("Other is null");
        }

        if(_dataType != DataTypeUtils::fromT<T>())
            throw std::runtime_error("NDArray::applyPairwiseLambda<T> method: wrong template parameter T, its type should be the same as type of this array!");
        if(_dataType != other->_dataType || _dataType != target->_dataType)
            throw std::runtime_error("NDArray::applyPairwiseLambda<T> method: all three arrays (this, other, target) must have the same type !");

        if (this->lengthOf() != other->lengthOf()) {
            nd4j_printf("applyPairwiseLambda requires both operands to have the same shape\n","");
            throw std::runtime_error("Shapes mismach");
        }

        auto f = this->bufferAsT<T>();
        auto s = other->bufferAsT<T>();
        auto z = target->bufferAsT<T>();

        if (this->ordering() == other->ordering() && this->ordering() == target->ordering() && (this->ews() == 1 && target->ews() == 1) && this->ews() == other->ews()) {
#pragma omp parallel for simd schedule(guided)
            for (int e = 0; e < this->lengthOf(); e++)
                z[e] = func(f[e], s[e]);
        } else {
            if (f == z) {

#pragma omp parallel for schedule(guided)
                for (int e = 0; e < this->lengthOf(); e++) {

                    auto xOffset = this->getOffset(e);
                    auto yOffset = other->getOffset(e);

                    f[xOffset] = func(f[xOffset], s[yOffset]);
                }
            } else {

#pragma omp parallel for schedule(guided)
                for (int e = 0; e < this->lengthOf(); e++) {

                    auto xOffset = this->getOffset(e);
                    auto yOffset = other->getOffset(e);
                    auto zOffset = target->getOffset(e);

                    z[zOffset] = func(f[xOffset], s[yOffset]);
                }
            }
        }
        target->tickWriteDevice();
    }
    template void NDArray::applyPairwiseLambda(NDArray* other, const std::function<double (double, double)>& func, NDArray* target);
    template void NDArray::applyPairwiseLambda(NDArray* other, const std::function<float (float, float)>& func, NDArray* target);
    template void NDArray::applyPairwiseLambda(NDArray* other, const std::function<float16 (float16, float16)>& func, NDArray* target);
    template void NDArray::applyPairwiseLambda(NDArray* other, const std::function<bfloat16 (bfloat16, bfloat16)>& func, NDArray* target);
    template void NDArray::applyPairwiseLambda(NDArray* other, const std::function<Nd4jLong (Nd4jLong, Nd4jLong)>& func, NDArray* target);
    template void NDArray::applyPairwiseLambda(NDArray* other, const std::function<int (int, int)>& func, NDArray* target);
    template void NDArray::applyPairwiseLambda(NDArray* other, const std::function<int16_t (int16_t, int16_t)>& func, NDArray* target);
    template void NDArray::applyPairwiseLambda(NDArray* other, const std::function<uint8_t (uint8_t, uint8_t)>& func, NDArray* target);
    template void NDArray::applyPairwiseLambda(NDArray* other, const std::function<int8_t (int8_t, int8_t)>& func, NDArray* target);
    template void NDArray::applyPairwiseLambda(NDArray* other, const std::function<bool (bool, bool)>& func, NDArray* target);


////////////////////////////////////////////////////////////////////////
    template<typename T>
    void NDArray::applyLambda(const std::function<T(T)>& func, NDArray* target) {
        if (target == nullptr)
            target = this;

        if(_dataType != DataTypeUtils::fromT<T>())
            throw std::runtime_error("NDArray::applyLambda<T> method: wrong template parameter T, its type should be the same as type of this array!");
        if(_dataType != target->_dataType)
            throw std::runtime_error("NDArray::applyLambda<T> method: types of this and target array should match !");

        auto f = this->bufferAsT<T>();
        auto z = target->bufferAsT<T>();

        if (this->ordering() == target->ordering() && (this->ews() == 1 && target->ews() == 1)) {
#pragma omp parallel for simd schedule(guided)
            for (int e = 0; e < this->lengthOf(); e++)
                z[e] = func(f[e]);
        } else {
            if (f == z) {

#pragma omp parallel for schedule(guided)
                for (int e = 0; e < this->lengthOf(); e++) {

                    auto xOffset = this->getOffset(e);

                    f[xOffset] = func(f[xOffset]);
                }
            } else {

#pragma omp parallel for schedule(guided)
                for (int e = 0; e < this->lengthOf(); e++) {

                    auto xOffset = this->getOffset(e);
                    auto zOffset = target->getOffset(e);

                    z[zOffset] = func(f[xOffset]);
                }
            }
        }
        target->tickWriteDevice();
    }
    template void NDArray::applyLambda(const std::function<double(double)>& func, NDArray* target);
    template void NDArray::applyLambda(const std::function<float(float)>& func, NDArray* target);
    template void NDArray::applyLambda(const std::function<float16(float16)>& func, NDArray* target);
    template void NDArray::applyLambda(const std::function<bfloat16(bfloat16)>& func, NDArray* target);
    template void NDArray::applyLambda(const std::function<Nd4jLong(Nd4jLong)>& func, NDArray* target);
    template void NDArray::applyLambda(const std::function<int16_t(int16_t)>& func, NDArray* target);
    template void NDArray::applyLambda(const std::function<int32_t(int32_t)>& func, NDArray* target);
    template void NDArray::applyLambda(const std::function<uint8_t(uint8_t)>& func, NDArray* target);
    template void NDArray::applyLambda(const std::function<int8_t(int8_t)>& func, NDArray* target);
    template void NDArray::applyLambda(const std::function<bool(bool)>& func, NDArray* target);

    template<typename T>
    void NDArray::applyIndexedLambda(const std::function<T(Nd4jLong, T)>& func, NDArray* target) {
        if (target == nullptr)
            target = this;

        if(_dataType != DataTypeUtils::fromT<T>())
            throw std::runtime_error("NDArray::applyIndexedLambda<T> method: wrong template parameter T, its type should be the same as type of this array!");
        if(_dataType != target->_dataType)
            throw std::runtime_error("NDArray::applyIndexedLambda<T> method: types of this and target array should match !");

        auto f = this->bufferAsT<T>();
        auto z = target->bufferAsT<T>();

        if (this->ordering() == target->ordering() && (this->ews() == 1 && target->ews() == 1)) {
#pragma omp parallel for simd schedule(guided)
            for (Nd4jLong e = 0; e < this->lengthOf(); e++)
                z[e] = func(e, f[e]);
        } else {
            if (f == z) {

#pragma omp parallel for schedule(guided)
                for (Nd4jLong e = 0; e < this->lengthOf(); e++) {

                    auto xOffset = this->getOffset(e);

                    f[xOffset] = func(e, f[xOffset]);
                }
            } else {

#pragma omp parallel for schedule(guided)
                for (Nd4jLong e = 0; e < this->lengthOf(); e++) {

                    auto xOffset = this->getOffset(e);
                    auto zOffset = target->getOffset(e);

                    z[zOffset] = func(e, f[xOffset]);
                }
            }
        }
        target->tickWriteDevice();
    }
    template void NDArray::applyIndexedLambda(const std::function<double(Nd4jLong, double)>& func, NDArray* target);
    template void NDArray::applyIndexedLambda(const std::function<float(Nd4jLong, float)>& func, NDArray* target);
    template void NDArray::applyIndexedLambda(const std::function<float16(Nd4jLong, float16)>& func, NDArray* target);
    template void NDArray::applyIndexedLambda(const std::function<bfloat16(Nd4jLong, bfloat16)>& func, NDArray* target);
    template void NDArray::applyIndexedLambda(const std::function<Nd4jLong(Nd4jLong, Nd4jLong)>& func, NDArray* target);
    template void NDArray::applyIndexedLambda(const std::function<int(Nd4jLong, int)>& func, NDArray* target);
    template void NDArray::applyIndexedLambda(const std::function<int16_t(Nd4jLong, int16_t)>& func, NDArray* target);
    template void NDArray::applyIndexedLambda(const std::function<uint8_t (Nd4jLong, uint8_t)>& func, NDArray* target);
    template void NDArray::applyIndexedLambda(const std::function<int8_t(Nd4jLong, int8_t)>& func, NDArray* target);
    template void NDArray::applyIndexedLambda(const std::function<bool(Nd4jLong, bool)>& func, NDArray* target);


    template<typename T>
    void NDArray::applyIndexedPairwiseLambda(NDArray* other, const std::function<T(Nd4jLong, T, T)>& func, NDArray* target) {
        if (target == nullptr)
            target = this;

        if (other == nullptr) {
            nd4j_printf("applyIndexedPairwiseLambda requires both operands to be valid NDArrays, but Y is NULL\n","");
            throw std::runtime_error("Other is null");
        }
        if(_dataType != DataTypeUtils::fromT<T>())
            throw std::runtime_error("NDArray::applyIndexedPairwiseLambda<T> method: wrong template parameter T, its type should be the same as type of this array!");
        if(_dataType != target->_dataType)
            throw std::runtime_error("NDArray::applyIndexedPairwiseLambda<T> method: types of this and target array should match !");
        if (this->lengthOf() != other->lengthOf()) {
            nd4j_printf("applyIndexedPairwiseLambda requires both operands to have the same shape\n","");
            throw std::runtime_error("Shapes mismach");
        }

        auto f = this->bufferAsT<T>();
        auto s = other->bufferAsT<T>();
        auto z = target->bufferAsT<T>();

        if (this->ordering() == other->ordering() && this->ordering() == target->ordering() && (this->ews() == 1 && target->ews() == 1) && this->ews() == other->ews()) {
#pragma omp parallel for simd schedule(guided)
            for (Nd4jLong e = 0; e < this->lengthOf(); e++)
                z[e] = func((Nd4jLong) e, f[e], s[e]);
        } else {
            if (f == z) {

#pragma omp parallel for schedule(guided)
                for (int e = 0; e < this->lengthOf(); e++) {

                    auto xOffset = this->getOffset(e);
                    auto yOffset = other->getOffset(e);

                    f[xOffset] = func((Nd4jLong) e, f[xOffset], s[yOffset]);
                }
            } else {

#pragma omp parallel for schedule(guided)
                for (int e = 0; e < this->lengthOf(); e++) {

                    auto xOffset = this->getOffset(e);
                    auto yOffset = other->getOffset(e);
                    auto zOffset = target->getOffset(e);

                    z[zOffset] = func((Nd4jLong) e, f[xOffset], s[yOffset]);
                }
            }
        }
        target->tickWriteDevice();
    }
    template void NDArray::applyIndexedPairwiseLambda(NDArray* other, const std::function<double (Nd4jLong, double, double)>& func, NDArray* target);
    template void NDArray::applyIndexedPairwiseLambda(NDArray* other, const std::function<float (Nd4jLong, float, float)>& func, NDArray* target);
    template void NDArray::applyIndexedPairwiseLambda(NDArray* other, const std::function<float16 (Nd4jLong, float16, float16)>& func, NDArray* target);
    template void NDArray::applyIndexedPairwiseLambda(NDArray* other, const std::function<bfloat16 (Nd4jLong, bfloat16, bfloat16)>& func, NDArray* target);
    template void NDArray::applyIndexedPairwiseLambda(NDArray* other, const std::function<Nd4jLong (Nd4jLong, Nd4jLong, Nd4jLong)>& func, NDArray* target);
    template void NDArray::applyIndexedPairwiseLambda(NDArray* other, const std::function<int (Nd4jLong, int, int)>& func, NDArray* target);
    template void NDArray::applyIndexedPairwiseLambda(NDArray* other, const std::function<int16_t (Nd4jLong, int16_t, int16_t)>& func, NDArray* target);
    template void NDArray::applyIndexedPairwiseLambda(NDArray* other, const std::function<uint8_t (Nd4jLong, uint8_t, uint8_t)>& func, NDArray* target);
    template void NDArray::applyIndexedPairwiseLambda(NDArray* other, const std::function<int8_t (Nd4jLong, int8_t, int8_t)>& func, NDArray* target);
    template void NDArray::applyIndexedPairwiseLambda(NDArray* other, const std::function<bool (Nd4jLong, bool, bool)>& func, NDArray* target);
#endif

//////////////////////////////////////////////////////////////////////////
// perform array transformation
    void NDArray::applyTransform(nd4j::transform::FloatOps op, NDArray *target, ExtraArguments *extraParams) {

        if (isS())
            throw std::runtime_error("NDArray::applyTransform FloatOps: you can't use this method on String array!");

        if (target == nullptr)
            target = this;

        if (!target->isR())
            throw std::runtime_error("NDArray::applyTransform FloatOps: target array must have one of FLOAT types");

        NativeOpExecutioner::execTransformFloat(_context, op, _buffer, _shapeInfo, _bufferD, _shapeInfoD, target->_buffer, target->_shapeInfo, target->_bufferD, target->_shapeInfoD, extraParams != nullptr ? extraParams->argumentAsT(target->dataType()) : nullptr, nullptr, nullptr);
        if (target != nullptr)
            target->tickWriteDevice();
        else
            this->tickWriteDevice();
    }

    void NDArray::applyTransform(nd4j::transform::AnyOps op, NDArray *target, ExtraArguments *extraParams) {
        nd4j_printf("Float op %i transform:\n", (int)op);

        if (isS())
            throw std::runtime_error("NDArray::applyTransform FloatOps: you can't use this method on String array!");

        if (target == nullptr)
            target = this;

        NDArray::registerSpecialUse({target}, {this});
        NativeOpExecutioner::execTransformFloat(_context, op, _buffer, _shapeInfo, _bufferD, _shapeInfoD, target->_buffer, target->_shapeInfo, target->_bufferD, target->_shapeInfoD, extraParams != nullptr ? extraParams->argumentAsT(target->dataType()) : nullptr, nullptr, nullptr);
        //target->tickWriteDevice();
    }

    void NDArray::applyTransform(nd4j::transform::SameOps op, NDArray *target, ExtraArguments *extraParams) {
        nd4j_printf("Same op %i transform:\n", (int)op);
        if (isS())
            throw std::runtime_error("NDArray::applyTransform SameOps: you can't use this method on String array!");

        if (target == nullptr)
            target = this;

        if (target->dataType() != this->dataType())
            throw std::runtime_error("NDArray::applyTransform SameOps: target array must have the same data type as original array");
        NDArray::registerSpecialUse({target}, {this});
        NativeOpExecutioner::execTransformSame(_context, op, this->_buffer, this->_shapeInfo, this->_bufferD, this->_shapeInfoD, target->_buffer, target->_shapeInfo, target->_bufferD, target->_shapeInfoD, extraParams != nullptr ? extraParams->argumentAsT(target->dataType()) : nullptr, nullptr, nullptr);
//        target->tickWriteDevice();
    }

    void NDArray::applyTransform(nd4j::transform::BoolOps op, NDArray *target, ExtraArguments *extraParams) {
        if (isS())
            throw std::runtime_error("NDArray::applyTransform BoolOps: you can't use this method on String array!");

        if (target == nullptr)
            target = this;

        if (!target->isB())
            throw std::runtime_error("NDArray::applyTransform BoolOps: target array must have one of BOOL types");

        NDArray::registerSpecialUse({target}, {this});
        NativeOpExecutioner::execTransformBool(_context, op, this->_buffer, this->_shapeInfo, _bufferD, _shapeInfoD, target->_buffer, target->_shapeInfo, target->_bufferD, target->_shapeInfoD, extraParams != nullptr ? extraParams->argumentAsT(this->dataType()) : nullptr, nullptr, nullptr);
    }

    void NDArray::applyTransform(nd4j::transform::StrictOps op, NDArray *target, ExtraArguments *extraParams) {
        if (isS())
            throw std::runtime_error("NDArray::applyTransform StrictOps: you can't use this method on String array!");

        if (target == nullptr)
            target = this;

        if (!this->isR() || !target->isR() || (this->dataType() != target->dataType()))
            throw std::runtime_error("NDArray::applyTransform StrictOps: both Source and Target array must have same FLOAT type !");

        NDArray::registerSpecialUse({target}, {this});
        NativeOpExecutioner::execTransformStrict(_context, op, this->_buffer, this->_shapeInfo, _bufferD, _shapeInfoD, target->_buffer, target->_shapeInfo, target->_bufferD, target->_shapeInfoD, extraParams != nullptr ? extraParams->argumentAsT(target->dataType()) : nullptr, nullptr, nullptr);
    }

//////////////////////////////////////////////////////////////////////////
// perform array transformation
    // void NDArray::applyTransform(nd4j::transform::FloatOps op, void *extraParams) {
    //     applyTransform(op, this, extraParams);
    // }

    // void NDArray::applyTransform(nd4j::transform::AnyOps op, void *extraParams) {
    //     applyTransform(op, this, extraParams);
    // }

    // void NDArray::applyTransform(nd4j::transform::SameOps op, void *extraParams) {
    //     applyTransform(op, this, extraParams);
    // }

    // void NDArray::applyTransform(nd4j::transform::BoolOps op, void *extraParams) {
    //     applyTransform(op, this, extraParams);
    // }

    // void NDArray::applyTransform(nd4j::transform::StrictOps op, void *extraParams) {
    //     applyTransform(op, this, extraParams);
    // }

    // perform array transformation
    NDArray NDArray::transform(nd4j::transform::FloatOps op, void *extraParams) const {
        if (isS())
            throw std::runtime_error("NDArray::transform FloatOps: you can't use this method on String array!");

        NDArray result(this->ordering(), getShapeAsVector(), DataTypeUtils::pickFloatingType(dataType()), this->_context);
        NativeOpExecutioner::execTransformFloat(_context, op, this->_buffer, this->_shapeInfo, this->_bufferD, this->_shapeInfoD, result._buffer, result._shapeInfo, result._bufferD, result._shapeInfoD, extraParams, nullptr, nullptr);
        return result;
    }

    NDArray NDArray::transform(nd4j::transform::SameOps op, void *extraParams) const {
        if (isS())
            throw std::runtime_error("NDArray::transform SameOps: you can't use this method on String array!");

        NDArray result(this->_shapeInfo, false, this->_context);
        NativeOpExecutioner::execTransformSame(_context, op, this->_buffer, this->_shapeInfo, this->_bufferD, this->_shapeInfoD, result._buffer, result._shapeInfo, result._bufferD, result._shapeInfoD, extraParams, nullptr, nullptr);
        return result;
    }

    NDArray NDArray::transform(nd4j::transform::StrictOps op, void *extraParams) const {
        if (!this->isR())
            throw std::runtime_error("Source array must have one of FLOAT types");

        NDArray result(this->_shapeInfo, false, this->_context);
        NativeOpExecutioner::execTransformStrict(_context, op, this->_buffer, this->_shapeInfo, this->_bufferD, this->_shapeInfoD, result._buffer, result._shapeInfo, result._bufferD, result._shapeInfoD, extraParams, nullptr, nullptr);
        return result;
    }

    NDArray NDArray::transform(nd4j::transform::BoolOps op, void *extraParams) const {
        if (isS())
            throw std::runtime_error("NDArray::transform BoolOps: you can't use this method on String array!");

        NDArray result(this->ordering(), getShapeAsVector(), nd4j::DataType::BOOL, this->_context);
        NativeOpExecutioner::execTransformBool(_context, op, this->_buffer, this->_shapeInfo, this->_bufferD, this->_shapeInfoD, result._buffer, result._shapeInfo, result._bufferD, result._shapeInfoD, extraParams, nullptr, nullptr);
        return result;
    }

//////////////////////////////////////////////////////////////////////////
// perform pairwise transformation
    void NDArray::applyPairwiseTransform(nd4j::pairwise::Ops op, const NDArray& other, ExtraArguments *extraParams) {
        applyPairwiseTransform(op, &other, this, extraParams);
    }

    void NDArray::applyPairwiseTransform(nd4j::pairwise::BoolOps op, const NDArray *other, NDArray *target, ExtraArguments *extraParams) const{
        if (isS())
            throw std::runtime_error("NDArray::applyPairwiseTransform BoolOps: you can't use this method on String array!");
        if (other->lengthOf() != target->lengthOf())
            throw std::invalid_argument("NDArray::applyPairwiseTransform BoolOps method - lengths of arrays are mismatched");
        if (!target->isB())
            throw std::invalid_argument("NDArray::applyPairwiseTransform BoolOps method - result must have bool type");
        if (_dataType != other->_dataType)
            throw std::invalid_argument("NDArray::applyPairwiseTransform BoolOps method - this and other arrays must have the same type !");

        NDArray::registerSpecialUse({target}, {this,other});
        NativeOpExecutioner::execPairwiseBoolTransform(_context, op, this->_buffer, this->_shapeInfo, this->_bufferD, this->_shapeInfoD, other->_buffer, other->_shapeInfo, other->_bufferD, other->_shapeInfoD, target->_buffer, target->_shapeInfo, target->_bufferD, target->_shapeInfoD, extraParams != nullptr ? extraParams->argumentAsT(target->dataType()) : nullptr);
    }

//////////////////////////////////////////////////////////////////////////
    void NDArray::applyScalarArr(nd4j::scalar::BoolOps op, const NDArray* scalar, NDArray *target, ExtraArguments *extraParams) const {
        if (isS())
            throw std::runtime_error("NDArray::applyScalarArr BoolOps: you can't use this method on String array!");
        if (target == nullptr || !target->isB())
            throw std::invalid_argument("NDArray::applyScalarArr bool method: target is nullptr or has not bool type!");
        if (_dataType != scalar->_dataType) {
            nd4j_printf("This dtype: [%i]; scalar dtype: [%i]\n", this->_dataType, scalar->_dataType);
            throw std::invalid_argument("NDArray::applyScalarArr bool method: this and scalar arrays must have the same type!");
        }
        if (!this->isActualOnDeviceSide())
            this->syncToDevice();

        if (!scalar->isActualOnDeviceSide())
            scalar->syncToDevice();
        NDArray::registerSpecialUse({target}, {this, scalar});
        NativeOpExecutioner::execScalarBool(_context, op, _buffer, _shapeInfo, _bufferD, _shapeInfoD, target->_buffer, target->_shapeInfo, target->_bufferD, target->_shapeInfoD, scalar->_buffer, scalar->_shapeInfo, scalar->_bufferD, scalar->_shapeInfoD, extraParams != nullptr ? extraParams->argumentAsT(target->dataType()): nullptr);
    }

    template <typename T>
    void NDArray::applyScalar(nd4j::scalar::BoolOps op, const T scalar, NDArray *target, ExtraArguments *extraParams) const {

        auto scalarArr = NDArrayFactory::create<T>(scalar, _context);
        applyScalarArr(op, &scalarArr, target, extraParams);
    }

    template <> void NDArray::applyScalar(nd4j::scalar::BoolOps op, const NDArray* scalar, NDArray *target, ExtraArguments *extraParams) const { throw std::runtime_error("NDArray::applyScalar<NDArray*> method: do not use me!");}
    template void NDArray::applyScalar<double>(nd4j::scalar::BoolOps op, const double scalar, NDArray *target, ExtraArguments *extraParams) const;
    template void NDArray::applyScalar<float>(nd4j::scalar::BoolOps op, const float scalar, NDArray *target, ExtraArguments *extraParams) const;
    template void NDArray::applyScalar<float16>(nd4j::scalar::BoolOps op, const float16 scalar, NDArray *target, ExtraArguments *extraParams) const;
    template void NDArray::applyScalar<bfloat16>(nd4j::scalar::BoolOps op, const bfloat16 scalar, NDArray *target, ExtraArguments *extraParams) const;
    template void NDArray::applyScalar<Nd4jLong>(nd4j::scalar::BoolOps op, const Nd4jLong scalar, NDArray *target, ExtraArguments *extraParams) const;
    template void NDArray::applyScalar<int>(nd4j::scalar::BoolOps op, const int scalar, NDArray *target, ExtraArguments *extraParams) const;
    template void NDArray::applyScalar<int16_t>(nd4j::scalar::BoolOps op, const int16_t scalar, NDArray *target, ExtraArguments *extraParams) const;
    template void NDArray::applyScalar<int8_t>(nd4j::scalar::BoolOps op, const int8_t scalar, NDArray *target, ExtraArguments *extraParams) const;
    template void NDArray::applyScalar<uint8_t>(nd4j::scalar::BoolOps op, const uint8_t scalar, NDArray *target, ExtraArguments *extraParams) const;
    template void NDArray::applyScalar<bool>(nd4j::scalar::BoolOps op, const bool scalar, NDArray *target, ExtraArguments *extraParams) const;

//////////////////////////////////////////////////////////////////////////
    void NDArray::applyScalarArr(nd4j::scalar::Ops op, const NDArray* scalar, NDArray* target, ExtraArguments *extraParams) {
        if (isS())
            throw std::runtime_error("NDArray::applyScalarArr: you can't use this method on String array!");
        if (!scalar->isScalar())
            throw std::invalid_argument("NDArray::applyScalarArr method: operand is not a scalar!");
        if(target == nullptr)
            target = this;
        if(target->_dataType != DataTypeUtils::pickPairwiseResultType(_shapeInfo, scalar->_shapeInfo) && !(target->_dataType == this->_dataType || target->_dataType == scalar->_dataType))
            throw std::invalid_argument("NDArray::applyScalarArr method: wrong type of target array!");

        if (!this->isActualOnDeviceSide())
            this->syncToDevice();

        if (!scalar->isActualOnDeviceSide())
            scalar->syncToDevice();
        NDArray::registerSpecialUse({target}, {this,scalar});
        NativeOpExecutioner::execScalar(_context, op, _buffer, _shapeInfo, _bufferD, _shapeInfoD, target->_buffer, target->_shapeInfo, target->_bufferD, target->_shapeInfoD, scalar->getBuffer(), scalar->getShapeInfo(), scalar->_bufferD, scalar->_shapeInfoD, extraParams != nullptr ? extraParams->argumentAsT(target->dataType()) : nullptr);
    }

    template <typename T>
    void NDArray::applyScalar(nd4j::scalar::Ops op, const T scalar, NDArray *target, ExtraArguments *extraParams) {

        auto scalarArr = NDArrayFactory::create<T>(this->dataType(), scalar, this->_context);
        applyScalarArr(op, &scalarArr, target, extraParams);
    }

    template <> void NDArray::applyScalar(nd4j::scalar::Ops op, const NDArray* scalar, NDArray *target, ExtraArguments *extraParams) { throw std::runtime_error("NDArray::applyScalar<NDArray*> method: do not use me!");}
    template void NDArray::applyScalar(nd4j::scalar::Ops op, const double scalar, NDArray *target, ExtraArguments *extraParams);
    template void NDArray::applyScalar(nd4j::scalar::Ops op, const float scalar, NDArray *target, ExtraArguments *extraParams);
    template void NDArray::applyScalar(nd4j::scalar::Ops op, const float16 scalar, NDArray *target, ExtraArguments *extraParams);
    template void NDArray::applyScalar(nd4j::scalar::Ops op, const bfloat16 scalar, NDArray *target, ExtraArguments *extraParams);
    template void NDArray::applyScalar(nd4j::scalar::Ops op, const Nd4jLong scalar, NDArray *target, ExtraArguments *extraParams);
    template void NDArray::applyScalar(nd4j::scalar::Ops op, const int scalar, NDArray *target, ExtraArguments *extraParams);
    template void NDArray::applyScalar(nd4j::scalar::Ops op, const int16_t scalar, NDArray *target, ExtraArguments *extraParams);
    template void NDArray::applyScalar(nd4j::scalar::Ops op, const int8_t scalar, NDArray *target, ExtraArguments *extraParams);
    template void NDArray::applyScalar(nd4j::scalar::Ops op, const uint8_t scalar, NDArray *target, ExtraArguments *extraParams);
    template void NDArray::applyScalar(nd4j::scalar::Ops op, const bool scalar, NDArray *target, ExtraArguments *extraParams);

    //////////////////////////////////////////////////////////////////////////
    void NDArray::applyBroadcast(nd4j::broadcast::Ops op, const std::vector<int>& dimensions, const NDArray* tadArray, NDArray* target, ExtraArguments* extraArgs) {
        if (isS())
            throw std::runtime_error("NDArray::applyBroadcast: you can't use this method on String array!");
        if(((op == broadcast::Divide || op == broadcast::FloorDiv || op == broadcast::FloorMod) && tadArray->isB()) || (op == broadcast::ReverseDivide && this->isB()))
            throw std::runtime_error("NDArray::applyBroadcast: you can't divide by array!");

        if (dimensions.size() == 0)
            return;
        auto result = target == nullptr ? this : target;

        if(result->_dataType != DataTypeUtils::pickPairwiseResultType(_shapeInfo, tadArray->_shapeInfo))
            throw std::invalid_argument("NDArray::applyBroadcast method: wrong type of target array !");
        if(!result->isSameShape(this))
            throw std::invalid_argument("NDArray::applyBroadcast method: this and target arrays must have the same shape !");

        std::vector<int> copy(dimensions);

        //if (dimensions.size() > 1)
        //    std::sort(copy.begin(), copy.end());

        Nd4jLong tadLength = shape::tadLength(this->_shapeInfo, copy.data(), (int) copy.size());
        if (tadLength != tadArray->lengthOf())
            throw std::runtime_error("NDArray::applyBroadcast method: tad length mismatch !");

        shape::TAD tad(this->_shapeInfo, copy.data(), copy.size());
        tad.createTadOnlyShapeInfo();
        tad.createOffsets();
        if (!this->isActualOnDeviceSide())
            this->syncToDevice();

        if (!tadArray->isActualOnDeviceSide())
            tadArray->syncToDevice();

        // prepare input arrays for prepareDataForCuda function
        std::vector<std::pair<void*,size_t>> hostData;
        hostData.emplace_back(copy.data(), copy.size() * sizeof(int));							// 0 -- dimensions
        hostData.emplace_back(tad.tadOnlyShapeInfo, shape::shapeInfoByteLength(tad.tadOnlyShapeInfo));	// 1 -- xTadShapeInfo
        hostData.emplace_back(tad.tadOffsets, tad.numTads * sizeof(Nd4jLong));							// 2 -- xTadOffsets
        std::vector<void*> devicePtrs(hostData.size(), nullptr);

        // create cuda stream and LaunchContext
        hipError_t cudaResult;
        //hipStream_t stream;
        //cudaResult = hipStreamCreate(&stream);	ASSERT_EQ(0, cudaResult);
        hipStream_t* stream = this->getContext()->getCudaStream();
        // allocate required amount of global device memory and copy host data to it
//    cudaResult = allocateDeviceMem(*pLc, devicePtrs, hostData);	ASSERT_EQ(0, cudaResult);
        for(int i = 0; i < devicePtrs.size(); ++i) {

            cudaResult = hipMalloc(reinterpret_cast<void **>(&devicePtrs[i]), hostData[i].second);
            if(cudaResult != 0) throw cuda_exception::build("Cannot allocate memory for tads on device", cudaResult);
            hipMemcpyAsync(devicePtrs[i], hostData[i].first, hostData[i].second, hipMemcpyHostToDevice, *stream);
        }

        // call cuda kernel which calculates result

        NDArray::registerSpecialUse({target}, {this, const_cast<NDArray*>(tadArray)});

        // TODO: eventually we want separate tads here
        NativeOpExecutioner::execBroadcast(_context, op, this->_buffer, this->_shapeInfo, this->_bufferD, this->_shapeInfoD, tadArray->_buffer, tadArray->_shapeInfo, tadArray->_bufferD, tadArray->_shapeInfoD, result->_buffer, result->_shapeInfo, result->_bufferD, result->_shapeInfoD, (int*)devicePtrs[0], (int)copy.size(), (Nd4jLong*)devicePtrs[1], (Nd4jLong*)devicePtrs[2], nullptr, nullptr);
    }

    //////////////////////////////////////////////////////////////////////////
    void NDArray::applyBroadcast(nd4j::broadcast::BoolOps op, const std::vector<int>& dimensions, const NDArray* tadArray, NDArray* target, ExtraArguments* extraArgs) {
        if (isS())
            throw std::runtime_error("NDArray::applyBroadcast BoolOps: you can't use this method on String array!");

        if (dimensions.size() == 0)
            return;

        auto result = target == nullptr ? this : target;

        if(result->_dataType != DataType::BOOL)
            throw std::invalid_argument("NDArray::applyBroadcast bool method: type of target array must be BOOL!");
        if(!result->isSameShape(this))
            throw std::invalid_argument("NDArray::applyBroadcast bool method: this and other arrays must have the same shape !");
        if(_dataType != tadArray->_dataType)
            throw std::invalid_argument("NDArray::applyBroadcast bool method: this and tad arrays must have the same type !");

        std::vector<int> copy(dimensions);

        if (dimensions.size() > 1)
            std::sort(copy.begin(), copy.end());

        Nd4jLong tadLength = shape::tadLength(this->_shapeInfo, copy.data(), (int) copy.size());
        if (tadLength != tadArray->lengthOf())
            throw std::runtime_error("Tad length mismatch");

        shape::TAD tad(this->_shapeInfo, copy.data(), copy.size());
        tad.createTadOnlyShapeInfo();
        tad.createOffsets();
        if (!this->isActualOnDeviceSide())
            this->syncToDevice();

        if (!tadArray->isActualOnDeviceSide())
            tadArray->syncToDevice();

        NDArray::registerSpecialUse({target}, {this, tadArray});
        // TODO: eventually we want separate tads here
        NativeOpExecutioner::execBroadcastBool(_context, op, this->_buffer, this->_shapeInfo, this->_bufferD, this->_shapeInfoD,
                                               tadArray->_buffer, tadArray->_shapeInfo, tadArray->_bufferD, tadArray->_shapeInfoD,
                                               result->_buffer, result->_shapeInfo, result->_bufferD, result->_shapeInfoD, copy.data(), (int)copy.size(), tad.tadOnlyShapeInfo, tad.tadOffsets, nullptr, nullptr);
    }

    //////////////////////////////////////////////////////////////////////////
    NDArray NDArray::applyTrueBroadcast(nd4j::BroadcastOpsTuple op, const NDArray& other, ExtraArguments *extraArgs) const {
        Nd4jLong* newShapeInfo = nullptr;
        if(!ShapeUtils::evalBroadcastShapeInfo(*this, &other, true, newShapeInfo, _context->getWorkspace()))          // the rank of new array = max->rankOf)()
            throw std::runtime_error("NDArray::applyTrueBroadcast method: the shapes of this and other arrays are not suitable for broadcast operation !");
        NDArray result(newShapeInfo, true, this->_context);

        // if workspace is not null - do not call delete.
        if (_context->getWorkspace() == nullptr)
            delete[] newShapeInfo;

        this->applyTrueBroadcast(op, &other, &result, false, extraArgs);

        return result;
    }
  
    ////////////////////////////////////////////////////////////////////////
    void NDArray::applyIndexReduce(nd4j::indexreduce::Ops op, NDArray* target, const std::vector<int>& dimensions, const ExtraArguments *extraParams) const {
        if (isS())
            throw std::runtime_error("NDArray::applyIndexReduce: you can't use this method on String array!");

        if (target->dataType() != nd4j::DataType::INT64)
            throw std::runtime_error("NDArray::applyIndexReduce operations return INT64");
        
        void* params = extraParams != nullptr ? const_cast<ExtraArguments*>(extraParams)->argumentAsT(this->dataType()) : nullptr;

        if (target->isScalar()) {
            //target->_buffer[0] = functions::indexreduce::IndexReduce<T>::template execScalar<OpName>(_buffer, _shapeInfo, const_cast<T*>(extraParams));
            NativeOpExecutioner::execIndexReduceScalar(_context, op, _buffer, _shapeInfo, _bufferD, _shapeInfoD, params, target->_buffer, target->_shapeInfo, target->_bufferD, target->_shapeInfoD);

            auto cudaResult = hipStreamSynchronize(*_context->getCudaStream());        
            if (cudaResult != 0) throw cuda_exception::build("NDArray::applyIndexReduce cuda failed !", cudaResult);            
        } 
        else {

            std::vector<int> copy(dimensions);
            shape::checkDimensions(rankOf(), copy);

            shape::TAD xTad(_shapeInfo, copy.data(), copy.size());
            xTad.createTadOnlyShapeInfo();
            xTad.createOffsets();

            // device memory allocation for tads
            Nd4jLong *xTadShapeInfo, *xTadOffsets;
            int* dims;
            
            auto cudaResult = hipMalloc(reinterpret_cast<void **>(&dims), copy.size() * sizeof(int));
            if (cudaResult != 0) throw cuda_exception::build("NDArray::applyIndexReduce cuda memory allocation failed !", cudaResult);
            
            cudaResult = hipMalloc(reinterpret_cast<void **>(&xTadShapeInfo), shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));
            if (cudaResult != 0) throw cuda_exception::build("NDArray::applyIndexReduce cuda memory allocation failed !", cudaResult);
            
            cudaResult = hipMalloc(reinterpret_cast<void **>(&xTadOffsets), xTad.numTads * sizeof(Nd4jLong));
            if (cudaResult != 0) throw cuda_exception::build("NDArray::applyIndexReduce cuda memory allocation failed !", cudaResult);

            hipMemcpyAsync(dims, copy.data(), copy.size() * sizeof(int), hipMemcpyHostToDevice, *_context->getCudaStream());
            hipMemcpyAsync(xTadShapeInfo, xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo), hipMemcpyHostToDevice, *_context->getCudaStream());
            hipMemcpyAsync(xTadOffsets, xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong), hipMemcpyHostToDevice, *_context->getCudaStream());

            NativeOpExecutioner::execIndexReduce(_context, op, _buffer, _shapeInfo, _bufferD, _shapeInfoD, params,target->_buffer, target->_shapeInfo, target->_bufferD, target->_shapeInfoD,dims, copy.size(),xTadShapeInfo, xTadOffsets);

            cudaResult = hipStreamSynchronize(*_context->getCudaStream());        
            if (cudaResult != 0) throw cuda_exception::build("NDArray::applyIndexReduce cuda failed !", cudaResult);

            hipFree(dims); hipFree(xTadShapeInfo); hipFree(xTadOffsets);
        }
        NDArray::registerSpecialUse({target}, {this});
    }
    
    ////////////////////////////////////////////////////////////////////////
    // reduce dimensions in this array relying on index operations
    NDArray* NDArray::applyIndexReduce(nd4j::indexreduce::Ops op,const std::vector<int>& dimensions, const ExtraArguments* extraParams ) const {
        if (isS())
            throw std::runtime_error("NDArray::applyIndexReduce: you can't use this method on String array!");

        std::vector<int> copy(dimensions);
        shape::checkDimensions(rankOf(), copy);

        auto newShape = ShapeUtils::evalReduceShapeInfo('c', copy, *this, false, false, _context->getWorkspace());
        ArrayOptions::setDataType(newShape, nd4j::INT64);
        auto result = new NDArray(newShape, true, _context, true);

        void* params = extraParams != nullptr ? const_cast<ExtraArguments*>(extraParams)->argumentAsT(this->dataType()) : nullptr;        

        if (rankOf() == copy.size()) {
            NativeOpExecutioner::execIndexReduceScalar(_context, op, _buffer, _shapeInfo, _bufferD, _shapeInfoD, params, result->_buffer, result->_shapeInfo, result->_bufferD, result->_shapeInfoD);

            auto cudaResult = hipStreamSynchronize(*_context->getCudaStream());        
            if (cudaResult != 0) throw cuda_exception::build("NDArray::applyIndexReduce cuda failed !", cudaResult);            
        } 
        else {            

            shape::TAD xTad(_shapeInfo, copy.data(), copy.size());
            xTad.createTadOnlyShapeInfo();
            xTad.createOffsets();

            // device memory allocation for tads
            Nd4jLong *xTadShapeInfo, *xTadOffsets;
            int* dims;
            
            auto cudaResult = hipMalloc(reinterpret_cast<void **>(&dims), copy.size() * sizeof(int));
            if (cudaResult != 0) throw cuda_exception::build("NDArray::applyIndexReduce cuda memory allocation failed !", cudaResult);
            
            cudaResult = hipMalloc(reinterpret_cast<void **>(&xTadShapeInfo), shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));
            if (cudaResult != 0) throw cuda_exception::build("NDArray::applyIndexReduce cuda memory allocation failed !", cudaResult);
            
            cudaResult = hipMalloc(reinterpret_cast<void **>(&xTadOffsets), xTad.numTads * sizeof(Nd4jLong));
            if (cudaResult != 0) throw cuda_exception::build("NDArray::applyIndexReduce cuda memory allocation failed !", cudaResult);

            hipMemcpyAsync(dims, copy.data(), copy.size() * sizeof(int), hipMemcpyHostToDevice, *_context->getCudaStream());
            hipMemcpyAsync(xTadShapeInfo, xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo), hipMemcpyHostToDevice, *_context->getCudaStream());
            hipMemcpyAsync(xTadOffsets, xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong), hipMemcpyHostToDevice, *_context->getCudaStream());

            NativeOpExecutioner::execIndexReduce(_context, op, _buffer, _shapeInfo, _bufferD, _shapeInfoD,
                                                params,
                                                result->_buffer, result->_shapeInfo, result->_bufferD, result->_shapeInfoD,
                                                dims, copy.size(),
                                                xTadShapeInfo, xTadOffsets);

            cudaResult = hipStreamSynchronize(*_context->getCudaStream());        
            if (cudaResult != 0) throw cuda_exception::build("NDArray::applyIndexReduce cuda failed !", cudaResult);

            hipFree(dims); hipFree(xTadShapeInfo); hipFree(xTadOffsets);
        }
        
        NDArray::registerSpecialUse({result}, {this});
        
        return result;
    }

    ////////////////////////////////////////////////////////////////////////
    // apply reduce3 operations to this and other array, return result in new output array
    NDArray* NDArray::applyReduce3(nd4j::reduce3::Ops op, const NDArray* other, const ExtraArguments* extraParams) const {
        if (isS())
            throw std::runtime_error("NDArray::applyReduce3 method: you can't use this method on String array!");
        if(_dataType != other->_dataType)
            throw std::runtime_error("NDArray::applyReduce3 method: the types of this and other arrays must be the same !");
        // check shapes consistency
        if(!isSameShape(other))
            throw std::runtime_error("NDArray::applyReduce3 method: the shapes of this and other arrays must be the same !");
        // create shapeInfo for scalar
        auto newShape = ShapeBuilders::createScalarShapeInfo(DataTypeUtils::pickFloatingType(_dataType), _context->getWorkspace());
        // create output array (scalar)
        auto result = new NDArray(newShape, true, _context, true);
        // create dynamic array of extra parameters if array extraParams is empty (==nullptr)
        void* params = extraParams != nullptr ? const_cast<ExtraArguments*>(extraParams)->argumentAsT(this->dataType()) : nullptr;        

        NativeOpExecutioner::execReduce3Scalar(_context, op, _buffer, _shapeInfo, _bufferD, _shapeInfoD, params, other->_buffer, other->_shapeInfo, other->_bufferD, other->_shapeInfoD, result->_buffer, result->_shapeInfo, result->_bufferD, result->_shapeInfoD);

        auto cudaResult = hipStreamSynchronize(*_context->getCudaStream());        
        if (cudaResult != 0) throw cuda_exception::build("NDArray::applyReduce3 cuda failed !", cudaResult);

        NDArray::registerSpecialUse({result}, {this, other});

        return result;
    }

    ////////////////////////////////////////////////////////////////////////
    // apply reduce3 (exec) operations to this and other array, return result in new output array
    NDArray* NDArray::applyReduce3(nd4j::reduce3::Ops op, const NDArray* other, const std::vector<int>& dimensions, const ExtraArguments* extraParams) const {
        
        if (isS())
            throw std::runtime_error("NDArray::applyReduce3: you can't use this method on String array!");
        if(_dataType != other->_dataType)
            throw std::runtime_error("NDArray::applyReduce3 method: the types of this and other arrays must be the same !");

        std::vector<int> copy(dimensions);
        shape::checkDimensions(rankOf(), copy);
        shape::checkDimensions(other->rankOf(), copy);

        if(!isActualOnDeviceSide()) 
            syncToDevice();

        if(!other->isActualOnDeviceSide())
            other->syncToDevice();

        auto newShape = ShapeUtils::evalReduceShapeInfo('c', copy, *this, false, false, _context->getWorkspace());
        ArrayOptions::setDataType(newShape, DataTypeUtils::pickFloatingType(_dataType));
        auto result = new NDArray(newShape, true, _context, true);
        // create temporary dynamic array of extra parameters if array extraParams is empty (==nullptr)
        void* params = extraParams != nullptr ? const_cast<ExtraArguments*>(extraParams)->argumentAsT(this->dataType()) : nullptr;

        // perform calculations
        if(rankOf() == copy.size() && other->rankOf() == copy.size()) {
            NativeOpExecutioner::execReduce3Scalar(_context, op, _buffer, _shapeInfo, _bufferD, _shapeInfoD, params, other->_buffer, other->_shapeInfo, other->_bufferD, other->_shapeInfoD, result->_buffer, result->shapeInfo(), result->specialBuffer(), result->specialShapeInfo());
            auto cudaResult = hipStreamSynchronize(*_context->getCudaStream());        
            if (cudaResult != 0) throw cuda_exception::build("NDArray::applyReduce3 cuda failed !", cudaResult);
        }
        else {
            
            // evaluate xTad data 
            shape::TAD xTad(_shapeInfo, copy.data(), copy.size());
            xTad.createTadOnlyShapeInfo();
            xTad.createOffsets();

            // evaluate yTad data
            shape::TAD yTad(other->_shapeInfo, copy.data(), copy.size());         
            yTad.createTadOnlyShapeInfo();
            yTad.createOffsets();

            if(!shape::equalsSoft(xTad.tadOnlyShapeInfo, yTad.tadOnlyShapeInfo) || (xTad.numTads != yTad.numTads && xTad.numTads != 1 && yTad.numTads != 1))
                throw std::runtime_error("NDArray::applyReduce3 cuda method: arrays tads are inconsistent !");

            // device memory allocation for tads
            Nd4jLong *xTadShapeInfo, *xTadOffsets, *yTadShapeInfo, *yTadOffsets;
            int *dims;

            auto cudaResult = hipMalloc(reinterpret_cast<void **>(&dims), copy.size() * sizeof(int));
            if (cudaResult != 0) throw cuda_exception::build("NDArray::applyReduce3 cuda memory allocation failed !", cudaResult);
            
            cudaResult = hipMalloc(reinterpret_cast<void **>(&xTadShapeInfo), shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));
            if (cudaResult != 0) throw cuda_exception::build("NDArray::applyReduce3 cuda memory allocation failed !", cudaResult);

            cudaResult = hipMalloc(reinterpret_cast<void **>(&yTadShapeInfo), shape::shapeInfoByteLength(yTad.tadOnlyShapeInfo));
            if (cudaResult != 0) throw cuda_exception::build("NDArray::applyReduce3 cuda memory allocation failed !", cudaResult);

            cudaResult = hipMalloc(reinterpret_cast<void **>(&xTadOffsets), xTad.numTads * sizeof(Nd4jLong));
            if (cudaResult != 0) throw cuda_exception::build("NDArray::applyReduce3 cuda memory allocation failed !", cudaResult);

            cudaResult = hipMalloc(reinterpret_cast<void **>(&yTadOffsets), yTad.numTads * sizeof(Nd4jLong));
            if (cudaResult != 0) throw cuda_exception::build("NDArray::applyReduce3 cuda memory allocation failed !", cudaResult);

            hipMemcpyAsync(dims, copy.data(), copy.size() * sizeof(int), hipMemcpyHostToDevice, *_context->getCudaStream());
            hipMemcpyAsync(xTadShapeInfo, xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo), hipMemcpyHostToDevice, *_context->getCudaStream());
            hipMemcpyAsync(yTadShapeInfo, yTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(yTad.tadOnlyShapeInfo), hipMemcpyHostToDevice, *_context->getCudaStream());
            hipMemcpyAsync(xTadOffsets, xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong), hipMemcpyHostToDevice, *_context->getCudaStream());
            hipMemcpyAsync(yTadOffsets, yTad.tadOffsets, yTad.numTads * sizeof(Nd4jLong), hipMemcpyHostToDevice, *_context->getCudaStream());

            NativeOpExecutioner::execReduce3(_context, op, _buffer, _shapeInfo, _bufferD, _shapeInfoD, params, other->_buffer, other->_shapeInfo, other->_bufferD, other->_shapeInfoD, result->_buffer, result->_shapeInfo, result->_bufferD, result->_shapeInfoD, dims, copy.size(), xTadShapeInfo, xTadOffsets,  yTadShapeInfo, yTadOffsets);

            cudaResult = hipStreamSynchronize(*_context->getCudaStream());        
            if (cudaResult != 0) throw cuda_exception::build("NDArray::applyReduce3 cuda failed !", cudaResult);

            hipFree(dims); hipFree(xTadShapeInfo); hipFree(xTadOffsets); hipFree(yTadShapeInfo); hipFree(yTadOffsets);            
        }        

        NDArray::registerSpecialUse({result}, {this, other});

        return result;
    }

    ////////////////////////////////////////////////////////////////////////
    // apply reduce3 (execAll) operations to this and other array, return result in new output array
    NDArray* NDArray::applyAllReduce3(nd4j::reduce3::Ops op, const NDArray *other, const std::vector<int>& dimensions, const ExtraArguments* extraParams) const {
        if (isS())
            throw std::runtime_error("NDArray::applyAllReduce3: you can't use this method on String array!");
        if(_dataType != other->_dataType)
            throw std::runtime_error("NDArray::applyAllReduce3 method: the types of this and other arrays must be the same !");

        // be careful, copy array may undergo changes (sort, transformation of negative dimensions to positive, duplicates removing )
        std::vector<int> copy(dimensions);
        shape::checkDimensions(rankOf(), copy);
        shape::checkDimensions(other->rankOf(), copy);
        
        // create tads
        shape::TAD xTad(_shapeInfo, copy.data(), copy.size());
        xTad.createTadOnlyShapeInfo();
        xTad.createOffsets();

        shape::TAD yTad(other->_shapeInfo, copy.data(), copy.size());
        yTad.createTadOnlyShapeInfo();
        yTad.createOffsets();

        // check tads shapes
        if(!shape::equalsSoft(xTad.tadOnlyShapeInfo, yTad.tadOnlyShapeInfo))
            throw std::runtime_error("NDArray::applyAllReduce3 method: the shapes of array tads are different !");

        // set newShape for output array
        Nd4jLong *newShape = nullptr;
        ALLOCATE(newShape, _context->getWorkspace(), 8, Nd4jLong);
        newShape[0] = 2;        // output rank is always equal to 2 for execAll case
        newShape[1] = xTad.numTads;
        newShape[2] = yTad.numTads;
        ShapeUtils::updateStridesAndType(newShape, DataTypeUtils::pickFloatingType(_dataType), 'c');
        // create output array
        auto result = new NDArray(newShape, true, _context, true);

        // create dynamic array of extra parameters if array extraParams is empty (==nullptr)
        void* params = extraParams != nullptr ? const_cast<ExtraArguments*>(extraParams)->argumentAsT(this->dataType()) : nullptr;

         // device memory allocation for tads
        Nd4jLong *xTadShapeInfo, *xTadOffsets, *yTadShapeInfo, *yTadOffsets;
        int *dims;     
        
        auto cudaResult = hipMalloc(reinterpret_cast<void **>(&dims), copy.size() * sizeof(int));
        if (cudaResult != 0) throw cuda_exception::build("NDArray::applyAllReduce3 cuda memory allocation failed !", cudaResult);

        cudaResult = hipMalloc(reinterpret_cast<void **>(&xTadShapeInfo), shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));
        if (cudaResult != 0) throw cuda_exception::build("NDArray::applyAllReduce3 cuda memory allocation failed !", cudaResult);

        cudaResult = hipMalloc(reinterpret_cast<void **>(&yTadShapeInfo), shape::shapeInfoByteLength(yTad.tadOnlyShapeInfo));
        if (cudaResult != 0) throw cuda_exception::build("NDArray::applyAllReduce3 cuda memory allocation failed !", cudaResult);

        cudaResult = hipMalloc(reinterpret_cast<void **>(&xTadOffsets), xTad.numTads * sizeof(Nd4jLong));
        if (cudaResult != 0) throw cuda_exception::build("NDArray::applyAllReduce3 cuda memory allocation failed !", cudaResult);

        cudaResult = hipMalloc(reinterpret_cast<void **>(&yTadOffsets), yTad.numTads * sizeof(Nd4jLong));
        if (cudaResult != 0) throw cuda_exception::build("NDArray::applyAllReduce3 cuda memory allocation failed !", cudaResult);

        hipMemcpyAsync(dims, copy.data(), copy.size() * sizeof(int), hipMemcpyHostToDevice, *_context->getCudaStream());  
        hipMemcpyAsync(xTadShapeInfo, xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo), hipMemcpyHostToDevice, *_context->getCudaStream());
        hipMemcpyAsync(yTadShapeInfo, yTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(yTad.tadOnlyShapeInfo), hipMemcpyHostToDevice, *_context->getCudaStream());
        hipMemcpyAsync(xTadOffsets, xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong), hipMemcpyHostToDevice, *_context->getCudaStream());
        hipMemcpyAsync(yTadOffsets, yTad.tadOffsets, yTad.numTads * sizeof(Nd4jLong), hipMemcpyHostToDevice, *_context->getCudaStream());

        NativeOpExecutioner::execReduce3All(_context, op, _buffer, _shapeInfo, _bufferD, _shapeInfoD, params,other->_buffer, other->_shapeInfo, other->_bufferD, other->_shapeInfoD,result->_buffer,result->_shapeInfo, result->_bufferD, result->_shapeInfoD,dims, copy.size(), xTadShapeInfo, xTadOffsets, yTadShapeInfo, yTadOffsets);

        cudaResult = hipStreamSynchronize(*_context->getCudaStream());        
        if (cudaResult != 0) throw cuda_exception::build("NDArray::applyAllReduce3 cuda failed !", cudaResult);

        hipFree(dims); hipFree(xTadShapeInfo); hipFree(xTadOffsets); hipFree(yTadShapeInfo); hipFree(yTadOffsets);            

        NDArray::registerSpecialUse({result}, {this, other});
        
        return result;
    }

    
////////////////////////////////////////////////////////////////////////
// default destructor
NDArray::~NDArray() noexcept {

    if (_isBuffAlloc && _context->getWorkspace() == nullptr && _buffer != nullptr) {
        if (!isS()) {
            delete[] _buffer;
        } else {
            for (int e = 0; e < lengthOf(); e++) {
                auto t = reinterpret_cast<utf8string**>(_buffer);
                delete t[e];
            };
            delete[] _buffer;
        }
    }

    if (_isShapeAlloc  && _context->getWorkspace() == nullptr && _shapeInfo != nullptr)
        delete[] _shapeInfo;

    if (_isShapeDAlloc)
        RELEASE_SPECIAL(_shapeInfoD, _context->getWorkspace());

    if (_isBuffDAlloc)
        RELEASE_SPECIAL(_bufferD, _context->getWorkspace());
}

//////////////////////////////////////////////////////////////////////////
void NDArray::setShapeInfo(Nd4jLong *shapeInfo) {

    if(_isShapeAlloc && _context->getWorkspace() == nullptr)                
        delete []_shapeInfo;

    if(_isShapeDAlloc)
        hipFree(_shapeInfoD);
        
    _shapeInfo = shapeInfo;

    if (shapeInfo != nullptr) {

        if(ArrayOptions::arrayType(shapeInfo) == ArrayType::EMPTY)
            _length = 0;
        else
            _length = shape::length(shapeInfo);
        
        _dataType = ArrayOptions::dataType(shapeInfo);
        ALLOCATE_SPECIAL(_shapeInfoD, _context->getWorkspace(), shape::shapeInfoLength(_shapeInfo), Nd4jLong);
        _isShapeDAlloc = true;
        syncShape();
    } 
    else {
        this->_dataType = nd4j::DataType::INHERIT;    
        _shapeInfoD = nullptr;
        _isShapeDAlloc = false;
    }
}

////////////////////////////////////////////////////////////////////////
    // This method returns true if two arrays are equal, with custom or default Eps value of 1e-5, false otherwise
    bool NDArray::equalsTo(const NDArray *other, double eps) const {
        if (this->dataType() != other->dataType() || lengthOf() != other->lengthOf())
            return false;

        // we need to be able to compare [1, len] to [len]
        if ((rankOf() == 1 && other->rankOf() == 2) || (rankOf() == 2 && other->rankOf() == 1)) {
            // FIXME: do something here?
        } else if (!shape::equalsSoft(_shapeInfo, other->_shapeInfo))
            return false;

        NDArray tmp(nd4j::DataType::FLOAT32, _context); // scalar = 0

        if(!isActualOnDeviceSide()) 
            syncToDevice();

        if(!other->isActualOnDeviceSide())
            other->syncToDevice();

        ExtraArguments extras({eps}); 
        NativeOpExecutioner::execReduce3Scalar(_context, reduce3::EqualsWithEps, _buffer, _shapeInfo, _bufferD, _shapeInfoD, extras.argumentAsT(DataType::FLOAT32), other->_buffer, other->_shapeInfo, other->_bufferD, other->_shapeInfoD, tmp.buffer(), tmp.shapeInfo(), tmp._bufferD, tmp._shapeInfoD);

        auto res = hipStreamSynchronize(*_context->getCudaStream());
        if (res != 0)
            throw cuda_exception::build("NDArray::equalsTo failed", res);

        if (tmp.e<int>(0) > 0)
            return false;

        return true;
    }

    //////////////////////////////////////////////////////////////////////////
    bool NDArray::permutei(const int* dimensions, const int rank) {

        // check if current object is _shapeInfo owner
        if (!_isShapeAlloc) {             // if _shapeInfo is not its own
            _shapeInfo = ShapeUtils::evalPermShapeInfo(dimensions, rank, *this, _context->getWorkspace());
            _isShapeAlloc = true;
        } else {
            if (!nonNull() || rank != rankOf())
                throw std::runtime_error("NDArray::permutei method: wrong arguments in permutei method: either array is nullptr or rank is not suitable!");
            shape::doPermuteShapeInfo(_shapeInfo, dimensions);
        }

        syncShape();

        return true;
    }

    //////////////////////////////////////////////////////////////////////////
    bool NDArray::permutei(const Nd4jLong* dimensions, const int rank) {

        // check if current object is _shapeInfo owner
        if (!_isShapeAlloc) {             // if _shapeInfo is not its own
            _shapeInfo = ShapeUtils::evalPermShapeInfo(dimensions, rank, *this, _context->getWorkspace());
            _isShapeAlloc = true;
        } else {
            if (!nonNull() || rank != rankOf())
                throw std::runtime_error("NDArray::permutei method: wrong arguments in permutei method: either array is nullptr or rank is not suitable!");
            shape::doPermuteShapeInfo(_shapeInfo, dimensions);
        }

        syncShape();
        
        return true;
    }

//////////////////////////////////////////////////////////////////////////
// method reduces array by excluding its shapes along axes present in dimensions vector
void NDArray::reduceAlongDimension(nd4j::reduce::FloatOps op, NDArray* target, const std::vector<int>& dimensions, const bool keepDims, const bool supportOldShapes, const bool checkTargetShape) const {

    if (isS())
        throw std::runtime_error("NDArray::reduceAlongDimension FloatOps: you can't use this method on String array!");
    if (target == nullptr || !target->isR())
        throw std::invalid_argument("NDArray::reduceAlongDimension FloatOps: requires target array to be present and have type form real space!");

    std::vector<int> copy(dimensions);
    shape::checkDimensions(rankOf(), copy);

    if(checkTargetShape) {
        auto newShape = ShapeUtils::evalReduceShapeInfo(target->ordering(), copy, *this, keepDims, supportOldShapes, _context->getWorkspace());
        if(!shape::shapeEquals(newShape, target->getShapeInfo()))
            throw std::runtime_error("NDArray::reduceAlongDimension FloatOps: wrong target shape!");
        RELEASE(newShape, _context->getWorkspace());
    }

    if(rankOf() == copy.size() || copy.empty()) {        
        NativeOpExecutioner::execReduceFloatScalar(_context, op, _buffer, _shapeInfo, _bufferD, _shapeInfoD,nullptr, target->_buffer, target->_shapeInfo, target->_bufferD, target->_shapeInfoD);        

        auto cudaResult = hipStreamSynchronize(*_context->getCudaStream());
        if (cudaResult != 0) throw cuda_exception::build("NDArray::applyAllReduce3 cuda failed !", cudaResult);
    }
    else {

        shape::TAD xTad(_shapeInfo, copy.data(), copy.size());
        xTad.createTadOnlyShapeInfo();
        xTad.createOffsets();

         // device memory allocation for tads
        Nd4jLong *xTadShapeInfo, *xTadOffsets;
        int *dims;     
        
        auto cudaResult = hipMalloc(reinterpret_cast<void **>(&dims), copy.size() * sizeof(int));
        if (cudaResult != 0) throw cuda_exception::build("NDArray::applyAllReduce3 cuda memory allocation failed !", cudaResult);

        cudaResult = hipMalloc(reinterpret_cast<void **>(&xTadShapeInfo), shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo));
        if (cudaResult != 0) throw cuda_exception::build("NDArray::applyAllReduce3 cuda memory allocation failed !", cudaResult);

        cudaResult = hipMalloc(reinterpret_cast<void **>(&xTadOffsets), xTad.numTads * sizeof(Nd4jLong));
        if (cudaResult != 0) throw cuda_exception::build("NDArray::applyAllReduce3 cuda memory allocation failed !", cudaResult);

        hipMemcpyAsync(dims, copy.data(), copy.size() * sizeof(int), hipMemcpyHostToDevice, *_context->getCudaStream());  
        hipMemcpyAsync(xTadShapeInfo, xTad.tadOnlyShapeInfo, shape::shapeInfoByteLength(xTad.tadOnlyShapeInfo), hipMemcpyHostToDevice, *_context->getCudaStream());
        hipMemcpyAsync(xTadOffsets, xTad.tadOffsets, xTad.numTads * sizeof(Nd4jLong), hipMemcpyHostToDevice, *_context->getCudaStream());

        NativeOpExecutioner::execReduceFloat(_context, op, _buffer, _shapeInfo, _bufferD, _shapeInfoD, nullptr, target->_buffer, target->_shapeInfo, target->_bufferD, target->_shapeInfoD, dims, copy.size(), xTadShapeInfo, xTadOffsets);

        cudaResult = hipStreamSynchronize(*_context->getCudaStream());
        if (cudaResult != 0) throw cuda_exception::build("NDArray::applyAllReduce3 cuda failed !", cudaResult);

        hipFree(dims); hipFree(xTadShapeInfo); hipFree(xTadOffsets);
    }
    NDArray::registerSpecialUse({target}, {this});
}

//////////////////////////////////////////////////////////////////////////
// This method sets value in linear buffer to position i
    template <typename T>
    void NDArray::p(const Nd4jLong i, const T value) {
        lazyAllocateBuffer();
        if (isActualOnDeviceSide() && !isActualOnHostSide())
            syncToHost();
        if (i >= _length)
            throw std::invalid_argument("NDArray::p(i, value): input index is out of array length !");

        auto rp = getOffset(i);
        const void *pV = reinterpret_cast<const void*>(const_cast<T *>(&value));
        BUILD_SINGLE_PARTIAL_SELECTOR(this->dataType(), templatedSet<, T>(this->_buffer, rp, pV), LIBND4J_TYPES);
        syncToDevice();
    }
    template void NDArray::p(const Nd4jLong i, const double value);
    template void NDArray::p(const Nd4jLong i, const float value);
    template void NDArray::p(const Nd4jLong i, const float16 value);
    template void NDArray::p(const Nd4jLong i, const bfloat16 value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong value);
    template void NDArray::p(const Nd4jLong i, const int value);
    template void NDArray::p(const Nd4jLong i, const int8_t value);
    template void NDArray::p(const Nd4jLong i, const uint8_t value);
    template void NDArray::p(const Nd4jLong i, const int16_t value);
    template void NDArray::p(const Nd4jLong i, const bool value);

    void NDArray::p(const Nd4jLong i, const NDArray& scalar) {
        if(!scalar.isScalar())
            throw std::invalid_argument("NDArray::p method: input array must be scalar!");
        if (i >= _length)
            throw std::invalid_argument("NDArray::p(i, NDArray_scalar): input index is out of array length !");
        // probably wrong args order
        lazyAllocateBuffer();
        auto rp = getOffset(i);
        BUILD_SINGLE_SELECTOR(scalar.dataType(), templatedSet, (_buffer, rp, scalar.dataType(), scalar.getBuffer()), LIBND4J_TYPES);
        // void NDArray::templatedSet(void *buffer, const Nd4jLong xOfsset, nd4j::DataType dtype, void *value)
        syncToDevice();
    }


//////////////////////////////////////////////////////////////////////////
// This method sets value in 2D matrix to position i, j

    template <typename T>
    void NDArray::p(const Nd4jLong i, const Nd4jLong j, const T value) {
        //(*this)(i,j) = value;
        if (rankOf() != 2 || i >= shapeOf()[0] || j >= shapeOf()[1])
            throw std::invalid_argument("NDArray:pe(i,j, value): one of input indexes is out of array length or rank!=2 !");

        lazyAllocateBuffer();
        void *p = reinterpret_cast<void *>(const_cast<T *>(&value));
        auto xType = this->dataType();
        Nd4jLong coords[2] = {i, j};
        auto xOffset = shape::getOffset(0, shapeOf(), stridesOf(), coords, rankOf());
        BUILD_SINGLE_PARTIAL_SELECTOR(xType, templatedSet<, T>(this->_buffer, xOffset, p), LIBND4J_TYPES);
        syncToDevice();
    }
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const double value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const float value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const float16 value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const bfloat16 value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const int value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const int8_t value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const uint8_t value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const int16_t value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const bool value);
    // template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const utf8string value);

//////////////////////////////////////////////////////////////////////////
// This method sets value in 3D matrix to position i,j,k
    template <typename T>
    void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const T value) {
        //(*this)(i,j,k) = value;
        if (rankOf() != 3 || i >= shapeOf()[0] || j >= shapeOf()[1] || k >= shapeOf()[2])
            throw std::invalid_argument("NDArray:pe(i,j,k, value): one of input indexes is out of array length or rank!=3 !");
        void *p = reinterpret_cast<void *>(const_cast<T *>(&value));
        auto xType = this->dataType();
        Nd4jLong coords[3] = {i, j, k};
        lazyAllocateBuffer();
        auto xOffset = shape::getOffset(0, shapeOf(), stridesOf(), coords, rankOf());
        BUILD_SINGLE_PARTIAL_SELECTOR(xType, templatedSet<, T>(this->_buffer, xOffset, p), LIBND4J_TYPES);
        syncToDevice();
    }
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const double value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const float value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const float16 value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const bfloat16 value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const Nd4jLong value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const int value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const int8_t value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const uint8_t value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const int16_t value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const bool value);

//////////////////////////////////////////////////////////////////////////
    template <typename T>
    void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const Nd4jLong l, const T value) {
        //(*this)(i,j,k) = value;
        if (rankOf() != 4 || i >= shapeOf()[0] || j >= shapeOf()[1] || k >= shapeOf()[2] || l >= shapeOf()[3])
            throw std::invalid_argument("NDArray::p(i,j,k,l, value): one of input indexes is out of array length or rank!=4 !");
        void *p = reinterpret_cast<void *>(const_cast<T *>(&value));
        auto xType = this->dataType();
        Nd4jLong coords[4] = {i, j, k, l};
        lazyAllocateBuffer();
        auto xOffset = shape::getOffset(0, shapeOf(), stridesOf(), coords, rankOf());
        BUILD_SINGLE_PARTIAL_SELECTOR(xType, templatedSet<, T>(this->_buffer, xOffset, p), LIBND4J_TYPES);
        syncToDevice();
    }
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const Nd4jLong l, const double value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const Nd4jLong l, const float value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const Nd4jLong l, const float16 value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const Nd4jLong l, const bfloat16 value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const Nd4jLong l, const Nd4jLong value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const Nd4jLong l, const int value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const Nd4jLong l, const int8_t value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const Nd4jLong l, const uint8_t value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const Nd4jLong l, const int16_t value);
    template void NDArray::p(const Nd4jLong i, const Nd4jLong j, const Nd4jLong k, const Nd4jLong l, const bool value);

//////////////////////////////////////////////////////////////////////////
    void* NDArray::specialBufferWithOffset(Nd4jLong offset) const {
        return _bufferD + (offset * sizeOfT());
    }

//////////////////////////////////////////////////////////////////////////    
    NDArray* NDArray::tensorAlongDimension(Nd4jLong index, const std::vector<int>& dimensions) const {
        std::vector<int> copy(dimensions);
        shape::checkDimensions(rankOf(), copy);

        Nd4jLong tadLength = shape::tadLength(this->_shapeInfo, copy.data(), copy.size());
        Nd4jLong numTads = this->lengthOf() / tadLength;

        if (index >= numTads)
            throw std::runtime_error("Can't get index higher than total number of TADs");

        shape::TAD tad(this->_shapeInfo, copy.data(), copy.size());
        tad.createTadOnlyShapeInfo();
        tad.createOffsets();

        Nd4jLong* shapeInfo;
        if (_context->getWorkspace() == nullptr) {
            shapeInfo = new Nd4jLong[shape::shapeInfoLength(tad.tadOnlyShapeInfo)];
        } else {
            shapeInfo = reinterpret_cast<Nd4jLong *>(_context->getWorkspace()->allocateBytes(shape::shapeInfoByteLength(tad.tadOnlyShapeInfo)));
        }
        std::memcpy(shapeInfo, tad.tadOnlyShapeInfo, shape::shapeInfoByteLength(tad.tadOnlyShapeInfo));
        auto array = new NDArray(shapeInfo, _context);
        //hipFree(array->_bufferD);
        array->_bufferD = (int8_t*)specialBufferWithOffset(tad.tadOffsets[index]); //, array->lengthOf() * DataTypeUtils::sizeOf(dataType()), hipMemcpyDeviceToDevice);
        array->_isBuffAlloc = false;
        array->_isBuffDAlloc = false;
        array->_isShapeAlloc = true;
        array->_isShapeDAlloc = true;
        array->_isView = true;

        return array;
    }




}



#endif

